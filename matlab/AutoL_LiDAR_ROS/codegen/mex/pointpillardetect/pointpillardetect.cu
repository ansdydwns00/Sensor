#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// pointpillardetect.cu
//
// Code generation for function 'pointpillardetect'
//

// Include files
#include "pointpillardetect.h"
#include "PointCloudImpl.h"
#include "dlnetwork.h"
#include "handle.h"
#include "pointPillarsObjectDetector.h"
#include "pointpillardetect_data.h"
#include "pointpillardetect_emxutil.h"
#include "pointpillardetect_internal_types.h"
#include "pointpillardetect_mexutil.h"
#include "pointpillardetect_types.h"
#include "predict.h"
#include "rt_nonfinite.h"
#include "selectStrongestBboxMulticlass.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWCudnnCustomLayerBase.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include "MWSortFunctors.h"
#include "MWSortWithIndexUtility.h"
#include "MWTensorBase.hpp"
#include "createPillars.hpp"
#include <cmath>
#include <cstring>

// Type Definitions
struct cell_wrap_5 {
  real_T f1[10];
};

// Variable Definitions
static pretrained0_0 gobj_0;

static coder::internal::pointPillarsObjectDetector pointPillarObj;

static boolean_T pointPillarObj_not_empty;

static emlrtMCInfo emlrtMCI{
    57,                                                              // lineNo
    5,                                                               // colNo
    "repmat",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/repmat.m" // pName
};

static emlrtRTEInfo emlrtRTEI{
    6,                   // lineNo
    12,                  // colNo
    "pointpillardetect", // fName
    "/home/user/ros2_ws/src/YongJun_ws/matlab/AutoL_LiDAR_ROS/"
    "pointpillardetect.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    85,               // lineNo
    13,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    144,              // lineNo
    13,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    11,                          // lineNo
    15,                          // colNo
    "createAnchorsPointPillars", // fName
    "/usr/local/MATLAB/R2024a/toolbox/lidar/lidar/+lidar/+internal/+cnn/"
    "createAnchorsPointPillars.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    11,                          // lineNo
    5,                           // colNo
    "createAnchorsPointPillars", // fName
    "/usr/local/MATLAB/R2024a/toolbox/lidar/lidar/+lidar/+internal/+cnn/"
    "createAnchorsPointPillars.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    107,                                                              // lineNo
    20,                                                               // colNo
    "repelem",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/repelem.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    28,                          // lineNo
    1,                           // colNo
    "createAnchorsPointPillars", // fName
    "/usr/local/MATLAB/R2024a/toolbox/lidar/lidar/+lidar/+internal/+cnn/"
    "createAnchorsPointPillars.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    134,              // lineNo
    13,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    139,              // lineNo
    12,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    69,                                                              // lineNo
    28,                                                              // colNo
    "repmat",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/repmat.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    770,              // lineNo
    17,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    36,                          // lineNo
    1,                           // colNo
    "createAnchorsPointPillars", // fName
    "/usr/local/MATLAB/R2024a/toolbox/lidar/lidar/+lidar/+internal/+cnn/"
    "createAnchorsPointPillars.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    1,                            // lineNo
    1,                            // colNo
    "pointPillarsObjectDetector", // fName
    "/home/user/Documents/MATLAB/SupportPackages/R2024a/toolbox/shared/"
    "dlcoder_base/supportpackages/shared_dl_targets/+coder/+interna"
    "l/pointPillarsObjectDetector.p" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    37,                          // lineNo
    1,                           // colNo
    "createAnchorsPointPillars", // fName
    "/usr/local/MATLAB/R2024a/toolbox/lidar/lidar/+lidar/+internal/+cnn/"
    "createAnchorsPointPillars.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    38,                          // lineNo
    1,                           // colNo
    "createAnchorsPointPillars", // fName
    "/usr/local/MATLAB/R2024a/toolbox/lidar/lidar/+lidar/+internal/+cnn/"
    "createAnchorsPointPillars.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    105,                                                          // lineNo
    24,                                                           // colNo
    "cat",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/cat.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    60,                                                               // lineNo
    20,                                                               // colNo
    "permute",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/permute.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    1,                   // lineNo
    35,                  // colNo
    "pointpillardetect", // fName
    "/home/user/ros2_ws/src/YongJun_ws/matlab/AutoL_LiDAR_ROS/"
    "pointpillardetect.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    12,                  // lineNo
    1,                   // colNo
    "pointpillardetect", // fName
    "/home/user/ros2_ws/src/YongJun_ws/matlab/AutoL_LiDAR_ROS/"
    "pointpillardetect.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    9,                           // lineNo
    1,                           // colNo
    "createAnchorsPointPillars", // fName
    "/usr/local/MATLAB/R2024a/toolbox/lidar/lidar/+lidar/+internal/+cnn/"
    "createAnchorsPointPillars.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    25,                          // lineNo
    1,                           // colNo
    "createAnchorsPointPillars", // fName
    "/usr/local/MATLAB/R2024a/toolbox/lidar/lidar/+lidar/+internal/+cnn/"
    "createAnchorsPointPillars.m" // pName
};

static emlrtRTEInfo v_emlrtRTEI{
    35,                          // lineNo
    1,                           // colNo
    "createAnchorsPointPillars", // fName
    "/usr/local/MATLAB/R2024a/toolbox/lidar/lidar/+lidar/+internal/+cnn/"
    "createAnchorsPointPillars.m" // pName
};

static emlrtRTEInfo w_emlrtRTEI{
    30,                          // lineNo
    1,                           // colNo
    "createAnchorsPointPillars", // fName
    "/usr/local/MATLAB/R2024a/toolbox/lidar/lidar/+lidar/+internal/+cnn/"
    "createAnchorsPointPillars.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    770,              // lineNo
    25,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static boolean_T c_gpuConstsCopied_pointpillarde;

// Function Declarations
static void b_error(const mxArray *m, emlrtMCInfo *location);

static uint64_T computeNumIters(int32_T ub, int32_T b_ub, int32_T c_ub);

static int32_T div_s32(int32_T numerator, int32_T denominator);

static
#ifdef __HIPCC__
    __device__
#endif
        int32_T
        div_s32_device(int32_T numerator, int32_T denominator);

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu);

static void gpuEmxMemcpyGpuToCpu_boolean_T(emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu);

static void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu);

static void gpuEmxMemcpyGpuToCpu_uint8_T(emxArray_uint8_T *cpu,
                                         emxArray_uint8_T *gpu);

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu);

static __global__ void pointpillardetect_kernel1(
    cell_wrap_5 *r1, cell_wrap_5 *r, real_T dv1_dim0, real_T dv1_dim1,
    real_T dv1_dim2, real_T dv1_dim3, real_T dv1_dim4, real_T dv1_dim5,
    real_T dv1_dim6, real_T dv1_dim7, real_T dv1_dim8, real_T dv1_dim9,
    real_T dv_dim0, real_T dv_dim1, real_T dv_dim2, real_T dv_dim3,
    real_T dv_dim4, real_T dv_dim5, real_T dv_dim6, real_T dv_dim7,
    real_T dv_dim8, real_T dv_dim9);

static __global__ void
pointpillardetect_kernel10(const emxArray_real_T anchorsDim, const int32_T b,
                           emxArray_real_T anchors3dMod,
                           int32_T anchors3dMod_dim0, int32_T anchorsDim_dim0);

static __global__ void pointpillardetect_kernel11(
    const emxArray_real_T anchors3dMod, const int32_T rowreps, const int32_T i,
    const int32_T sizes, emxArray_real_T b_anchors3dMod,
    int32_T anchors3dMod_dim0, int32_T b_anchors3dMod_dim0);

static __global__ void pointpillardetect_kernel12(
    const emxArray_real_T anchors3dMod, const int32_T ibcol,
    const int32_T rowIdx, const int32_T b_anchors3dMod,
    emxArray_real_T c_anchors3dMod, int32_T anchors3dMod_dim0,
    int32_T b_anchors3dMod_dim0);

static __global__ void
pointpillardetect_kernel13(const emxArray_real_T anchors3dMod,
                           const int32_T b_anchors3dMod, emxArray_real_T tmp2,
                           emxArray_real_T tmp1, int32_T tmp1_dim0,
                           int32_T anchors3dMod_dim0, int32_T tmp2_dim0);

static __global__ void pointpillardetect_kernel14(const emxArray_uint8_T b,
                                                  const int32_T b_b,
                                                  emxArray_uint8_T b_value);

static __global__ void
pointpillardetect_kernel15(real32_T pillarIndices[24000]);

static __global__ void
pointpillardetect_kernel16(real32_T dlPillarFeatures_Data[10800000]);

static __global__ void pointpillardetect_kernel17(
    cell_wrap_5 *r1, cell_wrap_5 *r, real_T dv1_dim0, real_T dv1_dim1,
    real_T dv1_dim2, real_T dv1_dim3, real_T dv1_dim4, real_T dv1_dim5,
    real_T dv1_dim6, real_T dv1_dim7, real_T dv1_dim8, real_T dv1_dim9,
    real_T dv_dim0, real_T dv_dim1, real_T dv_dim2, real_T dv_dim3,
    real_T dv_dim4, real_T dv_dim5, real_T dv_dim6, real_T dv_dim7,
    real_T dv_dim8, real_T dv_dim9);

static __global__ void
pointpillardetect_kernel18(real_T sparseVoxelMap[214272]);

static __global__ void
pointpillardetect_kernel19(const real32_T pillarIndices[24000], const int32_T b,
                           int32_T iv2_data[12000]);

static __global__ void pointpillardetect_kernel2(const emxArray_real_T anchors,
                                                 const int32_T b_anchors,
                                                 emxArray_real_T c_anchors,
                                                 int32_T anchors_dim0,
                                                 int32_T b_anchors_dim0);

static __global__ void
pointpillardetect_kernel20(const int32_T iv2_data[12000], const int32_T b,
                           real_T sparseVoxelMap[214272]);

static __global__ void
pointpillardetect_kernel21(const emxArray_real_T anchorMask,
                           const int32_T b_anchorMask,
                           emxArray_boolean_T c_anchorMask);

static __global__ void
pointpillardetect_kernel22(const emxArray_real_T anchorMask, const int32_T b,
                           emxArray_boolean_T b_anchorMask);

static __global__ void
pointpillardetect_kernel23(const emxArray_boolean_T anchorMask,
                           const int32_T b_anchorMask,
                           emxArray_boolean_T c_anchorMask);

static __global__ void
pointpillardetect_kernel24(const emxArray_boolean_T anchorMask,
                           const real_T threshold,
                           const real32_T predOcc[214272], boolean_T x[214272]);

static __global__ void pointpillardetect_kernel25(const int32_T ii_data[214272],
                                                  const int32_T b,
                                                  int32_T v1_data[214272]);

static __global__ void pointpillardetect_kernel26(const int32_T v1_data[214272],
                                                  const int32_T b,
                                                  int16_T vk_data[214272]);

static __global__ void
pointpillardetect_kernel27(const int16_T vk_data[214272], const int32_T b,
                           int32_T v1_data[214272],
                           int8_T varargout_6_data[214272]);

static __global__ void pointpillardetect_kernel28(const int32_T v1_data[214272],
                                                  const int32_T b,
                                                  int16_T vk_data[214272]);

static __global__ void pointpillardetect_kernel29(const int16_T vk_data[214272],
                                                  const int32_T b,
                                                  int32_T v1_data[214272]);

static __global__ void pointpillardetect_kernel3(const emxArray_real_T anchors,
                                                 const int32_T b_anchors,
                                                 emxArray_real_T c_anchors);

static __global__ void pointpillardetect_kernel30(const int16_T vk_data[214272],
                                                  const int32_T v1_data[214272],
                                                  const int32_T b,
                                                  int16_T col_data[214272],
                                                  int32_T ii_data[214272]);

static __global__ void pointpillardetect_kernel31(
    const real32_T predOcc[214272], const int8_T varargout_6_data[214272],
    const int16_T col_data[214272], const int32_T ii_data[214272],
    const int32_T b, real_T yCen_data[214272], real_T xCen_data[214272],
    real32_T confScore_data[214272]);

static __global__ void pointpillardetect_kernel32(
    const real32_T this_workspace_predHeading[214272],
    const int8_T varargout_6_data[214272], const int16_T col_data[214272],
    const int32_T ii_data[214272], const int32_T b, real32_T hdGt_data[214272]);

static __global__ void pointpillardetect_kernel33(const int32_T rowIdx,
                                                  real32_T hdGt_data[214272]);

static __global__ void pointpillardetect_kernel34(const int32_T rowIdx,
                                                  real32_T hdGt_data[214272]);

static __global__ void pointpillardetect_kernel35(real32_T predAngle[214272]);

static __global__ void pointpillardetect_kernel36(const int32_T rowIdx,
                                                  real32_T angGt_data[214272]);

static __global__ void pointpillardetect_kernel37(const int32_T rowIdx,
                                                  real32_T angGt_data[214272]);

static __global__ void pointpillardetect_kernel38(const int32_T b,
                                                  emxArray_real_T clsMat);

static __global__ void pointpillardetect_kernel39(
    const real32_T predClassification[428544], const int32_T ibcol,
    const int8_T varargout_6_data[214272], const int16_T col_data[214272],
    const int32_T ii_data[214272], const int32_T b, emxArray_real_T clsMat,
    int32_T clsMat_dim1, int32_T clsMat_dim0);

static __global__ void pointpillardetect_kernel4(const real_T dv3[248],
                                                 real_T xx[53568],
                                                 real_T yy[53568]);

static __global__ void pointpillardetect_kernel40(const int32_T clsMat,
                                                  int32_T ii_data[214272]);

static __global__ void
pointpillardetect_kernel41(const int32_T rowreps, const emxArray_real_T clsMat,
                           const int32_T rowIdx, const int32_T colIdx,
                           const int32_T b, int32_T ii_data[214272],
                           real_T xCen_data[214272]);

static __global__ void pointpillardetect_kernel42(const int32_T ii_data[214272],
                                                  const int32_T b,
                                                  real_T yCen_data[214272]);

static __global__ void pointpillardetect_kernel43(
    const real_T yCen_data[214272], const real32_T confScore_data[214272],
    const real32_T angGt_data[214272], const real32_T hGt_data[214272],
    const real32_T wGt_data[214272], const real32_T lGt_data[214272],
    const real32_T zGt_data[214272], const real32_T yGt_data[214272],
    const int32_T boxPreds_size_dim0, const real32_T xGt_data[214272],
    const int32_T b, real32_T boxPreds_data[1928448]);

static __global__ void pointpillardetect_kernel44(
    const real32_T boxPreds_data[1928448], const int32_T boxPreds_size_dim0,
    const int32_T boxPreds_size, boolean_T x_data[214272]);

static __global__ void
pointpillardetect_kernel45(const int32_T boxPreds_size_dim0, const int32_T b,
                           real32_T boxPreds_data[1928448]);

static __global__ void
pointpillardetect_kernel46(const real32_T boxPreds_data[1928448],
                           const int32_T boxPreds_size_dim0,
                           const int32_T ii_data[214272], const int32_T b,
                           real32_T hdGt_data[214272], real32_T scores_data[]);

static __global__ void
pointpillardetect_kernel47(const real32_T scores_data[],
                           const int32_T scores_size, real_T xCen_data[214272],
                           real32_T varargout_1_data[214272]);

static __global__ void pointpillardetect_kernel48(
    const real32_T boxPreds_data[1928448],
    const int32_T boxPreds_size_dim0_dup0, const int32_T ii_data[214272],
    const real_T xCen_data[214272], const int32_T boxPreds_size_dim0,
    const int32_T idx_size, real32_T b_boxPreds_data[1071360], int8_T iv23_dim0,
    int8_T iv23_dim1, int8_T iv23_dim2, int8_T iv23_dim3, int8_T iv23_dim4);

static __global__ void pointpillardetect_kernel49(
    const real32_T hdGt_data[214272], const real_T xCen_data[214272],
    const int32_T idx_size, real32_T varargout_1_data[214272]);

static __global__ void pointpillardetect_kernel5(const real_T yy[53568],
                                                 const real_T xx[53568],
                                                 real_T tmp[107136]);

static __global__ void
pointpillardetect_kernel50(const boolean_T selectedIndex_data[214272],
                           const int32_T selectedIndex_size,
                           boolean_T x_data[214272]);

static __global__ void
pointpillardetect_kernel51(const boolean_T x_data[214272],
                           const real_T xCen_data[214272], const int32_T b,
                           boolean_T selectedIndex_data[214272]);

static __global__ void pointpillardetect_kernel52(const int32_T b,
                                                  real_T bboxes_data[]);

static __global__ void pointpillardetect_kernel53(
    const real32_T boxPreds_data[1928448], const int32_T boxPreds_size_dim0,
    const int32_T ii_data[214272], const int32_T sizes, real_T bboxes_data[],
    int8_T iv27_dim0, int8_T iv27_dim1, int8_T iv27_dim2, int8_T iv27_dim3,
    int8_T iv27_dim4, int8_T iv27_dim5, int8_T iv27_dim6,
    int32_T bboxes_size_dim0);

static __global__ void
pointpillardetect_kernel54(const real32_T hdGt_data[214272],
                           const int32_T tmp_data[214272], const int32_T b,
                           real32_T varargout_1_data[214272]);

static __global__ void
pointpillardetect_kernel55(const real32_T predClassification[428544],
                           const int8_T varargout_6_data[214272],
                           const int16_T col_data[214272],
                           const int32_T ii_data[214272], const int32_T b,
                           real32_T varargout_1_data[214272]);

static __global__ void
pointpillardetect_kernel56(const real32_T varargout_1_data[214272],
                           const int32_T b, real_T xCen_data[214272]);

static __global__ void pointpillardetect_kernel57(const int32_T b,
                                                  real_T xCen_data[214272]);

static __global__ void
pointpillardetect_kernel58(const real_T xCen_data[214272], const int32_T b,
                           real_T yCen_data[214272]);

static __global__ void pointpillardetect_kernel59(const real_T threshold,
                                                  const int32_T rowIdx,
                                                  real_T yCen_data[214272]);

static __global__ void pointpillardetect_kernel6(const int32_T b,
                                                 emxArray_real_T anchors3dMod);

static __global__ void pointpillardetect_kernel60(const real_T threshold,
                                                  const int32_T rowIdx,
                                                  real_T yCen_data[214272]);

static __global__ void
pointpillardetect_kernel61(const emxArray_boolean_T anchorMask,
                           const uint8_T dv6_dim0, const int32_T dv6,
                           emxArray_boolean_T b_anchorMask);

static __global__ void pointpillardetect_kernel7(const emxArray_real_T tmp,
                                                 const int32_T sizes,
                                                 emxArray_real_T anchors3dMod,
                                                 int32_T anchors3dMod_dim0,
                                                 int32_T tmp_dim0);

static __global__ void
pointpillardetect_kernel8(const emxArray_real_T anchorsDim, const int32_T b,
                          emxArray_real_T anchors3dMod,
                          int32_T anchors3dMod_dim0, int32_T anchorsDim_dim0);

static __global__ void
pointpillardetect_kernel9(const emxArray_real_T anchorsDim, const int32_T sizes,
                          emxArray_real_T anchors3dMod,
                          int32_T anchors3dMod_dim0, int32_T anchorsDim_dim0);

// Function Definitions
static void b_error(const mxArray *m, emlrtMCInfo *location)
{
  const mxArray *pArray;
  pArray = m;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, nullptr, 1, &pArray, "error",
                        true, location);
}

static uint64_T computeNumIters(int32_T ub, int32_T b_ub, int32_T c_ub)
{
  uint64_T n;
  uint64_T numIters;
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
  }
  numIters *= n;
  n = 0UL;
  if (c_ub >= 0) {
    n = static_cast<uint64_T>(c_ub + 1);
  }
  numIters *= n;
  return numIters;
}

static int32_T div_s32(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  if (denominator == 0) {
    emlrtDivisionByZeroErrorR2012b(nullptr, emlrtRootTLSGlobal);
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if ((numerator < 0) != (denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  return quotient;
}

static __device__ int32_T div_s32_device(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  if (denominator == 0) {
    if (numerator >= 0) {
      quotient = MAX_int32_T;
    } else {
      quotient = MIN_int32_T;
    }
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if (static_cast<int32_T>(numerator < 0) !=
        static_cast<int32_T>(denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  return quotient;
}

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyGpuToCpu_boolean_T(emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(
      hipMemcpy(cpu->data, gpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(boolean_T),
                 hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
}

static void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(
      hipMemcpy(cpu->data, gpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(real32_T),
                 hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
}

static void gpuEmxMemcpyGpuToCpu_uint8_T(emxArray_uint8_T *cpu,
                                         emxArray_uint8_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(uint8_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
}

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_boolean_T));
}

static __global__ __launch_bounds__(32, 1) void pointpillardetect_kernel1(
    cell_wrap_5 *r1, cell_wrap_5 *r, real_T dv1_dim0, real_T dv1_dim1,
    real_T dv1_dim2, real_T dv1_dim3, real_T dv1_dim4, real_T dv1_dim5,
    real_T dv1_dim6, real_T dv1_dim7, real_T dv1_dim8, real_T dv1_dim9,
    real_T dv_dim0, real_T dv_dim1, real_T dv_dim2, real_T dv_dim3,
    real_T dv_dim4, real_T dv_dim5, real_T dv_dim6, real_T dv_dim7,
    real_T dv_dim8, real_T dv_dim9)
{
  __shared__ real_T dv1_shared[10];
  __shared__ real_T dv_shared[10];
  int32_T i;
  if (mwGetThreadIndexWithinBlock() == 0U) {
    dv_shared[0] = dv_dim0;
    dv_shared[1] = dv_dim1;
    dv_shared[2] = dv_dim2;
    dv_shared[3] = dv_dim3;
    dv_shared[4] = dv_dim4;
    dv_shared[5] = dv_dim5;
    dv_shared[6] = dv_dim6;
    dv_shared[7] = dv_dim7;
    dv_shared[8] = dv_dim8;
    dv_shared[9] = dv_dim9;
    dv1_shared[0] = dv1_dim0;
    dv1_shared[1] = dv1_dim1;
    dv1_shared[2] = dv1_dim2;
    dv1_shared[3] = dv1_dim3;
    dv1_shared[4] = dv1_dim4;
    dv1_shared[5] = dv1_dim5;
    dv1_shared[6] = dv1_dim6;
    dv1_shared[7] = dv1_dim7;
    dv1_shared[8] = dv1_dim8;
    dv1_shared[9] = dv1_dim9;
  }
  __syncthreads();
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 10) {
    r->f1[i] = dv_shared[i];
    r1->f1[i] = dv1_shared[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel10(
    const emxArray_real_T anchorsDim, const int32_T b,
    emxArray_real_T anchors3dMod, int32_T anchors3dMod_dim0,
    int32_T anchorsDim_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    anchors3dMod.data[i + anchors3dMod_dim0 * 6] =
        anchorsDim.data[i + anchorsDim_dim0 * 4];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel11(
    const emxArray_real_T anchors3dMod, const int32_T rowreps, const int32_T i,
    const int32_T sizes, emxArray_real_T b_anchors3dMod,
    int32_T anchors3dMod_dim0, int32_T b_anchors3dMod_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 2UL * (static_cast<uint64_T>(sizes) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_i;
    int32_T ix;
    ix = static_cast<int32_T>(idx % (static_cast<uint64_T>(sizes) + 1UL));
    b_i = static_cast<int32_T>((idx - static_cast<uint64_T>(ix)) /
                               (static_cast<uint64_T>(sizes) + 1UL));
    b_anchors3dMod.data[ix + anchors3dMod_dim0 * b_i] =
        anchors3dMod.data[(i + rowreps * ix) + b_anchors3dMod_dim0 * (4 - b_i)];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel12(
    const emxArray_real_T anchors3dMod, const int32_T ibcol,
    const int32_T rowIdx, const int32_T b_anchors3dMod,
    emxArray_real_T c_anchors3dMod, int32_T anchors3dMod_dim0,
    int32_T b_anchors3dMod_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 2UL * (static_cast<uint64_T>(b_anchors3dMod) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T ix;
    ix = static_cast<int32_T>(idx %
                              (static_cast<uint64_T>(b_anchors3dMod) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(ix)) /
                             (static_cast<uint64_T>(b_anchors3dMod) + 1UL));
    c_anchors3dMod
        .data[((rowIdx + ibcol * ix) + anchors3dMod_dim0 * (i + 3)) - 1] =
        anchors3dMod.data[ix + b_anchors3dMod_dim0 * i];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel13(
    const emxArray_real_T anchors3dMod, const int32_T b_anchors3dMod,
    emxArray_real_T tmp2, emxArray_real_T tmp1, int32_T tmp1_dim0,
    int32_T anchors3dMod_dim0, int32_T tmp2_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 2UL * (static_cast<uint64_T>(b_anchors3dMod) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T ix;
    ix = static_cast<int32_T>(idx %
                              (static_cast<uint64_T>(b_anchors3dMod) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(ix)) /
                             (static_cast<uint64_T>(b_anchors3dMod) + 1UL));
    tmp1.data[ix + tmp1_dim0 * i] =
        anchors3dMod.data[ix + anchors3dMod_dim0 * i] -
        anchors3dMod.data[ix + anchors3dMod_dim0 * (i + 3)] / 2.0;
    tmp2.data[ix + tmp2_dim0 * i] =
        anchors3dMod.data[ix + anchors3dMod_dim0 * i] +
        anchors3dMod.data[ix + anchors3dMod_dim0 * (i + 3)] / 2.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel14(
    const emxArray_uint8_T b, const int32_T b_b, emxArray_uint8_T b_value)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    b_value.data[i] = b.data[i];
  }
}

static __global__ __launch_bounds__(224, 1) void pointpillardetect_kernel15(
    real32_T pillarIndices[24000])
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 24000) {
    pillarIndices[i] = 0.0F;
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillardetect_kernel16(
    real32_T dlPillarFeatures_Data[10800000])
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 10800000) {
    dlPillarFeatures_Data[i] = 0.0F;
  }
}

static __global__ __launch_bounds__(32, 1) void pointpillardetect_kernel17(
    cell_wrap_5 *r1, cell_wrap_5 *r, real_T dv1_dim0, real_T dv1_dim1,
    real_T dv1_dim2, real_T dv1_dim3, real_T dv1_dim4, real_T dv1_dim5,
    real_T dv1_dim6, real_T dv1_dim7, real_T dv1_dim8, real_T dv1_dim9,
    real_T dv_dim0, real_T dv_dim1, real_T dv_dim2, real_T dv_dim3,
    real_T dv_dim4, real_T dv_dim5, real_T dv_dim6, real_T dv_dim7,
    real_T dv_dim8, real_T dv_dim9)
{
  __shared__ real_T dv1_shared[10];
  __shared__ real_T dv_shared[10];
  int32_T i;
  if (mwGetThreadIndexWithinBlock() == 0U) {
    dv_shared[0] = dv_dim0;
    dv_shared[1] = dv_dim1;
    dv_shared[2] = dv_dim2;
    dv_shared[3] = dv_dim3;
    dv_shared[4] = dv_dim4;
    dv_shared[5] = dv_dim5;
    dv_shared[6] = dv_dim6;
    dv_shared[7] = dv_dim7;
    dv_shared[8] = dv_dim8;
    dv_shared[9] = dv_dim9;
    dv1_shared[0] = dv1_dim0;
    dv1_shared[1] = dv1_dim1;
    dv1_shared[2] = dv1_dim2;
    dv1_shared[3] = dv1_dim3;
    dv1_shared[4] = dv1_dim4;
    dv1_shared[5] = dv1_dim5;
    dv1_shared[6] = dv1_dim6;
    dv1_shared[7] = dv1_dim7;
    dv1_shared[8] = dv1_dim8;
    dv1_shared[9] = dv1_dim9;
  }
  __syncthreads();
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 10) {
    r->f1[i] = dv_shared[i];
    r1->f1[i] = dv1_shared[i];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillardetect_kernel18(
    real_T sparseVoxelMap[214272])
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 214272) {
    sparseVoxelMap[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel19(
    const real32_T pillarIndices[24000], const int32_T b,
    int32_T iv2_data[12000])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    iv2_data[i] = static_cast<int32_T>(pillarIndices[i]) +
                  432 * (static_cast<int32_T>(pillarIndices[i + 12000]) - 1);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel2(
    const emxArray_real_T anchors, const int32_T b_anchors,
    emxArray_real_T c_anchors, int32_T anchors_dim0, int32_T b_anchors_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 5UL * (static_cast<uint64_T>(b_anchors) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T ix;
    ix = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_anchors) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(ix)) /
                             (static_cast<uint64_T>(b_anchors) + 1UL));
    c_anchors.data[ix + anchors_dim0 * i] =
        anchors.data[ix + b_anchors_dim0 * i];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel20(
    const int32_T iv2_data[12000], const int32_T b,
    real_T sparseVoxelMap[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    sparseVoxelMap[iv2_data[i] - 1] = 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel21(
    const emxArray_real_T anchorMask, const int32_T b_anchorMask,
    emxArray_boolean_T c_anchorMask)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_anchorMask);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_anchorMask.data[i] = (anchorMask.data[i] > 1.0);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel22(
    const emxArray_real_T anchorMask, const int32_T b,
    emxArray_boolean_T b_anchorMask)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    b_anchorMask.data[i] = (anchorMask.data[i] > 1.0);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel23(
    const emxArray_boolean_T anchorMask, const int32_T b_anchorMask,
    emxArray_boolean_T c_anchorMask)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_anchorMask);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_anchorMask.data[i] = anchorMask.data[i];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillardetect_kernel24(
    const emxArray_boolean_T anchorMask, const real_T threshold,
    const real32_T predOcc[214272], boolean_T x[214272])
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 214272) {
    x[i] =
        ((static_cast<real_T>(predOcc[i]) > threshold) && anchorMask.data[i]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel25(
    const int32_T ii_data[214272], const int32_T b, int32_T v1_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    v1_data[i] = ii_data[i] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel26(
    const int32_T v1_data[214272], const int32_T b, int16_T vk_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    vk_data[i] = static_cast<int16_T>(v1_data[i] / 53568);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel27(
    const int16_T vk_data[214272], const int32_T b, int32_T v1_data[214272],
    int8_T varargout_6_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int16_T i1;
    i = static_cast<int32_T>(idx);
    i1 = vk_data[i];
    varargout_6_data[i] = static_cast<int8_T>(static_cast<int32_T>(i1) + 1);
    v1_data[i] -= static_cast<int32_T>(i1) * 53568;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel28(
    const int32_T v1_data[214272], const int32_T b, int16_T vk_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    vk_data[i] = static_cast<int16_T>(v1_data[i] / 216);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel29(
    const int16_T vk_data[214272], const int32_T b, int32_T v1_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    v1_data[i] -= static_cast<int32_T>(vk_data[i]) * 216;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel3(
    const emxArray_real_T anchors, const int32_T b_anchors,
    emxArray_real_T c_anchors)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_anchors);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_anchors.data[i] = anchors.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel30(
    const int16_T vk_data[214272], const int32_T v1_data[214272],
    const int32_T b, int16_T col_data[214272], int32_T ii_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    ii_data[i] = v1_data[i] + 1;
    col_data[i] = static_cast<int16_T>(static_cast<int32_T>(vk_data[i]) + 1);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel31(
    const real32_T predOcc[214272], const int8_T varargout_6_data[214272],
    const int16_T col_data[214272], const int32_T ii_data[214272],
    const int32_T b, real_T yCen_data[214272], real_T xCen_data[214272],
    real32_T confScore_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T j;
    int16_T i1;
    j = static_cast<int32_T>(idx);
    i = ii_data[j];
    i1 = col_data[j];
    confScore_data[j] =
        predOcc[((i + 216 * (static_cast<int32_T>(i1) - 1)) +
                 53568 * (static_cast<int32_T>(varargout_6_data[j]) - 1)) -
                1];
    xCen_data[j] = 0.32 * (static_cast<real_T>(i) - 1.0) + 0.16;
    yCen_data[j] = (0.32 * (static_cast<real_T>(i1) - 1.0) - 39.68) + 0.16;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel32(
    const real32_T this_workspace_predHeading[214272],
    const int8_T varargout_6_data[214272], const int16_T col_data[214272],
    const int32_T ii_data[214272], const int32_T b, real32_T hdGt_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    hdGt_data[j] = this_workspace_predHeading
        [((ii_data[j] + 216 * (static_cast<int32_T>(col_data[j]) - 1)) +
          53568 * (static_cast<int32_T>(varargout_6_data[j]) - 1)) -
         1];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel33(
    const int32_T rowIdx, real32_T hdGt_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    if (hdGt_data[i] >= 0.5F) {
      hdGt_data[i] = 1.0F;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel34(
    const int32_T rowIdx, real32_T hdGt_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    if (hdGt_data[i] < 0.5F) {
      hdGt_data[i] = -1.0F;
    }
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillardetect_kernel35(
    real32_T predAngle[214272])
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 214272) {
    real32_T f;
    f = predAngle[i];
    if ((f > 1.0F) || (f < -1.0F)) {
      predAngle[i] = 0.0F;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel36(
    const int32_T rowIdx, real32_T angGt_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    real32_T f;
    i = static_cast<int32_T>(idx);
    f = angGt_data[i];
    if (static_cast<real_T>(f) > 3.1415926535897931) {
      f -= 6.28318548F;
      angGt_data[i] = f;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel37(
    const int32_T rowIdx, real32_T angGt_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    real32_T f;
    i = static_cast<int32_T>(idx);
    f = angGt_data[i];
    if (static_cast<real_T>(f) < -3.1415926535897931) {
      f += 6.28318548F;
      angGt_data[i] = f;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel38(
    const int32_T b, emxArray_real_T clsMat)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    clsMat.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel39(
    const real32_T predClassification[428544], const int32_T ibcol,
    const int8_T varargout_6_data[214272], const int16_T col_data[214272],
    const int32_T ii_data[214272], const int32_T b, emxArray_real_T clsMat,
    int32_T clsMat_dim1, int32_T clsMat_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T iv1[2];
    int32_T i;
    int32_T iv0;
    int32_T ix;
    int32_T j;
    i = static_cast<int32_T>(idx);
    iv0 = ii_data[i];
    ix = static_cast<int32_T>(col_data[i]);
    j = static_cast<int32_T>(varargout_6_data[i]);
    iv1[1] = clsMat_dim1;
    for (int32_T b_i{0}; b_i < iv1[1]; b_i++) {
      clsMat.data[i + clsMat_dim0 * b_i] = static_cast<real_T>(
          predClassification[(((iv0 + 216 * (ix - 1)) + 53568 * (j - 1)) +
                              53568 * ibcol * b_i) -
                             1]);
    }
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillardetect_kernel4(
    const real_T dv3[248], real_T xx[53568], real_T yy[53568])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T j;
  gThreadId = mwGetGlobalThreadIndex();
  i = static_cast<int32_T>(gThreadId % 216UL);
  j = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(i)) / 216UL);
  if ((j < 248) && (i < 216)) {
    yy[i + 216 * j] = dv3[j];
    xx[i + 216 * j] = 0.32000000000000006 * static_cast<real_T>(i) + 0.16;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel40(
    const int32_T clsMat, int32_T ii_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(clsMat);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    ii_data[i] = 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel41(
    const int32_T rowreps, const emxArray_real_T clsMat, const int32_T rowIdx,
    const int32_T colIdx, const int32_T b, int32_T ii_data[214272],
    real_T xCen_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T iv0;
    int32_T j;
    j = static_cast<int32_T>(idx);
    iv0 = (div_s32_device(j, colIdx) * rowIdx + j) + 1;
    xCen_data[j] = clsMat.data[iv0 - 1];
    for (int32_T i{0}; i <= rowreps - 2; i++) {
      int32_T ix;
      boolean_T p;
      ix = iv0 + (i + 1) * colIdx;
      if (isnan(clsMat.data[ix - 1])) {
        p = false;
      } else {
        real_T d;
        d = xCen_data[j];
        if (isnan(d)) {
          p = true;
        } else {
          p = (d < clsMat.data[ix - 1]);
        }
      }
      if (p) {
        xCen_data[j] = clsMat.data[ix - 1];
        ii_data[j] = i + 2;
      }
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel42(
    const int32_T ii_data[214272], const int32_T b, real_T yCen_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    yCen_data[i] = static_cast<real_T>(ii_data[i]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel43(
    const real_T yCen_data[214272], const real32_T confScore_data[214272],
    const real32_T angGt_data[214272], const real32_T hGt_data[214272],
    const real32_T wGt_data[214272], const real32_T lGt_data[214272],
    const real32_T zGt_data[214272], const real32_T yGt_data[214272],
    const int32_T boxPreds_size_dim0, const real32_T xGt_data[214272],
    const int32_T b, real32_T boxPreds_data[1928448])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    boxPreds_data[i] = xGt_data[i];
    boxPreds_data[i + boxPreds_size_dim0] = yGt_data[i];
    boxPreds_data[i + boxPreds_size_dim0 * 2] = 0.5F * zGt_data[i];
    boxPreds_data[i + boxPreds_size_dim0 * 3] = lGt_data[i];
    boxPreds_data[i + boxPreds_size_dim0 * 4] = wGt_data[i];
    boxPreds_data[i + boxPreds_size_dim0 * 5] = hGt_data[i];
    boxPreds_data[i + boxPreds_size_dim0 * 6] = angGt_data[i];
    boxPreds_data[i + boxPreds_size_dim0 * 7] = confScore_data[i];
    boxPreds_data[i + boxPreds_size_dim0 * 8] =
        static_cast<real32_T>(yCen_data[i]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel44(
    const real32_T boxPreds_data[1928448], const int32_T boxPreds_size_dim0,
    const int32_T boxPreds_size, boolean_T x_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(boxPreds_size);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x_data[i] =
        (static_cast<real_T>(boxPreds_data[i + boxPreds_size_dim0 * 8]) > 0.15);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel45(
    const int32_T boxPreds_size_dim0, const int32_T b,
    real32_T boxPreds_data[1928448])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    boxPreds_data[i + boxPreds_size_dim0 * 6] *= 57.2957802F;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel46(
    const real32_T boxPreds_data[1928448], const int32_T boxPreds_size_dim0,
    const int32_T ii_data[214272], const int32_T b, real32_T hdGt_data[214272],
    real32_T scores_data[])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T ix;
    i = static_cast<int32_T>(idx);
    ix = ii_data[i];
    scores_data[i] = boxPreds_data[(ix + boxPreds_size_dim0 * 7) - 1];
    hdGt_data[i] = boxPreds_data[(ix + boxPreds_size_dim0 * 8) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel47(
    const real32_T scores_data[], const int32_T scores_size,
    real_T xCen_data[214272], real32_T varargout_1_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(scores_size);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    varargout_1_data[i] = scores_data[i];
    xCen_data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel48(
    const real32_T boxPreds_data[1928448],
    const int32_T boxPreds_size_dim0_dup0, const int32_T ii_data[214272],
    const real_T xCen_data[214272], const int32_T boxPreds_size_dim0,
    const int32_T idx_size, real32_T b_boxPreds_data[1071360], int8_T iv23_dim0,
    int8_T iv23_dim1, int8_T iv23_dim2, int8_T iv23_dim3, int8_T iv23_dim4)
{
  __shared__ int8_T iv23_shared[5];
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  if (mwGetThreadIndexWithinBlock() == 0U) {
    iv23_shared[0] = iv23_dim0;
    iv23_shared[1] = iv23_dim1;
    iv23_shared[2] = iv23_dim2;
    iv23_shared[3] = iv23_dim3;
    iv23_shared[4] = iv23_dim4;
  }
  __syncthreads();
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 5UL * (static_cast<uint64_T>(idx_size) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T ix;
    ix = static_cast<int32_T>(idx % (static_cast<uint64_T>(idx_size) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(ix)) /
                             (static_cast<uint64_T>(idx_size) + 1UL));
    b_boxPreds_data[ix + boxPreds_size_dim0 * i] =
        boxPreds_data[(ii_data[static_cast<int32_T>(xCen_data[ix]) - 1] +
                       boxPreds_size_dim0_dup0 *
                           static_cast<int32_T>(iv23_shared[i])) -
                      1];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel49(
    const real32_T hdGt_data[214272], const real_T xCen_data[214272],
    const int32_T idx_size, real32_T varargout_1_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(idx_size);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    varargout_1_data[i] = hdGt_data[static_cast<int32_T>(xCen_data[i]) - 1];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillardetect_kernel5(
    const real_T yy[53568], const real_T xx[53568], real_T tmp[107136])
{
  int32_T j;
  j = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (j < 53568) {
    tmp[j] = xx[j];
    tmp[j + 53568] = yy[j];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel50(
    const boolean_T selectedIndex_data[214272],
    const int32_T selectedIndex_size, boolean_T x_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(selectedIndex_size);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x_data[i] = selectedIndex_data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel51(
    const boolean_T x_data[214272], const real_T xCen_data[214272],
    const int32_T b, boolean_T selectedIndex_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    selectedIndex_data[static_cast<int32_T>(xCen_data[i]) - 1] = x_data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel52(
    const int32_T b, real_T bboxes_data[])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    bboxes_data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel53(
    const real32_T boxPreds_data[1928448], const int32_T boxPreds_size_dim0,
    const int32_T ii_data[214272], const int32_T sizes, real_T bboxes_data[],
    int8_T iv27_dim0, int8_T iv27_dim1, int8_T iv27_dim2, int8_T iv27_dim3,
    int8_T iv27_dim4, int8_T iv27_dim5, int8_T iv27_dim6,
    int32_T bboxes_size_dim0)
{
  __shared__ int8_T iv27_shared[7];
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  if (mwGetThreadIndexWithinBlock() == 0U) {
    iv27_shared[0] = iv27_dim0;
    iv27_shared[1] = iv27_dim1;
    iv27_shared[2] = iv27_dim2;
    iv27_shared[3] = iv27_dim3;
    iv27_shared[4] = iv27_dim4;
    iv27_shared[5] = iv27_dim5;
    iv27_shared[6] = iv27_dim6;
  }
  __syncthreads();
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 7UL * (static_cast<uint64_T>(sizes) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T ix;
    ix = static_cast<int32_T>(idx % (static_cast<uint64_T>(sizes) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(ix)) /
                             (static_cast<uint64_T>(sizes) + 1UL));
    bboxes_data[ix + bboxes_size_dim0 * static_cast<int32_T>(iv27_shared[i])] =
        static_cast<real_T>(
            boxPreds_data[(ii_data[ix] + boxPreds_size_dim0 * i) - 1]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel54(
    const real32_T hdGt_data[214272], const int32_T tmp_data[214272],
    const int32_T b, real32_T varargout_1_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    varargout_1_data[i] = hdGt_data[tmp_data[i]];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel55(
    const real32_T predClassification[428544],
    const int8_T varargout_6_data[214272], const int16_T col_data[214272],
    const int32_T ii_data[214272], const int32_T b,
    real32_T varargout_1_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    varargout_1_data[j] = predClassification
        [((ii_data[j] + 216 * (static_cast<int32_T>(col_data[j]) - 1)) +
          53568 * (static_cast<int32_T>(varargout_6_data[j]) - 1)) -
         1];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel56(
    const real32_T varargout_1_data[214272], const int32_T b,
    real_T xCen_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    xCen_data[i] = -static_cast<real_T>(varargout_1_data[i]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel57(
    const int32_T b, real_T xCen_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    xCen_data[j] = exp(xCen_data[j]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel58(
    const real_T xCen_data[214272], const int32_T b, real_T yCen_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    yCen_data[i] = 1.0 / (xCen_data[i] + 1.0);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel59(
    const real_T threshold, const int32_T rowIdx, real_T yCen_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    if (yCen_data[i] >= threshold) {
      yCen_data[i] = 1.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel6(
    const int32_T b, emxArray_real_T anchors3dMod)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    anchors3dMod.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel60(
    const real_T threshold, const int32_T rowIdx, real_T yCen_data[214272])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    if (yCen_data[i] < threshold) {
      yCen_data[i] = 0.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel61(
    const emxArray_boolean_T anchorMask, const uint8_T dv6_dim0,
    const int32_T dv6, emxArray_boolean_T b_anchorMask)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 53568UL * (static_cast<uint64_T>(dv6) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    uint64_T tmpIndex;
    int32_T iv0;
    int32_T ix;
    int32_T j;
    ix = static_cast<int32_T>(idx % (static_cast<uint64_T>(dv6) + 1UL));
    tmpIndex =
        (idx - static_cast<uint64_T>(ix)) / (static_cast<uint64_T>(dv6) + 1UL);
    iv0 = static_cast<int32_T>(tmpIndex % 216UL);
    tmpIndex = (tmpIndex - static_cast<uint64_T>(iv0)) / 216UL;
    j = static_cast<int32_T>(tmpIndex);
    b_anchorMask.data[(iv0 + 216 * j) + 53568 * ix] =
        anchorMask.data[(ix + static_cast<int32_T>(dv6_dim0) * iv0) +
                        static_cast<int32_T>(dv6_dim0) * 216 * j];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel7(
    const emxArray_real_T tmp, const int32_T sizes,
    emxArray_real_T anchors3dMod, int32_T anchors3dMod_dim0, int32_T tmp_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 2UL * (static_cast<uint64_T>(sizes) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T ix;
    ix = static_cast<int32_T>(idx % (static_cast<uint64_T>(sizes) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(ix)) /
                             (static_cast<uint64_T>(sizes) + 1UL));
    anchors3dMod.data[ix + anchors3dMod_dim0 * i] = tmp.data[ix + tmp_dim0 * i];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel8(
    const emxArray_real_T anchorsDim, const int32_T b,
    emxArray_real_T anchors3dMod, int32_T anchors3dMod_dim0,
    int32_T anchorsDim_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    anchors3dMod.data[i + anchors3dMod_dim0 * 2] =
        anchorsDim.data[i + anchorsDim_dim0 * 3];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillardetect_kernel9(
    const emxArray_real_T anchorsDim, const int32_T sizes,
    emxArray_real_T anchors3dMod, int32_T anchors3dMod_dim0,
    int32_T anchorsDim_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * (static_cast<uint64_T>(sizes) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T ix;
    ix = static_cast<int32_T>(idx % (static_cast<uint64_T>(sizes) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(ix)) /
                             (static_cast<uint64_T>(sizes) + 1UL));
    anchors3dMod.data[ix + anchors3dMod_dim0 * (i + 3)] =
        anchorsDim.data[ix + anchorsDim_dim0 * i];
  }
}

void MWScatterLayer_pretrained0_01::cleanupLayer()
{
}

void MWResize2DLayer_pretrained0_01::cleanupLayer()
{
}

void MWResize2DLayer_pretrained0_02::cleanupLayer()
{
}

void MWScatterLayer_pretrained0_01::createCustomLayer(
    MWCudnnTarget::MWTargetNetworkImpl *targetImpl, MWTensorBase *b,
    MWTensorBase *c, const char_T *d, int32_T e)
{
  setupTensors(2, 1, b, c, d, e);
  setupLayer(targetImpl);
}

void MWResize2DLayer_pretrained0_01::createCustomLayer(
    MWCudnnTarget::MWTargetNetworkImpl *targetImpl, MWTensorBase *b,
    const char_T *c, int32_T d)
{
  setupTensors(1, 1, b, c, d);
  setupLayer(targetImpl);
}

void MWResize2DLayer_pretrained0_02::createCustomLayer(
    MWCudnnTarget::MWTargetNetworkImpl *targetImpl, MWTensorBase *b,
    const char_T *c, int32_T d)
{
  setupTensors(1, 1, b, c, d);
  setupLayer(targetImpl);
}

void MWResize2DLayer_pretrained0_01::predict()
{
  reorderInputData(getInputTensor(0), 0);
  coder::internal::l_dlnetwork_layerPredictWithCol(
      m_ntwkImpl->getPermuteBuffer(0), m_ntwkImpl->getPermuteBuffer(1));
  reorderOutputData(getOutputTensor(0), 1);
}

void MWScatterLayer_pretrained0_01::predict()
{
  reorderInputData(getInputTensor(0), 0);
  reorderInputData(getInputTensor(1), 1);
  coder::internal::l_dlnetwork_layerPredictWithCol(
      m_ntwkImpl->getPermuteBuffer(0), m_ntwkImpl->getPermuteBuffer(1),
      m_ntwkImpl->getPermuteBuffer(2));
  reorderOutputData(getOutputTensor(0), 2);
}

void MWResize2DLayer_pretrained0_02::predict()
{
  reorderInputData(getInputTensor(0), 0);
  coder::internal::m_dlnetwork_layerPredictWithCol(
      m_ntwkImpl->getPermuteBuffer(0), m_ntwkImpl->getPermuteBuffer(1));
  reorderOutputData(getOutputTensor(0), 1);
}

void MWScatterLayer_pretrained0_01::propagateSize()
{
  int32_T batchSize;
  MWTensorBase::DIMSLABEL b_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL c_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL d_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL layout[5]{MWTensorBase::S, MWTensorBase::N,
                                    MWTensorBase::C, MWTensorBase::W,
                                    MWTensorBase::H};
  batchSize = getInputTensor(0)->getBatchSize();
  resizeOutputTensor(432, 496, 64, batchSize, 1, 0);
  setupInputDescriptors(getInputTensor(0), b_layout, layout);
  setupInputDescriptors(getInputTensor(1), c_layout, layout);
  setupOutputDescriptors(getOutputTensor(0), layout, d_layout);
}

void MWResize2DLayer_pretrained0_01::propagateSize()
{
  int32_T batchSize;
  MWTensorBase::DIMSLABEL b_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL c_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL layout[5]{MWTensorBase::S, MWTensorBase::N,
                                    MWTensorBase::C, MWTensorBase::W,
                                    MWTensorBase::H};
  batchSize = getInputTensor(0)->getBatchSize();
  resizeOutputTensor(216, 248, 128, batchSize, 1, 0);
  setupInputDescriptors(getInputTensor(0), b_layout, layout);
  setupOutputDescriptors(getOutputTensor(0), layout, c_layout);
}

void MWResize2DLayer_pretrained0_02::propagateSize()
{
  int32_T batchSize;
  MWTensorBase::DIMSLABEL b_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL c_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL layout[5]{MWTensorBase::S, MWTensorBase::N,
                                    MWTensorBase::C, MWTensorBase::W,
                                    MWTensorBase::H};
  batchSize = getInputTensor(0)->getBatchSize();
  resizeOutputTensor(216, 248, 128, batchSize, 1, 0);
  setupInputDescriptors(getInputTensor(0), b_layout, layout);
  setupOutputDescriptors(getOutputTensor(0), layout, c_layout);
}

void pointpillardetect(pointpillardetectStackData *SD,
                       const emxArray_real32_T *dataLoc,
                       const emxArray_real32_T *dataInt, real_T threshold,
                       real_T cpu_bboxes_data[], int32_T bboxes_size[2],
                       real32_T cpu_scores_data[], int32_T scores_size[1],
                       coder::categorical *labels)
{
  static const real_T cpu_dv3[248]{-39.52,
                                   -39.2,
                                   -38.88,
                                   -38.56,
                                   -38.24,
                                   -37.92,
                                   -37.6,
                                   -37.28,
                                   -36.96,
                                   -36.64,
                                   -36.32,
                                   -36.0,
                                   -35.68,
                                   -35.36,
                                   -35.04,
                                   -34.72,
                                   -34.4,
                                   -34.08,
                                   -33.76,
                                   -33.44,
                                   -33.12,
                                   -32.8,
                                   -32.480000000000004,
                                   -32.160000000000004,
                                   -31.84,
                                   -31.52,
                                   -31.2,
                                   -30.88,
                                   -30.560000000000002,
                                   -30.240000000000002,
                                   -29.92,
                                   -29.6,
                                   -29.28,
                                   -28.96,
                                   -28.64,
                                   -28.32,
                                   -28.0,
                                   -27.68,
                                   -27.36,
                                   -27.04,
                                   -26.72,
                                   -26.400000000000002,
                                   -26.080000000000002,
                                   -25.76,
                                   -25.44,
                                   -25.12,
                                   -24.8,
                                   -24.48,
                                   -24.16,
                                   -23.84,
                                   -23.52,
                                   -23.2,
                                   -22.88,
                                   -22.56,
                                   -22.240000000000002,
                                   -21.92,
                                   -21.6,
                                   -21.28,
                                   -20.96,
                                   -20.64,
                                   -20.32,
                                   -20.0,
                                   -19.68,
                                   -19.36,
                                   -19.04,
                                   -18.72,
                                   -18.400000000000002,
                                   -18.080000000000002,
                                   -17.76,
                                   -17.44,
                                   -17.12,
                                   -16.8,
                                   -16.48,
                                   -16.16,
                                   -15.84,
                                   -15.52,
                                   -15.200000000000001,
                                   -14.88,
                                   -14.56,
                                   -14.24,
                                   -13.92,
                                   -13.6,
                                   -13.280000000000001,
                                   -12.96,
                                   -12.64,
                                   -12.32,
                                   -12.0,
                                   -11.68,
                                   -11.36,
                                   -11.040000000000001,
                                   -10.72,
                                   -10.4,
                                   -10.08,
                                   -9.76,
                                   -9.44,
                                   -9.120000000000001,
                                   -8.8,
                                   -8.48,
                                   -8.16,
                                   -7.84,
                                   -7.5200000000000005,
                                   -7.2,
                                   -6.88,
                                   -6.5600000000000005,
                                   -6.24,
                                   -5.92,
                                   -5.6000000000000005,
                                   -5.28,
                                   -4.96,
                                   -4.64,
                                   -4.32,
                                   -4.0,
                                   -3.68,
                                   -3.36,
                                   -3.04,
                                   -2.72,
                                   -2.4,
                                   -2.08,
                                   -1.76,
                                   -1.44,
                                   -1.12,
                                   -0.8,
                                   -0.48,
                                   -0.16,
                                   0.16,
                                   0.48,
                                   0.8,
                                   1.12,
                                   1.44,
                                   1.76,
                                   2.08,
                                   2.4,
                                   2.72,
                                   3.04,
                                   3.36,
                                   3.68,
                                   4.0,
                                   4.32,
                                   4.64,
                                   4.96,
                                   5.28,
                                   5.6000000000000005,
                                   5.92,
                                   6.24,
                                   6.5600000000000005,
                                   6.88,
                                   7.2,
                                   7.5200000000000005,
                                   7.84,
                                   8.16,
                                   8.48,
                                   8.8,
                                   9.120000000000001,
                                   9.44,
                                   9.76,
                                   10.08,
                                   10.4,
                                   10.72,
                                   11.040000000000001,
                                   11.36,
                                   11.68,
                                   12.0,
                                   12.32,
                                   12.64,
                                   12.96,
                                   13.280000000000001,
                                   13.6,
                                   13.92,
                                   14.24,
                                   14.56,
                                   14.88,
                                   15.200000000000001,
                                   15.52,
                                   15.84,
                                   16.16,
                                   16.48,
                                   16.8,
                                   17.12,
                                   17.44,
                                   17.76,
                                   18.080000000000002,
                                   18.400000000000002,
                                   18.72,
                                   19.04,
                                   19.36,
                                   19.68,
                                   20.0,
                                   20.32,
                                   20.64,
                                   20.96,
                                   21.28,
                                   21.6,
                                   21.92,
                                   22.240000000000002,
                                   22.56,
                                   22.88,
                                   23.2,
                                   23.52,
                                   23.84,
                                   24.16,
                                   24.48,
                                   24.8,
                                   25.12,
                                   25.44,
                                   25.76,
                                   26.080000000000002,
                                   26.400000000000002,
                                   26.72,
                                   27.04,
                                   27.36,
                                   27.68,
                                   28.0,
                                   28.32,
                                   28.64,
                                   28.96,
                                   29.28,
                                   29.6,
                                   29.92,
                                   30.240000000000002,
                                   30.560000000000002,
                                   30.88,
                                   31.2,
                                   31.52,
                                   31.84,
                                   32.160000000000004,
                                   32.480000000000004,
                                   32.8,
                                   33.12,
                                   33.44,
                                   33.76,
                                   34.08,
                                   34.4,
                                   34.72,
                                   35.04,
                                   35.36,
                                   35.68,
                                   36.0,
                                   36.32,
                                   36.64,
                                   36.96,
                                   37.28,
                                   37.6,
                                   37.92,
                                   38.24,
                                   38.56,
                                   38.88,
                                   39.2,
                                   39.52};
  static const real_T dv[10]{1.912377555486092,
                             1.912377555486092,
                             4.540055829946871,
                             4.540055829946871,
                             1.7084437636761454,
                             1.7084437636761454,
                             -1.78,
                             -1.78,
                             0.0,
                             1.5707963267948966};
  static const real_T dv1[10]{2.1634333333333342,
                              2.1634333333333342,
                              5.901875,
                              5.901875,
                              2.2756435185185189,
                              2.2756435185185189,
                              -1.78,
                              -1.78,
                              0.0,
                              1.5707963267948966};
  static const int32_T iv[2]{1, 15};
  static const int32_T iv1[2]{1, 15};
  static const char_T u[15]{'M', 'A', 'T', 'L', 'A', 'B', ':', 'p',
                            'm', 'a', 'x', 's', 'i', 'z', 'e'};
  static const int8_T iv27[7]{0, 1, 2, 3, 4, 5, 8};
  static const int8_T iv23[5]{0, 1, 3, 4, 6};
  static const int8_T dv8[3]{2, 3, 1};
  cell_wrap_5 rv[2];
  cell_wrap_5 cpu_r;
  cell_wrap_5 cpu_r1;
  cell_wrap_5 *gpu_r;
  cell_wrap_5 *gpu_r1;
  coder::b_pointCloud ptCloud;
  coder::pointCloud ptCloudUnorg;
  dim3 block;
  dim3 grid;
  emxArray_boolean_T b_gpu_anchorMask;
  emxArray_boolean_T c_gpu_anchorMask;
  emxArray_boolean_T d_gpu_anchorMask;
  emxArray_boolean_T gpu_indices;
  emxArray_boolean_T *b_cpu_anchorMask;
  emxArray_boolean_T *c_cpu_anchorMask;
  emxArray_boolean_T *cpu_indices;
  emxArray_boolean_T *d_cpu_anchorMask;
  emxArray_cell_wrap_40 *newAnchors;
  emxArray_real32_T gpu_intensity;
  emxArray_real32_T gpu_location;
  emxArray_real32_T gpu_normals;
  emxArray_real32_T gpu_rangeData;
  emxArray_real32_T *cpu_intensity;
  emxArray_real32_T *cpu_location;
  emxArray_real32_T *cpu_normals;
  emxArray_real32_T *cpu_rangeData;
  emxArray_real32_T *dataIn;
  emxArray_real_T b_gpu_anchors;
  emxArray_real_T b_gpu_anchors3dMod;
  emxArray_real_T b_gpu_tmp;
  emxArray_real_T gpu_anchorMask;
  emxArray_real_T gpu_anchors;
  emxArray_real_T gpu_anchors3dMod;
  emxArray_real_T gpu_anchorsDim;
  emxArray_real_T gpu_clsMat;
  emxArray_real_T gpu_tmp1;
  emxArray_real_T gpu_tmp2;
  emxArray_real_T *b_cpu_anchors;
  emxArray_real_T *b_cpu_anchors3dMod;
  emxArray_real_T *cpu_anchorMask;
  emxArray_real_T *cpu_anchors;
  emxArray_real_T *cpu_anchors3dMod;
  emxArray_real_T *cpu_anchorsDim;
  emxArray_real_T *cpu_clsMat;
  emxArray_real_T *cpu_tmp;
  emxArray_real_T *cpu_tmp1;
  emxArray_real_T *cpu_tmp2;
  emxArray_uint8_T gpu_b;
  emxArray_uint8_T gpu_value;
  emxArray_uint8_T *cpu_b;
  emxArray_uint8_T *cpu_value;
  const mxArray *b_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *y;
  real_T(*gpu_sparseVoxelMap)[214272];
  real_T(*gpu_xCen_data)[214272];
  real_T(*gpu_yCen_data)[214272];
  real_T(*gpu_tmp)[107136];
  real_T(*gpu_xx)[53568];
  real_T(*gpu_yy)[53568];
  real_T numPillars;
  real_T *gpu_bboxes_data;
  int32_T(*gpu_ii_data)[214272];
  int32_T(*gpu_tmp_data)[214272];
  int32_T(*gpu_v1_data)[214272];
  int32_T(*gpu_iv2_data)[12000];
  int32_T boxPreds_size[2];
  int32_T sizes[2];
  int32_T sz[2];
  int32_T colIdx;
  int32_T i;
  int32_T ibcol;
  int32_T itilerow;
  int32_T k;
  int32_T ntilerows;
  int32_T rowIdx;
  int32_T rowreps;
  real32_T(*gpu_dlPillarFeatures_Data)[10800000];
  real32_T(*gpu_boxPreds_data)[1928448];
  real32_T(*b_gpu_boxPreds_data)[1071360];
  real32_T(*gpu_fv1)[642816];
  real32_T(*gpu_fv2)[642816];
  real32_T(*gpu_predClassification)[428544];
  real32_T(*gpu_angGt_data)[214272];
  real32_T(*gpu_confScore_data)[214272];
  real32_T(*gpu_hGt_data)[214272];
  real32_T(*gpu_hdGt_data)[214272];
  real32_T(*gpu_lGt_data)[214272];
  real32_T(*gpu_predAngle)[214272];
  real32_T(*gpu_predOcc)[214272];
  real32_T(*gpu_this_workspace_predHeading)[214272];
  real32_T(*gpu_varargout_1_data)[214272];
  real32_T(*gpu_wGt_data)[214272];
  real32_T(*gpu_xGt_data)[214272];
  real32_T(*gpu_yGt_data)[214272];
  real32_T(*gpu_zGt_data)[214272];
  real32_T cpu_pillarIndices[24000];
  real32_T(*gpu_pillarIndices)[24000];
  real32_T *gpu_scores_data;
  uint32_T numPoints[2];
  int16_T(*gpu_col_data)[214272];
  int16_T(*gpu_vk_data)[214272];
  int8_T(*gpu_varargout_6_data)[214272];
  int8_T input_sizes_idx_1;
  boolean_T(*gpu_selectedIndex_data)[214272];
  boolean_T(*gpu_x)[214272];
  boolean_T(*gpu_x_data)[214272];
  boolean_T angGt_data_outdatedOnGpu;
  boolean_T empty_non_axis_sizes;
  boolean_T exitg1;
  boolean_T hGt_data_outdatedOnGpu;
  boolean_T indices_outdatedOnCpu;
  boolean_T indices_outdatedOnGpu;
  boolean_T intensity_outdatedOnCpu;
  boolean_T lGt_data_outdatedOnGpu;
  boolean_T location_outdatedOnCpu;
  boolean_T location_outdatedOnGpu;
  boolean_T normals_outdatedOnCpu;
  boolean_T normals_outdatedOnGpu;
  boolean_T rangeData_outdatedOnCpu;
  boolean_T rangeData_outdatedOnGpu;
  boolean_T tmp_data_outdatedOnGpu;
  boolean_T validLaunchParams;
  boolean_T value_outdatedOnCpu;
  boolean_T value_outdatedOnGpu;
  boolean_T wGt_data_outdatedOnGpu;
  boolean_T xGt_data_outdatedOnGpu;
  boolean_T yGt_data_outdatedOnGpu;
  boolean_T zGt_data_outdatedOnGpu;
  if (!c_gpuConstsCopied_pointpillarde) {
    c_gpuConstsCopied_pointpillarde = true;
    checkCudaError(hipMemcpy(*global_gpu_dv3, cpu_dv3, sizeof(real_T[248]),
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  checkCudaError(mwCudaMalloc(&gpu_tmp_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_selectedIndex_data, 214272UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_boxPreds_data, 4285440UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_x_data, 214272UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_boxPreds_data, 7713792UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_varargout_1_data, 857088UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_angGt_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_hdGt_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_hGt_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_wGt_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_lGt_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_zGt_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_yGt_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_xGt_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_confScore_data, 857088UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_yCen_data, 1714176UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_xCen_data, 1714176UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_col_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_varargout_6_data, 214272UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_vk_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_v1_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_ii_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_x, 214272UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_iv2_data, 48000UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_sparseVoxelMap, 1714176UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_predAngle, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_this_workspace_predHeading, 857088UL),
                 __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_predClassification, 1714176UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_predOcc, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_fv2, 2571264UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_fv1, 2571264UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_dlPillarFeatures_Data, 43200000UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_pillarIndices, 96000UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_tmp, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_xx, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_yy, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_r1, 80UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_r, 80UL), __FILE__, __LINE__);
  gpuEmxReset_real_T(&b_gpu_anchors3dMod);
  gpuEmxReset_real_T(&b_gpu_anchors);
  gpuEmxReset_real_T(&gpu_clsMat);
  gpuEmxReset_boolean_T(&d_gpu_anchorMask);
  gpuEmxReset_boolean_T(&c_gpu_anchorMask);
  gpuEmxReset_boolean_T(&b_gpu_anchorMask);
  gpuEmxReset_real_T(&gpu_anchorMask);
  gpuEmxReset_real_T(&gpu_tmp2);
  gpuEmxReset_real_T(&gpu_tmp1);
  gpuEmxReset_uint8_T(&gpu_b);
  gpuEmxReset_real_T(&gpu_anchorsDim);
  gpuEmxReset_real_T(&gpu_anchors3dMod);
  gpuEmxReset_real_T(&b_gpu_tmp);
  gpuEmxReset_real32_T(&gpu_rangeData);
  gpuEmxReset_real32_T(&gpu_intensity);
  gpuEmxReset_real32_T(&gpu_normals);
  gpuEmxReset_uint8_T(&gpu_value);
  gpuEmxReset_real32_T(&gpu_location);
  gpuEmxReset_boolean_T(&gpu_indices);
  gpuEmxReset_real_T(&gpu_anchors);
  checkCudaError(mwCudaMalloc(&gpu_scores_data, 214272U * sizeof(real32_T)),
                 __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_bboxes_data, 1928448U * sizeof(real_T)),
                 __FILE__, __LINE__);
  tmp_data_outdatedOnGpu = false;
  angGt_data_outdatedOnGpu = false;
  hGt_data_outdatedOnGpu = false;
  wGt_data_outdatedOnGpu = false;
  lGt_data_outdatedOnGpu = false;
  zGt_data_outdatedOnGpu = false;
  yGt_data_outdatedOnGpu = false;
  xGt_data_outdatedOnGpu = false;
  rangeData_outdatedOnCpu = false;
  rangeData_outdatedOnGpu = false;
  intensity_outdatedOnCpu = false;
  normals_outdatedOnCpu = false;
  normals_outdatedOnGpu = false;
  value_outdatedOnCpu = false;
  value_outdatedOnGpu = false;
  location_outdatedOnCpu = false;
  location_outdatedOnGpu = false;
  indices_outdatedOnCpu = false;
  indices_outdatedOnGpu = false;
  empty_non_axis_sizes = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInitStruct_pointCloud(&ptCloud, &s_emlrtRTEI, true);
  emxInit_real_T(&cpu_anchors, 2, &t_emlrtRTEI, true);
  emxInitStruct_pointCloud1(&ptCloudUnorg, &m_emlrtRTEI, true);
  emxInit_boolean_T(&cpu_indices, 1, &r_emlrtRTEI, true);
  emxInit_real32_T(&cpu_location, 2, &r_emlrtRTEI, true);
  emxInit_uint8_T(&cpu_value, 2, &k_emlrtRTEI, true);
  emxInit_real32_T(&cpu_normals, 2, &r_emlrtRTEI, true);
  emxInit_real32_T(&cpu_intensity, 2, &r_emlrtRTEI, true);
  emxInit_real32_T(&cpu_rangeData, 2, &r_emlrtRTEI, true);
  emxInit_real_T(&cpu_tmp, 2, &u_emlrtRTEI, true);
  emxInit_real_T(&cpu_anchors3dMod, 2, &v_emlrtRTEI, true);
  emxInit_real_T(&cpu_anchorsDim, 2, &w_emlrtRTEI, true);
  emxInit_uint8_T(&cpu_b, 2, &x_emlrtRTEI, true);
  emxInit_real_T(&cpu_tmp1, 2, &n_emlrtRTEI, true);
  emxInit_real_T(&cpu_tmp2, 2, &o_emlrtRTEI, true);
  emxInit_real32_T(&dataIn, 2, &m_emlrtRTEI, true);
  emxInit_real_T(&cpu_anchorMask, 1, &m_emlrtRTEI, true);
  emxInit_boolean_T(&b_cpu_anchorMask, 1, &m_emlrtRTEI, true);
  emxInit_cell_wrap_40(&newAnchors, 2, &m_emlrtRTEI, true);
  emxInit_boolean_T(&c_cpu_anchorMask, 3, &m_emlrtRTEI, true);
  emxInit_boolean_T(&d_cpu_anchorMask, 3, &m_emlrtRTEI, true);
  emxInit_real_T(&cpu_clsMat, 2, &m_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_anchors, 2, &d_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_anchors3dMod, 2, &l_emlrtRTEI, true);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emlrtPushHeapReferenceStackR2021a(
      emlrtRootTLSGlobal, false, &ptCloud,
      (void *)static_cast<void (*)(coder::b_pointCloud *)>(
          &coder::handle_matlabCodegenDestructor),
      nullptr, nullptr, nullptr);
  ptCloud.matlabCodegenIsDeleted = true;
  //  Predict the output of network and extract the confidence, x, y,
  //  width, height, and class.
  //  load the deep learning network for prediction
  if (!pointPillarObj_not_empty) {
    pointPillarObj.AnchorsBEV->size[0] = 0;
    pointPillarObj.AnchorsBEV->size[1] = 4;
    coder::internal::dlnetwork_setup(&gobj_0);
    gobj_0.matlabCodegenIsDeleted = false;
    pointPillarObj.Network = &gobj_0;
    pointpillardetect_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        gpu_r1, gpu_r, dv1[0], dv1[1], dv1[2], dv1[3], dv1[4], dv1[5], dv1[6],
        dv1[7], dv1[8], dv1[9], dv[0], dv[1], dv[2], dv[3], dv[4], dv[5], dv[6],
        dv[7], dv[8], dv[9]);
    checkCudaError(hipMemcpy(&cpu_r, gpu_r, 80UL, hipMemcpyDeviceToHost),
                   __FILE__, __LINE__);
    rv[0] = cpu_r;
    checkCudaError(hipMemcpy(&cpu_r1, gpu_r1, 80UL, hipMemcpyDeviceToHost),
                   __FILE__, __LINE__);
    rv[1] = cpu_r1;
    normals_outdatedOnGpu = false;
    cpu_anchors->size[0] = 0;
    cpu_anchors->size[1] = 5;
    for (i = 0; i < 2; i++) {
      ntilerows = b_cpu_anchors->size[0] * b_cpu_anchors->size[1];
      b_cpu_anchors->size[0] = cpu_anchors->size[0] + 2;
      b_cpu_anchors->size[1] = 5;
      emxEnsureCapacity_real_T(b_cpu_anchors, ntilerows, &d_emlrtRTEI);
      rowreps = cpu_anchors->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(4, rowreps), &grid, &block,
                              2147483647U);
      if (empty_non_axis_sizes) {
        gpuEmxEnsureCapacity_real_T(cpu_anchors, &gpu_anchors, true);
      }
      gpuEmxEnsureCapacity_real_T(b_cpu_anchors, &b_gpu_anchors, true);
      empty_non_axis_sizes = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel2<<<grid, block>>>(
            gpu_anchors, rowreps, b_gpu_anchors, b_cpu_anchors->size[0U],
            cpu_anchors->size[0U]);
      }
      rangeData_outdatedOnCpu = false;
      normals_outdatedOnGpu = true;
      for (ntilerows = 0; ntilerows < 5; ntilerows++) {
        if (normals_outdatedOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(b_cpu_anchors, &b_gpu_anchors);
        }
        b_cpu_anchors
            ->data[cpu_anchors->size[0] + b_cpu_anchors->size[0] * ntilerows] =
            rv[i].f1[ntilerows << 1];
        b_cpu_anchors
            ->data[(cpu_anchors->size[0] + b_cpu_anchors->size[0] * ntilerows) +
                   1] = rv[i].f1[(ntilerows << 1) + 1];
        normals_outdatedOnGpu = false;
        rangeData_outdatedOnCpu = true;
        empty_non_axis_sizes = true;
      }
      ntilerows = cpu_anchors->size[0] * cpu_anchors->size[1];
      cpu_anchors->size[0] = b_cpu_anchors->size[0];
      cpu_anchors->size[1] = 5;
      emxEnsureCapacity_real_T(cpu_anchors, ntilerows, &e_emlrtRTEI);
      rowreps = b_cpu_anchors->size[0] * 5 - 1;
      mwGetLaunchParameters1D(computeNumIters(rowreps), &grid, &block,
                              2147483647U);
      if (empty_non_axis_sizes) {
        gpuEmxEnsureCapacity_real_T(b_cpu_anchors, &b_gpu_anchors,
                                    !rangeData_outdatedOnCpu);
      }
      gpuEmxEnsureCapacity_real_T(cpu_anchors, &gpu_anchors, true);
      empty_non_axis_sizes = false;
      if (rangeData_outdatedOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_anchors, b_cpu_anchors);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel3<<<grid, block>>>(b_gpu_anchors, rowreps,
                                                   gpu_anchors);
      }
      normals_outdatedOnGpu = true;
    }
    pointpillardetect_kernel4<<<dim3(105U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *global_gpu_dv3, *gpu_xx, *gpu_yy);
    pointpillardetect_kernel5<<<dim3(105U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_yy, *gpu_xx, *gpu_tmp);
    rangeData_outdatedOnGpu = true;
    empty_non_axis_sizes = false;
    ntilerows = cpu_tmp->size[0] * cpu_tmp->size[1];
    cpu_tmp->size[0] = 53568 * cpu_anchors->size[0];
    cpu_tmp->size[1] = 2;
    emxEnsureCapacity_real_T(cpu_tmp, ntilerows, &f_emlrtRTEI);
    if (53568 * cpu_anchors->size[0] != 0) {
      rowreps = cpu_anchors->size[0];
      for (ntilerows = 0; ntilerows < 2; ntilerows++) {
        rowIdx = -1;
        for (i = 0; i < 53568; i++) {
          for (k = 0; k < rowreps; k++) {
            if (rangeData_outdatedOnGpu) {
              checkCudaError(hipMemcpy(SD->f3.cpu_tmp, *gpu_tmp, 857088UL,
                                        hipMemcpyDeviceToHost),
                             __FILE__, __LINE__);
            }
            rangeData_outdatedOnGpu = false;
            cpu_tmp->data[((rowIdx + k) + cpu_tmp->size[0] * ntilerows) + 1] =
                SD->f3.cpu_tmp[i + 53568 * ntilerows];
            empty_non_axis_sizes = true;
          }
          if (rowreps - 1 >= 0) {
            rowIdx += rowreps;
          }
        }
      }
    }
    numPillars = 53568.0 * static_cast<real_T>(cpu_anchors->size[0]);
    ntilerows = cpu_anchors3dMod->size[0] * cpu_anchors3dMod->size[1];
    cpu_anchors3dMod->size[0] = static_cast<int32_T>(numPillars);
    cpu_anchors3dMod->size[1] = 7;
    emxEnsureCapacity_real_T(cpu_anchors3dMod, ntilerows, &g_emlrtRTEI);
    rowreps = static_cast<int32_T>(numPillars) * 7 - 1;
    mwGetLaunchParameters1D(computeNumIters(rowreps), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_anchors3dMod, &gpu_anchors3dMod, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel6<<<grid, block>>>(rowreps, gpu_anchors3dMod);
    }
    mwGetLaunchParameters1D(
        computeNumIters(1, static_cast<int32_T>(numPillars) - 1), &grid, &block,
        2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_tmp, &b_gpu_tmp, !empty_non_axis_sizes);
    if (empty_non_axis_sizes) {
      gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_tmp, cpu_tmp);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel7<<<grid, block>>>(
          b_gpu_tmp, static_cast<int32_T>(numPillars) - 1, gpu_anchors3dMod,
          cpu_anchors3dMod->size[0U], cpu_tmp->size[0U]);
    }
    colIdx = cpu_anchors->size[0] * 53568;
    if (!(colIdx == static_cast<real_T>(cpu_anchors->size[0]) * 53568.0)) {
      y = nullptr;
      m = emlrtCreateCharArray(2, &iv[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 15, m, &u[0]);
      emlrtAssign(&y, m);
      b_error(y, &emlrtMCI);
    }
    empty_non_axis_sizes = false;
    ntilerows = cpu_anchorsDim->size[0] * cpu_anchorsDim->size[1];
    cpu_anchorsDim->size[0] = colIdx;
    cpu_anchorsDim->size[1] = 5;
    emxEnsureCapacity_real_T(cpu_anchorsDim, ntilerows, &j_emlrtRTEI);
    colIdx = cpu_anchors->size[0];
    for (i = 0; i < 5; i++) {
      rowreps = i * colIdx;
      rowIdx = i * (colIdx * 53568);
      for (itilerow = 0; itilerow < 53568; itilerow++) {
        ibcol = rowIdx + itilerow * colIdx;
        for (k = 0; k < colIdx; k++) {
          if (normals_outdatedOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_anchors, &gpu_anchors);
          }
          normals_outdatedOnGpu = false;
          cpu_anchorsDim->data[ibcol + k] = cpu_anchors->data[rowreps + k];
          empty_non_axis_sizes = true;
        }
      }
    }
    rowreps = cpu_anchors3dMod->size[0];
    mwGetLaunchParameters1D(computeNumIters(rowreps - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_anchorsDim, &gpu_anchorsDim,
                                !empty_non_axis_sizes);
    if (empty_non_axis_sizes) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_anchorsDim, cpu_anchorsDim);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel8<<<grid, block>>>(
          gpu_anchorsDim, rowreps - 1, gpu_anchors3dMod,
          cpu_anchors3dMod->size[0U], cpu_anchorsDim->size[0U]);
    }
    sizes[0] = cpu_anchors3dMod->size[0];
    mwGetLaunchParameters1D(computeNumIters(2, sizes[0] - 1), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel9<<<grid, block>>>(
          gpu_anchorsDim, sizes[0] - 1, gpu_anchors3dMod,
          cpu_anchors3dMod->size[0U], cpu_anchorsDim->size[0U]);
    }
    rowreps = cpu_anchors3dMod->size[0];
    mwGetLaunchParameters1D(computeNumIters(rowreps - 1), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel10<<<grid, block>>>(
          gpu_anchorsDim, rowreps - 1, gpu_anchors3dMod,
          cpu_anchors3dMod->size[0U], cpu_anchorsDim->size[0U]);
    }
    if (cpu_anchors3dMod->size[0] < 2) {
      i = 0;
      rowreps = 1;
      rowIdx = 1;
      ibcol = 1;
      colIdx = 0;
    } else {
      i = 1;
      rowreps = 2;
      rowIdx = 2;
      ibcol = 2;
      colIdx = cpu_anchors3dMod->size[0];
    }
    sizes[0] = div_s32(colIdx - rowIdx, ibcol) + 1;
    ntilerows = b_cpu_anchors3dMod->size[0] * b_cpu_anchors3dMod->size[1];
    b_cpu_anchors3dMod->size[0] = sizes[0];
    b_cpu_anchors3dMod->size[1] = 2;
    emxEnsureCapacity_real_T(b_cpu_anchors3dMod, ntilerows, &l_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(1, sizes[0] - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real_T(b_cpu_anchors3dMod, &b_gpu_anchors3dMod, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel11<<<grid, block>>>(
          gpu_anchors3dMod, rowreps, i, sizes[0] - 1, b_gpu_anchors3dMod,
          b_cpu_anchors3dMod->size[0U], cpu_anchors3dMod->size[0U]);
    }
    rowreps = b_cpu_anchors3dMod->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(1, rowreps), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel12<<<grid, block>>>(
          b_gpu_anchors3dMod, ibcol, rowIdx, rowreps, gpu_anchors3dMod,
          cpu_anchors3dMod->size[0U], b_cpu_anchors3dMod->size[0U]);
    }
    ntilerows = cpu_tmp1->size[0] * cpu_tmp1->size[1];
    cpu_tmp1->size[0] = cpu_anchors3dMod->size[0];
    cpu_tmp1->size[1] = 2;
    emxEnsureCapacity_real_T(cpu_tmp1, ntilerows, &n_emlrtRTEI);
    ntilerows = cpu_tmp2->size[0] * cpu_tmp2->size[1];
    cpu_tmp2->size[0] = cpu_anchors3dMod->size[0];
    cpu_tmp2->size[1] = 2;
    emxEnsureCapacity_real_T(cpu_tmp2, ntilerows, &o_emlrtRTEI);
    rowreps = cpu_anchors3dMod->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(1, rowreps), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_tmp2, &gpu_tmp2, true);
    gpuEmxEnsureCapacity_real_T(cpu_tmp1, &gpu_tmp1, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel13<<<grid, block>>>(
          gpu_anchors3dMod, rowreps, gpu_tmp2, gpu_tmp1, cpu_tmp1->size[0U],
          cpu_anchors3dMod->size[0U], cpu_tmp2->size[0U]);
    }
    rangeData_outdatedOnCpu = true;
    rangeData_outdatedOnGpu = true;
    ntilerows =
        pointPillarObj.AnchorsBEV->size[0] * pointPillarObj.AnchorsBEV->size[1];
    pointPillarObj.AnchorsBEV->size[0] = cpu_tmp1->size[0];
    pointPillarObj.AnchorsBEV->size[1] = 4;
    emxEnsureCapacity_real_T(pointPillarObj.AnchorsBEV, ntilerows,
                             &p_emlrtRTEI);
    rowreps = cpu_tmp1->size[0] << 1;
    for (ntilerows = 0; ntilerows < rowreps; ntilerows++) {
      if (rangeData_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_tmp1, &gpu_tmp1);
      }
      rangeData_outdatedOnCpu = false;
      pointPillarObj.AnchorsBEV->data[ntilerows] = cpu_tmp1->data[ntilerows];
    }
    if (rowreps - 1 < 0) {
      colIdx = -1;
    } else {
      colIdx = rowreps + -1;
    }
    rowreps = cpu_tmp2->size[0] << 1;
    for (ntilerows = 0; ntilerows < rowreps; ntilerows++) {
      if (rangeData_outdatedOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_tmp2, &gpu_tmp2);
      }
      rangeData_outdatedOnGpu = false;
      pointPillarObj.AnchorsBEV->data[(colIdx + ntilerows) + 1] =
          cpu_tmp2->data[ntilerows];
    }
    pointPillarObj_not_empty = true;
  }
  ptCloud.HasKdtreeConstructed = false;
  ptCloud.HasLocationHandleAllocated = false;
  ntilerows = ptCloud.Location->size[0] * ptCloud.Location->size[1];
  ptCloud.Location->size[0] = dataLoc->size[0];
  ptCloud.Location->size[1] = 3;
  emxEnsureCapacity_real32_T(ptCloud.Location, ntilerows, &b_emlrtRTEI);
  for (ntilerows = 0; ntilerows < dataLoc->size[0] * 3; ntilerows++) {
    ptCloud.Location->data[ntilerows] = dataLoc->data[ntilerows];
  }
  ptCloud.Color->size[0] = 0;
  ptCloud.Color->size[1] = 0;
  ptCloud.Normal->size[0] = 0;
  ptCloud.Normal->size[1] = 0;
  ntilerows = ptCloud.Intensity->size[0];
  ptCloud.Intensity->size[0] = dataInt->size[0];
  emxEnsureCapacity_real32_T(ptCloud.Intensity, ntilerows, &c_emlrtRTEI);
  for (ntilerows = 0; ntilerows < dataInt->size[0]; ntilerows++) {
    ptCloud.Intensity->data[ntilerows] = dataInt->data[ntilerows];
  }
  ptCloud.RangeData->size[0] = 0;
  ptCloud.RangeData->size[1] = 0;
  ptCloud.PointCloudArrayData->size[0] = 1;
  ptCloud.PointCloudArrayData->size[1] = 1;
  ptCloud.Kdtree = nullptr;
  ptCloud.LocationHandle = nullptr;
  ptCloud.matlabCodegenIsDeleted = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emlrtPushHeapReferenceStackR2021a(
      emlrtRootTLSGlobal, false, &ptCloudUnorg,
      (void *)static_cast<void (*)(coder::pointCloud *)>(
          &coder::handle_matlabCodegenDestructor),
      nullptr, nullptr, nullptr);
  ptCloudUnorg.matlabCodegenIsDeleted = true;
  coder::vision::internal::codegen::gpu::d_PointCloudImpl_extractValidPo(
      ptCloud.Location, cpu_indices, &indices_outdatedOnCpu, &gpu_indices,
      &indices_outdatedOnGpu);
  coder::vision::internal::codegen::gpu::PointCloudImpl_subsetImpl(
      ptCloud.Location, ptCloud.Color, ptCloud.Normal, ptCloud.Intensity,
      ptCloud.RangeData, cpu_indices, &gpu_indices, &indices_outdatedOnGpu,
      cpu_location, &location_outdatedOnCpu, &gpu_location,
      &location_outdatedOnGpu, cpu_value, &value_outdatedOnCpu, &gpu_value,
      &value_outdatedOnGpu, cpu_normals, &normals_outdatedOnCpu, &gpu_normals,
      &normals_outdatedOnGpu, cpu_intensity, &intensity_outdatedOnCpu,
      &gpu_intensity, &empty_non_axis_sizes, cpu_rangeData,
      &rangeData_outdatedOnCpu, &gpu_rangeData, &rangeData_outdatedOnGpu);
  ptCloudUnorg.HasKdtreeConstructed = false;
  ptCloudUnorg.HasLocationHandleAllocated = false;
  ntilerows = ptCloudUnorg.Location->size[0] * ptCloudUnorg.Location->size[1];
  ptCloudUnorg.Location->size[0] = cpu_location->size[0];
  ptCloudUnorg.Location->size[1] = 3;
  emxEnsureCapacity_real32_T(ptCloudUnorg.Location, ntilerows, &b_emlrtRTEI);
  for (ntilerows = 0; ntilerows < cpu_location->size[0] * 3; ntilerows++) {
    if (location_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_location, &gpu_location);
    }
    location_outdatedOnCpu = false;
    ptCloudUnorg.Location->data[ntilerows] = cpu_location->data[ntilerows];
  }
  for (rowreps = 0; rowreps < 2; rowreps++) {
    numPoints[rowreps] = static_cast<uint32_T>(cpu_value->size[rowreps]);
  }
  empty_non_axis_sizes = true;
  k = 0;
  exitg1 = false;
  while ((!exitg1) && (k < 2)) {
    if (static_cast<int32_T>(numPoints[k]) != (k << 1) + 1) {
      empty_non_axis_sizes = false;
      exitg1 = true;
    } else {
      k++;
    }
  }
  if (empty_non_axis_sizes) {
    numPoints[0] = static_cast<uint32_T>(ptCloudUnorg.Location->size[0]);
    colIdx = cpu_value->size[0] * static_cast<int32_T>(numPoints[0]);
    if (!(colIdx == static_cast<real_T>(cpu_value->size[0]) *
                        static_cast<real_T>(numPoints[0]))) {
      b_y = nullptr;
      m1 = emlrtCreateCharArray(2, &iv1[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 15, m1, &u[0]);
      emlrtAssign(&b_y, m1);
      b_error(b_y, &emlrtMCI);
    }
    empty_non_axis_sizes = false;
    ntilerows = cpu_b->size[0] * cpu_b->size[1];
    cpu_b->size[0] = colIdx;
    cpu_b->size[1] = 3;
    emxEnsureCapacity_uint8_T(cpu_b, ntilerows, &j_emlrtRTEI);
    colIdx = cpu_value->size[0];
    ntilerows = static_cast<int32_T>(numPoints[0]);
    for (i = 0; i < 3; i++) {
      rowreps = i * colIdx;
      rowIdx = i * (colIdx * ntilerows);
      for (itilerow = 0; itilerow < ntilerows; itilerow++) {
        ibcol = rowIdx + itilerow * colIdx;
        for (k = 0; k < colIdx; k++) {
          if (value_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_uint8_T(cpu_value, &gpu_value);
          }
          value_outdatedOnCpu = false;
          cpu_b->data[ibcol + k] = cpu_value->data[rowreps + k];
          empty_non_axis_sizes = true;
        }
      }
    }
    ntilerows = cpu_value->size[0] * cpu_value->size[1];
    cpu_value->size[0] = cpu_b->size[0];
    cpu_value->size[1] = 3;
    emxEnsureCapacity_uint8_T(cpu_value, ntilerows, &k_emlrtRTEI);
    rowreps = cpu_b->size[0] * 3 - 1;
    mwGetLaunchParameters1D(computeNumIters(rowreps), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_uint8_T(cpu_b, &gpu_b, !empty_non_axis_sizes);
    gpuEmxEnsureCapacity_uint8_T(cpu_value, &gpu_value, true);
    if (empty_non_axis_sizes) {
      gpuEmxMemcpyCpuToGpu_uint8_T(&gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel14<<<grid, block>>>(gpu_b, rowreps, gpu_value);
    }
    value_outdatedOnCpu = true;
  }
  ntilerows = ptCloudUnorg.Color->size[0] * ptCloudUnorg.Color->size[1];
  ptCloudUnorg.Color->size[0] = cpu_value->size[0];
  ptCloudUnorg.Color->size[1] = 3;
  emxEnsureCapacity_uint8_T(ptCloudUnorg.Color, ntilerows, &h_emlrtRTEI);
  for (ntilerows = 0; ntilerows < cpu_value->size[0] * 3; ntilerows++) {
    if (value_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_uint8_T(cpu_value, &gpu_value);
    }
    value_outdatedOnCpu = false;
    ptCloudUnorg.Color->data[ntilerows] = cpu_value->data[ntilerows];
  }
  ntilerows = ptCloudUnorg.Normal->size[0] * ptCloudUnorg.Normal->size[1];
  ptCloudUnorg.Normal->size[0] = cpu_normals->size[0];
  ptCloudUnorg.Normal->size[1] = 3;
  emxEnsureCapacity_real32_T(ptCloudUnorg.Normal, ntilerows, &i_emlrtRTEI);
  for (ntilerows = 0; ntilerows < cpu_normals->size[0] * 3; ntilerows++) {
    if (normals_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_normals, &gpu_normals);
    }
    normals_outdatedOnCpu = false;
    ptCloudUnorg.Normal->data[ntilerows] = cpu_normals->data[ntilerows];
  }
  ntilerows = ptCloudUnorg.Intensity->size[0] * ptCloudUnorg.Intensity->size[1];
  ptCloudUnorg.Intensity->size[0] = cpu_intensity->size[0];
  ptCloudUnorg.Intensity->size[1] = 1;
  emxEnsureCapacity_real32_T(ptCloudUnorg.Intensity, ntilerows, &c_emlrtRTEI);
  for (ntilerows = 0; ntilerows < cpu_intensity->size[0]; ntilerows++) {
    if (intensity_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_intensity, &gpu_intensity);
    }
    intensity_outdatedOnCpu = false;
    ptCloudUnorg.Intensity->data[ntilerows] = cpu_intensity->data[ntilerows];
  }
  ptCloudUnorg.Kdtree = nullptr;
  ptCloudUnorg.LocationHandle = nullptr;
  ptCloudUnorg.matlabCodegenIsDeleted = false;
  if (ptCloudUnorg.Location->size[0] != 0) {
    colIdx = ptCloudUnorg.Location->size[0];
  } else if ((ptCloudUnorg.Intensity->size[0] != 0) &&
             (ptCloudUnorg.Intensity->size[1] != 0)) {
    colIdx = ptCloudUnorg.Intensity->size[0];
  } else {
    colIdx = 0;
    if (ptCloudUnorg.Intensity->size[0] > 0) {
      colIdx = ptCloudUnorg.Intensity->size[0];
    }
  }
  empty_non_axis_sizes = (colIdx == 0);
  if (empty_non_axis_sizes || (ptCloudUnorg.Location->size[0] != 0)) {
    input_sizes_idx_1 = 3;
  } else {
    input_sizes_idx_1 = 0;
  }
  if (empty_non_axis_sizes || ((ptCloudUnorg.Intensity->size[0] != 0) &&
                               (ptCloudUnorg.Intensity->size[1] != 0))) {
    sizes[1] = ptCloudUnorg.Intensity->size[1];
  } else {
    sizes[1] = 0;
  }
  ntilerows = dataIn->size[0] * dataIn->size[1];
  dataIn->size[0] = colIdx;
  dataIn->size[1] = input_sizes_idx_1 + sizes[1];
  emxEnsureCapacity_real32_T(dataIn, ntilerows, &m_emlrtRTEI);
  rowreps = input_sizes_idx_1;
  for (ntilerows = 0; ntilerows < rowreps; ntilerows++) {
    for (rowIdx = 0; rowIdx < colIdx; rowIdx++) {
      dataIn->data[rowIdx + dataIn->size[0] * ntilerows] =
          ptCloudUnorg.Location->data[rowIdx + colIdx * ntilerows];
    }
  }
  rowreps = sizes[1];
  for (ntilerows = 0; ntilerows < rowreps; ntilerows++) {
    for (rowIdx = 0; rowIdx < colIdx; rowIdx++) {
      dataIn->data[rowIdx + dataIn->size[0] * (ntilerows + input_sizes_idx_1)] =
          ptCloudUnorg.Intensity->data[rowIdx + colIdx * ntilerows];
    }
  }
  pointpillardetect_kernel15<<<dim3(108U, 1U, 1U), dim3(224U, 1U, 1U)>>>(
      *gpu_pillarIndices);
  pointpillardetect_kernel16<<<dim3(21094U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_dlPillarFeatures_Data);
  checkCudaError(hipMemcpy(SD->f3.cpu_dlPillarFeatures_Data,
                            *gpu_dlPillarFeatures_Data, 43200000UL,
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(cpu_pillarIndices, *gpu_pillarIndices, 96000UL,
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  createPillarsImplSingle(&SD->f3.cpu_dlPillarFeatures_Data[0],
                          &cpu_pillarIndices[0], &dataIn->data[0],
                          dataIn->size[0], 100, 12000, 0.16F, 0.16F, 0.0F,
                          69.12F, -39.68F, 39.68F, -5.0F, 5.0F);
  checkCudaError(hipMemcpy(*gpu_pillarIndices, cpu_pillarIndices, 96000UL,
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(*gpu_dlPillarFeatures_Data,
                            SD->f3.cpu_dlPillarFeatures_Data, 43200000UL,
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  coder::internal::dlnetwork_predict(
      pointPillarObj.Network, *gpu_pillarIndices, *gpu_dlPillarFeatures_Data,
      *gpu_fv1, *gpu_fv2, *gpu_predOcc, *gpu_predClassification,
      *gpu_this_workspace_predHeading, *gpu_predAngle);
  location_outdatedOnGpu = true;
  indices_outdatedOnCpu = true;
  pointpillardetect_kernel17<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      gpu_r1, gpu_r, dv1[0], dv1[1], dv1[2], dv1[3], dv1[4], dv1[5], dv1[6],
      dv1[7], dv1[8], dv1[9], dv[0], dv[1], dv[2], dv[3], dv[4], dv[5], dv[6],
      dv[7], dv[8], dv[9]);
  checkCudaError(hipMemcpy(&cpu_r, gpu_r, 80UL, hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  rv[0] = cpu_r;
  checkCudaError(hipMemcpy(&cpu_r1, gpu_r1, 80UL, hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  rv[1] = cpu_r1;
  pointpillardetect_kernel18<<<dim3(419U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_sparseVoxelMap);
  rowreps = 0;
  for (k = 0; k < 24000; k++) {
    if (cpu_pillarIndices[k] != 0.0F) {
      rowreps++;
    }
  }
  numPillars = static_cast<real_T>(rowreps) / 2.0;
  if (numPillars < 1.0) {
    rowreps = 0;
  } else {
    rowreps = static_cast<int32_T>(numPillars);
  }
  mwGetLaunchParameters1D(computeNumIters(rowreps - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel19<<<grid, block>>>(*gpu_pillarIndices, rowreps - 1,
                                                *gpu_iv2_data);
  }
  mwGetLaunchParameters1D(computeNumIters(rowreps - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel20<<<grid, block>>>(*gpu_iv2_data, rowreps - 1,
                                                *gpu_sparseVoxelMap);
  }
  mwCallThrustScanNDEdge(&(*gpu_sparseVoxelMap)[0], 432, false, 214272);
  for (rowreps = 0; rowreps < 2; rowreps++) {
    sz[rowreps] = (rowreps << 6) + 432;
  }
  mwCallThrustScanNDOther(&(*gpu_sparseVoxelMap)[0], 2, &sz[0], 1, false,
                          214272, false);
  empty_non_axis_sizes = true;
  rangeData_outdatedOnCpu = false;
  ntilerows = cpu_anchorMask->size[0];
  cpu_anchorMask->size[0] = pointPillarObj.AnchorsBEV->size[0];
  emxEnsureCapacity_real_T(cpu_anchorMask, ntilerows, &m_emlrtRTEI);
  for (ntilerows = 0; ntilerows < pointPillarObj.AnchorsBEV->size[0];
       ntilerows++) {
    cpu_anchorMask->data[ntilerows] = 0.0;
    rangeData_outdatedOnCpu = true;
  }
  colIdx = pointPillarObj.AnchorsBEV->size[0];
  for (k = 0; k < colIdx; k++) {
    real_T xmaxGrid;
    real_T ymaxGrid;
    real_T yminGrid;
    numPillars =
        std::fmax(std::floor(pointPillarObj.AnchorsBEV->data[k] / 0.16), 1.0);
    yminGrid = std::fmax(
        std::floor((pointPillarObj.AnchorsBEV
                        ->data[k + pointPillarObj.AnchorsBEV->size[0]] -
                    -39.68) /
                   0.16),
        1.0);
    xmaxGrid = std::fmin(
        std::floor(pointPillarObj.AnchorsBEV
                       ->data[k + pointPillarObj.AnchorsBEV->size[0] * 2] /
                   0.16),
        432.0);
    ymaxGrid = std::fmin(
        std::floor((pointPillarObj.AnchorsBEV
                        ->data[k + pointPillarObj.AnchorsBEV->size[0] * 3] -
                    -39.68) /
                   0.16),
        496.0);
    if (empty_non_axis_sizes) {
      checkCudaError(hipMemcpy(SD->f3.cpu_sparseVoxelMap, *gpu_sparseVoxelMap,
                                1714176UL, hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    empty_non_axis_sizes = false;
    cpu_anchorMask->data[k] =
        ((SD->f3
              .cpu_sparseVoxelMap[(static_cast<int32_T>(xmaxGrid) +
                                   432 * (static_cast<int32_T>(ymaxGrid) - 1)) -
                                  1] -
          SD->f3
              .cpu_sparseVoxelMap[(static_cast<int32_T>(xmaxGrid) +
                                   432 * (static_cast<int32_T>(yminGrid) - 1)) -
                                  1]) -
         SD->f3
             .cpu_sparseVoxelMap[(static_cast<int32_T>(numPillars) +
                                  432 * (static_cast<int32_T>(ymaxGrid) - 1)) -
                                 1]) +
        SD->f3.cpu_sparseVoxelMap[(static_cast<int32_T>(numPillars) +
                                   432 * (static_cast<int32_T>(yminGrid) - 1)) -
                                  1];
    rangeData_outdatedOnCpu = true;
  }
  ntilerows = b_cpu_anchorMask->size[0];
  b_cpu_anchorMask->size[0] = cpu_anchorMask->size[0];
  emxEnsureCapacity_boolean_T(b_cpu_anchorMask, ntilerows, &m_emlrtRTEI);
  rowreps = cpu_anchorMask->size[0] - 1;
  mwGetLaunchParameters1D(computeNumIters(rowreps), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_anchorMask, &gpu_anchorMask,
                              !rangeData_outdatedOnCpu);
  gpuEmxEnsureCapacity_boolean_T(b_cpu_anchorMask, &b_gpu_anchorMask, true);
  if (rangeData_outdatedOnCpu) {
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_anchorMask, cpu_anchorMask);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel21<<<grid, block>>>(gpu_anchorMask, rowreps,
                                                b_gpu_anchorMask);
  }
  ibcol = 0;
  for (i = 0; i < 2; i++) {
    ibcol += 2;
  }
  ntilerows = newAnchors->size[0] * newAnchors->size[1];
  newAnchors->size[0] = 1;
  newAnchors->size[1] = ibcol;
  emxEnsureCapacity_cell_wrap_40(newAnchors, ntilerows, &m_emlrtRTEI);
  for (i = 0; i < 2; i++) {
    for (ntilerows = 0; ntilerows < 2; ntilerows++) {
      numPillars = static_cast<real_T>(ntilerows + i * 2) + 1.0;
      for (rowreps = 0; rowreps < 5; rowreps++) {
        newAnchors->data[static_cast<int32_T>(numPillars) - 1].f1[rowreps] =
            rv[i].f1[ntilerows + (rowreps << 1)];
      }
    }
  }
  ntilerows = c_cpu_anchorMask->size[0] * c_cpu_anchorMask->size[1] *
              c_cpu_anchorMask->size[2];
  c_cpu_anchorMask->size[0] = ibcol;
  c_cpu_anchorMask->size[1] = 216;
  c_cpu_anchorMask->size[2] = 248;
  emxEnsureCapacity_boolean_T(c_cpu_anchorMask, ntilerows, &m_emlrtRTEI);
  rowreps = ibcol * 216 * 248 - 1;
  mwGetLaunchParameters1D(computeNumIters(rowreps), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_boolean_T(c_cpu_anchorMask, &c_gpu_anchorMask, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel22<<<grid, block>>>(gpu_anchorMask, rowreps,
                                                c_gpu_anchorMask);
  }
  empty_non_axis_sizes = true;
  if (c_cpu_anchorMask->size[0] != 0) {
    rowreps = 0;
    k = 0;
    exitg1 = false;
    while ((!exitg1) && (k + 1 < 4)) {
      if (c_cpu_anchorMask->size[dv8[k] - 1] != 1) {
        if (rowreps > dv8[k]) {
          empty_non_axis_sizes = false;
          exitg1 = true;
        } else {
          rowreps = dv8[k];
          k++;
        }
      } else {
        k++;
      }
    }
  }
  if (empty_non_axis_sizes) {
    ntilerows = d_cpu_anchorMask->size[0] * d_cpu_anchorMask->size[1] *
                d_cpu_anchorMask->size[2];
    d_cpu_anchorMask->size[0] = 216;
    d_cpu_anchorMask->size[1] = 248;
    d_cpu_anchorMask->size[2] = c_cpu_anchorMask->size[0];
    emxEnsureCapacity_boolean_T(d_cpu_anchorMask, ntilerows, &m_emlrtRTEI);
    rowreps = 53568 * c_cpu_anchorMask->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(rowreps), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_boolean_T(d_cpu_anchorMask, &d_gpu_anchorMask, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel23<<<grid, block>>>(b_gpu_anchorMask, rowreps,
                                                  d_gpu_anchorMask);
    }
  } else {
    ntilerows = d_cpu_anchorMask->size[0] * d_cpu_anchorMask->size[1] *
                d_cpu_anchorMask->size[2];
    d_cpu_anchorMask->size[0] = 216;
    d_cpu_anchorMask->size[1] = 248;
    d_cpu_anchorMask->size[2] = c_cpu_anchorMask->size[0];
    emxEnsureCapacity_boolean_T(d_cpu_anchorMask, ntilerows, &q_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(247, 215, ibcol - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_boolean_T(d_cpu_anchorMask, &d_gpu_anchorMask, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel61<<<grid, block>>>(b_gpu_anchorMask,
                                                  static_cast<uint8_T>(ibcol),
                                                  ibcol - 1, d_gpu_anchorMask);
    }
  }
  if (d_cpu_anchorMask->size[2] == 4) {
    pointpillardetect_kernel24<<<dim3(419U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        d_gpu_anchorMask, threshold, *gpu_predOcc, *gpu_x);
    empty_non_axis_sizes = true;
  } else {
    checkCudaError(hipMemcpy(SD->f3.cpu_predOcc, *gpu_predOcc, 857088UL,
                              hipMemcpyDeviceToHost),
                   __FILE__, __LINE__);
    gpuEmxMemcpyGpuToCpu_boolean_T(d_cpu_anchorMask, &d_gpu_anchorMask);
    binary_expand_op(SD->f3.cpu_x, SD->f3.cpu_predOcc, threshold,
                     d_cpu_anchorMask);
    empty_non_axis_sizes = false;
  }
  rowIdx = 0;
  normals_outdatedOnGpu = false;
  colIdx = 1;
  exitg1 = false;
  while ((!exitg1) && (colIdx < 214273)) {
    if (empty_non_axis_sizes) {
      checkCudaError(
          hipMemcpy(SD->f3.cpu_x, *gpu_x, 214272UL, hipMemcpyDeviceToHost),
          __FILE__, __LINE__);
    }
    empty_non_axis_sizes = false;
    if (SD->f3.cpu_x[colIdx - 1]) {
      rowIdx++;
      SD->f3.cpu_ii_data[rowIdx - 1] = colIdx;
      normals_outdatedOnGpu = true;
      if (rowIdx >= 214272) {
        exitg1 = true;
      } else {
        colIdx++;
      }
    } else {
      colIdx++;
    }
  }
  if (rowIdx < 1) {
    k = 0;
  } else {
    k = rowIdx;
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  if (normals_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_ii_data, SD->f3.cpu_ii_data, 857088UL,
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel25<<<grid, block>>>(*gpu_ii_data, k - 1,
                                                *gpu_v1_data);
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel26<<<grid, block>>>(*gpu_v1_data, k - 1,
                                                *gpu_vk_data);
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel27<<<grid, block>>>(
        *gpu_vk_data, k - 1, *gpu_v1_data, *gpu_varargout_6_data);
  }
  normals_outdatedOnGpu = true;
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel28<<<grid, block>>>(*gpu_v1_data, k - 1,
                                                *gpu_vk_data);
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel29<<<grid, block>>>(*gpu_vk_data, k - 1,
                                                *gpu_v1_data);
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel30<<<grid, block>>>(
        *gpu_vk_data, *gpu_v1_data, k - 1, *gpu_col_data, *gpu_ii_data);
  }
  value_outdatedOnGpu = true;
  location_outdatedOnCpu = true;
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel31<<<grid, block>>>(
        *gpu_predOcc, *gpu_varargout_6_data, *gpu_col_data, *gpu_ii_data, k - 1,
        *gpu_yCen_data, *gpu_xCen_data, *gpu_confScore_data);
  }
  rangeData_outdatedOnCpu = true;
  rangeData_outdatedOnGpu = true;
  for (itilerow = 0; itilerow < k; itilerow++) {
    if (normals_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_varargout_6_data,
                                *gpu_varargout_6_data, 214272UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    normals_outdatedOnGpu = false;
    input_sizes_idx_1 = SD->f3.cpu_varargout_6_data[itilerow];
    if (location_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_fv2, *gpu_fv2, 2571264UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    location_outdatedOnGpu = false;
    if (value_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_ii_data, *gpu_ii_data, 857088UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    value_outdatedOnGpu = false;
    if (location_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_col_data, *gpu_col_data, 428544UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    location_outdatedOnCpu = false;
    if (rangeData_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_xCen_data, *gpu_xCen_data, 1714176UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    rangeData_outdatedOnCpu = false;
    SD->f3.cpu_xGt_data[itilerow] =
        SD->f3.cpu_fv2[((SD->f3.cpu_ii_data[itilerow] +
                         216 * (SD->f3.cpu_col_data[itilerow] - 1)) +
                        53568 * (input_sizes_idx_1 - 1)) -
                       1] *
            static_cast<real32_T>(
                std::sqrt(newAnchors->data[input_sizes_idx_1 - 1].f1[0] *
                              newAnchors->data[input_sizes_idx_1 - 1].f1[0] +
                          newAnchors->data[input_sizes_idx_1 - 1].f1[1] *
                              newAnchors->data[input_sizes_idx_1 - 1].f1[1])) +
        static_cast<real32_T>(SD->f3.cpu_xCen_data[itilerow]);
    xGt_data_outdatedOnGpu = true;
  }
  for (itilerow = 0; itilerow < k; itilerow++) {
    if (normals_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_varargout_6_data,
                                *gpu_varargout_6_data, 214272UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    normals_outdatedOnGpu = false;
    input_sizes_idx_1 = SD->f3.cpu_varargout_6_data[itilerow];
    if (location_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_fv2, *gpu_fv2, 2571264UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    location_outdatedOnGpu = false;
    if (value_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_ii_data, *gpu_ii_data, 857088UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    value_outdatedOnGpu = false;
    if (location_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_col_data, *gpu_col_data, 428544UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    location_outdatedOnCpu = false;
    if (rangeData_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_yCen_data, *gpu_yCen_data, 1714176UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    rangeData_outdatedOnGpu = false;
    SD->f3.cpu_yGt_data[itilerow] =
        SD->f3.cpu_fv2[(((SD->f3.cpu_ii_data[itilerow] +
                          216 * (SD->f3.cpu_col_data[itilerow] - 1)) +
                         53568 * (input_sizes_idx_1 - 1)) +
                        53568 * ibcol) -
                       1] *
            static_cast<real32_T>(
                std::sqrt(newAnchors->data[input_sizes_idx_1 - 1].f1[0] *
                              newAnchors->data[input_sizes_idx_1 - 1].f1[0] +
                          newAnchors->data[input_sizes_idx_1 - 1].f1[1] *
                              newAnchors->data[input_sizes_idx_1 - 1].f1[1])) +
        static_cast<real32_T>(SD->f3.cpu_yCen_data[itilerow]);
    yGt_data_outdatedOnGpu = true;
  }
  for (itilerow = 0; itilerow < k; itilerow++) {
    if (normals_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_varargout_6_data,
                                *gpu_varargout_6_data, 214272UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    normals_outdatedOnGpu = false;
    input_sizes_idx_1 = SD->f3.cpu_varargout_6_data[itilerow];
    if (location_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_fv2, *gpu_fv2, 2571264UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    location_outdatedOnGpu = false;
    if (value_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_ii_data, *gpu_ii_data, 857088UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    value_outdatedOnGpu = false;
    if (location_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_col_data, *gpu_col_data, 428544UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    location_outdatedOnCpu = false;
    SD->f3.cpu_zGt_data[itilerow] =
        SD->f3.cpu_fv2[(((SD->f3.cpu_ii_data[itilerow] +
                          216 * (SD->f3.cpu_col_data[itilerow] - 1)) +
                         53568 * (input_sizes_idx_1 - 1)) +
                        53568 * ibcol * 2) -
                       1] *
            static_cast<real32_T>(
                newAnchors->data[input_sizes_idx_1 - 1].f1[2]) +
        static_cast<real32_T>(newAnchors->data[input_sizes_idx_1 - 1].f1[3]);
    zGt_data_outdatedOnGpu = true;
  }
  for (itilerow = 0; itilerow < k; itilerow++) {
    if (normals_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_varargout_6_data,
                                *gpu_varargout_6_data, 214272UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    normals_outdatedOnGpu = false;
    input_sizes_idx_1 = SD->f3.cpu_varargout_6_data[itilerow];
    if (indices_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_fv1, *gpu_fv1, 2571264UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    indices_outdatedOnCpu = false;
    if (value_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_ii_data, *gpu_ii_data, 857088UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    value_outdatedOnGpu = false;
    if (location_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_col_data, *gpu_col_data, 428544UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    location_outdatedOnCpu = false;
    SD->f3.cpu_lGt_data[itilerow] =
        std::exp(SD->f3.cpu_fv1[((SD->f3.cpu_ii_data[itilerow] +
                                  216 * (SD->f3.cpu_col_data[itilerow] - 1)) +
                                 53568 * (input_sizes_idx_1 - 1)) -
                                1]) *
        static_cast<real32_T>(newAnchors->data[input_sizes_idx_1 - 1].f1[0]);
    lGt_data_outdatedOnGpu = true;
  }
  for (itilerow = 0; itilerow < k; itilerow++) {
    if (normals_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_varargout_6_data,
                                *gpu_varargout_6_data, 214272UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    normals_outdatedOnGpu = false;
    input_sizes_idx_1 = SD->f3.cpu_varargout_6_data[itilerow];
    if (indices_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_fv1, *gpu_fv1, 2571264UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    indices_outdatedOnCpu = false;
    if (value_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_ii_data, *gpu_ii_data, 857088UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    value_outdatedOnGpu = false;
    if (location_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_col_data, *gpu_col_data, 428544UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    location_outdatedOnCpu = false;
    SD->f3.cpu_wGt_data[itilerow] =
        std::exp(SD->f3.cpu_fv1[(((SD->f3.cpu_ii_data[itilerow] +
                                   216 * (SD->f3.cpu_col_data[itilerow] - 1)) +
                                  53568 * (input_sizes_idx_1 - 1)) +
                                 53568 * ibcol) -
                                1]) *
        static_cast<real32_T>(newAnchors->data[input_sizes_idx_1 - 1].f1[1]);
    wGt_data_outdatedOnGpu = true;
  }
  for (itilerow = 0; itilerow < k; itilerow++) {
    if (normals_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_varargout_6_data,
                                *gpu_varargout_6_data, 214272UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    normals_outdatedOnGpu = false;
    input_sizes_idx_1 = SD->f3.cpu_varargout_6_data[itilerow];
    if (indices_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_fv1, *gpu_fv1, 2571264UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    indices_outdatedOnCpu = false;
    if (value_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_ii_data, *gpu_ii_data, 857088UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    value_outdatedOnGpu = false;
    if (location_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_col_data, *gpu_col_data, 428544UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    location_outdatedOnCpu = false;
    SD->f3.cpu_hGt_data[itilerow] =
        std::exp(SD->f3.cpu_fv1[(((SD->f3.cpu_ii_data[itilerow] +
                                   216 * (SD->f3.cpu_col_data[itilerow] - 1)) +
                                  53568 * (input_sizes_idx_1 - 1)) +
                                 53568 * ibcol * 2) -
                                1]) *
        static_cast<real32_T>(newAnchors->data[input_sizes_idx_1 - 1].f1[2]);
    hGt_data_outdatedOnGpu = true;
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel32<<<grid, block>>>(
        *gpu_this_workspace_predHeading, *gpu_varargout_6_data, *gpu_col_data,
        *gpu_ii_data, k - 1, *gpu_hdGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel33<<<grid, block>>>(k - 1, *gpu_hdGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel34<<<grid, block>>>(k - 1, *gpu_hdGt_data);
  }
  empty_non_axis_sizes = true;
  pointpillardetect_kernel35<<<dim3(419U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_predAngle);
  rangeData_outdatedOnCpu = true;
  for (itilerow = 0; itilerow < k; itilerow++) {
    if (normals_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_varargout_6_data,
                                *gpu_varargout_6_data, 214272UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    normals_outdatedOnGpu = false;
    input_sizes_idx_1 = SD->f3.cpu_varargout_6_data[itilerow];
    if (empty_non_axis_sizes) {
      checkCudaError(hipMemcpy(SD->f3.cpu_hdGt_data, *gpu_hdGt_data, 857088UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    empty_non_axis_sizes = false;
    if (rangeData_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_predAngle, *gpu_predAngle, 857088UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    rangeData_outdatedOnCpu = false;
    if (value_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_ii_data, *gpu_ii_data, 857088UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    value_outdatedOnGpu = false;
    if (location_outdatedOnCpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_col_data, *gpu_col_data, 428544UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    location_outdatedOnCpu = false;
    SD->f3.cpu_angGt_data[itilerow] =
        SD->f3.cpu_hdGt_data[itilerow] *
            std::asin(
                SD->f3.cpu_predAngle[((SD->f3.cpu_ii_data[itilerow] +
                                       216 * (SD->f3.cpu_col_data[itilerow] -
                                              1)) +
                                      53568 * (input_sizes_idx_1 - 1)) -
                                     1]) +
        static_cast<real32_T>(newAnchors->data[input_sizes_idx_1 - 1].f1[4]);
    angGt_data_outdatedOnGpu = true;
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  if (angGt_data_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_angGt_data, SD->f3.cpu_angGt_data, 857088UL,
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel36<<<grid, block>>>(k - 1, *gpu_angGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel37<<<grid, block>>>(k - 1, *gpu_angGt_data);
  }
  colIdx = 53568 * ibcol;
  if (colIdx > 0) {
    colIdx = 428544 / colIdx;
    ntilerows = cpu_clsMat->size[0] * cpu_clsMat->size[1];
    cpu_clsMat->size[0] = k;
    cpu_clsMat->size[1] = colIdx;
    emxEnsureCapacity_real_T(cpu_clsMat, ntilerows, &m_emlrtRTEI);
    rowreps = k * colIdx - 1;
    mwGetLaunchParameters1D(computeNumIters(rowreps), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_clsMat, &gpu_clsMat, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel38<<<grid, block>>>(rowreps, gpu_clsMat);
    }
    mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel39<<<grid, block>>>(
          *gpu_predClassification, ibcol, *gpu_varargout_6_data, *gpu_col_data,
          *gpu_ii_data, k - 1, gpu_clsMat, cpu_clsMat->size[1U],
          cpu_clsMat->size[0U]);
    }
    ibcol = cpu_clsMat->size[0];
    rowreps = cpu_clsMat->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(rowreps), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel40<<<grid, block>>>(rowreps, *gpu_ii_data);
    }
    if (cpu_clsMat->size[0] != 0) {
      rowreps = cpu_clsMat->size[1];
      colIdx = cpu_clsMat->size[0];
      rowIdx = cpu_clsMat->size[0] * (cpu_clsMat->size[1] - 1);
      mwGetLaunchParameters1D(computeNumIters(colIdx - 1), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel41<<<grid, block>>>(
            rowreps, gpu_clsMat, rowIdx, colIdx, colIdx - 1, *gpu_ii_data,
            *gpu_xCen_data);
      }
    }
    mwGetLaunchParameters1D(computeNumIters(ibcol - 1), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel42<<<grid, block>>>(*gpu_ii_data, ibcol - 1,
                                                  *gpu_yCen_data);
    }
  } else {
    mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel55<<<grid, block>>>(
          *gpu_predClassification, *gpu_varargout_6_data, *gpu_col_data,
          *gpu_ii_data, k - 1, *gpu_varargout_1_data);
    }
    mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel56<<<grid, block>>>(*gpu_varargout_1_data, k - 1,
                                                  *gpu_xCen_data);
    }
    mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel57<<<grid, block>>>(k - 1, *gpu_xCen_data);
    }
    mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel58<<<grid, block>>>(*gpu_xCen_data, k - 1,
                                                  *gpu_yCen_data);
    }
    mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel59<<<grid, block>>>(threshold, k - 1,
                                                  *gpu_yCen_data);
    }
    mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel60<<<grid, block>>>(threshold, k - 1,
                                                  *gpu_yCen_data);
    }
  }
  mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
  if (hGt_data_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_hGt_data, SD->f3.cpu_hGt_data, 857088UL,
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  if (wGt_data_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_wGt_data, SD->f3.cpu_wGt_data, 857088UL,
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  if (lGt_data_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_lGt_data, SD->f3.cpu_lGt_data, 857088UL,
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  if (zGt_data_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_zGt_data, SD->f3.cpu_zGt_data, 857088UL,
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  if (yGt_data_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_yGt_data, SD->f3.cpu_yGt_data, 857088UL,
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  if (xGt_data_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_xGt_data, SD->f3.cpu_xGt_data, 857088UL,
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillardetect_kernel43<<<grid, block>>>(
        *gpu_yCen_data, *gpu_confScore_data, *gpu_angGt_data, *gpu_hGt_data,
        *gpu_wGt_data, *gpu_lGt_data, *gpu_zGt_data, *gpu_yGt_data, k,
        *gpu_xGt_data, k - 1, *gpu_boxPreds_data);
  }
  if (k != 0) {
    mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel44<<<grid, block>>>(*gpu_boxPreds_data, k, k - 1,
                                                  *gpu_x_data);
    }
    empty_non_axis_sizes = true;
    rowIdx = 0;
    normals_outdatedOnGpu = false;
    itilerow = k;
    colIdx = 1;
    exitg1 = false;
    while ((!exitg1) && (colIdx <= k)) {
      if (empty_non_axis_sizes) {
        checkCudaError(hipMemcpy(SD->f3.cpu_x_data, *gpu_x_data, 214272UL,
                                  hipMemcpyDeviceToHost),
                       __FILE__, __LINE__);
      }
      empty_non_axis_sizes = false;
      if (SD->f3.cpu_x_data[colIdx - 1]) {
        rowIdx++;
        SD->f3.cpu_ii_data[rowIdx - 1] = colIdx;
        normals_outdatedOnGpu = true;
        if (rowIdx >= k) {
          exitg1 = true;
        } else {
          colIdx++;
        }
      } else {
        colIdx++;
      }
    }
    if (k == 1) {
      if (rowIdx == 0) {
        normals_outdatedOnGpu = false;
        itilerow = 0;
      }
    } else if (rowIdx < 1) {
      itilerow = 0;
    } else {
      itilerow = rowIdx;
    }
    mwGetLaunchParameters1D(computeNumIters(k - 1), &grid, &block, 2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillardetect_kernel45<<<grid, block>>>(k, k - 1, *gpu_boxPreds_data);
    }
    if (itilerow != 0) {
      int32_T selectedIndex_size[1];
      scores_size[0] = itilerow;
      mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                              2147483647U);
      if (normals_outdatedOnGpu) {
        checkCudaError(hipMemcpy(*gpu_ii_data, SD->f3.cpu_ii_data, 857088UL,
                                  hipMemcpyHostToDevice),
                       __FILE__, __LINE__);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel46<<<grid, block>>>(
            *gpu_boxPreds_data, k, *gpu_ii_data, itilerow - 1, *gpu_hdGt_data,
            gpu_scores_data);
      }
      rangeData_outdatedOnCpu = true;
      colIdx = 2;
      if (itilerow != 1) {
        colIdx = 1;
      }
      sz[0] = itilerow;
      sz[1] = 1;
      rowreps = scores_size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(rowreps), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel47<<<grid, block>>>(
            gpu_scores_data, rowreps, *gpu_xCen_data, *gpu_varargout_1_data);
      }
      gpu_thrust_sort_with_index(&(*gpu_varargout_1_data)[0],
                                 &(*gpu_xCen_data)[0], 2, &sz[0], colIdx, 'd',
                                 false);
      boxPreds_size[0] = itilerow;
      boxPreds_size[1] = 5;
      mwGetLaunchParameters1D(computeNumIters(4, itilerow - 1), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel48<<<grid, block>>>(
            *gpu_boxPreds_data, k, *gpu_ii_data, *gpu_xCen_data, itilerow,
            itilerow - 1, *b_gpu_boxPreds_data, iv23[0], iv23[1], iv23[2],
            iv23[3], iv23[4]);
      }
      mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel49<<<grid, block>>>(
            *gpu_hdGt_data, *gpu_xCen_data, itilerow - 1,
            *gpu_varargout_1_data);
      }
      checkCudaError(hipMemcpy(SD->f3.cpu_boxPreds_data, *b_gpu_boxPreds_data,
                                4285440UL, hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
      checkCudaError(hipMemcpy(SD->f3.cpu_xGt_data, *gpu_varargout_1_data,
                                857088UL, hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
      coder::b_selectStrongestBboxCodegen(
          SD, SD->f3.cpu_boxPreds_data, boxPreds_size, SD->f3.cpu_xGt_data,
          SD->f3.cpu_selectedIndex_data, selectedIndex_size);
      mwGetLaunchParameters1D(computeNumIters(selectedIndex_size[0] - 1), &grid,
                              &block, 2147483647U);
      checkCudaError(hipMemcpy(*gpu_selectedIndex_data,
                                SD->f3.cpu_selectedIndex_data, 214272UL,
                                hipMemcpyHostToDevice),
                     __FILE__, __LINE__);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel50<<<grid, block>>>(
            *gpu_selectedIndex_data, selectedIndex_size[0] - 1, *gpu_x_data);
      }
      mwGetLaunchParameters1D(computeNumIters(selectedIndex_size[0] - 1), &grid,
                              &block, 2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel51<<<grid, block>>>(*gpu_x_data, *gpu_xCen_data,
                                                    selectedIndex_size[0] - 1,
                                                    *gpu_selectedIndex_data);
      }
      rangeData_outdatedOnGpu = true;
      rowIdx = selectedIndex_size[0] - 1;
      rowreps = 0;
      colIdx = 0;
      for (i = 0; i <= rowIdx; i++) {
        if (rangeData_outdatedOnGpu) {
          checkCudaError(hipMemcpy(SD->f3.cpu_selectedIndex_data,
                                    *gpu_selectedIndex_data, 214272UL,
                                    hipMemcpyDeviceToHost),
                         __FILE__, __LINE__);
        }
        rangeData_outdatedOnGpu = false;
        if (SD->f3.cpu_selectedIndex_data[i]) {
          rowreps++;
          if (rangeData_outdatedOnCpu) {
            checkCudaError(hipMemcpy(cpu_scores_data, gpu_scores_data,
                                      static_cast<uint32_T>(scores_size[0]) *
                                          sizeof(real32_T),
                                      hipMemcpyDeviceToHost),
                           __FILE__, __LINE__);
          }
          cpu_scores_data[colIdx] = cpu_scores_data[i];
          rangeData_outdatedOnCpu = false;
          colIdx++;
        }
      }
      scores_size[0] = rowreps;
      ibcol = selectedIndex_size[0];
      rowIdx = 0;
      normals_outdatedOnGpu = false;
      itilerow = selectedIndex_size[0];
      colIdx = 1;
      exitg1 = false;
      while ((!exitg1) && (colIdx <= ibcol)) {
        if (rangeData_outdatedOnGpu) {
          checkCudaError(hipMemcpy(SD->f3.cpu_selectedIndex_data,
                                    *gpu_selectedIndex_data, 214272UL,
                                    hipMemcpyDeviceToHost),
                         __FILE__, __LINE__);
        }
        rangeData_outdatedOnGpu = false;
        if (SD->f3.cpu_selectedIndex_data[colIdx - 1]) {
          rowIdx++;
          SD->f3.cpu_ii_data[rowIdx - 1] = colIdx;
          normals_outdatedOnGpu = true;
          if (rowIdx >= ibcol) {
            exitg1 = true;
          } else {
            colIdx++;
          }
        } else {
          colIdx++;
        }
      }
      if (selectedIndex_size[0] == 1) {
        if (rowIdx == 0) {
          normals_outdatedOnGpu = false;
          itilerow = 0;
        }
      } else if (rowIdx < 1) {
        itilerow = 0;
      } else {
        itilerow = rowIdx;
      }
      bboxes_size[0] = itilerow;
      bboxes_size[1] = 9;
      rowreps = itilerow * 9 - 1;
      mwGetLaunchParameters1D(computeNumIters(rowreps), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel52<<<grid, block>>>(rowreps, gpu_bboxes_data);
      }
      mwGetLaunchParameters1D(computeNumIters(6, itilerow - 1), &grid, &block,
                              2147483647U);
      if (normals_outdatedOnGpu) {
        checkCudaError(hipMemcpy(*gpu_ii_data, SD->f3.cpu_ii_data, 857088UL,
                                  hipMemcpyHostToDevice),
                       __FILE__, __LINE__);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel53<<<grid, block>>>(
            *gpu_boxPreds_data, k, *gpu_ii_data, itilerow - 1, gpu_bboxes_data,
            iv27[0], iv27[1], iv27[2], iv27[3], iv27[4], iv27[5], iv27[6],
            bboxes_size[0]);
      }
      empty_non_axis_sizes = true;
      rowIdx = selectedIndex_size[0] - 1;
      rowreps = 0;
      for (i = 0; i <= rowIdx; i++) {
        if (rangeData_outdatedOnGpu) {
          checkCudaError(hipMemcpy(SD->f3.cpu_selectedIndex_data,
                                    *gpu_selectedIndex_data, 214272UL,
                                    hipMemcpyDeviceToHost),
                         __FILE__, __LINE__);
        }
        rangeData_outdatedOnGpu = false;
        if (SD->f3.cpu_selectedIndex_data[i]) {
          rowreps++;
        }
      }
      colIdx = 0;
      for (i = 0; i <= rowIdx; i++) {
        if (rangeData_outdatedOnGpu) {
          checkCudaError(hipMemcpy(SD->f3.cpu_selectedIndex_data,
                                    *gpu_selectedIndex_data, 214272UL,
                                    hipMemcpyDeviceToHost),
                         __FILE__, __LINE__);
        }
        rangeData_outdatedOnGpu = false;
        if (SD->f3.cpu_selectedIndex_data[i]) {
          SD->f3.cpu_tmp_data[colIdx] = i;
          tmp_data_outdatedOnGpu = true;
          colIdx++;
        }
      }
      mwGetLaunchParameters1D(computeNumIters(rowreps - 1), &grid, &block,
                              2147483647U);
      if (tmp_data_outdatedOnGpu) {
        checkCudaError(hipMemcpy(*gpu_tmp_data, SD->f3.cpu_tmp_data, 857088UL,
                                  hipMemcpyHostToDevice),
                       __FILE__, __LINE__);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillardetect_kernel54<<<grid, block>>>(
            *gpu_hdGt_data, *gpu_tmp_data, rowreps - 1, *gpu_varargout_1_data);
      }
      checkCudaError(hipMemcpy(SD->f3.cpu_xGt_data, *gpu_varargout_1_data,
                                857088UL, hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
      coder::internal::c_returnCategoricalLabels(
          SD, static_cast<real_T>(bboxes_size[0]), SD->f3.cpu_xGt_data,
          labels->codes.data, labels->codes.size, labels->categoryNames.data,
          labels->categoryNames.size);
    } else {
      empty_non_axis_sizes = false;
      bboxes_size[0] = 0;
      bboxes_size[1] = 9;
      rangeData_outdatedOnCpu = false;
      scores_size[0] = 0;
      coder::internal::c_returnCategoricalLabels(SD, labels->codes.size,
                                                 labels->categoryNames.data,
                                                 labels->categoryNames.size);
    }
  } else {
    empty_non_axis_sizes = false;
    bboxes_size[0] = 0;
    bboxes_size[1] = 9;
    rangeData_outdatedOnCpu = false;
    scores_size[0] = 0;
    coder::internal::c_returnCategoricalLabels(SD, labels->codes.size,
                                               labels->categoryNames.data,
                                               labels->categoryNames.size);
  }
  coder::handle_matlabCodegenDestructor(&ptCloudUnorg);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  coder::handle_matlabCodegenDestructor(&ptCloud);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  emxFree_real_T(&b_cpu_anchors3dMod);
  emxFree_real_T(&b_cpu_anchors);
  emxFree_real_T(&cpu_clsMat);
  emxFree_boolean_T(&d_cpu_anchorMask);
  emxFree_boolean_T(&c_cpu_anchorMask);
  emxFree_cell_wrap_40(&newAnchors);
  emxFree_boolean_T(&b_cpu_anchorMask);
  emxFree_real_T(&cpu_anchorMask);
  emxFree_real32_T(&dataIn);
  emxFree_real_T(&cpu_tmp2);
  emxFree_real_T(&cpu_tmp1);
  emxFree_uint8_T(&cpu_b);
  emxFree_real_T(&cpu_anchorsDim);
  emxFree_real_T(&cpu_anchors3dMod);
  emxFree_real_T(&cpu_tmp);
  emxFree_real32_T(&cpu_rangeData);
  emxFree_real32_T(&cpu_intensity);
  emxFree_real32_T(&cpu_normals);
  emxFree_uint8_T(&cpu_value);
  emxFree_real32_T(&cpu_location);
  emxFree_boolean_T(&cpu_indices);
  emxFreeStruct_pointCloud1(&ptCloudUnorg);
  emxFree_real_T(&cpu_anchors);
  emxFreeStruct_pointCloud(&ptCloud);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (empty_non_axis_sizes) {
    checkCudaError(
        hipMemcpy(cpu_bboxes_data, gpu_bboxes_data,
                   static_cast<uint32_T>(bboxes_size[0] * 9) * sizeof(real_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  if (rangeData_outdatedOnCpu) {
    checkCudaError(
        hipMemcpy(cpu_scores_data, gpu_scores_data,
                   static_cast<uint32_T>(scores_size[0]) * sizeof(real32_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  checkCudaError(mwCudaFree(gpu_bboxes_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_scores_data), __FILE__, __LINE__);
  gpuEmxFree_real_T(&gpu_anchors);
  gpuEmxFree_boolean_T(&gpu_indices);
  gpuEmxFree_real32_T(&gpu_location);
  gpuEmxFree_uint8_T(&gpu_value);
  gpuEmxFree_real32_T(&gpu_normals);
  gpuEmxFree_real32_T(&gpu_intensity);
  gpuEmxFree_real32_T(&gpu_rangeData);
  gpuEmxFree_real_T(&b_gpu_tmp);
  gpuEmxFree_real_T(&gpu_anchors3dMod);
  gpuEmxFree_real_T(&gpu_anchorsDim);
  gpuEmxFree_uint8_T(&gpu_b);
  gpuEmxFree_real_T(&gpu_tmp1);
  gpuEmxFree_real_T(&gpu_tmp2);
  gpuEmxFree_real_T(&gpu_anchorMask);
  gpuEmxFree_boolean_T(&b_gpu_anchorMask);
  gpuEmxFree_boolean_T(&c_gpu_anchorMask);
  gpuEmxFree_boolean_T(&d_gpu_anchorMask);
  gpuEmxFree_real_T(&gpu_clsMat);
  gpuEmxFree_real_T(&b_gpu_anchors);
  gpuEmxFree_real_T(&b_gpu_anchors3dMod);
  checkCudaError(mwCudaFree(gpu_r), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_r1), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_yy), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_xx), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_tmp), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_pillarIndices), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_dlPillarFeatures_Data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_fv1), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_fv2), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_predOcc), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_predClassification), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_this_workspace_predHeading), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaFree(*gpu_predAngle), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_sparseVoxelMap), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_iv2_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_x), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_ii_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_v1_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_vk_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_varargout_6_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_col_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_xCen_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_yCen_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_confScore_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_xGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_yGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_zGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_lGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_wGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_hGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_hdGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_angGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_varargout_1_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_boxPreds_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_x_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_boxPreds_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_selectedIndex_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_tmp_data), __FILE__, __LINE__);
}

void pointpillardetect_delete()
{
  if (!gobj_0.matlabCodegenIsDeleted) {
    gobj_0.matlabCodegenIsDeleted = true;
    coder::internal::dlnetwork_delete(&gobj_0);
  }
}

void pointpillardetect_emx_free()
{
  c_emxFreeStruct_pointPillarsObj(&pointPillarObj);
}

void pointpillardetect_emx_init()
{
  c_emxInitStruct_pointPillarsObj(&pointPillarObj, &emlrtRTEI, false);
}

void pointpillardetect_init()
{
  pointPillarObj_not_empty = false;
}

void pointpillardetect_new()
{
  gobj_0.matlabCodegenIsDeleted = true;
}

// End of code generation (pointpillardetect.cu)
