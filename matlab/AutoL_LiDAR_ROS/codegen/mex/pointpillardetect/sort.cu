#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// sort.cu
//
// Code generation for function 'sort'
//

// Include files
#include "sort.h"
#include "pointpillardetect_mexutil.h"
#include "rt_nonfinite.h"
#include <cmath>

// Function Definitions
namespace coder {
namespace internal {
void sort(real_T x_data[], const int32_T x_size[2], int32_T idx_data[],
          int32_T idx_size[2])
{
  int32_T i3;
  idx_size[0] = 1;
  idx_size[1] = x_size[1];
  for (i3 = 0; i3 < x_size[1]; i3++) {
    idx_data[i3] = 0;
  }
  if (x_size[1] != 0) {
    real_T xwork_data[24];
    real_T x4[4];
    int32_T bLen;
    int32_T i1;
    int32_T i2;
    int32_T i4;
    int32_T ib;
    int32_T n;
    int32_T nNonNaN;
    int32_T nPairs;
    int8_T iwork_data[24];
    int8_T idx4[4];
    for (i3 = 0; i3 < x_size[1]; i3++) {
      iwork_data[i3] = 0;
    }
    n = x_size[1] - 1;
    x4[0] = 0.0;
    idx4[0] = 0;
    x4[1] = 0.0;
    idx4[1] = 0;
    x4[2] = 0.0;
    idx4[2] = 0;
    x4[3] = 0.0;
    idx4[3] = 0;
    nPairs = 0;
    ib = 0;
    for (int32_T k{0}; k <= n; k++) {
      if (std::isnan(x_data[k])) {
        idx_data[n - nPairs] = k + 1;
        xwork_data[n - nPairs] = x_data[k];
        nPairs++;
      } else {
        ib++;
        idx4[ib - 1] = static_cast<int8_T>(k + 1);
        x4[ib - 1] = x_data[k];
        if (ib == 4) {
          ib = k - nPairs;
          if (x4[0] <= x4[1]) {
            i1 = 1;
            i2 = 2;
          } else {
            i1 = 2;
            i2 = 1;
          }
          if (x4[2] <= x4[3]) {
            i3 = 3;
            i4 = 4;
          } else {
            i3 = 4;
            i4 = 3;
          }
          if (x4[i1 - 1] <= x4[i3 - 1]) {
            if (x4[i2 - 1] <= x4[i3 - 1]) {
              nNonNaN = i1;
              bLen = i2;
              i1 = i3;
              i2 = i4;
            } else if (x4[i2 - 1] <= x4[i4 - 1]) {
              nNonNaN = i1;
              bLen = i3;
              i1 = i2;
              i2 = i4;
            } else {
              nNonNaN = i1;
              bLen = i3;
              i1 = i4;
            }
          } else if (x4[i1 - 1] <= x4[i4 - 1]) {
            if (x4[i2 - 1] <= x4[i4 - 1]) {
              nNonNaN = i3;
              bLen = i1;
              i1 = i2;
              i2 = i4;
            } else {
              nNonNaN = i3;
              bLen = i1;
              i1 = i4;
            }
          } else {
            nNonNaN = i3;
            bLen = i4;
          }
          idx_data[ib - 3] = idx4[nNonNaN - 1];
          idx_data[ib - 2] = idx4[bLen - 1];
          idx_data[ib - 1] = idx4[i1 - 1];
          idx_data[ib] = idx4[i2 - 1];
          x_data[ib - 3] = x4[nNonNaN - 1];
          x_data[ib - 2] = x4[bLen - 1];
          x_data[ib - 1] = x4[i1 - 1];
          x_data[ib] = x4[i2 - 1];
          ib = 0;
        }
      }
    }
    i2 = x_size[1] - nPairs;
    if (ib > 0) {
      int8_T perm[4];
      perm[1] = 0;
      perm[2] = 0;
      perm[3] = 0;
      if (ib == 1) {
        perm[0] = 1;
      } else if (ib == 2) {
        if (x4[0] <= x4[1]) {
          perm[0] = 1;
          perm[1] = 2;
        } else {
          perm[0] = 2;
          perm[1] = 1;
        }
      } else if (x4[0] <= x4[1]) {
        if (x4[1] <= x4[2]) {
          perm[0] = 1;
          perm[1] = 2;
          perm[2] = 3;
        } else if (x4[0] <= x4[2]) {
          perm[0] = 1;
          perm[1] = 3;
          perm[2] = 2;
        } else {
          perm[0] = 3;
          perm[1] = 1;
          perm[2] = 2;
        }
      } else if (x4[0] <= x4[2]) {
        perm[0] = 2;
        perm[1] = 1;
        perm[2] = 3;
      } else if (x4[1] <= x4[2]) {
        perm[0] = 2;
        perm[1] = 3;
        perm[2] = 1;
      } else {
        perm[0] = 3;
        perm[1] = 2;
        perm[2] = 1;
      }
      for (int32_T k{0}; k < ib; k++) {
        i3 = perm[k];
        idx_data[(i2 - ib) + k] = idx4[i3 - 1];
        x_data[(i2 - ib) + k] = x4[i3 - 1];
      }
    }
    i1 = nPairs >> 1;
    for (int32_T k{0}; k < i1; k++) {
      ib = idx_data[i2 + k];
      idx_data[i2 + k] = idx_data[n - k];
      idx_data[n - k] = ib;
      x_data[i2 + k] = xwork_data[n - k];
      x_data[n - k] = xwork_data[i2 + k];
    }
    if ((nPairs & 1) != 0) {
      x_data[i2 + i1] = xwork_data[i2 + i1];
    }
    nNonNaN = x_size[1] - nPairs;
    if (nNonNaN > 1) {
      int64_T b;
      int32_T exitg1;
      nPairs = nNonNaN >> 2;
      bLen = 4;
      while (nPairs > 1) {
        if ((nPairs & 1) != 0) {
          nPairs--;
          i2 = bLen * nPairs;
          ib = nNonNaN - i2;
          if (ib > bLen) {
            ib -= bLen;
            if (ib != 0) {
              n = bLen + ib;
              for (i1 = 0; i1 < n; i1++) {
                iwork_data[i1] = static_cast<int8_T>(idx_data[i2 + i1]);
                xwork_data[i1] = x_data[i2 + i1];
              }
              i4 = 1;
              i3 = bLen;
              i1 = bLen + ib;
              ib = i2 - 1;
              do {
                exitg1 = 0;
                ib++;
                if (xwork_data[i4 - 1] <= xwork_data[i3]) {
                  idx_data[ib] = iwork_data[i4 - 1];
                  x_data[ib] = xwork_data[i4 - 1];
                  if (i4 < bLen) {
                    i4++;
                  } else {
                    exitg1 = 1;
                  }
                } else {
                  idx_data[ib] = iwork_data[i3];
                  x_data[ib] = xwork_data[i3];
                  if (i3 + 1 < i1) {
                    i3++;
                  } else {
                    ib = (ib - i4) + 1;
                    b = computeEndIdx(static_cast<int64_T>(i4),
                                      static_cast<int64_T>(bLen), 1L);
                    for (int64_T j{0L}; j <= b; j++) {
                      idx_data[ib + static_cast<int32_T>(i4 + j)] =
                          iwork_data[static_cast<int32_T>(i4 + j) - 1];
                      x_data[ib + static_cast<int32_T>(i4 + j)] =
                          xwork_data[static_cast<int32_T>(i4 + j) - 1];
                    }
                    exitg1 = 1;
                  }
                }
              } while (exitg1 == 0);
            }
          }
        }
        i2 = bLen << 1;
        nPairs >>= 1;
        for (int32_T k{0}; k < nPairs; k++) {
          ib = k * i2;
          if (bLen != 0) {
            n = bLen + bLen;
            for (i1 = 0; i1 < n; i1++) {
              iwork_data[i1] = static_cast<int8_T>(idx_data[ib + i1]);
              xwork_data[i1] = x_data[ib + i1];
            }
            i4 = 1;
            i3 = bLen;
            i1 = bLen + bLen;
            ib--;
            do {
              exitg1 = 0;
              ib++;
              if (xwork_data[i4 - 1] <= xwork_data[i3]) {
                idx_data[ib] = iwork_data[i4 - 1];
                x_data[ib] = xwork_data[i4 - 1];
                if (i4 < bLen) {
                  i4++;
                } else {
                  exitg1 = 1;
                }
              } else {
                idx_data[ib] = iwork_data[i3];
                x_data[ib] = xwork_data[i3];
                if (i3 + 1 < i1) {
                  i3++;
                } else {
                  ib = (ib - i4) + 1;
                  b = computeEndIdx(static_cast<int64_T>(i4),
                                    static_cast<int64_T>(bLen), 1L);
                  for (int64_T j{0L}; j <= b; j++) {
                    idx_data[ib + static_cast<int32_T>(i4 + j)] =
                        iwork_data[static_cast<int32_T>(i4 + j) - 1];
                    x_data[ib + static_cast<int32_T>(i4 + j)] =
                        xwork_data[static_cast<int32_T>(i4 + j) - 1];
                  }
                  exitg1 = 1;
                }
              }
            } while (exitg1 == 0);
          }
        }
        bLen = i2;
      }
      if (nNonNaN > bLen) {
        ib = nNonNaN - bLen;
        if (ib != 0) {
          n = bLen + ib;
          for (i1 = 0; i1 < n; i1++) {
            iwork_data[i1] = static_cast<int8_T>(idx_data[i1]);
            xwork_data[i1] = x_data[i1];
          }
          i4 = 1;
          i3 = bLen;
          i1 = bLen + ib;
          ib = -1;
          do {
            exitg1 = 0;
            ib++;
            if (xwork_data[i4 - 1] <= xwork_data[i3]) {
              idx_data[ib] = iwork_data[i4 - 1];
              x_data[ib] = xwork_data[i4 - 1];
              if (i4 < bLen) {
                i4++;
              } else {
                exitg1 = 1;
              }
            } else {
              idx_data[ib] = iwork_data[i3];
              x_data[ib] = xwork_data[i3];
              if (i3 + 1 < i1) {
                i3++;
              } else {
                ib = (ib - i4) + 1;
                b = computeEndIdx(static_cast<int64_T>(i4),
                                  static_cast<int64_T>(bLen), 1L);
                for (int64_T j{0L}; j <= b; j++) {
                  idx_data[ib + static_cast<int32_T>(i4 + j)] =
                      iwork_data[static_cast<int32_T>(i4 + j) - 1];
                  x_data[ib + static_cast<int32_T>(i4 + j)] =
                      xwork_data[static_cast<int32_T>(i4 + j) - 1];
                }
                exitg1 = 1;
              }
            }
          } while (exitg1 == 0);
        }
      }
    }
  }
}

} // namespace internal
} // namespace coder

// End of code generation (sort.cu)
