//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// pointpillardetect_terminate.cu
//
// Code generation for function 'pointpillardetect_terminate'
//

// Include files
#include "pointpillardetect_terminate.h"
#include "_coder_pointpillardetect_mex.h"
#include "pointpillardetect.h"
#include "pointpillardetect_data.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"

// Function Declarations
static void c_pointpillardetect_emx_free_dt(const void *r);

static void emlrtExitTimeCleanupDtorFcn(const void *r);

// Function Definitions
static void c_pointpillardetect_emx_free_dt(const void *r)
{
  pointpillardetect_emx_free();
}

static void emlrtExitTimeCleanupDtorFcn(const void *r)
{
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void pointpillardetect_atexit()
{
  mexFunctionCreateRootTLS();
  try {
    emlrtPushHeapReferenceStackR2021a(emlrtRootTLSGlobal, false, nullptr,
                                      (void *)&emlrtExitTimeCleanupDtorFcn,
                                      nullptr, nullptr, nullptr);
    emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
    emlrtPushHeapReferenceStackR2021a(emlrtRootTLSGlobal, false, nullptr,
                                      (void *)&c_pointpillardetect_emx_free_dt,
                                      nullptr, nullptr, nullptr);
    pointpillardetect_delete();
    pointpillardetect_emx_free();
    emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
    emlrtExitTimeCleanup(&emlrtContextGlobal);
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
  mwCudaFree(*global_gpu_dv3);
  mwCudaFree(*global_gpu_xq);
  mwCudaFree(*b_global_gpu_xq);
  mwCudaFree(*c_global_gpu_xq);
  mwCudaFree(*d_global_gpu_xq);
}

void pointpillardetect_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (pointpillardetect_terminate.cu)
