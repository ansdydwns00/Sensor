//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// pointpillardetect_initialize.cu
//
// Code generation for function 'pointpillardetect_initialize'
//

// Include files
#include "pointpillardetect_initialize.h"
#include "_coder_pointpillardetect_mex.h"
#include "pointpillardetect.h"
#include "pointpillardetect_data.h"
#include "pointpillardetect_mexutil.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"

// Function Declarations
static void pointpillardetect_once();

// Function Definitions
static void pointpillardetect_once()
{
  mex_InitInfAndNan();
  pointpillardetect_emx_init();
  pointpillardetect_new();
  pointpillardetect_init();
  checkCudaError(mwCudaMalloc(&global_gpu_dv3, sizeof(real_T[248])), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&global_gpu_xq, sizeof(real_T[216])), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&b_global_gpu_xq, sizeof(real_T[248])), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&c_global_gpu_xq, sizeof(real_T[216])), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&d_global_gpu_xq, sizeof(real_T[248])), __FILE__,
                 __LINE__);
}

void pointpillardetect_initialize()
{
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "neural_network_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "lidar_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "video_and_image_blockset", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    pointpillardetect_once();
  }
}

// End of code generation (pointpillardetect_initialize.cu)
