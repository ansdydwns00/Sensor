#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// introsort.cu
//
// Code generation for function 'introsort'
//

// Include files
#include "introsort.h"
#include "heapsort.h"
#include "insertionsort.h"
#include "pointpillardetect_types.h"
#include "rt_nonfinite.h"
#include <cmath>

// Type Definitions
struct struct_T {
  int32_T xstart;
  int32_T xend;
  int32_T depth;
};

// Function Definitions
namespace coder {
namespace internal {
void introsort(int32_T x_data[], int32_T xend,
               const cell_wrap_6 cmp_workspace_c_data[])
{
  if (xend > 1) {
    if (xend <= 32) {
      insertionsort(x_data, 1, xend, cmp_workspace_c_data);
    } else {
      struct_T st_d_data[120];
      int32_T MAXDEPTH;
      int32_T pmax;
      int32_T pmin;
      int32_T pow2p;
      int32_T st_n;
      int32_T xmid;
      boolean_T exitg1;
      pmax = 31;
      pmin = 0;
      exitg1 = false;
      while ((!exitg1) && (pmax - pmin > 1)) {
        xmid = (pmin + pmax) >> 1;
        pow2p = 1 << xmid;
        if (pow2p == xend) {
          pmax = xmid;
          exitg1 = true;
        } else if (pow2p > xend) {
          pmax = xmid;
        } else {
          pmin = xmid;
        }
      }
      MAXDEPTH = (pmax - 1) << 1;
      st_d_data[0].xstart = 1;
      st_d_data[0].xend = xend;
      st_d_data[0].depth = 0;
      st_n = 1;
      while (st_n > 0) {
        struct_T expl_temp;
        expl_temp = st_d_data[st_n - 1];
        pmin = st_d_data[st_n - 1].xstart - 1;
        xend = st_d_data[st_n - 1].xend;
        st_n--;
        if ((expl_temp.xend - expl_temp.xstart) + 1 <= 32) {
          insertionsort(x_data, expl_temp.xstart, expl_temp.xend,
                        cmp_workspace_c_data);
        } else if (expl_temp.depth == MAXDEPTH) {
          b_heapsort(x_data, expl_temp.xstart, expl_temp.xend,
                     cmp_workspace_c_data);
        } else {
          int32_T exitg2;
          int32_T i;
          int32_T j;
          int32_T k;
          boolean_T varargout_1;
          xmid =
              (expl_temp.xstart + (expl_temp.xend - expl_temp.xstart) / 2) - 1;
          i = x_data[xmid];
          j = x_data[expl_temp.xstart - 1];
          pow2p = static_cast<int32_T>(std::fmin(
              static_cast<real_T>(
                  cmp_workspace_c_data[x_data[xmid] - 1].f1.size[1]),
              static_cast<real_T>(
                  cmp_workspace_c_data[x_data[pmin] - 1].f1.size[1])));
          varargout_1 = (cmp_workspace_c_data[x_data[xmid] - 1].f1.size[1] <
                         cmp_workspace_c_data[x_data[expl_temp.xstart - 1] - 1]
                             .f1.size[1]);
          k = 0;
          do {
            exitg2 = 0;
            if (k <= pow2p - 1) {
              if (cmp_workspace_c_data[x_data[xmid] - 1].f1.data[k] !=
                  cmp_workspace_c_data[x_data[pmin] - 1].f1.data[k]) {
                varargout_1 =
                    (cmp_workspace_c_data[x_data[xmid] - 1].f1.data[k] <
                     cmp_workspace_c_data[x_data[pmin] - 1].f1.data[k]);
                exitg2 = 1;
              } else {
                k++;
              }
            } else {
              if (cmp_workspace_c_data[i - 1].f1.size[1] ==
                  cmp_workspace_c_data[j - 1].f1.size[1]) {
                varargout_1 = (i < j);
              }
              exitg2 = 1;
            }
          } while (exitg2 == 0);
          if (varargout_1) {
            pmax = x_data[expl_temp.xstart - 1];
            x_data[expl_temp.xstart - 1] = x_data[xmid];
            x_data[xmid] = pmax;
          }
          i = x_data[expl_temp.xend - 1];
          j = x_data[expl_temp.xstart - 1];
          pow2p = static_cast<int32_T>(std::fmin(
              static_cast<real_T>(
                  cmp_workspace_c_data[x_data[xend - 1] - 1].f1.size[1]),
              static_cast<real_T>(
                  cmp_workspace_c_data[x_data[pmin] - 1].f1.size[1])));
          varargout_1 =
              (cmp_workspace_c_data[x_data[expl_temp.xend - 1] - 1].f1.size[1] <
               cmp_workspace_c_data[x_data[expl_temp.xstart - 1] - 1]
                   .f1.size[1]);
          k = 0;
          do {
            exitg2 = 0;
            if (k <= pow2p - 1) {
              if (cmp_workspace_c_data[x_data[xend - 1] - 1].f1.data[k] !=
                  cmp_workspace_c_data[x_data[pmin] - 1].f1.data[k]) {
                varargout_1 =
                    (cmp_workspace_c_data[x_data[xend - 1] - 1].f1.data[k] <
                     cmp_workspace_c_data[x_data[pmin] - 1].f1.data[k]);
                exitg2 = 1;
              } else {
                k++;
              }
            } else {
              if (cmp_workspace_c_data[i - 1].f1.size[1] ==
                  cmp_workspace_c_data[j - 1].f1.size[1]) {
                varargout_1 = (i < j);
              }
              exitg2 = 1;
            }
          } while (exitg2 == 0);
          if (varargout_1) {
            pmax = x_data[expl_temp.xstart - 1];
            x_data[expl_temp.xstart - 1] = x_data[expl_temp.xend - 1];
            x_data[expl_temp.xend - 1] = pmax;
          }
          i = x_data[expl_temp.xend - 1];
          j = x_data[xmid];
          pow2p = static_cast<int32_T>(std::fmin(
              static_cast<real_T>(
                  cmp_workspace_c_data[x_data[xend - 1] - 1].f1.size[1]),
              static_cast<real_T>(
                  cmp_workspace_c_data[x_data[xmid] - 1].f1.size[1])));
          varargout_1 =
              (cmp_workspace_c_data[x_data[expl_temp.xend - 1] - 1].f1.size[1] <
               cmp_workspace_c_data[x_data[xmid] - 1].f1.size[1]);
          k = 0;
          do {
            exitg2 = 0;
            if (k <= pow2p - 1) {
              if (cmp_workspace_c_data[x_data[xend - 1] - 1].f1.data[k] !=
                  cmp_workspace_c_data[x_data[xmid] - 1].f1.data[k]) {
                varargout_1 =
                    (cmp_workspace_c_data[x_data[xend - 1] - 1].f1.data[k] <
                     cmp_workspace_c_data[x_data[xmid] - 1].f1.data[k]);
                exitg2 = 1;
              } else {
                k++;
              }
            } else {
              if (cmp_workspace_c_data[i - 1].f1.size[1] ==
                  cmp_workspace_c_data[j - 1].f1.size[1]) {
                varargout_1 = (i < j);
              }
              exitg2 = 1;
            }
          } while (exitg2 == 0);
          if (varargout_1) {
            pmax = x_data[xmid];
            x_data[xmid] = x_data[expl_temp.xend - 1];
            x_data[expl_temp.xend - 1] = pmax;
          }
          pmin = x_data[xmid] - 1;
          x_data[xmid] = x_data[expl_temp.xend - 2];
          x_data[expl_temp.xend - 2] = pmin + 1;
          i = expl_temp.xstart - 1;
          j = expl_temp.xend - 2;
          do {
            int32_T exitg3;
            int32_T exitg4;
            exitg2 = 0;
            i++;
            do {
              exitg3 = 0;
              pmax = x_data[i];
              pow2p = static_cast<int32_T>(std::fmin(
                  static_cast<real_T>(
                      cmp_workspace_c_data[x_data[i] - 1].f1.size[1]),
                  static_cast<real_T>(cmp_workspace_c_data[pmin].f1.size[1])));
              varargout_1 = (cmp_workspace_c_data[x_data[i] - 1].f1.size[1] <
                             cmp_workspace_c_data[pmin].f1.size[1]);
              k = 0;
              do {
                exitg4 = 0;
                if (k <= pow2p - 1) {
                  if (cmp_workspace_c_data[x_data[i] - 1].f1.data[k] !=
                      cmp_workspace_c_data[pmin].f1.data[k]) {
                    varargout_1 =
                        (cmp_workspace_c_data[x_data[i] - 1].f1.data[k] <
                         cmp_workspace_c_data[pmin].f1.data[k]);
                    exitg4 = 1;
                  } else {
                    k++;
                  }
                } else {
                  if (cmp_workspace_c_data[pmax - 1].f1.size[1] ==
                      cmp_workspace_c_data[pmin].f1.size[1]) {
                    varargout_1 = (pmax < pmin + 1);
                  }
                  exitg4 = 1;
                }
              } while (exitg4 == 0);
              if (varargout_1) {
                i++;
              } else {
                exitg3 = 1;
              }
            } while (exitg3 == 0);
            j--;
            do {
              exitg3 = 0;
              pmax = x_data[j];
              pow2p = static_cast<int32_T>(std::fmin(
                  static_cast<real_T>(cmp_workspace_c_data[pmin].f1.size[1]),
                  static_cast<real_T>(
                      cmp_workspace_c_data[x_data[j] - 1].f1.size[1])));
              varargout_1 = (cmp_workspace_c_data[pmin].f1.size[1] <
                             cmp_workspace_c_data[x_data[j] - 1].f1.size[1]);
              k = 0;
              do {
                exitg4 = 0;
                if (k <= pow2p - 1) {
                  if (cmp_workspace_c_data[pmin].f1.data[k] !=
                      cmp_workspace_c_data[x_data[j] - 1].f1.data[k]) {
                    varargout_1 =
                        (cmp_workspace_c_data[pmin].f1.data[k] <
                         cmp_workspace_c_data[x_data[j] - 1].f1.data[k]);
                    exitg4 = 1;
                  } else {
                    k++;
                  }
                } else {
                  if (cmp_workspace_c_data[pmin].f1.size[1] ==
                      cmp_workspace_c_data[pmax - 1].f1.size[1]) {
                    varargout_1 = (pmin + 1 < pmax);
                  }
                  exitg4 = 1;
                }
              } while (exitg4 == 0);
              if (varargout_1) {
                j--;
              } else {
                exitg3 = 1;
              }
            } while (exitg3 == 0);
            if (i + 1 >= j + 1) {
              exitg2 = 1;
            } else {
              pmax = x_data[i];
              x_data[i] = x_data[j];
              x_data[j] = pmax;
            }
          } while (exitg2 == 0);
          x_data[expl_temp.xend - 2] = x_data[i];
          x_data[i] = pmin + 1;
          if (i + 2 < expl_temp.xend) {
            st_d_data[st_n].xstart = i + 2;
            st_d_data[st_n].xend = expl_temp.xend;
            st_d_data[st_n].depth = expl_temp.depth + 1;
            st_n++;
          }
          if (expl_temp.xstart < i + 1) {
            st_d_data[st_n].xstart = expl_temp.xstart;
            st_d_data[st_n].xend = i + 1;
            st_d_data[st_n].depth = expl_temp.depth + 1;
            st_n++;
          }
        }
      }
    }
  }
}

} // namespace internal
} // namespace coder

// End of code generation (introsort.cu)
