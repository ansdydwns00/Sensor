#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// insertionsort.cu
//
// Code generation for function 'insertionsort'
//

// Include files
#include "insertionsort.h"
#include "pointpillardetect_mexutil.h"
#include "pointpillardetect_types.h"
#include "rt_nonfinite.h"
#include <cmath>

// Function Definitions
namespace coder {
namespace internal {
void insertionsort(int32_T x_data[], int32_T xstart, int32_T xend,
                   const cell_wrap_6 cmp_workspace_c_data[])
{
  int64_T b;
  b = computeEndIdx(static_cast<int64_T>(xstart + 1),
                    static_cast<int64_T>(xend), 1L);
  for (int64_T k{0L}; k <= b; k++) {
    int32_T idx;
    int32_T xc;
    boolean_T exitg1;
    xc = x_data[static_cast<int32_T>((xstart + 1) + k) - 1] - 1;
    idx = static_cast<int32_T>((xstart + 1) + k) - 2;
    exitg1 = false;
    while ((!exitg1) && (idx + 1 >= xstart)) {
      int32_T b_k;
      int32_T j;
      int32_T n;
      boolean_T varargout_1;
      j = x_data[idx];
      n = static_cast<int32_T>(
          std::fmin(static_cast<real_T>(cmp_workspace_c_data[xc].f1.size[1]),
                    static_cast<real_T>(
                        cmp_workspace_c_data[x_data[idx] - 1].f1.size[1])));
      varargout_1 = (cmp_workspace_c_data[xc].f1.size[1] <
                     cmp_workspace_c_data[x_data[idx] - 1].f1.size[1]);
      b_k = 0;
      int32_T exitg2;
      do {
        exitg2 = 0;
        if (b_k <= n - 1) {
          if (cmp_workspace_c_data[xc].f1.data[b_k] !=
              cmp_workspace_c_data[x_data[idx] - 1].f1.data[b_k]) {
            varargout_1 = (cmp_workspace_c_data[xc].f1.data[b_k] <
                           cmp_workspace_c_data[x_data[idx] - 1].f1.data[b_k]);
            exitg2 = 1;
          } else {
            b_k++;
          }
        } else {
          if (cmp_workspace_c_data[xc].f1.size[1] ==
              cmp_workspace_c_data[j - 1].f1.size[1]) {
            varargout_1 = (xc + 1 < j);
          }
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      if (varargout_1) {
        x_data[idx + 1] = x_data[idx];
        idx--;
      } else {
        exitg1 = true;
      }
    }
    x_data[idx + 1] = xc + 1;
  }
}

} // namespace internal
} // namespace coder

// End of code generation (insertionsort.cu)
