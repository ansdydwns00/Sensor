#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// overlapPolygonRotatedRects.cu
//
// Code generation for function 'overlapPolygonRotatedRects'
//

// Include files
#include "overlapPolygonRotatedRects.h"
#include "rt_nonfinite.h"
#include "sort.h"
#include "rt_defines.h"
#include <cmath>
#include <cstring>

// Function Declarations
static real_T rt_atan2d_snf(real_T u0, real_T u1);

// Function Definitions
static real_T rt_atan2d_snf(real_T u0, real_T u1)
{
  real_T y;
  if (std::isnan(u0) || std::isnan(u1)) {
    y = rtNaN;
  } else if (std::isinf(u0) && std::isinf(u1)) {
    int32_T b;
    int32_T c;
    if (u1 > 0.0) {
      b = 1;
    } else {
      b = -1;
    }
    if (u0 > 0.0) {
      c = 1;
    } else {
      c = -1;
    }
    y = std::atan2(static_cast<real_T>(c), static_cast<real_T>(b));
  } else if (u1 == 0.0) {
    if (u0 > 0.0) {
      y = RT_PI / 2.0;
    } else if (u0 < 0.0) {
      y = -(RT_PI / 2.0);
    } else {
      y = 0.0;
    }
  } else {
    y = std::atan2(u0, u1);
  }
  return y;
}

namespace coder {
namespace vision {
namespace internal {
namespace detector {
void c_overlapPolygonRotatedRects_fi(const real32_T xIndices1_data[],
                                     const real32_T yIndices1_data[],
                                     const real32_T xIndices2_data[],
                                     const real32_T yIndices2_data[],
                                     real_T overlapedPolyPts_data[],
                                     int32_T overlapedPolyPts_size[2])
{
  real_T b_pointsInsidePoly[16];
  real_T overlapedPolyPts[16];
  real_T pointsInsidePoly[16];
  int32_T b_count;
  int32_T count;
  int32_T i;
  int32_T neg;
  int32_T pos;
  real32_T a1;
  real32_T a2;
  real32_T b1;
  real32_T b2;
  real32_T c1;
  real32_T c2;
  real32_T det;
  real32_T intersectPt_idx_0;
  std::memset(&overlapedPolyPts[0], 0, 16U * sizeof(real_T));
  count = -1;
  a1 = yIndices1_data[1] - yIndices1_data[0];
  b1 = xIndices1_data[0] - xIndices1_data[1];
  c1 = a1 * xIndices1_data[0] + b1 * yIndices1_data[0];
  a2 = yIndices2_data[1] - yIndices2_data[0];
  b2 = xIndices2_data[0] - xIndices2_data[1];
  c2 = a2 * xIndices2_data[0] + b2 * yIndices2_data[0];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[0], xIndices1_data[1])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[0], xIndices1_data[1])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[0], yIndices1_data[1])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[0], yIndices1_data[1])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[0], xIndices2_data[1])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[0], xIndices2_data[1])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[0], yIndices2_data[1])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[0], yIndices2_data[1]))) {
      count = 0;
      overlapedPolyPts[0] = intersectPt_idx_0;
      overlapedPolyPts[1] = b2;
    }
  }
  a2 = yIndices2_data[2] - yIndices2_data[1];
  b2 = xIndices2_data[1] - xIndices2_data[2];
  c2 = a2 * xIndices2_data[1] + b2 * yIndices2_data[1];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[0], xIndices1_data[1])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[0], xIndices1_data[1])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[0], yIndices1_data[1])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[0], yIndices1_data[1])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[1], xIndices2_data[2])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[1], xIndices2_data[2])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[1], yIndices2_data[2])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[1], yIndices2_data[2]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[3] - yIndices2_data[2];
  b2 = xIndices2_data[2] - xIndices2_data[3];
  c2 = a2 * xIndices2_data[2] + b2 * yIndices2_data[2];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[0], xIndices1_data[1])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[0], xIndices1_data[1])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[0], yIndices1_data[1])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[0], yIndices1_data[1])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[2], xIndices2_data[3])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[2], xIndices2_data[3])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[2], yIndices2_data[3])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[2], yIndices2_data[3]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[0] - yIndices2_data[3];
  b2 = xIndices2_data[3] - xIndices2_data[0];
  c2 = a2 * xIndices2_data[3] + b2 * yIndices2_data[3];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[0], xIndices1_data[1])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[0], xIndices1_data[1])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[0], yIndices1_data[1])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[0], yIndices1_data[1])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[3], xIndices2_data[0])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[3], xIndices2_data[0])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[3], yIndices2_data[0])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[3], yIndices2_data[0]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a1 = yIndices1_data[2] - yIndices1_data[1];
  b1 = xIndices1_data[1] - xIndices1_data[2];
  c1 = a1 * xIndices1_data[1] + b1 * yIndices1_data[1];
  a2 = yIndices2_data[1] - yIndices2_data[0];
  b2 = xIndices2_data[0] - xIndices2_data[1];
  c2 = a2 * xIndices2_data[0] + b2 * yIndices2_data[0];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[1], xIndices1_data[2])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[1], xIndices1_data[2])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[1], yIndices1_data[2])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[1], yIndices1_data[2])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[0], xIndices2_data[1])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[0], xIndices2_data[1])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[0], yIndices2_data[1])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[0], yIndices2_data[1]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[2] - yIndices2_data[1];
  b2 = xIndices2_data[1] - xIndices2_data[2];
  c2 = a2 * xIndices2_data[1] + b2 * yIndices2_data[1];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[1], xIndices1_data[2])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[1], xIndices1_data[2])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[1], yIndices1_data[2])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[1], yIndices1_data[2])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[1], xIndices2_data[2])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[1], xIndices2_data[2])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[1], yIndices2_data[2])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[1], yIndices2_data[2]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[3] - yIndices2_data[2];
  b2 = xIndices2_data[2] - xIndices2_data[3];
  c2 = a2 * xIndices2_data[2] + b2 * yIndices2_data[2];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[1], xIndices1_data[2])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[1], xIndices1_data[2])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[1], yIndices1_data[2])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[1], yIndices1_data[2])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[2], xIndices2_data[3])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[2], xIndices2_data[3])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[2], yIndices2_data[3])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[2], yIndices2_data[3]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[0] - yIndices2_data[3];
  b2 = xIndices2_data[3] - xIndices2_data[0];
  c2 = a2 * xIndices2_data[3] + b2 * yIndices2_data[3];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[1], xIndices1_data[2])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[1], xIndices1_data[2])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[1], yIndices1_data[2])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[1], yIndices1_data[2])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[3], xIndices2_data[0])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[3], xIndices2_data[0])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[3], yIndices2_data[0])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[3], yIndices2_data[0]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a1 = yIndices1_data[3] - yIndices1_data[2];
  b1 = xIndices1_data[2] - xIndices1_data[3];
  c1 = a1 * xIndices1_data[2] + b1 * yIndices1_data[2];
  a2 = yIndices2_data[1] - yIndices2_data[0];
  b2 = xIndices2_data[0] - xIndices2_data[1];
  c2 = a2 * xIndices2_data[0] + b2 * yIndices2_data[0];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[2], xIndices1_data[3])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[2], xIndices1_data[3])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[2], yIndices1_data[3])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[2], yIndices1_data[3])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[0], xIndices2_data[1])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[0], xIndices2_data[1])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[0], yIndices2_data[1])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[0], yIndices2_data[1]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[2] - yIndices2_data[1];
  b2 = xIndices2_data[1] - xIndices2_data[2];
  c2 = a2 * xIndices2_data[1] + b2 * yIndices2_data[1];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[2], xIndices1_data[3])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[2], xIndices1_data[3])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[2], yIndices1_data[3])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[2], yIndices1_data[3])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[1], xIndices2_data[2])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[1], xIndices2_data[2])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[1], yIndices2_data[2])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[1], yIndices2_data[2]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[3] - yIndices2_data[2];
  b2 = xIndices2_data[2] - xIndices2_data[3];
  c2 = a2 * xIndices2_data[2] + b2 * yIndices2_data[2];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[2], xIndices1_data[3])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[2], xIndices1_data[3])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[2], yIndices1_data[3])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[2], yIndices1_data[3])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[2], xIndices2_data[3])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[2], xIndices2_data[3])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[2], yIndices2_data[3])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[2], yIndices2_data[3]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[0] - yIndices2_data[3];
  b2 = xIndices2_data[3] - xIndices2_data[0];
  c2 = a2 * xIndices2_data[3] + b2 * yIndices2_data[3];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[2], xIndices1_data[3])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[2], xIndices1_data[3])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[2], yIndices1_data[3])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[2], yIndices1_data[3])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[3], xIndices2_data[0])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[3], xIndices2_data[0])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[3], yIndices2_data[0])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[3], yIndices2_data[0]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a1 = yIndices1_data[0] - yIndices1_data[3];
  b1 = xIndices1_data[3] - xIndices1_data[0];
  c1 = a1 * xIndices1_data[3] + b1 * yIndices1_data[3];
  a2 = yIndices2_data[1] - yIndices2_data[0];
  b2 = xIndices2_data[0] - xIndices2_data[1];
  c2 = a2 * xIndices2_data[0] + b2 * yIndices2_data[0];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[3], xIndices1_data[0])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[3], xIndices1_data[0])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[3], yIndices1_data[0])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[3], yIndices1_data[0])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[0], xIndices2_data[1])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[0], xIndices2_data[1])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[0], yIndices2_data[1])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[0], yIndices2_data[1]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[2] - yIndices2_data[1];
  b2 = xIndices2_data[1] - xIndices2_data[2];
  c2 = a2 * xIndices2_data[1] + b2 * yIndices2_data[1];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[3], xIndices1_data[0])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[3], xIndices1_data[0])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[3], yIndices1_data[0])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[3], yIndices1_data[0])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[1], xIndices2_data[2])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[1], xIndices2_data[2])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[1], yIndices2_data[2])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[1], yIndices2_data[2]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[3] - yIndices2_data[2];
  b2 = xIndices2_data[2] - xIndices2_data[3];
  c2 = a2 * xIndices2_data[2] + b2 * yIndices2_data[2];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[3], xIndices1_data[0])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[3], xIndices1_data[0])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[3], yIndices1_data[0])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[3], yIndices1_data[0])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[2], xIndices2_data[3])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[2], xIndices2_data[3])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[2], yIndices2_data[3])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[2], yIndices2_data[3]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  a2 = yIndices2_data[0] - yIndices2_data[3];
  b2 = xIndices2_data[3] - xIndices2_data[0];
  c2 = a2 * xIndices2_data[3] + b2 * yIndices2_data[3];
  det = a1 * b2 - a2 * b1;
  if (!(det == 0.0F)) {
    intersectPt_idx_0 = (b2 * c1 - b1 * c2) / det;
    b2 = (a1 * c2 - a2 * c1) / det;
    if ((intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices1_data[3], xIndices1_data[0])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices1_data[3], xIndices1_data[0])) &&
        (b2 - 0.001F <= std::fmax(yIndices1_data[3], yIndices1_data[0])) &&
        (b2 + 0.001F >= std::fmin(yIndices1_data[3], yIndices1_data[0])) &&
        (intersectPt_idx_0 - 0.001F <=
         std::fmax(xIndices2_data[3], xIndices2_data[0])) &&
        (intersectPt_idx_0 + 0.001F >=
         std::fmin(xIndices2_data[3], xIndices2_data[0])) &&
        (b2 - 0.001F <= std::fmax(yIndices2_data[3], yIndices2_data[0])) &&
        (b2 + 0.001F >= std::fmin(yIndices2_data[3], yIndices2_data[0]))) {
      count++;
      overlapedPolyPts[count << 1] = intersectPt_idx_0;
      overlapedPolyPts[(count << 1) + 1] = b2;
    }
  }
  std::memset(&pointsInsidePoly[0], 0, 16U * sizeof(real_T));
  b_count = -1;
  neg = 0;
  pos = 0;
  b2 = (yIndices1_data[0] - yIndices2_data[0]) *
           (xIndices2_data[1] - xIndices2_data[0]) -
       (xIndices1_data[0] - xIndices2_data[0]) *
           (yIndices2_data[1] - yIndices2_data[0]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[0] - 0.001F <=
         std::fmax(xIndices2_data[0], xIndices2_data[1])) &&
        (xIndices1_data[0] + 0.001F >=
         std::fmin(xIndices2_data[0], xIndices2_data[1])) &&
        (yIndices1_data[0] - 0.001F <=
         std::fmax(yIndices2_data[0], yIndices2_data[1])) &&
        (yIndices1_data[0] + 0.001F >=
         std::fmin(yIndices2_data[0], yIndices2_data[1]))) {
      b_count = 0;
      pointsInsidePoly[0] = xIndices1_data[0];
      pointsInsidePoly[1] = yIndices1_data[0];
    }
  } else if (b2 < 0.0F) {
    neg = 1;
  } else if (b2 > 0.0F) {
    pos = 1;
  }
  b2 = (yIndices1_data[0] - yIndices2_data[1]) *
           (xIndices2_data[2] - xIndices2_data[1]) -
       (xIndices1_data[0] - xIndices2_data[1]) *
           (yIndices2_data[2] - yIndices2_data[1]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[0] - 0.001F <=
         std::fmax(xIndices2_data[1], xIndices2_data[2])) &&
        (xIndices1_data[0] + 0.001F >=
         std::fmin(xIndices2_data[1], xIndices2_data[2])) &&
        (yIndices1_data[0] - 0.001F <=
         std::fmax(yIndices2_data[1], yIndices2_data[2])) &&
        (yIndices1_data[0] + 0.001F >=
         std::fmin(yIndices2_data[1], yIndices2_data[2]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[0];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[0];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices1_data[0] - yIndices2_data[2]) *
           (xIndices2_data[3] - xIndices2_data[2]) -
       (xIndices1_data[0] - xIndices2_data[2]) *
           (yIndices2_data[3] - yIndices2_data[2]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[0] - 0.001F <=
         std::fmax(xIndices2_data[2], xIndices2_data[3])) &&
        (xIndices1_data[0] + 0.001F >=
         std::fmin(xIndices2_data[2], xIndices2_data[3])) &&
        (yIndices1_data[0] - 0.001F <=
         std::fmax(yIndices2_data[2], yIndices2_data[3])) &&
        (yIndices1_data[0] + 0.001F >=
         std::fmin(yIndices2_data[2], yIndices2_data[3]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[0];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[0];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices1_data[0] - yIndices2_data[3]) *
           (xIndices2_data[0] - xIndices2_data[3]) -
       (xIndices1_data[0] - xIndices2_data[3]) *
           (yIndices2_data[0] - yIndices2_data[3]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[0] - 0.001F <=
         std::fmax(xIndices2_data[3], xIndices2_data[0])) &&
        (xIndices1_data[0] + 0.001F >=
         std::fmin(xIndices2_data[3], xIndices2_data[0])) &&
        (yIndices1_data[0] - 0.001F <=
         std::fmax(yIndices2_data[3], yIndices2_data[0])) &&
        (yIndices1_data[0] + 0.001F >=
         std::fmin(yIndices2_data[3], yIndices2_data[0]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[0];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[0];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  if ((static_cast<real_T>(neg) / 4.0 == 1.0) ||
      (static_cast<real_T>(pos) / 4.0 == 1.0)) {
    b_count++;
    pointsInsidePoly[b_count << 1] = xIndices1_data[0];
    pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[0];
  }
  neg = 0;
  pos = 0;
  b2 = (yIndices1_data[1] - yIndices2_data[0]) *
           (xIndices2_data[1] - xIndices2_data[0]) -
       (xIndices1_data[1] - xIndices2_data[0]) *
           (yIndices2_data[1] - yIndices2_data[0]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[1] - 0.001F <=
         std::fmax(xIndices2_data[0], xIndices2_data[1])) &&
        (xIndices1_data[1] + 0.001F >=
         std::fmin(xIndices2_data[0], xIndices2_data[1])) &&
        (yIndices1_data[1] - 0.001F <=
         std::fmax(yIndices2_data[0], yIndices2_data[1])) &&
        (yIndices1_data[1] + 0.001F >=
         std::fmin(yIndices2_data[0], yIndices2_data[1]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[1];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[1];
    }
  } else if (b2 < 0.0F) {
    neg = 1;
  } else if (b2 > 0.0F) {
    pos = 1;
  }
  b2 = (yIndices1_data[1] - yIndices2_data[1]) *
           (xIndices2_data[2] - xIndices2_data[1]) -
       (xIndices1_data[1] - xIndices2_data[1]) *
           (yIndices2_data[2] - yIndices2_data[1]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[1] - 0.001F <=
         std::fmax(xIndices2_data[1], xIndices2_data[2])) &&
        (xIndices1_data[1] + 0.001F >=
         std::fmin(xIndices2_data[1], xIndices2_data[2])) &&
        (yIndices1_data[1] - 0.001F <=
         std::fmax(yIndices2_data[1], yIndices2_data[2])) &&
        (yIndices1_data[1] + 0.001F >=
         std::fmin(yIndices2_data[1], yIndices2_data[2]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[1];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[1];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices1_data[1] - yIndices2_data[2]) *
           (xIndices2_data[3] - xIndices2_data[2]) -
       (xIndices1_data[1] - xIndices2_data[2]) *
           (yIndices2_data[3] - yIndices2_data[2]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[1] - 0.001F <=
         std::fmax(xIndices2_data[2], xIndices2_data[3])) &&
        (xIndices1_data[1] + 0.001F >=
         std::fmin(xIndices2_data[2], xIndices2_data[3])) &&
        (yIndices1_data[1] - 0.001F <=
         std::fmax(yIndices2_data[2], yIndices2_data[3])) &&
        (yIndices1_data[1] + 0.001F >=
         std::fmin(yIndices2_data[2], yIndices2_data[3]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[1];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[1];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices1_data[1] - yIndices2_data[3]) *
           (xIndices2_data[0] - xIndices2_data[3]) -
       (xIndices1_data[1] - xIndices2_data[3]) *
           (yIndices2_data[0] - yIndices2_data[3]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[1] - 0.001F <=
         std::fmax(xIndices2_data[3], xIndices2_data[0])) &&
        (xIndices1_data[1] + 0.001F >=
         std::fmin(xIndices2_data[3], xIndices2_data[0])) &&
        (yIndices1_data[1] - 0.001F <=
         std::fmax(yIndices2_data[3], yIndices2_data[0])) &&
        (yIndices1_data[1] + 0.001F >=
         std::fmin(yIndices2_data[3], yIndices2_data[0]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[1];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[1];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  if ((static_cast<real_T>(neg) / 4.0 == 1.0) ||
      (static_cast<real_T>(pos) / 4.0 == 1.0)) {
    b_count++;
    pointsInsidePoly[b_count << 1] = xIndices1_data[1];
    pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[1];
  }
  neg = 0;
  pos = 0;
  b2 = (yIndices1_data[2] - yIndices2_data[0]) *
           (xIndices2_data[1] - xIndices2_data[0]) -
       (xIndices1_data[2] - xIndices2_data[0]) *
           (yIndices2_data[1] - yIndices2_data[0]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[2] - 0.001F <=
         std::fmax(xIndices2_data[0], xIndices2_data[1])) &&
        (xIndices1_data[2] + 0.001F >=
         std::fmin(xIndices2_data[0], xIndices2_data[1])) &&
        (yIndices1_data[2] - 0.001F <=
         std::fmax(yIndices2_data[0], yIndices2_data[1])) &&
        (yIndices1_data[2] + 0.001F >=
         std::fmin(yIndices2_data[0], yIndices2_data[1]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[2];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[2];
    }
  } else if (b2 < 0.0F) {
    neg = 1;
  } else if (b2 > 0.0F) {
    pos = 1;
  }
  b2 = (yIndices1_data[2] - yIndices2_data[1]) *
           (xIndices2_data[2] - xIndices2_data[1]) -
       (xIndices1_data[2] - xIndices2_data[1]) *
           (yIndices2_data[2] - yIndices2_data[1]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[2] - 0.001F <=
         std::fmax(xIndices2_data[1], xIndices2_data[2])) &&
        (xIndices1_data[2] + 0.001F >=
         std::fmin(xIndices2_data[1], xIndices2_data[2])) &&
        (yIndices1_data[2] - 0.001F <=
         std::fmax(yIndices2_data[1], yIndices2_data[2])) &&
        (yIndices1_data[2] + 0.001F >=
         std::fmin(yIndices2_data[1], yIndices2_data[2]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[2];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[2];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices1_data[2] - yIndices2_data[2]) *
           (xIndices2_data[3] - xIndices2_data[2]) -
       (xIndices1_data[2] - xIndices2_data[2]) *
           (yIndices2_data[3] - yIndices2_data[2]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[2] - 0.001F <=
         std::fmax(xIndices2_data[2], xIndices2_data[3])) &&
        (xIndices1_data[2] + 0.001F >=
         std::fmin(xIndices2_data[2], xIndices2_data[3])) &&
        (yIndices1_data[2] - 0.001F <=
         std::fmax(yIndices2_data[2], yIndices2_data[3])) &&
        (yIndices1_data[2] + 0.001F >=
         std::fmin(yIndices2_data[2], yIndices2_data[3]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[2];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[2];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices1_data[2] - yIndices2_data[3]) *
           (xIndices2_data[0] - xIndices2_data[3]) -
       (xIndices1_data[2] - xIndices2_data[3]) *
           (yIndices2_data[0] - yIndices2_data[3]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[2] - 0.001F <=
         std::fmax(xIndices2_data[3], xIndices2_data[0])) &&
        (xIndices1_data[2] + 0.001F >=
         std::fmin(xIndices2_data[3], xIndices2_data[0])) &&
        (yIndices1_data[2] - 0.001F <=
         std::fmax(yIndices2_data[3], yIndices2_data[0])) &&
        (yIndices1_data[2] + 0.001F >=
         std::fmin(yIndices2_data[3], yIndices2_data[0]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[2];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[2];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  if ((static_cast<real_T>(neg) / 4.0 == 1.0) ||
      (static_cast<real_T>(pos) / 4.0 == 1.0)) {
    b_count++;
    pointsInsidePoly[b_count << 1] = xIndices1_data[2];
    pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[2];
  }
  neg = 0;
  pos = 0;
  b2 = (yIndices1_data[3] - yIndices2_data[0]) *
           (xIndices2_data[1] - xIndices2_data[0]) -
       (xIndices1_data[3] - xIndices2_data[0]) *
           (yIndices2_data[1] - yIndices2_data[0]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[3] - 0.001F <=
         std::fmax(xIndices2_data[0], xIndices2_data[1])) &&
        (xIndices1_data[3] + 0.001F >=
         std::fmin(xIndices2_data[0], xIndices2_data[1])) &&
        (yIndices1_data[3] - 0.001F <=
         std::fmax(yIndices2_data[0], yIndices2_data[1])) &&
        (yIndices1_data[3] + 0.001F >=
         std::fmin(yIndices2_data[0], yIndices2_data[1]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[3];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[3];
    }
  } else if (b2 < 0.0F) {
    neg = 1;
  } else if (b2 > 0.0F) {
    pos = 1;
  }
  b2 = (yIndices1_data[3] - yIndices2_data[1]) *
           (xIndices2_data[2] - xIndices2_data[1]) -
       (xIndices1_data[3] - xIndices2_data[1]) *
           (yIndices2_data[2] - yIndices2_data[1]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[3] - 0.001F <=
         std::fmax(xIndices2_data[1], xIndices2_data[2])) &&
        (xIndices1_data[3] + 0.001F >=
         std::fmin(xIndices2_data[1], xIndices2_data[2])) &&
        (yIndices1_data[3] - 0.001F <=
         std::fmax(yIndices2_data[1], yIndices2_data[2])) &&
        (yIndices1_data[3] + 0.001F >=
         std::fmin(yIndices2_data[1], yIndices2_data[2]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[3];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[3];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices1_data[3] - yIndices2_data[2]) *
           (xIndices2_data[3] - xIndices2_data[2]) -
       (xIndices1_data[3] - xIndices2_data[2]) *
           (yIndices2_data[3] - yIndices2_data[2]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[3] - 0.001F <=
         std::fmax(xIndices2_data[2], xIndices2_data[3])) &&
        (xIndices1_data[3] + 0.001F >=
         std::fmin(xIndices2_data[2], xIndices2_data[3])) &&
        (yIndices1_data[3] - 0.001F <=
         std::fmax(yIndices2_data[2], yIndices2_data[3])) &&
        (yIndices1_data[3] + 0.001F >=
         std::fmin(yIndices2_data[2], yIndices2_data[3]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[3];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[3];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices1_data[3] - yIndices2_data[3]) *
           (xIndices2_data[0] - xIndices2_data[3]) -
       (xIndices1_data[3] - xIndices2_data[3]) *
           (yIndices2_data[0] - yIndices2_data[3]);
  if (b2 == 0.0F) {
    if ((xIndices1_data[3] - 0.001F <=
         std::fmax(xIndices2_data[3], xIndices2_data[0])) &&
        (xIndices1_data[3] + 0.001F >=
         std::fmin(xIndices2_data[3], xIndices2_data[0])) &&
        (yIndices1_data[3] - 0.001F <=
         std::fmax(yIndices2_data[3], yIndices2_data[0])) &&
        (yIndices1_data[3] + 0.001F >=
         std::fmin(yIndices2_data[3], yIndices2_data[0]))) {
      b_count++;
      pointsInsidePoly[b_count << 1] = xIndices1_data[3];
      pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[3];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  if ((static_cast<real_T>(neg) / 4.0 == 1.0) ||
      (static_cast<real_T>(pos) / 4.0 == 1.0)) {
    b_count++;
    pointsInsidePoly[b_count << 1] = xIndices1_data[3];
    pointsInsidePoly[(b_count << 1) + 1] = yIndices1_data[3];
  }
  if (b_count + 1 < 1) {
    i = -1;
  } else {
    i = b_count;
  }
  std::memset(&b_pointsInsidePoly[0], 0, 16U * sizeof(real_T));
  b_count = -1;
  neg = 0;
  pos = 0;
  b2 = (yIndices2_data[0] - yIndices1_data[0]) *
           (xIndices1_data[1] - xIndices1_data[0]) -
       (xIndices2_data[0] - xIndices1_data[0]) *
           (yIndices1_data[1] - yIndices1_data[0]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[0] - 0.001F <=
         std::fmax(xIndices1_data[0], xIndices1_data[1])) &&
        (xIndices2_data[0] + 0.001F >=
         std::fmin(xIndices1_data[0], xIndices1_data[1])) &&
        (yIndices2_data[0] - 0.001F <=
         std::fmax(yIndices1_data[0], yIndices1_data[1])) &&
        (yIndices2_data[0] + 0.001F >=
         std::fmin(yIndices1_data[0], yIndices1_data[1]))) {
      b_count = 0;
      b_pointsInsidePoly[0] = xIndices2_data[0];
      b_pointsInsidePoly[1] = yIndices2_data[0];
    }
  } else if (b2 < 0.0F) {
    neg = 1;
  } else if (b2 > 0.0F) {
    pos = 1;
  }
  b2 = (yIndices2_data[0] - yIndices1_data[1]) *
           (xIndices1_data[2] - xIndices1_data[1]) -
       (xIndices2_data[0] - xIndices1_data[1]) *
           (yIndices1_data[2] - yIndices1_data[1]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[0] - 0.001F <=
         std::fmax(xIndices1_data[1], xIndices1_data[2])) &&
        (xIndices2_data[0] + 0.001F >=
         std::fmin(xIndices1_data[1], xIndices1_data[2])) &&
        (yIndices2_data[0] - 0.001F <=
         std::fmax(yIndices1_data[1], yIndices1_data[2])) &&
        (yIndices2_data[0] + 0.001F >=
         std::fmin(yIndices1_data[1], yIndices1_data[2]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[0];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[0];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices2_data[0] - yIndices1_data[2]) *
           (xIndices1_data[3] - xIndices1_data[2]) -
       (xIndices2_data[0] - xIndices1_data[2]) *
           (yIndices1_data[3] - yIndices1_data[2]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[0] - 0.001F <=
         std::fmax(xIndices1_data[2], xIndices1_data[3])) &&
        (xIndices2_data[0] + 0.001F >=
         std::fmin(xIndices1_data[2], xIndices1_data[3])) &&
        (yIndices2_data[0] - 0.001F <=
         std::fmax(yIndices1_data[2], yIndices1_data[3])) &&
        (yIndices2_data[0] + 0.001F >=
         std::fmin(yIndices1_data[2], yIndices1_data[3]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[0];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[0];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices2_data[0] - yIndices1_data[3]) *
           (xIndices1_data[0] - xIndices1_data[3]) -
       (xIndices2_data[0] - xIndices1_data[3]) *
           (yIndices1_data[0] - yIndices1_data[3]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[0] - 0.001F <=
         std::fmax(xIndices1_data[3], xIndices1_data[0])) &&
        (xIndices2_data[0] + 0.001F >=
         std::fmin(xIndices1_data[3], xIndices1_data[0])) &&
        (yIndices2_data[0] - 0.001F <=
         std::fmax(yIndices1_data[3], yIndices1_data[0])) &&
        (yIndices2_data[0] + 0.001F >=
         std::fmin(yIndices1_data[3], yIndices1_data[0]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[0];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[0];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  if ((static_cast<real_T>(neg) / 4.0 == 1.0) ||
      (static_cast<real_T>(pos) / 4.0 == 1.0)) {
    b_count++;
    b_pointsInsidePoly[b_count << 1] = xIndices2_data[0];
    b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[0];
  }
  neg = 0;
  pos = 0;
  b2 = (yIndices2_data[1] - yIndices1_data[0]) *
           (xIndices1_data[1] - xIndices1_data[0]) -
       (xIndices2_data[1] - xIndices1_data[0]) *
           (yIndices1_data[1] - yIndices1_data[0]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[1] - 0.001F <=
         std::fmax(xIndices1_data[0], xIndices1_data[1])) &&
        (xIndices2_data[1] + 0.001F >=
         std::fmin(xIndices1_data[0], xIndices1_data[1])) &&
        (yIndices2_data[1] - 0.001F <=
         std::fmax(yIndices1_data[0], yIndices1_data[1])) &&
        (yIndices2_data[1] + 0.001F >=
         std::fmin(yIndices1_data[0], yIndices1_data[1]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[1];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[1];
    }
  } else if (b2 < 0.0F) {
    neg = 1;
  } else if (b2 > 0.0F) {
    pos = 1;
  }
  b2 = (yIndices2_data[1] - yIndices1_data[1]) *
           (xIndices1_data[2] - xIndices1_data[1]) -
       (xIndices2_data[1] - xIndices1_data[1]) *
           (yIndices1_data[2] - yIndices1_data[1]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[1] - 0.001F <=
         std::fmax(xIndices1_data[1], xIndices1_data[2])) &&
        (xIndices2_data[1] + 0.001F >=
         std::fmin(xIndices1_data[1], xIndices1_data[2])) &&
        (yIndices2_data[1] - 0.001F <=
         std::fmax(yIndices1_data[1], yIndices1_data[2])) &&
        (yIndices2_data[1] + 0.001F >=
         std::fmin(yIndices1_data[1], yIndices1_data[2]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[1];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[1];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices2_data[1] - yIndices1_data[2]) *
           (xIndices1_data[3] - xIndices1_data[2]) -
       (xIndices2_data[1] - xIndices1_data[2]) *
           (yIndices1_data[3] - yIndices1_data[2]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[1] - 0.001F <=
         std::fmax(xIndices1_data[2], xIndices1_data[3])) &&
        (xIndices2_data[1] + 0.001F >=
         std::fmin(xIndices1_data[2], xIndices1_data[3])) &&
        (yIndices2_data[1] - 0.001F <=
         std::fmax(yIndices1_data[2], yIndices1_data[3])) &&
        (yIndices2_data[1] + 0.001F >=
         std::fmin(yIndices1_data[2], yIndices1_data[3]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[1];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[1];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices2_data[1] - yIndices1_data[3]) *
           (xIndices1_data[0] - xIndices1_data[3]) -
       (xIndices2_data[1] - xIndices1_data[3]) *
           (yIndices1_data[0] - yIndices1_data[3]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[1] - 0.001F <=
         std::fmax(xIndices1_data[3], xIndices1_data[0])) &&
        (xIndices2_data[1] + 0.001F >=
         std::fmin(xIndices1_data[3], xIndices1_data[0])) &&
        (yIndices2_data[1] - 0.001F <=
         std::fmax(yIndices1_data[3], yIndices1_data[0])) &&
        (yIndices2_data[1] + 0.001F >=
         std::fmin(yIndices1_data[3], yIndices1_data[0]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[1];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[1];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  if ((static_cast<real_T>(neg) / 4.0 == 1.0) ||
      (static_cast<real_T>(pos) / 4.0 == 1.0)) {
    b_count++;
    b_pointsInsidePoly[b_count << 1] = xIndices2_data[1];
    b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[1];
  }
  neg = 0;
  pos = 0;
  b2 = (yIndices2_data[2] - yIndices1_data[0]) *
           (xIndices1_data[1] - xIndices1_data[0]) -
       (xIndices2_data[2] - xIndices1_data[0]) *
           (yIndices1_data[1] - yIndices1_data[0]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[2] - 0.001F <=
         std::fmax(xIndices1_data[0], xIndices1_data[1])) &&
        (xIndices2_data[2] + 0.001F >=
         std::fmin(xIndices1_data[0], xIndices1_data[1])) &&
        (yIndices2_data[2] - 0.001F <=
         std::fmax(yIndices1_data[0], yIndices1_data[1])) &&
        (yIndices2_data[2] + 0.001F >=
         std::fmin(yIndices1_data[0], yIndices1_data[1]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[2];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[2];
    }
  } else if (b2 < 0.0F) {
    neg = 1;
  } else if (b2 > 0.0F) {
    pos = 1;
  }
  b2 = (yIndices2_data[2] - yIndices1_data[1]) *
           (xIndices1_data[2] - xIndices1_data[1]) -
       (xIndices2_data[2] - xIndices1_data[1]) *
           (yIndices1_data[2] - yIndices1_data[1]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[2] - 0.001F <=
         std::fmax(xIndices1_data[1], xIndices1_data[2])) &&
        (xIndices2_data[2] + 0.001F >=
         std::fmin(xIndices1_data[1], xIndices1_data[2])) &&
        (yIndices2_data[2] - 0.001F <=
         std::fmax(yIndices1_data[1], yIndices1_data[2])) &&
        (yIndices2_data[2] + 0.001F >=
         std::fmin(yIndices1_data[1], yIndices1_data[2]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[2];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[2];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices2_data[2] - yIndices1_data[2]) *
           (xIndices1_data[3] - xIndices1_data[2]) -
       (xIndices2_data[2] - xIndices1_data[2]) *
           (yIndices1_data[3] - yIndices1_data[2]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[2] - 0.001F <=
         std::fmax(xIndices1_data[2], xIndices1_data[3])) &&
        (xIndices2_data[2] + 0.001F >=
         std::fmin(xIndices1_data[2], xIndices1_data[3])) &&
        (yIndices2_data[2] - 0.001F <=
         std::fmax(yIndices1_data[2], yIndices1_data[3])) &&
        (yIndices2_data[2] + 0.001F >=
         std::fmin(yIndices1_data[2], yIndices1_data[3]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[2];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[2];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices2_data[2] - yIndices1_data[3]) *
           (xIndices1_data[0] - xIndices1_data[3]) -
       (xIndices2_data[2] - xIndices1_data[3]) *
           (yIndices1_data[0] - yIndices1_data[3]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[2] - 0.001F <=
         std::fmax(xIndices1_data[3], xIndices1_data[0])) &&
        (xIndices2_data[2] + 0.001F >=
         std::fmin(xIndices1_data[3], xIndices1_data[0])) &&
        (yIndices2_data[2] - 0.001F <=
         std::fmax(yIndices1_data[3], yIndices1_data[0])) &&
        (yIndices2_data[2] + 0.001F >=
         std::fmin(yIndices1_data[3], yIndices1_data[0]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[2];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[2];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  if ((static_cast<real_T>(neg) / 4.0 == 1.0) ||
      (static_cast<real_T>(pos) / 4.0 == 1.0)) {
    b_count++;
    b_pointsInsidePoly[b_count << 1] = xIndices2_data[2];
    b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[2];
  }
  neg = 0;
  pos = 0;
  b2 = (yIndices2_data[3] - yIndices1_data[0]) *
           (xIndices1_data[1] - xIndices1_data[0]) -
       (xIndices2_data[3] - xIndices1_data[0]) *
           (yIndices1_data[1] - yIndices1_data[0]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[3] - 0.001F <=
         std::fmax(xIndices1_data[0], xIndices1_data[1])) &&
        (xIndices2_data[3] + 0.001F >=
         std::fmin(xIndices1_data[0], xIndices1_data[1])) &&
        (yIndices2_data[3] - 0.001F <=
         std::fmax(yIndices1_data[0], yIndices1_data[1])) &&
        (yIndices2_data[3] + 0.001F >=
         std::fmin(yIndices1_data[0], yIndices1_data[1]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[3];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[3];
    }
  } else if (b2 < 0.0F) {
    neg = 1;
  } else if (b2 > 0.0F) {
    pos = 1;
  }
  b2 = (yIndices2_data[3] - yIndices1_data[1]) *
           (xIndices1_data[2] - xIndices1_data[1]) -
       (xIndices2_data[3] - xIndices1_data[1]) *
           (yIndices1_data[2] - yIndices1_data[1]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[3] - 0.001F <=
         std::fmax(xIndices1_data[1], xIndices1_data[2])) &&
        (xIndices2_data[3] + 0.001F >=
         std::fmin(xIndices1_data[1], xIndices1_data[2])) &&
        (yIndices2_data[3] - 0.001F <=
         std::fmax(yIndices1_data[1], yIndices1_data[2])) &&
        (yIndices2_data[3] + 0.001F >=
         std::fmin(yIndices1_data[1], yIndices1_data[2]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[3];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[3];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices2_data[3] - yIndices1_data[2]) *
           (xIndices1_data[3] - xIndices1_data[2]) -
       (xIndices2_data[3] - xIndices1_data[2]) *
           (yIndices1_data[3] - yIndices1_data[2]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[3] - 0.001F <=
         std::fmax(xIndices1_data[2], xIndices1_data[3])) &&
        (xIndices2_data[3] + 0.001F >=
         std::fmin(xIndices1_data[2], xIndices1_data[3])) &&
        (yIndices2_data[3] - 0.001F <=
         std::fmax(yIndices1_data[2], yIndices1_data[3])) &&
        (yIndices2_data[3] + 0.001F >=
         std::fmin(yIndices1_data[2], yIndices1_data[3]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[3];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[3];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  b2 = (yIndices2_data[3] - yIndices1_data[3]) *
           (xIndices1_data[0] - xIndices1_data[3]) -
       (xIndices2_data[3] - xIndices1_data[3]) *
           (yIndices1_data[0] - yIndices1_data[3]);
  if (b2 == 0.0F) {
    if ((xIndices2_data[3] - 0.001F <=
         std::fmax(xIndices1_data[3], xIndices1_data[0])) &&
        (xIndices2_data[3] + 0.001F >=
         std::fmin(xIndices1_data[3], xIndices1_data[0])) &&
        (yIndices2_data[3] - 0.001F <=
         std::fmax(yIndices1_data[3], yIndices1_data[0])) &&
        (yIndices2_data[3] + 0.001F >=
         std::fmin(yIndices1_data[3], yIndices1_data[0]))) {
      b_count++;
      b_pointsInsidePoly[b_count << 1] = xIndices2_data[3];
      b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[3];
    }
  } else if (b2 < 0.0F) {
    neg++;
  } else if (b2 > 0.0F) {
    pos++;
  }
  if ((static_cast<real_T>(neg) / 4.0 == 1.0) ||
      (static_cast<real_T>(pos) / 4.0 == 1.0)) {
    b_count++;
    b_pointsInsidePoly[b_count << 1] = xIndices2_data[3];
    b_pointsInsidePoly[(b_count << 1) + 1] = yIndices2_data[3];
  }
  if (b_count + 1 < 1) {
    neg = -1;
  } else {
    neg = b_count;
  }
  if (count + 1 < 1) {
    b_count = -1;
  } else {
    b_count = count;
  }
  overlapedPolyPts_size[0] = 2;
  overlapedPolyPts_size[1] = ((b_count + i) + neg) + 3;
  for (pos = 0; pos <= b_count; pos++) {
    overlapedPolyPts_data[2 * pos] = overlapedPolyPts[pos << 1];
    overlapedPolyPts_data[2 * pos + 1] = overlapedPolyPts[(pos << 1) + 1];
  }
  for (pos = 0; pos <= i; pos++) {
    overlapedPolyPts_data[2 * ((pos + b_count) + 1)] =
        pointsInsidePoly[pos << 1];
    overlapedPolyPts_data[2 * ((pos + b_count) + 1) + 1] =
        pointsInsidePoly[(pos << 1) + 1];
  }
  for (pos = 0; pos <= neg; pos++) {
    overlapedPolyPts_data[2 * (((pos + b_count) + i) + 2)] =
        b_pointsInsidePoly[pos << 1];
    overlapedPolyPts_data[2 * (((pos + b_count) + i) + 2) + 1] =
        b_pointsInsidePoly[(pos << 1) + 1];
  }
}

void d_overlapPolygonRotatedRects_fi(real_T overlapedPolyPts_data[],
                                     int32_T overlapedPolyPts_size[2],
                                     real_T overlapPoly_data[],
                                     int32_T overlapPoly_size[2])
{
  real_T distVec_data[48];
  real_T theta_data[24];
  real_T center_idx_0;
  real_T center_idx_1;
  int32_T iidx_data[24];
  int32_T iidx_size[2];
  int32_T theta_size[2];
  int32_T vlen;
  int32_T xoffset;
  vlen = overlapedPolyPts_size[1];
  if (overlapedPolyPts_size[1] == 0) {
    center_idx_0 = 0.0;
    center_idx_1 = 0.0;
  } else {
    center_idx_0 = overlapedPolyPts_data[0];
    center_idx_1 = overlapedPolyPts_data[1];
    for (int32_T k{0}; k <= vlen - 2; k++) {
      xoffset = (k + 1) << 1;
      center_idx_0 += overlapedPolyPts_data[xoffset];
      center_idx_1 += overlapedPolyPts_data[xoffset + 1];
    }
  }
  center_idx_0 /= static_cast<real_T>(overlapedPolyPts_size[1]);
  center_idx_1 /= static_cast<real_T>(overlapedPolyPts_size[1]);
  vlen = overlapedPolyPts_size[1];
  for (xoffset = 0; xoffset < overlapedPolyPts_size[1]; xoffset++) {
    distVec_data[2 * xoffset] =
        overlapedPolyPts_data[2 * xoffset] - center_idx_0;
    distVec_data[2 * xoffset + 1] =
        overlapedPolyPts_data[2 * xoffset + 1] - center_idx_1;
  }
  theta_size[0] = 1;
  theta_size[1] = overlapedPolyPts_size[1];
  for (int32_T k{0}; k < vlen; k++) {
    theta_data[k] = rt_atan2d_snf(distVec_data[2 * k + 1], distVec_data[2 * k]);
  }
  ::coder::internal::sort(theta_data, theta_size, iidx_data, iidx_size);
  vlen = iidx_size[1];
  for (xoffset = 0; xoffset < iidx_size[1]; xoffset++) {
    distVec_data[2 * xoffset] =
        overlapedPolyPts_data[2 * (iidx_data[xoffset] - 1)];
    distVec_data[2 * xoffset + 1] =
        overlapedPolyPts_data[2 * (iidx_data[xoffset] - 1) + 1];
  }
  for (xoffset = 0; xoffset < (vlen << 1); xoffset++) {
    overlapedPolyPts_data[xoffset] = distVec_data[xoffset];
  }
  overlapPoly_size[0] = 2;
  overlapPoly_size[1] = iidx_size[1] + 1;
  vlen = iidx_size[1];
  for (xoffset = 0; xoffset < vlen; xoffset++) {
    overlapPoly_data[2 * xoffset] = overlapedPolyPts_data[2 * xoffset];
    overlapPoly_data[2 * xoffset + 1] = overlapedPolyPts_data[2 * xoffset + 1];
  }
  overlapPoly_data[2 * iidx_size[1]] = overlapedPolyPts_data[0];
  overlapPoly_data[2 * iidx_size[1] + 1] = overlapedPolyPts_data[1];
}

} // namespace detector
} // namespace internal
} // namespace vision
} // namespace coder

// End of code generation (overlapPolygonRotatedRects.cu)
