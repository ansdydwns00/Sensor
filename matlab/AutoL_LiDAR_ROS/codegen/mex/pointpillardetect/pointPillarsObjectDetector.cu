#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// pointPillarsObjectDetector.cu
//
// Code generation for function 'pointPillarsObjectDetector'
//

// Include files
#include "pointPillarsObjectDetector.h"
#include "introsort.h"
#include "pointpillardetect_data.h"
#include "pointpillardetect_emxutil.h"
#include "pointpillardetect_mexutil.h"
#include "pointpillardetect_types.h"
#include "rt_nonfinite.h"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include <cmath>

// Type Definitions
struct emxArray_cell_wrap_6_1x2 {
  cell_wrap_6 data[2];
  int32_T size[2];
};

// Variable Definitions
static emlrtRTEInfo bb_emlrtRTEI{
    11,                                                                // lineNo
    5,                                                                 // colNo
    "cumsum",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/cumsum.m" // pName
};

static emlrtRTEInfo cb_emlrtRTEI{
    44,               // lineNo
    10,               // colNo
    "cellstr_unique", // fName
    "/usr/local/MATLAB/R2024a/toolbox/matlab/datatypes/shared/codegen/+matlab/"
    "+internal/+coder/+datatypes/cellstr_unique.m" // pName
};

static const char_T cv[10]{'C', 'T',    'a', 'r',    'r',
                           'u', '\x00', 'c', '\x00', 'k'};

static const boolean_T bv[128]{
    false, false, false, false, false, false, false, false, false, true,  true,
    true,  true,  true,  false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, true,  true,  true,  true,  true,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false};

// Function Definitions
void binary_expand_op(boolean_T in1[214272], const real32_T in2[214272],
                      real_T in3, const emxArray_boolean_T *in4)
{
  int32_T aux_0_2;
  int32_T stride_0_2;
  stride_0_2 = (in4->size[2] != 1);
  aux_0_2 = 0;
  for (int32_T i{0}; i < 4; i++) {
    for (int32_T i1{0}; i1 < 248; i1++) {
      for (int32_T i2{0}; i2 < 216; i2++) {
        in1[(i2 + 216 * i1) + 53568 * i] =
            ((in2[(i2 + 216 * i1) + 53568 * i] > in3) &&
             in4->data[(i2 + 216 * i1) + 53568 * aux_0_2]);
      }
    }
    aux_0_2 += stride_0_2;
  }
}

namespace coder {
namespace internal {
void c_returnCategoricalLabels(pointpillardetectStackData *SD,
                               int32_T labelNamesMod_codes_size[2],
                               cell_wrap_6 c_labelNamesMod_categoryNames_d[],
                               int32_T c_labelNamesMod_categoryNames_s[1])
{
  emxArray_cell_wrap_6_1x2 rv;
  emxArray_cell_wrap_6_1x2 valueset;
  emxArray_real_T b_gpu_y;
  emxArray_real_T gpu_y;
  emxArray_real_T *b_cpu_y;
  emxArray_real_T *cpu_y;
  int32_T b_i;
  int32_T b_j1;
  int32_T exitg1;
  int32_T i;
  int32_T j2;
  int32_T k;
  int32_T n;
  boolean_T d_data[2];
  boolean_T b;
  boolean_T b_bool;
  gpuEmxReset_real_T(&b_gpu_y);
  gpuEmxReset_real_T(&gpu_y);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  valueset.size[1] = 0;
  for (i = 0; i < 2; i++) {
    char_T v_data[5];
    n = 0;
    for (k = 0; k < 5; k++) {
      if (cv[i + (k << 1)] != 0) {
        n++;
      }
    }
    b_i = -1;
    for (k = 0; k < 5; k++) {
      char_T c;
      c = cv[i + (k << 1)];
      if (c != 0) {
        b_i++;
        v_data[b_i] = c;
      }
    }
    j2 = valueset.size[1] + 1;
    valueset.size[1]++;
    valueset.data[j2 - 1].f1.size[0] = 1;
    valueset.data[valueset.size[1] - 1].f1.size[1] = n;
    for (k = 0; k < n; k++) {
      valueset.data[valueset.size[1] - 1].f1.data[k] = v_data[k];
    }
  }
  i = valueset.size[1];
  for (b_i = 0; b_i < i; b_i++) {
    b_j1 = 0;
    while ((b_j1 + 1 <= valueset.data[b_i].f1.size[1]) &&
           bv[static_cast<uint8_T>(valueset.data[b_i].f1.data[b_j1]) & 127] &&
           (valueset.data[b_i].f1.data[b_j1] != '\x00')) {
      b_j1++;
    }
    j2 = valueset.data[b_i].f1.size[1] - 1;
    while ((j2 + 1 > 0) &&
           bv[static_cast<uint8_T>(valueset.data[b_i].f1.data[j2]) & 127] &&
           (valueset.data[b_i].f1.data[j2] != '\x00')) {
      j2--;
    }
    if (b_j1 + 1 > j2 + 1) {
      b_j1 = 0;
      j2 = -1;
    }
    rv.data[b_i].f1.size[0] = 1;
    rv.data[b_i].f1.size[1] = (j2 - b_j1) + 1;
    n = j2 - b_j1;
    for (k = 0; k <= n; k++) {
      rv.data[b_i].f1.data[k] = valueset.data[b_i].f1.data[b_j1 + k];
    }
  }
  if (valueset.size[1] != 0) {
    n = valueset.size[1];
    i = valueset.size[1];
    SD->f0.y_data[0] = 1;
    j2 = 1;
    for (k = 0; k <= n - 2; k++) {
      j2++;
      SD->f0.y_data[1] = static_cast<int8_T>(j2);
    }
    for (k = 0; k < i; k++) {
      SD->f0.idx_data[k] = SD->f0.y_data[k];
    }
    introsort(SD->f0.idx_data, valueset.size[1], rv.data);
    SD->f0.c.size[0] = valueset.size[1];
    j2 = valueset.size[1];
    for (i = 0; i < j2; i++) {
      SD->f0.c.data[i].f1.size[0] = 1;
      SD->f0.c.data[i].f1.size[1] = rv.data[SD->f0.idx_data[i] - 1].f1.size[1];
      n = rv.data[SD->f0.idx_data[i] - 1].f1.size[1];
      for (k = 0; k < n; k++) {
        SD->f0.c.data[i].f1.data[k] =
            rv.data[SD->f0.idx_data[i] - 1].f1.data[k];
      }
    }
    for (i = 0; i <= SD->f0.c.size[0] - 2; i++) {
      b_bool = false;
      b = (SD->f0.c.data[0].f1.size[1] == 0);
      if (b && (SD->f0.c.data[1].f1.size[1] == 0)) {
        b_bool = true;
      } else if (SD->f0.c.data[0].f1.size[1] == SD->f0.c.data[1].f1.size[1]) {
        j2 = 0;
        do {
          exitg1 = 0;
          if (j2 + 1 <= SD->f0.c.data[1].f1.size[1]) {
            if (SD->f0.c.data[0].f1.data[j2] != SD->f0.c.data[1].f1.data[j2]) {
              exitg1 = 1;
            } else {
              j2++;
            }
          } else {
            b_bool = true;
            exitg1 = 1;
          }
        } while (exitg1 == 0);
      }
      SD->f0.d_data[0] = !b_bool;
    }
    i = valueset.size[1];
    d_data[0] = true;
    n = valueset.size[1] - 1;
    for (k = 0; k < n; k++) {
      d_data[1] = SD->f0.d_data[0];
    }
    emxInit_real_T(&cpu_y, 1, &cb_emlrtRTEI, true);
    b_bool = false;
    k = cpu_y->size[0];
    cpu_y->size[0] = valueset.size[1];
    emxEnsureCapacity_real_T(cpu_y, k, &bb_emlrtRTEI);
    for (k = 0; k < i; k++) {
      cpu_y->data[k] = d_data[k];
      b_bool = true;
    }
    if (cpu_y->size[0] != 1) {
      gpuEmxEnsureCapacity_real_T(cpu_y, &gpu_y, !b_bool);
      if (b_bool) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_y, cpu_y);
      }
      mwCallThrustScan1D(&gpu_y.data[0], false, 2);
    }
    emxFree_real_T(&cpu_y);
  }
  if (valueset.size[1] != 0) {
    int8_T iv2_idx_0;
    iv2_idx_0 = static_cast<int8_T>(valueset.size[1]);
    c_labelNamesMod_categoryNames_s[0] = valueset.size[1];
    for (j2 = 0; j2 < iv2_idx_0; j2++) {
      c_labelNamesMod_categoryNames_d[j2].f1.size[0] = 1;
      c_labelNamesMod_categoryNames_d[j2].f1.size[1] = 0;
    }
    j2 = valueset.size[1];
    c_labelNamesMod_categoryNames_s[0] = valueset.size[1];
    for (i = 0; i < j2; i++) {
      c_labelNamesMod_categoryNames_d[i].f1.size[0] = 1;
      n = rv.data[i].f1.size[1];
      c_labelNamesMod_categoryNames_d[i].f1.size[1] = rv.data[i].f1.size[1];
      for (b_j1 = 0; b_j1 < n; b_j1++) {
        c_labelNamesMod_categoryNames_d[i].f1.data[b_j1] =
            rv.data[i].f1.data[b_j1];
      }
    }
    n = valueset.size[1];
    i = valueset.size[1];
    SD->f0.y_data[0] = 1;
    j2 = 1;
    for (k = 0; k <= n - 2; k++) {
      j2++;
      SD->f0.y_data[1] = static_cast<int8_T>(j2);
    }
    for (k = 0; k < i; k++) {
      SD->f0.idx_data[k] = SD->f0.y_data[k];
    }
    introsort(SD->f0.idx_data, valueset.size[1], rv.data);
    SD->f0.c.size[0] = valueset.size[1];
    j2 = valueset.size[1];
    for (i = 0; i < j2; i++) {
      SD->f0.c.data[i].f1.size[0] = 1;
      SD->f0.c.data[i].f1.size[1] = rv.data[SD->f0.idx_data[i] - 1].f1.size[1];
      n = rv.data[SD->f0.idx_data[i] - 1].f1.size[1];
      for (k = 0; k < n; k++) {
        SD->f0.c.data[i].f1.data[k] =
            rv.data[SD->f0.idx_data[i] - 1].f1.data[k];
      }
    }
    for (i = 0; i <= SD->f0.c.size[0] - 2; i++) {
      b_bool = false;
      b = (SD->f0.c.data[0].f1.size[1] == 0);
      if (b && (SD->f0.c.data[1].f1.size[1] == 0)) {
        b_bool = true;
      } else if (SD->f0.c.data[0].f1.size[1] == SD->f0.c.data[1].f1.size[1]) {
        j2 = 0;
        do {
          exitg1 = 0;
          if (j2 + 1 <= SD->f0.c.data[1].f1.size[1]) {
            if (SD->f0.c.data[0].f1.data[j2] != SD->f0.c.data[1].f1.data[j2]) {
              exitg1 = 1;
            } else {
              j2++;
            }
          } else {
            b_bool = true;
            exitg1 = 1;
          }
        } while (exitg1 == 0);
      }
      SD->f0.d_data[0] = !b_bool;
    }
    i = valueset.size[1];
    d_data[0] = true;
    n = valueset.size[1] - 1;
    for (k = 0; k < n; k++) {
      d_data[1] = SD->f0.d_data[0];
    }
    emxInit_real_T(&b_cpu_y, 1, &cb_emlrtRTEI, true);
    b_bool = false;
    k = b_cpu_y->size[0];
    b_cpu_y->size[0] = valueset.size[1];
    emxEnsureCapacity_real_T(b_cpu_y, k, &bb_emlrtRTEI);
    for (k = 0; k < i; k++) {
      b_cpu_y->data[k] = d_data[k];
      b_bool = true;
    }
    if (b_cpu_y->size[0] != 1) {
      gpuEmxEnsureCapacity_real_T(b_cpu_y, &b_gpu_y, !b_bool);
      if (b_bool) {
        gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_y, b_cpu_y);
      }
      mwCallThrustScan1D(&b_gpu_y.data[0], false, 2);
    }
    emxFree_real_T(&b_cpu_y);
  } else {
    c_labelNamesMod_categoryNames_s[0] = 0;
  }
  labelNamesMod_codes_size[0] = 1;
  labelNamesMod_codes_size[1] = 0;
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_y);
  gpuEmxFree_real_T(&b_gpu_y);
}

void c_returnCategoricalLabels(pointpillardetectStackData *SD, real_T numBBoxes,
                               const real32_T labels_data[],
                               uint32_T labelNamesMod_codes_data[],
                               int32_T labelNamesMod_codes_size[2],
                               cell_wrap_6 c_labelNamesMod_categoryNames_d[],
                               int32_T c_labelNamesMod_categoryNames_s[1])
{
  emxArray_cell_wrap_6_1x2 rv;
  emxArray_cell_wrap_6_1x2 valueset;
  emxArray_cell_wrap_6_2 uB;
  emxArray_real_T b_gpu_y;
  emxArray_real_T c_gpu_y;
  emxArray_real_T gpu_y;
  emxArray_real_T *b_cpu_y;
  emxArray_real_T *c_cpu_y;
  emxArray_real_T *cpu_y;
  int32_T b_idx_size[1];
  int32_T b_rv[1];
  int32_T idx_size[1];
  int32_T b_i;
  int32_T b_j1;
  int32_T c_i;
  int32_T d_size_idx_0;
  int32_T exitg1;
  int32_T i;
  int32_T ii_size_idx_0;
  int32_T j2;
  int32_T k;
  int32_T n;
  int32_T y_size_idx_1;
  char_T v_data[5];
  boolean_T d_data[2];
  boolean_T b;
  boolean_T b_bool;
  boolean_T exitg2;
  boolean_T y_outdatedOnCpu;
  gpuEmxReset_real_T(&b_gpu_y);
  gpuEmxReset_real_T(&gpu_y);
  gpuEmxReset_real_T(&c_gpu_y);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  i = static_cast<int32_T>(numBBoxes);
  for (b_i = 0; b_i < i; b_i++) {
    real32_T f;
    n = 0;
    f = labels_data[b_i];
    for (k = 0; k < 5; k++) {
      if (cv[(static_cast<int32_T>(f) + (k << 1)) - 1] != 0) {
        n++;
      }
    }
    c_i = -1;
    for (k = 0; k < 5; k++) {
      if (cv[(static_cast<int32_T>(f) + (k << 1)) - 1] != 0) {
        c_i++;
        v_data[c_i] =
            cv[(static_cast<int32_T>(labels_data[b_i]) + (k << 1)) - 1];
      }
    }
    SD->f2.labelCells.data[b_i].f1.size[0] = 1;
    SD->f2.labelCells.data[b_i].f1.size[1] = n;
    for (k = 0; k < n; k++) {
      SD->f2.labelCells.data[b_i].f1.data[k] = v_data[k];
    }
  }
  valueset.size[1] = 0;
  for (i = 0; i < 2; i++) {
    n = 0;
    for (k = 0; k < 5; k++) {
      if (cv[i + (k << 1)] != 0) {
        n++;
      }
    }
    b_i = -1;
    for (k = 0; k < 5; k++) {
      char_T c;
      c = cv[i + (k << 1)];
      if (c != 0) {
        b_i++;
        v_data[b_i] = c;
      }
    }
    j2 = valueset.size[1] + 1;
    valueset.size[1]++;
    valueset.data[j2 - 1].f1.size[0] = 1;
    valueset.data[valueset.size[1] - 1].f1.size[1] = n;
    for (k = 0; k < n; k++) {
      valueset.data[valueset.size[1] - 1].f1.data[k] = v_data[k];
    }
  }
  c_i = static_cast<int32_T>(numBBoxes);
  for (i = 0; i < c_i; i++) {
    b_j1 = 0;
    while ((b_j1 + 1 <= SD->f2.labelCells.data[i].f1.size[1]) &&
           bv[static_cast<int32_T>(SD->f2.labelCells.data[i].f1.data[b_j1])] &&
           (SD->f2.labelCells.data[i].f1.data[b_j1] != '\x00')) {
      b_j1++;
    }
    j2 = SD->f2.labelCells.data[i].f1.size[1] - 1;
    while ((j2 + 1 > 0) &&
           bv[static_cast<int32_T>(SD->f2.labelCells.data[i].f1.data[j2])] &&
           (SD->f2.labelCells.data[i].f1.data[j2] != '\x00')) {
      j2--;
    }
    if (b_j1 + 1 > j2 + 1) {
      b_j1 = 0;
      j2 = -1;
    }
    SD->f2.inData.data[i].f1.size[0] = 1;
    SD->f2.inData.data[i].f1.size[1] = (j2 - b_j1) + 1;
    n = j2 - b_j1;
    for (k = 0; k <= n; k++) {
      SD->f2.inData.data[i].f1.data[k] =
          SD->f2.labelCells.data[i].f1.data[b_j1 + k];
    }
  }
  c_i = valueset.size[1];
  for (i = 0; i < c_i; i++) {
    b_j1 = 0;
    while ((b_j1 + 1 <= valueset.data[i].f1.size[1]) &&
           bv[static_cast<uint8_T>(valueset.data[i].f1.data[b_j1]) & 127] &&
           (valueset.data[i].f1.data[b_j1] != '\x00')) {
      b_j1++;
    }
    j2 = valueset.data[i].f1.size[1] - 1;
    while ((j2 + 1 > 0) &&
           bv[static_cast<uint8_T>(valueset.data[i].f1.data[j2]) & 127] &&
           (valueset.data[i].f1.data[j2] != '\x00')) {
      j2--;
    }
    if (b_j1 + 1 > j2 + 1) {
      b_j1 = 0;
      j2 = -1;
    }
    rv.data[i].f1.size[0] = 1;
    rv.data[i].f1.size[1] = (j2 - b_j1) + 1;
    n = j2 - b_j1;
    for (k = 0; k <= n; k++) {
      rv.data[i].f1.data[k] = valueset.data[i].f1.data[b_j1 + k];
    }
  }
  if (valueset.size[1] != 0) {
    n = valueset.size[1];
    y_size_idx_1 = valueset.size[1];
    SD->f2.b_ii_data[0] = 1;
    b_j1 = 1;
    for (k = 0; k <= n - 2; k++) {
      b_j1++;
      SD->f2.b_ii_data[1] = static_cast<int8_T>(b_j1);
    }
    for (k = 0; k < y_size_idx_1; k++) {
      SD->f2.idx_data[k] = SD->f2.b_ii_data[k];
    }
    introsort(SD->f2.idx_data, valueset.size[1], rv.data);
    SD->f2.labelCells.size[0] = valueset.size[1];
    j2 = valueset.size[1];
    for (i = 0; i < j2; i++) {
      SD->f2.labelCells.data[i].f1.size[0] = 1;
      SD->f2.labelCells.data[i].f1.size[1] =
          rv.data[SD->f2.idx_data[i] - 1].f1.size[1];
      n = rv.data[SD->f2.idx_data[i] - 1].f1.size[1];
      for (k = 0; k < n; k++) {
        SD->f2.labelCells.data[i].f1.data[k] =
            rv.data[SD->f2.idx_data[i] - 1].f1.data[k];
      }
    }
    for (i = 0; i <= SD->f2.labelCells.size[0] - 2; i++) {
      b_bool = false;
      b = (SD->f2.labelCells.data[0].f1.size[1] == 0);
      if (b && (SD->f2.labelCells.data[1].f1.size[1] == 0)) {
        b_bool = true;
      } else if (SD->f2.labelCells.data[0].f1.size[1] ==
                 SD->f2.labelCells.data[1].f1.size[1]) {
        b_j1 = 0;
        do {
          exitg1 = 0;
          if (b_j1 + 1 <= SD->f2.labelCells.data[1].f1.size[1]) {
            if (SD->f2.labelCells.data[0].f1.data[b_j1] !=
                SD->f2.labelCells.data[1].f1.data[b_j1]) {
              exitg1 = 1;
            } else {
              b_j1++;
            }
          } else {
            b_bool = true;
            exitg1 = 1;
          }
        } while (exitg1 == 0);
      }
      SD->f2.b_d_data[0] = !b_bool;
    }
    d_size_idx_0 = valueset.size[1];
    d_data[0] = true;
    n = valueset.size[1] - 1;
    for (k = 0; k < n; k++) {
      d_data[1] = SD->f2.b_d_data[0];
    }
    emxInit_real_T(&b_cpu_y, 1, &cb_emlrtRTEI, true);
    b_bool = false;
    k = b_cpu_y->size[0];
    b_cpu_y->size[0] = valueset.size[1];
    emxEnsureCapacity_real_T(b_cpu_y, k, &bb_emlrtRTEI);
    for (k = 0; k < d_size_idx_0; k++) {
      b_cpu_y->data[k] = d_data[k];
      b_bool = true;
    }
    if (b_cpu_y->size[0] != 1) {
      gpuEmxEnsureCapacity_real_T(b_cpu_y, &gpu_y, !b_bool);
      if (b_bool) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_y, b_cpu_y);
      }
      mwCallThrustScan1D(&gpu_y.data[0], false, 2);
    }
    emxFree_real_T(&b_cpu_y);
  }
  if (valueset.size[1] != 0) {
    int8_T iv3_idx_0;
    iv3_idx_0 = static_cast<int8_T>(valueset.size[1]);
    c_labelNamesMod_categoryNames_s[0] = valueset.size[1];
    for (j2 = 0; j2 < iv3_idx_0; j2++) {
      c_labelNamesMod_categoryNames_d[j2].f1.size[0] = 1;
      c_labelNamesMod_categoryNames_d[j2].f1.size[1] = 0;
    }
    b_j1 = valueset.size[1];
    c_labelNamesMod_categoryNames_s[0] = valueset.size[1];
    for (i = 0; i < b_j1; i++) {
      c_labelNamesMod_categoryNames_d[i].f1.size[0] = 1;
      n = rv.data[i].f1.size[1];
      c_labelNamesMod_categoryNames_d[i].f1.size[1] = rv.data[i].f1.size[1];
      for (c_i = 0; c_i < n; c_i++) {
        c_labelNamesMod_categoryNames_d[i].f1.data[c_i] =
            rv.data[i].f1.data[c_i];
      }
    }
  } else {
    c_labelNamesMod_categoryNames_s[0] = 0;
  }
  emxInit_real_T(&cpu_y, 1, &cb_emlrtRTEI, true);
  if (static_cast<int32_T>(numBBoxes) == 0) {
    SD->f2.uA.size[0] = 0;
    y_outdatedOnCpu = false;
    cpu_y->size[0] = 0;
  } else {
    n = static_cast<int32_T>(numBBoxes);
    y_size_idx_1 = static_cast<int32_T>(numBBoxes);
    SD->f2.locbOfUA_data[0] = 1;
    b_j1 = 1;
    for (k = 0; k <= n - 2; k++) {
      b_j1++;
      SD->f2.locbOfUA_data[k + 1] = b_j1;
    }
    idx_size[0] = static_cast<int32_T>(numBBoxes);
    for (k = 0; k < y_size_idx_1; k++) {
      SD->f2.idx_data[k] = SD->f2.locbOfUA_data[k];
    }
    introsort(SD->f2.idx_data, static_cast<int32_T>(numBBoxes),
              SD->f2.inData.data);
    SD->f2.labelCells.size[0] = static_cast<int32_T>(numBBoxes);
    b_j1 = static_cast<int32_T>(numBBoxes);
    for (i = 0; i < b_j1; i++) {
      SD->f2.labelCells.data[i].f1.size[0] = 1;
      SD->f2.labelCells.data[i].f1.size[1] =
          SD->f2.inData.data[SD->f2.idx_data[i] - 1].f1.size[1];
      n = SD->f2.inData.data[SD->f2.idx_data[i] - 1].f1.size[1];
      for (k = 0; k < n; k++) {
        SD->f2.labelCells.data[i].f1.data[k] =
            SD->f2.inData.data[SD->f2.idx_data[i] - 1].f1.data[k];
      }
    }
    d_size_idx_0 = static_cast<int32_T>(numBBoxes) - 1;
    for (i = 0; i <= SD->f2.labelCells.size[0] - 2; i++) {
      SD->f2.b_d_data[i] = false;
      b_bool = false;
      b = (SD->f2.labelCells.data[i].f1.size[1] == 0);
      if (b && (SD->f2.labelCells.data[i + 1].f1.size[1] == 0)) {
        b_bool = true;
      } else if (SD->f2.labelCells.data[i].f1.size[1] ==
                 SD->f2.labelCells.data[i + 1].f1.size[1]) {
        b_j1 = 0;
        do {
          exitg1 = 0;
          if (b_j1 + 1 <= SD->f2.labelCells.data[i + 1].f1.size[1]) {
            if (SD->f2.labelCells.data[i].f1.data[b_j1] !=
                SD->f2.labelCells.data[i + 1].f1.data[b_j1]) {
              exitg1 = 1;
            } else {
              b_j1++;
            }
          } else {
            b_bool = true;
            exitg1 = 1;
          }
        } while (exitg1 == 0);
      }
      SD->f2.b_d_data[i] = !b_bool;
    }
    n = static_cast<int32_T>(numBBoxes);
    SD->f2.d_data[0] = true;
    for (k = 0; k < d_size_idx_0; k++) {
      SD->f2.d_data[k + 1] = SD->f2.b_d_data[k];
    }
    j2 = SD->f2.d_data[0];
    for (k = 0; k <= n - 2; k++) {
      j2 += SD->f2.d_data[k + 1];
    }
    SD->f2.uA.size[0] = static_cast<int32_T>(
        std::fmin(static_cast<real_T>(j2),
                  static_cast<real_T>(static_cast<int32_T>(numBBoxes))));
    c_i = 0;
    ii_size_idx_0 = static_cast<int32_T>(numBBoxes);
    j2 = 1;
    exitg2 = false;
    while ((!exitg2) && (j2 <= static_cast<int32_T>(numBBoxes))) {
      if (SD->f2.d_data[j2 - 1]) {
        c_i++;
        SD->f2.ii_data[c_i - 1] = j2;
        if (c_i >= static_cast<int32_T>(numBBoxes)) {
          exitg2 = true;
        } else {
          j2++;
        }
      } else {
        j2++;
      }
    }
    if (static_cast<int32_T>(numBBoxes) == 1) {
      if (c_i == 0) {
        ii_size_idx_0 = 0;
      }
    } else if (c_i < 1) {
      ii_size_idx_0 = 0;
    } else {
      ii_size_idx_0 = c_i;
    }
    for (i = 0; i < ii_size_idx_0; i++) {
      if (i + 1 != ii_size_idx_0) {
        if (SD->f2.ii_data[i + 1] - 1 < SD->f2.ii_data[i]) {
          y_size_idx_1 = 0;
        } else {
          k = SD->f2.ii_data[i];
          c_i = SD->f2.ii_data[i + 1];
          y_size_idx_1 = SD->f2.ii_data[i + 1] - SD->f2.ii_data[i];
          for (j2 = 0; j2 < c_i - k; j2++) {
            SD->f2.locbOfUA_data[j2] = k + j2;
          }
        }
        b_j1 = SD->f2.idx_data[SD->f2.locbOfUA_data[0] - 1];
        for (b_i = 0; b_i <= y_size_idx_1 - 2; b_i++) {
          k = SD->f2.locbOfUA_data[b_i + 1];
          if (b_j1 > SD->f2.idx_data[k - 1]) {
            b_j1 = SD->f2.idx_data[k - 1];
          }
        }
        SD->f2.a__2_data[i] = b_j1;
      } else {
        if (static_cast<int32_T>(numBBoxes) < SD->f2.ii_data[i]) {
          y_size_idx_1 = 0;
        } else {
          k = SD->f2.ii_data[i];
          y_size_idx_1 =
              (static_cast<int32_T>(numBBoxes) - SD->f2.ii_data[i]) + 1;
          for (c_i = 0; c_i <= idx_size[0] - k; c_i++) {
            SD->f2.locbOfUA_data[c_i] = k + c_i;
          }
        }
        b_j1 = SD->f2.idx_data[SD->f2.locbOfUA_data[0] - 1];
        for (b_i = 0; b_i <= y_size_idx_1 - 2; b_i++) {
          k = SD->f2.locbOfUA_data[b_i + 1];
          if (b_j1 > SD->f2.idx_data[k - 1]) {
            b_j1 = SD->f2.idx_data[k - 1];
          }
        }
        SD->f2.a__2_data[i] = b_j1;
      }
    }
    y_outdatedOnCpu = false;
    b_bool = false;
    k = cpu_y->size[0];
    cpu_y->size[0] = static_cast<int32_T>(numBBoxes);
    emxEnsureCapacity_real_T(cpu_y, k, &bb_emlrtRTEI);
    for (k = 0; k < n; k++) {
      cpu_y->data[k] = SD->f2.d_data[k];
      b_bool = true;
    }
    if (cpu_y->size[0] != 1) {
      gpuEmxEnsureCapacity_real_T(cpu_y, &c_gpu_y, !b_bool);
      if (b_bool) {
        gpuEmxMemcpyCpuToGpu_real_T(&c_gpu_y, cpu_y);
      }
      mwCallThrustScan1D(&c_gpu_y.data[0], false, cpu_y->size[0]);
      y_outdatedOnCpu = true;
    }
    n = cpu_y->size[0];
    for (k = 0; k < cpu_y->size[0]; k++) {
      if (y_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &c_gpu_y);
      }
      y_outdatedOnCpu = false;
      SD->f2.y_data[k] = cpu_y->data[k];
    }
    for (k = 0; k < n; k++) {
      if (y_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &c_gpu_y);
      }
      cpu_y->data[SD->f2.idx_data[k] - 1] = SD->f2.y_data[k];
      y_outdatedOnCpu = false;
    }
    for (i = 0; i < ii_size_idx_0; i++) {
      SD->f2.uA.data[i].f1.size[0] = 1;
      SD->f2.uA.data[i].f1.size[1] =
          SD->f2.inData.data[SD->f2.a__2_data[i] - 1].f1.size[1];
      n = SD->f2.inData.data[SD->f2.a__2_data[i] - 1].f1.size[1];
      for (k = 0; k < n; k++) {
        SD->f2.uA.data[i].f1.data[k] =
            SD->f2.inData.data[SD->f2.a__2_data[i] - 1].f1.data[k];
      }
    }
  }
  if (valueset.size[1] == 0) {
    uB.size[0] = 0;
  } else {
    n = valueset.size[1];
    y_size_idx_1 = valueset.size[1];
    SD->f2.b_ii_data[0] = 1;
    b_j1 = 1;
    for (k = 0; k <= n - 2; k++) {
      b_j1++;
      SD->f2.b_ii_data[1] = static_cast<int8_T>(b_j1);
    }
    b_idx_size[0] = valueset.size[1];
    for (k = 0; k < y_size_idx_1; k++) {
      SD->f2.idx_data[k] = SD->f2.b_ii_data[k];
    }
    introsort(SD->f2.idx_data, valueset.size[1], rv.data);
    SD->f2.labelCells.size[0] = valueset.size[1];
    j2 = valueset.size[1];
    for (i = 0; i < j2; i++) {
      SD->f2.labelCells.data[i].f1.size[0] = 1;
      SD->f2.labelCells.data[i].f1.size[1] =
          rv.data[SD->f2.idx_data[i] - 1].f1.size[1];
      n = rv.data[SD->f2.idx_data[i] - 1].f1.size[1];
      for (k = 0; k < n; k++) {
        SD->f2.labelCells.data[i].f1.data[k] =
            rv.data[SD->f2.idx_data[i] - 1].f1.data[k];
      }
    }
    for (i = 0; i <= SD->f2.labelCells.size[0] - 2; i++) {
      b_bool = false;
      b = (SD->f2.labelCells.data[0].f1.size[1] == 0);
      if (b && (SD->f2.labelCells.data[1].f1.size[1] == 0)) {
        b_bool = true;
      } else if (SD->f2.labelCells.data[0].f1.size[1] ==
                 SD->f2.labelCells.data[1].f1.size[1]) {
        b_j1 = 0;
        do {
          exitg1 = 0;
          if (b_j1 + 1 <= SD->f2.labelCells.data[1].f1.size[1]) {
            if (SD->f2.labelCells.data[0].f1.data[b_j1] !=
                SD->f2.labelCells.data[1].f1.data[b_j1]) {
              exitg1 = 1;
            } else {
              b_j1++;
            }
          } else {
            b_bool = true;
            exitg1 = 1;
          }
        } while (exitg1 == 0);
      }
      SD->f2.b_d_data[0] = !b_bool;
    }
    d_size_idx_0 = valueset.size[1];
    d_data[0] = true;
    n = valueset.size[1] - 1;
    for (k = 0; k < n; k++) {
      d_data[1] = SD->f2.b_d_data[0];
    }
    c_i = valueset.size[1];
    j2 = 1;
    for (k = 0; k <= c_i - 2; k++) {
      j2 += d_data[1];
    }
    uB.size[0] = static_cast<int32_T>(std::fmin(
        static_cast<real_T>(j2), static_cast<real_T>(valueset.size[1])));
    b_j1 = valueset.size[1];
    c_i = 0;
    ii_size_idx_0 = valueset.size[1];
    j2 = 1;
    exitg2 = false;
    while ((!exitg2) && (j2 <= b_j1)) {
      if (d_data[j2 - 1]) {
        c_i++;
        SD->f2.b_ii_data[c_i - 1] = static_cast<int8_T>(j2);
        if (c_i >= b_j1) {
          exitg2 = true;
        } else {
          j2++;
        }
      } else {
        j2++;
      }
    }
    if (valueset.size[1] == 1) {
      if (c_i == 0) {
        ii_size_idx_0 = 0;
      }
    } else if (c_i < 1) {
      ii_size_idx_0 = 0;
    } else {
      ii_size_idx_0 = c_i;
    }
    for (i = 0; i < ii_size_idx_0; i++) {
      if (i + 1 != ii_size_idx_0) {
        y_size_idx_1 = (SD->f2.b_ii_data[i + 1] - 1 >= SD->f2.b_ii_data[i]);
        for (k = 0; k < y_size_idx_1; k++) {
          SD->f2.locbOfUA_data[0] = SD->f2.idx_data[0];
        }
        SD->f2.ib_data[i] = SD->f2.locbOfUA_data[0];
      } else {
        if (b_idx_size[0] < SD->f2.b_ii_data[i]) {
          y_size_idx_1 = 0;
        } else {
          k = SD->f2.b_ii_data[i];
          y_size_idx_1 = (b_idx_size[0] - SD->f2.b_ii_data[i]) + 1;
          for (c_i = 0; c_i <= b_idx_size[0] - k; c_i++) {
            SD->f2.locbOfUA_data[c_i] = k + c_i;
          }
        }
        b_j1 = SD->f2.idx_data[SD->f2.locbOfUA_data[0] - 1];
        for (b_i = 0; b_i <= y_size_idx_1 - 2; b_i++) {
          if (b_j1 > SD->f2.idx_data[SD->f2.locbOfUA_data[1] - 1]) {
            b_j1 = SD->f2.idx_data[SD->f2.locbOfUA_data[1] - 1];
          }
        }
        SD->f2.ib_data[i] = b_j1;
      }
    }
    emxInit_real_T(&c_cpu_y, 1, &cb_emlrtRTEI, true);
    b_bool = false;
    k = c_cpu_y->size[0];
    c_cpu_y->size[0] = valueset.size[1];
    emxEnsureCapacity_real_T(c_cpu_y, k, &bb_emlrtRTEI);
    for (k = 0; k < d_size_idx_0; k++) {
      c_cpu_y->data[k] = d_data[k];
      b_bool = true;
    }
    if (c_cpu_y->size[0] != 1) {
      gpuEmxEnsureCapacity_real_T(c_cpu_y, &b_gpu_y, !b_bool);
      if (b_bool) {
        gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_y, c_cpu_y);
      }
      mwCallThrustScan1D(&b_gpu_y.data[0], false, 2);
    }
    emxFree_real_T(&c_cpu_y);
    for (i = 0; i < ii_size_idx_0; i++) {
      uB.data[i].f1.size[0] = 1;
      uB.data[i].f1.size[1] = rv.data[SD->f2.ib_data[i] - 1].f1.size[1];
      n = rv.data[SD->f2.ib_data[i] - 1].f1.size[1];
      for (k = 0; k < n; k++) {
        uB.data[i].f1.data[k] = rv.data[SD->f2.ib_data[i] - 1].f1.data[k];
      }
    }
  }
  for (k = 0; k < SD->f2.uA.size[0]; k++) {
    SD->f2.locbOfUA_data[k] = 0;
  }
  if (uB.size[0] > 0) {
    j2 = 0;
    i = 0;
    exitg2 = false;
    while ((!exitg2) && (i <= SD->f2.uA.size[0] - 1)) {
      c_i = SD->f2.uA.data[i].f1.size[1];
      b_j1 = uB.data[j2].f1.size[1];
      if (c_i <= b_j1) {
        b_j1 = c_i;
      }
      if (b_j1 == 0) {
        b = (SD->f2.uA.data[i].f1.size[1] < uB.data[j2].f1.size[1]);
      } else {
        b_i = 0;
        while ((b_i + 1 <= b_j1) &&
               (SD->f2.uA.data[i].f1.data[b_i] == uB.data[j2].f1.data[b_i])) {
          b_i++;
        }
        if (b_i + 1 == b_j1 + 1) {
          b = (SD->f2.uA.data[i].f1.size[1] < uB.data[j2].f1.size[1]);
        } else {
          b = (SD->f2.uA.data[i].f1.data[b_i] < uB.data[j2].f1.data[b_i]);
        }
      }
      if (!b) {
        b_bool = false;
        b = (SD->f2.uA.data[i].f1.size[1] == 0);
        if (b && (uB.data[j2].f1.size[1] == 0)) {
          b_bool = true;
        } else if (SD->f2.uA.data[i].f1.size[1] == uB.data[j2].f1.size[1]) {
          b_j1 = 0;
          do {
            exitg1 = 0;
            if (b_j1 + 1 <= uB.data[j2].f1.size[1]) {
              if (SD->f2.uA.data[i].f1.data[b_j1] !=
                  uB.data[j2].f1.data[b_j1]) {
                exitg1 = 1;
              } else {
                b_j1++;
              }
            } else {
              b_bool = true;
              exitg1 = 1;
            }
          } while (exitg1 == 0);
        }
        if (b_bool) {
          SD->f2.locbOfUA_data[i] = SD->f2.ib_data[j2];
          j2++;
        } else {
          boolean_T exitg3;
          exitg3 = false;
          while ((!exitg3) && (j2 + 1 <= uB.size[0])) {
            c_i = SD->f2.uA.data[i].f1.size[1];
            b_j1 = uB.data[j2].f1.size[1];
            if (c_i <= b_j1) {
              b_j1 = c_i;
            }
            if (b_j1 == 0) {
              b = (SD->f2.uA.data[i].f1.size[1] > uB.data[j2].f1.size[1]);
            } else {
              b_i = 0;
              while ((b_i + 1 <= b_j1) && (SD->f2.uA.data[i].f1.data[b_i] ==
                                           uB.data[j2].f1.data[b_i])) {
                b_i++;
              }
              if (b_i + 1 == b_j1 + 1) {
                b = (SD->f2.uA.data[i].f1.size[1] > uB.data[j2].f1.size[1]);
              } else {
                b = (SD->f2.uA.data[i].f1.data[b_i] > uB.data[j2].f1.data[b_i]);
              }
            }
            if (b) {
              j2++;
            } else {
              exitg3 = true;
            }
          }
          if (j2 + 1 <= uB.size[0]) {
            b = (SD->f2.uA.data[i].f1.size[1] == 0);
            if (b && (uB.data[j2].f1.size[1] == 0)) {
              b_bool = true;
            } else if (SD->f2.uA.data[i].f1.size[1] == uB.data[j2].f1.size[1]) {
              b_j1 = 0;
              do {
                exitg1 = 0;
                if (b_j1 + 1 <= uB.data[j2].f1.size[1]) {
                  if (SD->f2.uA.data[i].f1.data[b_j1] !=
                      uB.data[j2].f1.data[b_j1]) {
                    exitg1 = 1;
                  } else {
                    b_j1++;
                  }
                } else {
                  b_bool = true;
                  exitg1 = 1;
                }
              } while (exitg1 == 0);
            }
            if (b_bool) {
              SD->f2.locbOfUA_data[i] = SD->f2.ib_data[j2];
              j2++;
            }
          }
        }
      }
      if (j2 + 1 > uB.size[0]) {
        exitg2 = true;
      } else {
        i++;
      }
    }
  }
  b_rv[0] = cpu_y->size[0];
  labelNamesMod_codes_size[0] = 1;
  labelNamesMod_codes_size[1] = cpu_y->size[0];
  for (k = 0; k < b_rv[0]; k++) {
    if (y_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &c_gpu_y);
    }
    y_outdatedOnCpu = false;
    c_i = SD->f2.locbOfUA_data[static_cast<int32_T>(cpu_y->data[k]) - 1];
    if (c_i < 0) {
      c_i = 0;
    }
    labelNamesMod_codes_data[k] = static_cast<uint32_T>(c_i);
  }
  emxFree_real_T(&cpu_y);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&c_gpu_y);
  gpuEmxFree_real_T(&gpu_y);
  gpuEmxFree_real_T(&b_gpu_y);
}

} // namespace internal
} // namespace coder

// End of code generation (pointPillarsObjectDetector.cu)
