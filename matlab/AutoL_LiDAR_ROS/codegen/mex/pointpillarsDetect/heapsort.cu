#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// heapsort.cu
//
// Code generation for function 'heapsort'
//

// Include files
#include "heapsort.h"
#include "pointpillarsDetect_types.h"
#include "rt_nonfinite.h"
#include <cmath>

// Function Definitions
namespace coder {
namespace internal {
void b_heapsort(int32_T x_data[], int32_T xstart, int32_T xend,
                const cell_wrap_6 cmp_workspace_c_data[])
{
  int32_T b_n;
  int32_T cmpIdx;
  int32_T exitg2;
  int32_T extremum;
  int32_T extremumIdx;
  int32_T k;
  int32_T leftIdx;
  int32_T n;
  int32_T t;
  int32_T xcmp;
  int32_T xr;
  boolean_T changed;
  boolean_T exitg1;
  boolean_T varargout_1;
  n = (xend - xstart) - 1;
  for (t = 0; t <= n + 1; t++) {
    changed = true;
    extremumIdx = (n - t) + xstart;
    leftIdx = ((((n - t) + 2) << 1) + xstart) - 2;
    exitg1 = false;
    while ((!exitg1) && (leftIdx + 1 < xend)) {
      changed = false;
      extremum = x_data[extremumIdx];
      cmpIdx = leftIdx;
      xcmp = x_data[leftIdx] - 1;
      xr = x_data[leftIdx + 1];
      b_n = static_cast<int32_T>(std::fmin(
          static_cast<real_T>(
              cmp_workspace_c_data[x_data[leftIdx] - 1].f1.size[1]),
          static_cast<real_T>(
              cmp_workspace_c_data[x_data[leftIdx + 1] - 1].f1.size[1])));
      varargout_1 = (cmp_workspace_c_data[x_data[leftIdx] - 1].f1.size[1] <
                     cmp_workspace_c_data[x_data[leftIdx + 1] - 1].f1.size[1]);
      k = 0;
      do {
        exitg2 = 0;
        if (k <= b_n - 1) {
          if (cmp_workspace_c_data[x_data[leftIdx] - 1].f1.data[k] !=
              cmp_workspace_c_data[x_data[leftIdx + 1] - 1].f1.data[k]) {
            varargout_1 =
                (cmp_workspace_c_data[x_data[leftIdx] - 1].f1.data[k] <
                 cmp_workspace_c_data[x_data[leftIdx + 1] - 1].f1.data[k]);
            exitg2 = 1;
          } else {
            k++;
          }
        } else {
          if (cmp_workspace_c_data[xcmp].f1.size[1] ==
              cmp_workspace_c_data[xr - 1].f1.size[1]) {
            varargout_1 = (xcmp + 1 < xr);
          }
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      if (varargout_1) {
        cmpIdx = leftIdx + 1;
        xcmp = x_data[leftIdx + 1] - 1;
      }
      b_n = static_cast<int32_T>(std::fmin(
          static_cast<real_T>(
              cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.size[1]),
          static_cast<real_T>(cmp_workspace_c_data[xcmp].f1.size[1])));
      varargout_1 = (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.size[1] <
                     cmp_workspace_c_data[xcmp].f1.size[1]);
      k = 0;
      do {
        exitg2 = 0;
        if (k <= b_n - 1) {
          if (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.data[k] !=
              cmp_workspace_c_data[xcmp].f1.data[k]) {
            varargout_1 =
                (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.data[k] <
                 cmp_workspace_c_data[xcmp].f1.data[k]);
            exitg2 = 1;
          } else {
            k++;
          }
        } else {
          if (cmp_workspace_c_data[extremum - 1].f1.size[1] ==
              cmp_workspace_c_data[xcmp].f1.size[1]) {
            varargout_1 = (extremum < xcmp + 1);
          }
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      if (varargout_1) {
        x_data[extremumIdx] = xcmp + 1;
        x_data[cmpIdx] = extremum;
        extremumIdx = cmpIdx;
        leftIdx = ((((cmpIdx - xstart) + 2) << 1) + xstart) - 2;
        changed = true;
      } else {
        exitg1 = true;
      }
    }
    if (changed && (leftIdx + 1 <= xend)) {
      extremum = x_data[extremumIdx];
      xcmp = x_data[leftIdx];
      b_n = static_cast<int32_T>(std::fmin(
          static_cast<real_T>(
              cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.size[1]),
          static_cast<real_T>(
              cmp_workspace_c_data[x_data[leftIdx] - 1].f1.size[1])));
      varargout_1 = (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.size[1] <
                     cmp_workspace_c_data[x_data[leftIdx] - 1].f1.size[1]);
      k = 0;
      do {
        exitg2 = 0;
        if (k <= b_n - 1) {
          if (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.data[k] !=
              cmp_workspace_c_data[x_data[leftIdx] - 1].f1.data[k]) {
            varargout_1 =
                (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.data[k] <
                 cmp_workspace_c_data[x_data[leftIdx] - 1].f1.data[k]);
            exitg2 = 1;
          } else {
            k++;
          }
        } else {
          if (cmp_workspace_c_data[extremum - 1].f1.size[1] ==
              cmp_workspace_c_data[xcmp - 1].f1.size[1]) {
            varargout_1 = (extremum < xcmp);
          }
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      if (varargout_1) {
        x_data[extremumIdx] = x_data[leftIdx];
        x_data[leftIdx] = extremum;
      }
    }
  }
  for (k = 0; k <= n; k++) {
    t = x_data[(xend - k) - 1];
    x_data[(xend - k) - 1] = x_data[xstart - 1];
    x_data[xstart - 1] = t;
    changed = true;
    extremumIdx = xstart - 1;
    leftIdx = xstart;
    exitg1 = false;
    while ((!exitg1) && (leftIdx + 1 < (xend - k) - 1)) {
      changed = false;
      extremum = x_data[extremumIdx];
      cmpIdx = leftIdx;
      xcmp = x_data[leftIdx] - 1;
      xr = x_data[leftIdx + 1];
      b_n = static_cast<int32_T>(std::fmin(
          static_cast<real_T>(
              cmp_workspace_c_data[x_data[leftIdx] - 1].f1.size[1]),
          static_cast<real_T>(
              cmp_workspace_c_data[x_data[leftIdx + 1] - 1].f1.size[1])));
      varargout_1 = (cmp_workspace_c_data[x_data[leftIdx] - 1].f1.size[1] <
                     cmp_workspace_c_data[x_data[leftIdx + 1] - 1].f1.size[1]);
      t = 0;
      do {
        exitg2 = 0;
        if (t <= b_n - 1) {
          if (cmp_workspace_c_data[x_data[leftIdx] - 1].f1.data[t] !=
              cmp_workspace_c_data[x_data[leftIdx + 1] - 1].f1.data[t]) {
            varargout_1 =
                (cmp_workspace_c_data[x_data[leftIdx] - 1].f1.data[t] <
                 cmp_workspace_c_data[x_data[leftIdx + 1] - 1].f1.data[t]);
            exitg2 = 1;
          } else {
            t++;
          }
        } else {
          if (cmp_workspace_c_data[xcmp].f1.size[1] ==
              cmp_workspace_c_data[xr - 1].f1.size[1]) {
            varargout_1 = (xcmp + 1 < xr);
          }
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      if (varargout_1) {
        cmpIdx = leftIdx + 1;
        xcmp = x_data[leftIdx + 1] - 1;
      }
      b_n = static_cast<int32_T>(std::fmin(
          static_cast<real_T>(
              cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.size[1]),
          static_cast<real_T>(cmp_workspace_c_data[xcmp].f1.size[1])));
      varargout_1 = (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.size[1] <
                     cmp_workspace_c_data[xcmp].f1.size[1]);
      t = 0;
      do {
        exitg2 = 0;
        if (t <= b_n - 1) {
          if (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.data[t] !=
              cmp_workspace_c_data[xcmp].f1.data[t]) {
            varargout_1 =
                (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.data[t] <
                 cmp_workspace_c_data[xcmp].f1.data[t]);
            exitg2 = 1;
          } else {
            t++;
          }
        } else {
          if (cmp_workspace_c_data[extremum - 1].f1.size[1] ==
              cmp_workspace_c_data[xcmp].f1.size[1]) {
            varargout_1 = (extremum < xcmp + 1);
          }
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      if (varargout_1) {
        x_data[extremumIdx] = xcmp + 1;
        x_data[cmpIdx] = extremum;
        extremumIdx = cmpIdx;
        leftIdx = ((((cmpIdx - xstart) + 2) << 1) + xstart) - 2;
        changed = true;
      } else {
        exitg1 = true;
      }
    }
    if (changed && (leftIdx + 1 <= (xend - k) - 1)) {
      extremum = x_data[extremumIdx];
      xcmp = x_data[leftIdx];
      b_n = static_cast<int32_T>(std::fmin(
          static_cast<real_T>(
              cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.size[1]),
          static_cast<real_T>(
              cmp_workspace_c_data[x_data[leftIdx] - 1].f1.size[1])));
      varargout_1 = (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.size[1] <
                     cmp_workspace_c_data[x_data[leftIdx] - 1].f1.size[1]);
      t = 0;
      do {
        exitg2 = 0;
        if (t <= b_n - 1) {
          if (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.data[t] !=
              cmp_workspace_c_data[x_data[leftIdx] - 1].f1.data[t]) {
            varargout_1 =
                (cmp_workspace_c_data[x_data[extremumIdx] - 1].f1.data[t] <
                 cmp_workspace_c_data[x_data[leftIdx] - 1].f1.data[t]);
            exitg2 = 1;
          } else {
            t++;
          }
        } else {
          if (cmp_workspace_c_data[extremum - 1].f1.size[1] ==
              cmp_workspace_c_data[xcmp - 1].f1.size[1]) {
            varargout_1 = (extremum < xcmp);
          }
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      if (varargout_1) {
        x_data[extremumIdx] = x_data[leftIdx];
        x_data[leftIdx] = extremum;
      }
    }
  }
}

} // namespace internal
} // namespace coder

// End of code generation (heapsort.cu)
