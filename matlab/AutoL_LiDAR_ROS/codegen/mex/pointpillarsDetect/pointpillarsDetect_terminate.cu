//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// pointpillarsDetect_terminate.cu
//
// Code generation for function 'pointpillarsDetect_terminate'
//

// Include files
#include "pointpillarsDetect_terminate.h"
#include "_coder_pointpillarsDetect_mex.h"
#include "pointpillarsDetect.h"
#include "pointpillarsDetect_data.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"

// Function Declarations
static void c_pointpillarsDetect_emx_free_d(const void *r);

static void emlrtExitTimeCleanupDtorFcn(const void *r);

// Function Definitions
static void c_pointpillarsDetect_emx_free_d(const void *r)
{
}

static void emlrtExitTimeCleanupDtorFcn(const void *r)
{
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void pointpillarsDetect_atexit()
{
  mexFunctionCreateRootTLS();
  try {
    emlrtPushHeapReferenceStackR2021a(emlrtRootTLSGlobal, false, nullptr,
                                      (void *)&emlrtExitTimeCleanupDtorFcn,
                                      nullptr, nullptr, nullptr);
    emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
    emlrtPushHeapReferenceStackR2021a(emlrtRootTLSGlobal, false, nullptr,
                                      (void *)&c_pointpillarsDetect_emx_free_d,
                                      nullptr, nullptr, nullptr);
    pointpillarsDetect_delete();
    emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
    emlrtExitTimeCleanup(&emlrtContextGlobal);
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
  mwCudaFree(*global_gpu_dv2);
  mwCudaFree(*global_gpu_xq);
  mwCudaFree(*b_global_gpu_xq);
  mwCudaFree(*c_global_gpu_xq);
  mwCudaFree(*d_global_gpu_xq);
}

void pointpillarsDetect_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (pointpillarsDetect_terminate.cu)
