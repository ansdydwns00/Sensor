//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// pointpillarsDetect_initialize.cu
//
// Code generation for function 'pointpillarsDetect_initialize'
//

// Include files
#include "pointpillarsDetect_initialize.h"
#include "_coder_pointpillarsDetect_mex.h"
#include "pointpillarsDetect.h"
#include "pointpillarsDetect_data.h"
#include "pointpillarsDetect_mexutil.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"

// Function Declarations
static void pointpillarsDetect_once();

// Function Definitions
static void pointpillarsDetect_once()
{
  mex_InitInfAndNan();
  pointpillarsDetect_emx_init();
  pointpillarsDetect_new();
  pointpillarsDetect_init();
  checkCudaError(mwCudaMalloc(&global_gpu_dv2, sizeof(real_T[248])), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&global_gpu_xq, sizeof(real_T[216])), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&b_global_gpu_xq, sizeof(real_T[248])), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&c_global_gpu_xq, sizeof(real_T[216])), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&d_global_gpu_xq, sizeof(real_T[248])), __FILE__,
                 __LINE__);
}

void pointpillarsDetect_initialize()
{
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "neural_network_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "lidar_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "video_and_image_blockset", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    pointpillarsDetect_once();
  }
}

// End of code generation (pointpillarsDetect_initialize.cu)
