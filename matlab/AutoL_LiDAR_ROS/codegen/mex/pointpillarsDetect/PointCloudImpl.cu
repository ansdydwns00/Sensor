#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// PointCloudImpl.cu
//
// Code generation for function 'PointCloudImpl'
//

// Include files
#include "PointCloudImpl.h"
#include "pointpillarsDetect_data.h"
#include "pointpillarsDetect_emxutil.h"
#include "pointpillarsDetect_mexutil.h"
#include "pointpillarsDetect_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"

// Variable Definitions
static emlrtRTEInfo k_emlrtRTEI{
    1,                // lineNo
    1,                // colNo
    "PointCloudImpl", // fName
    "/usr/local/MATLAB/R2024a/toolbox/vision/vision/+vision/+internal/+codegen/"
    "+gpu/PointCloudImpl.p" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "getSubsetPoints", // fName
    "/usr/local/MATLAB/R2024a/toolbox/vision/vision/+vision/+internal/+codegen/"
    "+gpu/getSubsetPoints.p" // pName
};

// Function Declarations
static __global__ void c_PointCloudImpl_extractValidPo(
    const real_T numPoints, const emxArray_real32_T ptCloudCoords,
    const int32_T b, emxArray_boolean_T validCoords);

static __global__ void
c_PointCloudImpl_subsetImpl_ker(const emxArray_real32_T outIntensity,
                                const int32_T b_outIntensity,
                                emxArray_real32_T c_outIntensity);

static __global__ void
d_PointCloudImpl_subsetImpl_ker(const emxArray_boolean_T indices,
                                const int32_T b_indices,
                                emxArray_real_T outIndex);

static __global__ void
e_PointCloudImpl_subsetImpl_ker(const emxArray_real_T outIndex, real_T *outSize,
                                int32_T outIndex_dim0);

static __global__ void f_PointCloudImpl_subsetImpl_ker(
    const emxArray_real32_T rangeData, const real_T numElements,
    const real_T outSize, const emxArray_real_T outIndex,
    const emxArray_boolean_T indices, const int32_T b_indices,
    emxArray_real32_T outRangeData);

static __global__ void
g_PointCloudImpl_subsetImpl_ker(const emxArray_boolean_T indices,
                                const int32_T b_indices,
                                emxArray_real_T outIndex);

static void gpuEmxEnsureCapacity_boolean_T(const emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu,
                                           boolean_T needsCopy);

static void gpuEmxEnsureCapacity_real32_T(const emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu,
                                          boolean_T needsCopy);

static void gpuEmxMemcpyCpuToGpu_boolean_T(emxArray_boolean_T *gpu,
                                           const emxArray_boolean_T *cpu);

static void gpuEmxMemcpyCpuToGpu_real32_T(emxArray_real32_T *gpu,
                                          const emxArray_real32_T *cpu);

static __global__ void
h_PointCloudImpl_subsetImpl_ker(const emxArray_real_T outIndex, real_T *outSize,
                                int32_T outIndex_dim0);

static __global__ void i_PointCloudImpl_subsetImpl_ker(
    const emxArray_real32_T intensity, const real_T outSize,
    const emxArray_real_T outIndex, const emxArray_boolean_T indices,
    const int32_T b, emxArray_real32_T outIntensity);

static __global__ void
j_PointCloudImpl_subsetImpl_ker(const emxArray_boolean_T indices,
                                const int32_T b_indices,
                                emxArray_real_T outIndex);

static __global__ void
k_PointCloudImpl_subsetImpl_ker(const emxArray_real_T outIndex, real_T *outSize,
                                int32_T outIndex_dim0);

static __global__ void l_PointCloudImpl_subsetImpl_ker(
    const emxArray_real32_T normal, const real_T numElements,
    const real_T outSize, const emxArray_real_T outIndex,
    const emxArray_boolean_T indices, const int32_T b_indices,
    emxArray_real32_T outNorm);

static __global__ void
m_PointCloudImpl_subsetImpl_ker(const emxArray_boolean_T indices,
                                const int32_T b_indices,
                                emxArray_real_T outIndex);

static __global__ void
n_PointCloudImpl_subsetImpl_ker(const emxArray_real_T outIndex, real_T *outSize,
                                int32_T outIndex_dim0);

static __global__ void o_PointCloudImpl_subsetImpl_ker(
    const emxArray_uint8_T color, const real_T numElements,
    const real_T outSize, const emxArray_real_T outIndex,
    const emxArray_boolean_T indices, const int32_T b_indices,
    emxArray_uint8_T outCol);

static __global__ void
p_PointCloudImpl_subsetImpl_ker(const emxArray_boolean_T indices,
                                const int32_T b_indices,
                                emxArray_real_T outIndex);

static __global__ void
q_PointCloudImpl_subsetImpl_ker(const emxArray_real_T outIndex, real_T *outSize,
                                int32_T outIndex_dim0);

static __global__ void r_PointCloudImpl_subsetImpl_ker(
    const emxArray_real32_T location, const real_T numElements,
    const real_T outSize, const emxArray_real_T outIndex,
    const emxArray_boolean_T indices, const int32_T b_indices,
    emxArray_real32_T outLoc);

// Function Definitions
static __global__
    __launch_bounds__(1024, 1) void c_PointCloudImpl_extractValidPo(
        const real_T numPoints, const emxArray_real32_T ptCloudCoords,
        const int32_T b, emxArray_boolean_T validCoords)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    if ((static_cast<boolean_T>(
            !static_cast<int32_T>(isinf(ptCloudCoords.data[i])))) &&
        (static_cast<boolean_T>(
            !static_cast<int32_T>(isnan(ptCloudCoords.data[i]))))) {
      real32_T x;
      x = ptCloudCoords.data[i + static_cast<int32_T>(numPoints)];
      if ((static_cast<boolean_T>(!static_cast<int32_T>(isinf(x)))) &&
          (static_cast<boolean_T>(!static_cast<int32_T>(isnan(x))))) {
        x = ptCloudCoords.data[i + (static_cast<int32_T>(numPoints) << 1)];
        if ((static_cast<boolean_T>(!static_cast<int32_T>(isinf(x)))) &&
            (static_cast<boolean_T>(!static_cast<int32_T>(isnan(x))))) {
          validCoords.data[i] = true;
        } else {
          validCoords.data[i] = false;
        }
      } else {
        validCoords.data[i] = false;
      }
    } else {
      validCoords.data[i] = false;
    }
  }
}

static __global__
    __launch_bounds__(1024, 1) void c_PointCloudImpl_subsetImpl_ker(
        const emxArray_real32_T outIntensity, const int32_T b_outIntensity,
        emxArray_real32_T c_outIntensity)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_outIntensity);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T col;
    col = static_cast<int32_T>(idx);
    c_outIntensity.data[col] = outIntensity.data[col];
  }
}

static __global__
    __launch_bounds__(1024, 1) void d_PointCloudImpl_subsetImpl_ker(
        const emxArray_boolean_T indices, const int32_T b_indices,
        emxArray_real_T outIndex)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_indices);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T col;
    col = static_cast<int32_T>(idx);
    outIndex.data[col] = static_cast<real_T>(indices.data[col]);
  }
}

static __global__ __launch_bounds__(32, 1) void e_PointCloudImpl_subsetImpl_ker(
    const emxArray_real_T outIndex, real_T *outSize, int32_T outIndex_dim0)
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 2) {
    *outSize = outIndex.data[outIndex_dim0 - 1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void f_PointCloudImpl_subsetImpl_ker(
        const emxArray_real32_T rangeData, const real_T numElements,
        const real_T outSize, const emxArray_real_T outIndex,
        const emxArray_boolean_T indices, const int32_T b_indices,
        emxArray_real32_T outRangeData)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * (static_cast<uint64_T>(b_indices) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T col;
    int32_T itr;
    itr = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_indices) + 1UL));
    col = static_cast<int32_T>((idx - static_cast<uint64_T>(itr)) /
                               (static_cast<uint64_T>(b_indices) + 1UL));
    if (indices.data[itr]) {
      outRangeData
          .data[static_cast<int32_T>(outIndex.data[itr] +
                                     static_cast<real_T>(col) * outSize) -
                1] =
          rangeData.data[static_cast<int32_T>((static_cast<real_T>(itr) + 1.0) +
                                              static_cast<real_T>(col) *
                                                  numElements) -
                         1];
    }
  }
}

static __global__
    __launch_bounds__(1024, 1) void g_PointCloudImpl_subsetImpl_ker(
        const emxArray_boolean_T indices, const int32_T b_indices,
        emxArray_real_T outIndex)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_indices);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T col;
    col = static_cast<int32_T>(idx);
    outIndex.data[col] = static_cast<real_T>(indices.data[col]);
  }
}

static void gpuEmxEnsureCapacity_boolean_T(const emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu,
                                           boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  boolean_T *newData;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(boolean_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(boolean_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

static void gpuEmxEnsureCapacity_real32_T(const emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu,
                                          boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  real32_T *newData;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

static void gpuEmxMemcpyCpuToGpu_boolean_T(emxArray_boolean_T *gpu,
                                           const emxArray_boolean_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(
      hipMemcpy(gpu->data, cpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(boolean_T),
                 hipMemcpyHostToDevice),
      __FILE__, __LINE__);
}

static void gpuEmxMemcpyCpuToGpu_real32_T(emxArray_real32_T *gpu,
                                          const emxArray_real32_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(
      hipMemcpy(gpu->data, cpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(real32_T),
                 hipMemcpyHostToDevice),
      __FILE__, __LINE__);
}

static __global__ __launch_bounds__(32, 1) void h_PointCloudImpl_subsetImpl_ker(
    const emxArray_real_T outIndex, real_T *outSize, int32_T outIndex_dim0)
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 2) {
    *outSize = outIndex.data[outIndex_dim0 - 1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void i_PointCloudImpl_subsetImpl_ker(
        const emxArray_real32_T intensity, const real_T outSize,
        const emxArray_real_T outIndex, const emxArray_boolean_T indices,
        const int32_T b, emxArray_real32_T outIntensity)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T itr;
    itr = static_cast<int32_T>(idx);
    if (indices.data[itr]) {
      outIntensity
          .data[static_cast<int32_T>(outIndex.data[itr] + 0.0 * outSize) - 1] =
          intensity.data[itr];
    }
  }
}

static __global__
    __launch_bounds__(1024, 1) void j_PointCloudImpl_subsetImpl_ker(
        const emxArray_boolean_T indices, const int32_T b_indices,
        emxArray_real_T outIndex)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_indices);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T col;
    col = static_cast<int32_T>(idx);
    outIndex.data[col] = static_cast<real_T>(indices.data[col]);
  }
}

static __global__ __launch_bounds__(32, 1) void k_PointCloudImpl_subsetImpl_ker(
    const emxArray_real_T outIndex, real_T *outSize, int32_T outIndex_dim0)
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 2) {
    *outSize = outIndex.data[outIndex_dim0 - 1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void l_PointCloudImpl_subsetImpl_ker(
        const emxArray_real32_T normal, const real_T numElements,
        const real_T outSize, const emxArray_real_T outIndex,
        const emxArray_boolean_T indices, const int32_T b_indices,
        emxArray_real32_T outNorm)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * (static_cast<uint64_T>(b_indices) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T col;
    int32_T itr;
    itr = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_indices) + 1UL));
    col = static_cast<int32_T>((idx - static_cast<uint64_T>(itr)) /
                               (static_cast<uint64_T>(b_indices) + 1UL));
    if (indices.data[itr]) {
      outNorm.data[static_cast<int32_T>(outIndex.data[itr] +
                                        static_cast<real_T>(col) * outSize) -
                   1] =
          normal.data[static_cast<int32_T>((static_cast<real_T>(itr) + 1.0) +
                                           static_cast<real_T>(col) *
                                               numElements) -
                      1];
    }
  }
}

static __global__
    __launch_bounds__(1024, 1) void m_PointCloudImpl_subsetImpl_ker(
        const emxArray_boolean_T indices, const int32_T b_indices,
        emxArray_real_T outIndex)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_indices);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T col;
    col = static_cast<int32_T>(idx);
    outIndex.data[col] = static_cast<real_T>(indices.data[col]);
  }
}

static __global__ __launch_bounds__(32, 1) void n_PointCloudImpl_subsetImpl_ker(
    const emxArray_real_T outIndex, real_T *outSize, int32_T outIndex_dim0)
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 2) {
    *outSize = outIndex.data[outIndex_dim0 - 1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void o_PointCloudImpl_subsetImpl_ker(
        const emxArray_uint8_T color, const real_T numElements,
        const real_T outSize, const emxArray_real_T outIndex,
        const emxArray_boolean_T indices, const int32_T b_indices,
        emxArray_uint8_T outCol)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * (static_cast<uint64_T>(b_indices) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T col;
    int32_T itr;
    itr = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_indices) + 1UL));
    col = static_cast<int32_T>((idx - static_cast<uint64_T>(itr)) /
                               (static_cast<uint64_T>(b_indices) + 1UL));
    if (indices.data[itr]) {
      outCol.data[static_cast<int32_T>(outIndex.data[itr] +
                                       static_cast<real_T>(col) * outSize) -
                  1] =
          color.data[static_cast<int32_T>((static_cast<real_T>(itr) + 1.0) +
                                          static_cast<real_T>(col) *
                                              numElements) -
                     1];
    }
  }
}

static __global__
    __launch_bounds__(1024, 1) void p_PointCloudImpl_subsetImpl_ker(
        const emxArray_boolean_T indices, const int32_T b_indices,
        emxArray_real_T outIndex)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_indices);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T col;
    col = static_cast<int32_T>(idx);
    outIndex.data[col] = static_cast<real_T>(indices.data[col]);
  }
}

static __global__ __launch_bounds__(32, 1) void q_PointCloudImpl_subsetImpl_ker(
    const emxArray_real_T outIndex, real_T *outSize, int32_T outIndex_dim0)
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 2) {
    *outSize = outIndex.data[outIndex_dim0 - 1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void r_PointCloudImpl_subsetImpl_ker(
        const emxArray_real32_T location, const real_T numElements,
        const real_T outSize, const emxArray_real_T outIndex,
        const emxArray_boolean_T indices, const int32_T b_indices,
        emxArray_real32_T outLoc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * (static_cast<uint64_T>(b_indices) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T col;
    int32_T itr;
    itr = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_indices) + 1UL));
    col = static_cast<int32_T>((idx - static_cast<uint64_T>(itr)) /
                               (static_cast<uint64_T>(b_indices) + 1UL));
    if (indices.data[itr]) {
      outLoc.data[static_cast<int32_T>(outIndex.data[itr] +
                                       static_cast<real_T>(col) * outSize) -
                  1] =
          location.data[static_cast<int32_T>((static_cast<real_T>(itr) + 1.0) +
                                             static_cast<real_T>(col) *
                                                 numElements) -
                        1];
    }
  }
}

namespace coder {
namespace vision {
namespace internal {
namespace codegen {
namespace gpu {
void PointCloudImpl_subsetImpl(
    const emxArray_real32_T *cpu_location, const emxArray_uint8_T *cpu_color,
    const emxArray_real32_T *cpu_normal, const emxArray_real32_T *cpu_intensity,
    const emxArray_real32_T *cpu_rangeData, emxArray_boolean_T *cpu_indices,
    emxArray_boolean_T *gpu_indices, boolean_T *indices_outdatedOnGpu,
    emxArray_real32_T *cpu_outLoc, boolean_T *outLoc_outdatedOnCpu,
    emxArray_real32_T *gpu_outLoc, boolean_T *outLoc_outdatedOnGpu,
    emxArray_uint8_T *cpu_outCol, boolean_T *outCol_outdatedOnCpu,
    emxArray_uint8_T *gpu_outCol, boolean_T *outCol_outdatedOnGpu,
    emxArray_real32_T *cpu_outNorm, boolean_T *outNorm_outdatedOnCpu,
    emxArray_real32_T *gpu_outNorm, boolean_T *outNorm_outdatedOnGpu,
    emxArray_real32_T *cpu_outIntensity, boolean_T *outIntensity_outdatedOnCpu,
    emxArray_real32_T *gpu_outIntensity, boolean_T *outIntensity_outdatedOnGpu,
    emxArray_real32_T *cpu_outRangeData, boolean_T *outRangeData_outdatedOnCpu,
    emxArray_real32_T *gpu_outRangeData, boolean_T *outRangeData_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_real32_T b_gpu_outIntensity;
  emxArray_real32_T gpu_intensity;
  emxArray_real32_T gpu_location;
  emxArray_real32_T gpu_normal;
  emxArray_real32_T gpu_rangeData;
  emxArray_real32_T *b_cpu_outIntensity;
  emxArray_real_T b_gpu_outIndex;
  emxArray_real_T c_gpu_outIndex;
  emxArray_real_T d_gpu_outIndex;
  emxArray_real_T e_gpu_outIndex;
  emxArray_real_T gpu_outIndex;
  emxArray_real_T *b_cpu_outIndex;
  emxArray_real_T *c_cpu_outIndex;
  emxArray_real_T *cpu_outIndex;
  emxArray_real_T *d_cpu_outIndex;
  emxArray_real_T *e_cpu_outIndex;
  emxArray_uint8_T gpu_color;
  real_T b_cpu_outSize;
  real_T c_cpu_outSize;
  real_T cpu_outSize;
  real_T d_cpu_outSize;
  real_T e_cpu_outSize;
  real_T numElements;
  real_T *b_gpu_outSize;
  real_T *c_gpu_outSize;
  real_T *d_gpu_outSize;
  real_T *e_gpu_outSize;
  real_T *gpu_outSize;
  int32_T i;
  boolean_T c_outIntensity_needsGpuEnsureCa;
  boolean_T indices_needsGpuEnsureCapacity;
  boolean_T validLaunchParams;
  checkCudaError(mwCudaMalloc(&gpu_outSize, 8UL), __FILE__, __LINE__);
  gpuEmxReset_real_T(&gpu_outIndex);
  checkCudaError(mwCudaMalloc(&b_gpu_outSize, 8UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&c_gpu_outSize, 8UL), __FILE__, __LINE__);
  gpuEmxReset_real32_T(&b_gpu_outIntensity);
  gpuEmxReset_real_T(&b_gpu_outIndex);
  checkCudaError(mwCudaMalloc(&d_gpu_outSize, 8UL), __FILE__, __LINE__);
  gpuEmxReset_real_T(&c_gpu_outIndex);
  checkCudaError(mwCudaMalloc(&e_gpu_outSize, 8UL), __FILE__, __LINE__);
  gpuEmxReset_real_T(&d_gpu_outIndex);
  gpuEmxReset_real_T(&e_gpu_outIndex);
  gpuEmxReset_real32_T(&gpu_rangeData);
  gpuEmxReset_real32_T(&gpu_intensity);
  gpuEmxReset_real32_T(&gpu_normal);
  gpuEmxReset_uint8_T(&gpu_color);
  gpuEmxReset_real32_T(&gpu_location);
  c_outIntensity_needsGpuEnsureCa = true;
  indices_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&cpu_outIndex, 1, &l_emlrtRTEI, true);
  if ((cpu_location->size[0] == 0) || (cpu_indices->size[0] == 0)) {
    *outLoc_outdatedOnCpu = false;
    cpu_outLoc->size[0] = 0;
    cpu_outLoc->size[1] = 3;
  } else {
    numElements = static_cast<real_T>(cpu_location->size[0] * 3) / 3.0;
    i = cpu_outIndex->size[0];
    cpu_outIndex->size[0] = cpu_indices->size[0];
    emxEnsureCapacity_real_T(cpu_outIndex, i, &l_emlrtRTEI);
    i = cpu_indices->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_boolean_T(cpu_indices, gpu_indices,
                                   !*indices_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_outIndex, &e_gpu_outIndex, true);
    if (*indices_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_boolean_T(gpu_indices, cpu_indices);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      p_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(*gpu_indices, i,
                                                       e_gpu_outIndex);
    }
    if (cpu_outIndex->size[0] != 1) {
      mwCallThrustScan1D(&e_gpu_outIndex.data[0], false, cpu_outIndex->size[0]);
    }
    q_PointCloudImpl_subsetImpl_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        e_gpu_outIndex, e_gpu_outSize, cpu_outIndex->size[0U]);
    i = cpu_outLoc->size[0] * cpu_outLoc->size[1];
    checkCudaError(
        hipMemcpy(&cpu_outSize, e_gpu_outSize, 8UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    cpu_outLoc->size[0] = static_cast<int32_T>(cpu_outSize);
    cpu_outLoc->size[1] = 3;
    emxEnsureCapacity_real32_T(cpu_outLoc, i, &l_emlrtRTEI);
    i = cpu_indices->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(2, i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_location, &gpu_location, false);
    indices_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_real32_T(cpu_outLoc, gpu_outLoc, true);
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_location, cpu_location);
    *indices_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      r_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(
          gpu_location, numElements, cpu_outSize, e_gpu_outIndex, *gpu_indices,
          i, *gpu_outLoc);
    }
    *outLoc_outdatedOnCpu = true;
  }
  emxFree_real_T(&cpu_outIndex);
  emxInit_real_T(&b_cpu_outIndex, 1, &l_emlrtRTEI, true);
  if ((cpu_color->size[0] == 0) || (cpu_color->size[1] == 0) ||
      (cpu_indices->size[0] == 0)) {
    *outCol_outdatedOnCpu = false;
    cpu_outCol->size[0] = 0;
    cpu_outCol->size[1] = 3;
  } else {
    numElements =
        static_cast<real_T>(cpu_color->size[0] * cpu_color->size[1]) / 3.0;
    i = b_cpu_outIndex->size[0];
    b_cpu_outIndex->size[0] = cpu_indices->size[0];
    emxEnsureCapacity_real_T(b_cpu_outIndex, i, &l_emlrtRTEI);
    i = cpu_indices->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    if (indices_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_boolean_T(cpu_indices, gpu_indices,
                                     !*indices_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real_T(b_cpu_outIndex, &d_gpu_outIndex, true);
    if (*indices_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_boolean_T(gpu_indices, cpu_indices);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      m_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(*gpu_indices, i,
                                                       d_gpu_outIndex);
    }
    if (b_cpu_outIndex->size[0] != 1) {
      mwCallThrustScan1D(&d_gpu_outIndex.data[0], false,
                         b_cpu_outIndex->size[0]);
    }
    n_PointCloudImpl_subsetImpl_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        d_gpu_outIndex, d_gpu_outSize, b_cpu_outIndex->size[0U]);
    i = cpu_outCol->size[0] * cpu_outCol->size[1];
    checkCudaError(
        hipMemcpy(&b_cpu_outSize, d_gpu_outSize, 8UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    cpu_outCol->size[0] = static_cast<int32_T>(b_cpu_outSize);
    cpu_outCol->size[1] = 3;
    emxEnsureCapacity_uint8_T(cpu_outCol, i, &l_emlrtRTEI);
    i = cpu_indices->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(2, i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_uint8_T(cpu_color, &gpu_color, false);
    indices_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_uint8_T(cpu_outCol, gpu_outCol, true);
    gpuEmxMemcpyCpuToGpu_uint8_T(&gpu_color, cpu_color);
    *indices_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      o_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(
          gpu_color, numElements, b_cpu_outSize, d_gpu_outIndex, *gpu_indices,
          i, *gpu_outCol);
    }
    *outCol_outdatedOnCpu = true;
  }
  emxFree_real_T(&b_cpu_outIndex);
  emxInit_real_T(&c_cpu_outIndex, 1, &l_emlrtRTEI, true);
  if ((cpu_normal->size[0] == 0) || (cpu_normal->size[1] == 0) ||
      (cpu_indices->size[0] == 0)) {
    *outNorm_outdatedOnCpu = false;
    cpu_outNorm->size[0] = 0;
    cpu_outNorm->size[1] = 3;
  } else {
    numElements =
        static_cast<real_T>(cpu_normal->size[0] * cpu_normal->size[1]) / 3.0;
    i = c_cpu_outIndex->size[0];
    c_cpu_outIndex->size[0] = cpu_indices->size[0];
    emxEnsureCapacity_real_T(c_cpu_outIndex, i, &l_emlrtRTEI);
    i = cpu_indices->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    if (indices_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_boolean_T(cpu_indices, gpu_indices,
                                     !*indices_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real_T(c_cpu_outIndex, &c_gpu_outIndex, true);
    if (*indices_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_boolean_T(gpu_indices, cpu_indices);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      j_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(*gpu_indices, i,
                                                       c_gpu_outIndex);
    }
    if (c_cpu_outIndex->size[0] != 1) {
      mwCallThrustScan1D(&c_gpu_outIndex.data[0], false,
                         c_cpu_outIndex->size[0]);
    }
    k_PointCloudImpl_subsetImpl_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        c_gpu_outIndex, c_gpu_outSize, c_cpu_outIndex->size[0U]);
    i = cpu_outNorm->size[0] * cpu_outNorm->size[1];
    checkCudaError(
        hipMemcpy(&c_cpu_outSize, c_gpu_outSize, 8UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    cpu_outNorm->size[0] = static_cast<int32_T>(c_cpu_outSize);
    cpu_outNorm->size[1] = 3;
    emxEnsureCapacity_real32_T(cpu_outNorm, i, &l_emlrtRTEI);
    i = cpu_indices->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(2, i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_normal, &gpu_normal, false);
    indices_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_real32_T(cpu_outNorm, gpu_outNorm, true);
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_normal, cpu_normal);
    *indices_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      l_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(
          gpu_normal, numElements, c_cpu_outSize, c_gpu_outIndex, *gpu_indices,
          i, *gpu_outNorm);
    }
    *outNorm_outdatedOnCpu = true;
  }
  emxFree_real_T(&c_cpu_outIndex);
  emxInit_real_T(&d_cpu_outIndex, 1, &l_emlrtRTEI, true);
  emxInit_real32_T(&b_cpu_outIntensity, 1, &l_emlrtRTEI, true);
  if ((cpu_intensity->size[0] == 0) || (cpu_indices->size[0] == 0)) {
    b_cpu_outIntensity->size[0] = 0;
  } else {
    i = d_cpu_outIndex->size[0];
    d_cpu_outIndex->size[0] = cpu_indices->size[0];
    emxEnsureCapacity_real_T(d_cpu_outIndex, i, &l_emlrtRTEI);
    i = cpu_indices->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    if (indices_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_boolean_T(cpu_indices, gpu_indices,
                                     !*indices_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real_T(d_cpu_outIndex, &b_gpu_outIndex, true);
    if (*indices_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_boolean_T(gpu_indices, cpu_indices);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      g_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(*gpu_indices, i,
                                                       b_gpu_outIndex);
    }
    if (d_cpu_outIndex->size[0] != 1) {
      mwCallThrustScan1D(&b_gpu_outIndex.data[0], false,
                         d_cpu_outIndex->size[0]);
    }
    h_PointCloudImpl_subsetImpl_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        b_gpu_outIndex, b_gpu_outSize, d_cpu_outIndex->size[0U]);
    i = b_cpu_outIntensity->size[0];
    checkCudaError(
        hipMemcpy(&d_cpu_outSize, b_gpu_outSize, 8UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    b_cpu_outIntensity->size[0] = static_cast<int32_T>(d_cpu_outSize);
    emxEnsureCapacity_real32_T(b_cpu_outIntensity, i, &l_emlrtRTEI);
    i = cpu_indices->size[0];
    mwGetLaunchParameters1D(computeNumIters(i - 1), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_intensity, &gpu_intensity, false);
    indices_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_real32_T(b_cpu_outIntensity, &b_gpu_outIntensity,
                                  true);
    c_outIntensity_needsGpuEnsureCa = false;
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_intensity, cpu_intensity);
    *indices_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      i_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(
          gpu_intensity, d_cpu_outSize, b_gpu_outIndex, *gpu_indices, i - 1,
          b_gpu_outIntensity);
    }
  }
  emxFree_real_T(&d_cpu_outIndex);
  i = cpu_outIntensity->size[0] * cpu_outIntensity->size[1];
  cpu_outIntensity->size[0] = b_cpu_outIntensity->size[0];
  cpu_outIntensity->size[1] = 1;
  emxEnsureCapacity_real32_T(cpu_outIntensity, i, &l_emlrtRTEI);
  i = b_cpu_outIntensity->size[0] - 1;
  mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
  if (c_outIntensity_needsGpuEnsureCa) {
    gpuEmxEnsureCapacity_real32_T(b_cpu_outIntensity, &b_gpu_outIntensity,
                                  true);
  }
  gpuEmxEnsureCapacity_real32_T(cpu_outIntensity, gpu_outIntensity, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    c_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(b_gpu_outIntensity, i,
                                                     *gpu_outIntensity);
  }
  emxFree_real32_T(&b_cpu_outIntensity);
  emxInit_real_T(&e_cpu_outIndex, 1, &l_emlrtRTEI, true);
  if ((cpu_rangeData->size[0] == 0) || (cpu_rangeData->size[1] == 0) ||
      (cpu_indices->size[0] == 0)) {
    *outRangeData_outdatedOnCpu = false;
    cpu_outRangeData->size[0] = 0;
    cpu_outRangeData->size[1] = 3;
  } else {
    numElements =
        static_cast<real_T>(cpu_rangeData->size[0] * cpu_rangeData->size[1]) /
        3.0;
    i = e_cpu_outIndex->size[0];
    e_cpu_outIndex->size[0] = cpu_indices->size[0];
    emxEnsureCapacity_real_T(e_cpu_outIndex, i, &l_emlrtRTEI);
    i = cpu_indices->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    if (indices_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_boolean_T(cpu_indices, gpu_indices,
                                     !*indices_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real_T(e_cpu_outIndex, &gpu_outIndex, true);
    if (*indices_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_boolean_T(gpu_indices, cpu_indices);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      d_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(*gpu_indices, i,
                                                       gpu_outIndex);
    }
    if (e_cpu_outIndex->size[0] != 1) {
      mwCallThrustScan1D(&gpu_outIndex.data[0], false, e_cpu_outIndex->size[0]);
    }
    e_PointCloudImpl_subsetImpl_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        gpu_outIndex, gpu_outSize, e_cpu_outIndex->size[0U]);
    i = cpu_outRangeData->size[0] * cpu_outRangeData->size[1];
    checkCudaError(
        hipMemcpy(&e_cpu_outSize, gpu_outSize, 8UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    cpu_outRangeData->size[0] = static_cast<int32_T>(e_cpu_outSize);
    cpu_outRangeData->size[1] = 3;
    emxEnsureCapacity_real32_T(cpu_outRangeData, i, &l_emlrtRTEI);
    i = cpu_indices->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(2, i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_rangeData, &gpu_rangeData, false);
    gpuEmxEnsureCapacity_real32_T(cpu_outRangeData, gpu_outRangeData, true);
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_rangeData, cpu_rangeData);
    *indices_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      f_PointCloudImpl_subsetImpl_ker<<<grid, block>>>(
          gpu_rangeData, numElements, e_cpu_outSize, gpu_outIndex, *gpu_indices,
          i, *gpu_outRangeData);
    }
    *outRangeData_outdatedOnCpu = true;
  }
  emxFree_real_T(&e_cpu_outIndex);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&gpu_location);
  gpuEmxFree_uint8_T(&gpu_color);
  gpuEmxFree_real32_T(&gpu_normal);
  gpuEmxFree_real32_T(&gpu_intensity);
  gpuEmxFree_real32_T(&gpu_rangeData);
  gpuEmxFree_real_T(&e_gpu_outIndex);
  gpuEmxFree_real_T(&d_gpu_outIndex);
  checkCudaError(mwCudaFree(e_gpu_outSize), __FILE__, __LINE__);
  gpuEmxFree_real_T(&c_gpu_outIndex);
  checkCudaError(mwCudaFree(d_gpu_outSize), __FILE__, __LINE__);
  gpuEmxFree_real_T(&b_gpu_outIndex);
  gpuEmxFree_real32_T(&b_gpu_outIntensity);
  checkCudaError(mwCudaFree(c_gpu_outSize), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(b_gpu_outSize), __FILE__, __LINE__);
  gpuEmxFree_real_T(&gpu_outIndex);
  checkCudaError(mwCudaFree(gpu_outSize), __FILE__, __LINE__);
  *outLoc_outdatedOnGpu = false;
  *outCol_outdatedOnGpu = false;
  *outNorm_outdatedOnGpu = false;
  *outIntensity_outdatedOnCpu = true;
  *outIntensity_outdatedOnGpu = false;
  *outRangeData_outdatedOnGpu = false;
}

void d_PointCloudImpl_extractValidPo(const emxArray_real32_T *cpu_ptCloudCoords,
                                     emxArray_boolean_T *cpu_validCoords,
                                     boolean_T *validCoords_outdatedOnCpu,
                                     emxArray_boolean_T *gpu_validCoords,
                                     boolean_T *validCoords_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_real32_T gpu_ptCloudCoords;
  real_T numPoints;
  int32_T i;
  boolean_T validLaunchParams;
  gpuEmxReset_real32_T(&gpu_ptCloudCoords);
  numPoints = static_cast<real_T>(cpu_ptCloudCoords->size[0] * 3) / 3.0;
  i = cpu_validCoords->size[0];
  cpu_validCoords->size[0] = static_cast<int32_T>(numPoints);
  emxEnsureCapacity_boolean_T(cpu_validCoords, i, &k_emlrtRTEI);
  mwGetLaunchParameters1D(computeNumIters(static_cast<int32_T>(numPoints) - 1),
                          &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real32_T(cpu_ptCloudCoords, &gpu_ptCloudCoords, false);
  gpuEmxEnsureCapacity_boolean_T(cpu_validCoords, gpu_validCoords,
                                 !*validCoords_outdatedOnGpu);
  gpuEmxMemcpyCpuToGpu_real32_T(&gpu_ptCloudCoords, cpu_ptCloudCoords);
  if (*validCoords_outdatedOnGpu) {
    gpuEmxMemcpyCpuToGpu_boolean_T(gpu_validCoords, cpu_validCoords);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    c_PointCloudImpl_extractValidPo<<<grid, block>>>(
        numPoints, gpu_ptCloudCoords, static_cast<int32_T>(numPoints) - 1,
        *gpu_validCoords);
  }
  gpuEmxFree_real32_T(&gpu_ptCloudCoords);
  *validCoords_outdatedOnCpu = true;
  *validCoords_outdatedOnGpu = false;
}

} // namespace gpu
} // namespace codegen
} // namespace internal
} // namespace vision
} // namespace coder

// End of code generation (PointCloudImpl.cu)
