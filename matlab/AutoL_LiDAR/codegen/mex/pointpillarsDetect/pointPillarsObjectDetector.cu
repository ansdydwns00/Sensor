#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// pointPillarsObjectDetector.cu
//
// Code generation for function 'pointPillarsObjectDetector'
//

// Include files
#include "pointPillarsObjectDetector.h"
#include "introsort.h"
#include "pointpillarsDetect_data.h"
#include "pointpillarsDetect_emxutil.h"
#include "pointpillarsDetect_mexutil.h"
#include "pointpillarsDetect_types.h"
#include "rt_nonfinite.h"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include <cmath>

// Type Definitions
struct cell_wrap_63 {
  char_T f1[10];
};

struct emxArray_cell_wrap_6_1x1 {
  cell_wrap_6 data[1];
};

// Variable Definitions
static emlrtRTEInfo m_emlrtRTEI{
    11,                                                                // lineNo
    5,                                                                 // colNo
    "cumsum",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/cumsum.m" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    44,               // lineNo
    10,               // colNo
    "cellstr_unique", // fName
    "/usr/local/MATLAB/R2024a/toolbox/matlab/datatypes/shared/codegen/+matlab/"
    "+internal/+coder/+datatypes/cellstr_unique.m" // pName
};

static const char_T cv[10]{'P', 'e', 'd', 'e', 's', 't', 'r', 'a', 'i', 'n'};

static const boolean_T bv[128]{
    false, false, false, false, false, false, false, false, false, true,  true,
    true,  true,  true,  false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, true,  true,  true,  true,  true,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false, false, false, false, false,
    false, false, false, false, false, false, false};

// Function Declarations
static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu);

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

// Function Definitions
static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
}

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
}

namespace coder {
namespace internal {
void c_returnCategoricalLabels(pointpillarsDetectStackData *SD,
                               int32_T labelNamesMod_codes_size[2],
                               cell_wrap_6 c_labelNamesMod_categoryNames_d[],
                               int32_T c_labelNamesMod_categoryNames_s[1])
{
  cell_wrap_63 valueset_data[1];
  emxArray_cell_wrap_6_1x1 rv;
  int32_T b_j1;
  int32_T j2;
  char_T v[10];
  for (j2 = 0; j2 < 10; j2++) {
    v[j2] = cv[j2];
  }
  for (int32_T i{0}; i < 10; i++) {
    valueset_data[0].f1[i] = v[i];
  }
  b_j1 = 0;
  while ((b_j1 + 1 <= 10) &&
         bv[static_cast<int32_T>(valueset_data[0].f1[b_j1])]) {
    b_j1++;
  }
  j2 = 9;
  while ((j2 + 1 > 0) && bv[static_cast<int32_T>(valueset_data[0].f1[j2])]) {
    j2--;
  }
  if (b_j1 + 1 > j2 + 1) {
    b_j1 = 0;
    j2 = -1;
  }
  rv.data[0].f1.size[0] = 1;
  rv.data[0].f1.size[1] = (j2 - b_j1) + 1;
  j2 -= b_j1;
  for (int32_T i{0}; i <= j2; i++) {
    rv.data[0].f1.data[i] = valueset_data[0].f1[b_j1 + i];
  }
  SD->f0.idx_data[0] = 1;
  introsort(SD->f0.idx_data, 1, rv.data);
  c_labelNamesMod_categoryNames_s[0] = 1;
  j2 = rv.data[0].f1.size[1];
  c_labelNamesMod_categoryNames_d[0].f1.size[0] = 1;
  c_labelNamesMod_categoryNames_d[0].f1.size[1] = rv.data[0].f1.size[1];
  for (int32_T i{0}; i < j2; i++) {
    c_labelNamesMod_categoryNames_d[0].f1.data[i] = rv.data[0].f1.data[i];
  }
  SD->f0.idx_data[0] = 1;
  introsort(SD->f0.idx_data, 1, rv.data);
  labelNamesMod_codes_size[0] = 1;
  labelNamesMod_codes_size[1] = 0;
}

void c_returnCategoricalLabels(pointpillarsDetectStackData *SD,
                               real_T numBBoxes,
                               uint32_T labelNamesMod_codes_data[],
                               int32_T labelNamesMod_codes_size[2],
                               cell_wrap_6 c_labelNamesMod_categoryNames_d[],
                               int32_T c_labelNamesMod_categoryNames_s[1])
{
  cell_wrap_63 valueset_data[1];
  emxArray_cell_wrap_6_1 uB;
  emxArray_cell_wrap_6_1x1 rv;
  emxArray_real_T gpu_y;
  emxArray_real_T *cpu_y;
  int32_T idx_size[1];
  int32_T b_i;
  int32_T b_j1;
  int32_T c_i;
  int32_T exitg1;
  int32_T i;
  int32_T idx;
  int32_T j2;
  int32_T k;
  char_T v_data[10];
  boolean_T b;
  boolean_T b_bool;
  boolean_T exitg2;
  boolean_T y_outdatedOnCpu;
  gpuEmxReset_real_T(&gpu_y);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  i = static_cast<int32_T>(numBBoxes);
  for (b_i = 0; b_i < i; b_i++) {
    j2 = 0;
    for (k = 0; k < 10; k++) {
      j2 = k + 1;
    }
    for (k = 0; k < 10; k++) {
      v_data[k] = cv[k];
    }
    SD->f2.labelCells.data[b_i].f1.size[0] = 1;
    SD->f2.labelCells.data[b_i].f1.size[1] = j2;
    for (c_i = 0; c_i < j2; c_i++) {
      SD->f2.labelCells.data[b_i].f1.data[c_i] = v_data[c_i];
    }
  }
  for (k = 0; k < 10; k++) {
    v_data[k] = cv[k];
  }
  for (c_i = 0; c_i < 10; c_i++) {
    valueset_data[0].f1[c_i] = v_data[c_i];
  }
  idx = static_cast<int32_T>(numBBoxes);
  for (i = 0; i < idx; i++) {
    b_j1 = 0;
    while ((b_j1 + 1 <= SD->f2.labelCells.data[i].f1.size[1]) &&
           bv[static_cast<int32_T>(SD->f2.labelCells.data[i].f1.data[b_j1])]) {
      b_j1++;
    }
    j2 = SD->f2.labelCells.data[i].f1.size[1];
    while (
        (j2 > 0) &&
        bv[static_cast<int32_T>(SD->f2.labelCells.data[i].f1.data[j2 - 1])]) {
      j2--;
    }
    if (b_j1 + 1 > j2) {
      b_j1 = 0;
      j2 = 0;
    }
    SD->f2.inData.data[i].f1.size[0] = 1;
    SD->f2.inData.data[i].f1.size[1] = j2 - b_j1;
    k = j2 - b_j1;
    for (c_i = 0; c_i < k; c_i++) {
      SD->f2.inData.data[i].f1.data[c_i] =
          SD->f2.labelCells.data[i].f1.data[b_j1 + c_i];
    }
  }
  b_j1 = 0;
  while ((b_j1 + 1 <= 10) &&
         bv[static_cast<int32_T>(valueset_data[0].f1[b_j1])]) {
    b_j1++;
  }
  j2 = 9;
  while ((j2 + 1 > 0) && bv[static_cast<int32_T>(valueset_data[0].f1[j2])]) {
    j2--;
  }
  if (b_j1 + 1 > j2 + 1) {
    b_j1 = 0;
    j2 = -1;
  }
  rv.data[0].f1.size[0] = 1;
  rv.data[0].f1.size[1] = (j2 - b_j1) + 1;
  k = j2 - b_j1;
  for (c_i = 0; c_i <= k; c_i++) {
    rv.data[0].f1.data[c_i] = valueset_data[0].f1[b_j1 + c_i];
  }
  SD->f2.locbOfUA_data[0] = 1;
  introsort(SD->f2.locbOfUA_data, 1, rv.data);
  c_labelNamesMod_categoryNames_s[0] = 1;
  k = rv.data[0].f1.size[1];
  c_labelNamesMod_categoryNames_d[0].f1.size[0] = 1;
  c_labelNamesMod_categoryNames_d[0].f1.size[1] = rv.data[0].f1.size[1];
  for (c_i = 0; c_i < k; c_i++) {
    c_labelNamesMod_categoryNames_d[0].f1.data[c_i] = rv.data[0].f1.data[c_i];
  }
  emxInit_real_T(&cpu_y, 1, &n_emlrtRTEI, true);
  if (static_cast<int32_T>(numBBoxes) == 0) {
    SD->f2.uA.size[0] = 0;
    y_outdatedOnCpu = false;
    cpu_y->size[0] = 0;
  } else {
    int32_T d_size_idx_0;
    int32_T ii_size_idx_0;
    int32_T y_size_idx_1;
    j2 = static_cast<int32_T>(numBBoxes);
    y_size_idx_1 = static_cast<int32_T>(numBBoxes);
    SD->f2.b_y_data[0] = 1;
    b_j1 = 1;
    for (k = 0; k <= j2 - 2; k++) {
      b_j1++;
      SD->f2.b_y_data[k + 1] = b_j1;
    }
    idx_size[0] = static_cast<int32_T>(numBBoxes);
    for (c_i = 0; c_i < y_size_idx_1; c_i++) {
      SD->f2.locbOfUA_data[c_i] = SD->f2.b_y_data[c_i];
    }
    introsort(SD->f2.locbOfUA_data, static_cast<int32_T>(numBBoxes),
              SD->f2.inData.data);
    SD->f2.labelCells.size[0] = static_cast<int32_T>(numBBoxes);
    j2 = static_cast<int32_T>(numBBoxes);
    for (i = 0; i < j2; i++) {
      SD->f2.labelCells.data[i].f1.size[0] = 1;
      SD->f2.labelCells.data[i].f1.size[1] =
          SD->f2.inData.data[SD->f2.locbOfUA_data[i] - 1].f1.size[1];
      k = SD->f2.inData.data[SD->f2.locbOfUA_data[i] - 1].f1.size[1];
      for (c_i = 0; c_i < k; c_i++) {
        SD->f2.labelCells.data[i].f1.data[c_i] =
            SD->f2.inData.data[SD->f2.locbOfUA_data[i] - 1].f1.data[c_i];
      }
    }
    b_j1 = static_cast<int32_T>(numBBoxes) - 1;
    for (i = 0; i <= SD->f2.labelCells.size[0] - 2; i++) {
      SD->f2.b_d_data[i] = false;
      b_bool = false;
      b = (SD->f2.labelCells.data[i].f1.size[1] == 0);
      if (b && (SD->f2.labelCells.data[i + 1].f1.size[1] == 0)) {
        b_bool = true;
      } else if (SD->f2.labelCells.data[i].f1.size[1] ==
                 SD->f2.labelCells.data[i + 1].f1.size[1]) {
        j2 = 0;
        do {
          exitg1 = 0;
          if (j2 + 1 <= SD->f2.labelCells.data[i + 1].f1.size[1]) {
            if (SD->f2.labelCells.data[i].f1.data[j2] !=
                SD->f2.labelCells.data[i + 1].f1.data[j2]) {
              exitg1 = 1;
            } else {
              j2++;
            }
          } else {
            b_bool = true;
            exitg1 = 1;
          }
        } while (exitg1 == 0);
      }
      SD->f2.b_d_data[i] = !b_bool;
    }
    d_size_idx_0 = static_cast<int32_T>(numBBoxes);
    SD->f2.d_data[0] = true;
    for (c_i = 0; c_i < b_j1; c_i++) {
      SD->f2.d_data[c_i + 1] = SD->f2.b_d_data[c_i];
    }
    j2 = SD->f2.d_data[0];
    for (k = 0; k <= d_size_idx_0 - 2; k++) {
      j2 += SD->f2.d_data[k + 1];
    }
    SD->f2.uA.size[0] = static_cast<int32_T>(
        std::fmin(static_cast<real_T>(j2),
                  static_cast<real_T>(static_cast<int32_T>(numBBoxes))));
    idx = 0;
    ii_size_idx_0 = static_cast<int32_T>(numBBoxes);
    b_j1 = 1;
    exitg2 = false;
    while ((!exitg2) && (b_j1 <= static_cast<int32_T>(numBBoxes))) {
      if (SD->f2.d_data[b_j1 - 1]) {
        idx++;
        SD->f2.ii_data[idx - 1] = b_j1;
        if (idx >= static_cast<int32_T>(numBBoxes)) {
          exitg2 = true;
        } else {
          b_j1++;
        }
      } else {
        b_j1++;
      }
    }
    if (static_cast<int32_T>(numBBoxes) == 1) {
      if (idx == 0) {
        ii_size_idx_0 = 0;
      }
    } else if (idx < 1) {
      ii_size_idx_0 = 0;
    } else {
      ii_size_idx_0 = idx;
    }
    for (i = 0; i < ii_size_idx_0; i++) {
      if (i + 1 != ii_size_idx_0) {
        if (SD->f2.ii_data[i + 1] - 1 < SD->f2.ii_data[i]) {
          y_size_idx_1 = 0;
        } else {
          c_i = SD->f2.ii_data[i];
          j2 = SD->f2.ii_data[i + 1];
          y_size_idx_1 = SD->f2.ii_data[i + 1] - SD->f2.ii_data[i];
          for (idx = 0; idx < j2 - c_i; idx++) {
            SD->f2.b_y_data[idx] = c_i + idx;
          }
        }
        b_j1 = SD->f2.locbOfUA_data[SD->f2.b_y_data[0] - 1];
        for (b_i = 0; b_i <= y_size_idx_1 - 2; b_i++) {
          c_i = SD->f2.b_y_data[b_i + 1];
          if (b_j1 > SD->f2.locbOfUA_data[c_i - 1]) {
            b_j1 = SD->f2.locbOfUA_data[c_i - 1];
          }
        }
        SD->f2.a__2_data[i] = b_j1;
      } else {
        if (static_cast<int32_T>(numBBoxes) < SD->f2.ii_data[i]) {
          y_size_idx_1 = 0;
        } else {
          c_i = SD->f2.ii_data[i];
          y_size_idx_1 =
              (static_cast<int32_T>(numBBoxes) - SD->f2.ii_data[i]) + 1;
          for (j2 = 0; j2 <= idx_size[0] - c_i; j2++) {
            SD->f2.b_y_data[j2] = c_i + j2;
          }
        }
        b_j1 = SD->f2.locbOfUA_data[SD->f2.b_y_data[0] - 1];
        for (b_i = 0; b_i <= y_size_idx_1 - 2; b_i++) {
          c_i = SD->f2.b_y_data[b_i + 1];
          if (b_j1 > SD->f2.locbOfUA_data[c_i - 1]) {
            b_j1 = SD->f2.locbOfUA_data[c_i - 1];
          }
        }
        SD->f2.a__2_data[i] = b_j1;
      }
    }
    y_outdatedOnCpu = false;
    b_bool = false;
    c_i = cpu_y->size[0];
    cpu_y->size[0] = static_cast<int32_T>(numBBoxes);
    emxEnsureCapacity_real_T(cpu_y, c_i, &m_emlrtRTEI);
    for (c_i = 0; c_i < d_size_idx_0; c_i++) {
      cpu_y->data[c_i] = SD->f2.d_data[c_i];
      b_bool = true;
    }
    if (cpu_y->size[0] != 1) {
      gpuEmxEnsureCapacity_real_T(cpu_y, &gpu_y, !b_bool);
      if (b_bool) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_y, cpu_y);
      }
      mwCallThrustScan1D(&gpu_y.data[0], false, cpu_y->size[0]);
      y_outdatedOnCpu = true;
    }
    b_j1 = cpu_y->size[0];
    for (c_i = 0; c_i < cpu_y->size[0]; c_i++) {
      if (y_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &gpu_y);
      }
      y_outdatedOnCpu = false;
      SD->f2.y_data[c_i] = cpu_y->data[c_i];
    }
    for (c_i = 0; c_i < b_j1; c_i++) {
      if (y_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &gpu_y);
      }
      cpu_y->data[SD->f2.locbOfUA_data[c_i] - 1] = SD->f2.y_data[c_i];
      y_outdatedOnCpu = false;
    }
    for (i = 0; i < ii_size_idx_0; i++) {
      SD->f2.uA.data[i].f1.size[0] = 1;
      SD->f2.uA.data[i].f1.size[1] =
          SD->f2.inData.data[SD->f2.a__2_data[i] - 1].f1.size[1];
      k = SD->f2.inData.data[SD->f2.a__2_data[i] - 1].f1.size[1];
      for (c_i = 0; c_i < k; c_i++) {
        SD->f2.uA.data[i].f1.data[c_i] =
            SD->f2.inData.data[SD->f2.a__2_data[i] - 1].f1.data[c_i];
      }
    }
  }
  SD->f2.locbOfUA_data[0] = 1;
  introsort(SD->f2.locbOfUA_data, 1, rv.data);
  SD->f2.ib_data[0] = SD->f2.locbOfUA_data[0];
  k = rv.data[0].f1.size[1];
  uB.data[0].f1.size[1] = rv.data[0].f1.size[1];
  for (c_i = 0; c_i < k; c_i++) {
    uB.data[0].f1.data[c_i] = rv.data[0].f1.data[c_i];
  }
  for (c_i = 0; c_i < SD->f2.uA.size[0]; c_i++) {
    SD->f2.locbOfUA_data[c_i] = 0;
  }
  b_j1 = 1;
  i = 0;
  exitg2 = false;
  while ((!exitg2) && (i <= SD->f2.uA.size[0] - 1)) {
    j2 = SD->f2.uA.data[i].f1.size[1];
    idx = uB.data[0].f1.size[1];
    if (j2 <= uB.data[0].f1.size[1]) {
      idx = j2;
    }
    if (idx == 0) {
      b = (SD->f2.uA.data[i].f1.size[1] < uB.data[0].f1.size[1]);
    } else {
      b_i = 0;
      while ((b_i + 1 <= idx) &&
             (SD->f2.uA.data[i].f1.data[b_i] == uB.data[0].f1.data[b_i])) {
        b_i++;
      }
      if (b_i + 1 == idx + 1) {
        b = (SD->f2.uA.data[i].f1.size[1] < uB.data[0].f1.size[1]);
      } else {
        b = (SD->f2.uA.data[i].f1.data[b_i] < uB.data[0].f1.data[b_i]);
      }
    }
    if (!b) {
      b_bool = false;
      b = (SD->f2.uA.data[i].f1.size[1] == 0);
      if (b && (uB.data[0].f1.size[1] == 0)) {
        b_bool = true;
      } else if (SD->f2.uA.data[i].f1.size[1] == uB.data[0].f1.size[1]) {
        j2 = 0;
        do {
          exitg1 = 0;
          if (j2 + 1 <= uB.data[0].f1.size[1]) {
            if (SD->f2.uA.data[i].f1.data[j2] != uB.data[0].f1.data[j2]) {
              exitg1 = 1;
            } else {
              j2++;
            }
          } else {
            b_bool = true;
            exitg1 = 1;
          }
        } while (exitg1 == 0);
      }
      if (b_bool) {
        SD->f2.locbOfUA_data[i] = SD->f2.ib_data[0];
        b_j1 = 2;
      } else {
        boolean_T exitg3;
        exitg3 = false;
        while ((!exitg3) && (b_j1 <= 1)) {
          j2 = SD->f2.uA.data[i].f1.size[1];
          idx = uB.data[0].f1.size[1];
          if (j2 <= uB.data[0].f1.size[1]) {
            idx = j2;
          }
          if (idx == 0) {
            b = (SD->f2.uA.data[i].f1.size[1] > uB.data[0].f1.size[1]);
          } else {
            b_i = 0;
            while ((b_i + 1 <= idx) && (SD->f2.uA.data[i].f1.data[b_i] ==
                                        uB.data[0].f1.data[b_i])) {
              b_i++;
            }
            if (b_i + 1 == idx + 1) {
              b = (SD->f2.uA.data[i].f1.size[1] > uB.data[0].f1.size[1]);
            } else {
              b = (SD->f2.uA.data[i].f1.data[b_i] > uB.data[0].f1.data[b_i]);
            }
          }
          if (b) {
            b_j1 = 2;
          } else {
            exitg3 = true;
          }
        }
        if (b_j1 <= 1) {
          b = (SD->f2.uA.data[i].f1.size[1] == 0);
          if (b && (uB.data[0].f1.size[1] == 0)) {
            b_bool = true;
          } else if (SD->f2.uA.data[i].f1.size[1] == uB.data[0].f1.size[1]) {
            j2 = 0;
            do {
              exitg1 = 0;
              if (j2 + 1 <= uB.data[0].f1.size[1]) {
                if (SD->f2.uA.data[i].f1.data[j2] != uB.data[0].f1.data[j2]) {
                  exitg1 = 1;
                } else {
                  j2++;
                }
              } else {
                b_bool = true;
                exitg1 = 1;
              }
            } while (exitg1 == 0);
          }
          if (b_bool) {
            SD->f2.locbOfUA_data[i] = SD->f2.ib_data[0];
            b_j1 = 2;
          }
        }
      }
    }
    if (b_j1 > 1) {
      exitg2 = true;
    } else {
      i++;
    }
  }
  labelNamesMod_codes_size[0] = 1;
  labelNamesMod_codes_size[1] = cpu_y->size[0];
  k = cpu_y->size[0];
  for (c_i = 0; c_i < k; c_i++) {
    if (y_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &gpu_y);
    }
    y_outdatedOnCpu = false;
    j2 = SD->f2.locbOfUA_data[static_cast<int32_T>(cpu_y->data[c_i]) - 1];
    if (j2 < 0) {
      j2 = 0;
    }
    labelNamesMod_codes_data[c_i] = static_cast<uint32_T>(j2);
  }
  emxFree_real_T(&cpu_y);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_y);
}

} // namespace internal
} // namespace coder

// End of code generation (pointPillarsObjectDetector.cu)
