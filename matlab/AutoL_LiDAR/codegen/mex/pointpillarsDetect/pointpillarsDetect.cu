#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// pointpillarsDetect.cu
//
// Code generation for function 'pointpillarsDetect'
//

// Include files
#include "pointpillarsDetect.h"
#include "PointCloudImpl.h"
#include "dlnetwork.h"
#include "handle.h"
#include "pointPillarsObjectDetector.h"
#include "pointpillarsDetect_data.h"
#include "pointpillarsDetect_emxutil.h"
#include "pointpillarsDetect_internal_types.h"
#include "pointpillarsDetect_mexutil.h"
#include "pointpillarsDetect_types.h"
#include "predict.h"
#include "rt_nonfinite.h"
#include "selectStrongestBboxMulticlass.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWCudnnCustomLayerBase.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include "MWSortFunctors.h"
#include "MWSortWithIndexUtility.h"
#include "MWTensorBase.hpp"
#include "createPillars.hpp"
#include <cmath>
#include <cstring>

// Type Definitions
struct cell_wrap_40 {
  real_T f1[5];
};

// Variable Definitions
static detector0_0 gobj_0;

static coder::internal::pointPillarsObjectDetector pointPillarObj;

static boolean_T pointPillarObj_not_empty;

static emlrtMCInfo emlrtMCI{
    57,                                                              // lineNo
    5,                                                               // colNo
    "repmat",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/repmat.m" // pName
};

static emlrtRTEInfo emlrtRTEI{
    85,               // lineNo
    13,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    144,              // lineNo
    13,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    134,              // lineNo
    13,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    139,              // lineNo
    12,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    69,                                                              // lineNo
    28,                                                              // colNo
    "repmat",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/repmat.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    770,              // lineNo
    17,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    1,                            // lineNo
    1,                            // colNo
    "pointPillarsObjectDetector", // fName
    "/home/aiv/Documents/MATLAB/SupportPackages/R2024a/toolbox/shared/"
    "dlcoder_base/supportpackages/shared_dl_targets/+coder/+internal"
    "/pointPillarsObjectDetector.p" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    1,                                                                 // lineNo
    35,                                                                // colNo
    "pointpillarsDetect",                                              // fName
    "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/pointpillarsDetect.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    12,                                                                // lineNo
    1,                                                                 // colNo
    "pointpillarsDetect",                                              // fName
    "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/pointpillarsDetect.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    770,              // lineNo
    25,               // colNo
    "pointCloudBase", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/pointclouds/+pointclouds/"
    "+internal/pointCloudBase.m" // pName
};

static boolean_T c_gpuConstsCopied_pointpillarsD;

// Function Declarations
static void b_error(const mxArray *m, emlrtMCInfo *location);

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu);

static void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu);

static void gpuEmxMemcpyGpuToCpu_uint8_T(emxArray_uint8_T *cpu,
                                         emxArray_uint8_T *gpu);

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu);

static __global__ void
pointpillarsDetect_kernel1(real_T anchors_data[10], real_T dv1_dim0,
                           real_T dv1_dim1, real_T dv1_dim2, real_T dv1_dim3,
                           real_T dv1_dim4, real_T dv1_dim5, real_T dv1_dim6,
                           real_T dv1_dim7, real_T dv1_dim8, real_T dv1_dim9);

static __global__ void
pointpillarsDetect_kernel10(const real_T anchorsDim_data[535680],
                            real_T anchors3dMod_data[749952]);

static __global__ void
pointpillarsDetect_kernel11(const real_T anchors3dMod_data[749952],
                            real_T anchorMask_data[107136]);

static __global__ void
pointpillarsDetect_kernel12(const real_T anchorMask_data[107136],
                            real_T anchors3dMod_data[749952]);

static __global__ void
pointpillarsDetect_kernel13(const real_T anchors3dMod_data[749952],
                            real_T tmp2_data[214272], real_T tmp_data[214272]);

static __global__ void pointpillarsDetect_kernel14(const emxArray_uint8_T b,
                                                   const int32_T b_b,
                                                   emxArray_uint8_T b_value);

static __global__ void
pointpillarsDetect_kernel15(real32_T pillarIndices[24000]);

static __global__ void
pointpillarsDetect_kernel16(real32_T dlPillarFeatures_Data[10800000]);

static __global__ void
pointpillarsDetect_kernel17(real_T sparseVoxelMap[214272]);

static __global__ void
pointpillarsDetect_kernel18(const real32_T pillarIndices[24000],
                            const int32_T b, int32_T iv1_data[12000]);

static __global__ void
pointpillarsDetect_kernel19(const real_T anchorMask_data[107136],
                            boolean_T b_anchorMask_data[107136]);

static __global__ void pointpillarsDetect_kernel2(const real_T dv2[248],
                                                  real_T xx[53568],
                                                  real_T yy[53568]);

static __global__ void
pointpillarsDetect_kernel20(const boolean_T anchorMask_data[107136],
                            boolean_T b_anchorMask_data[107136]);

static __global__ void pointpillarsDetect_kernel21(
    const boolean_T anchorMask_data[107136], const real_T threshold,
    const real32_T predOcc[107136], boolean_T x[107136]);

static __global__ void
pointpillarsDetect_kernel22(const int32_T ii_data[107136], const int32_T b,
                            int32_T v1_data[107136]);

static __global__ void
pointpillarsDetect_kernel23(const int32_T v1_data[107136], const int32_T b,
                            int16_T vk_data[107136]);

static __global__ void
pointpillarsDetect_kernel24(const int16_T vk_data[107136], const int32_T b,
                            int32_T v1_data[107136],
                            int8_T varargout_6_data[107136]);

static __global__ void
pointpillarsDetect_kernel25(const int32_T v1_data[107136], const int32_T b,
                            int16_T vk_data[107136]);

static __global__ void
pointpillarsDetect_kernel26(const int16_T vk_data[107136], const int32_T b,
                            int32_T v1_data[107136]);

static __global__ void
pointpillarsDetect_kernel27(const int16_T vk_data[107136],
                            const int32_T v1_data[107136], const int32_T b,
                            int16_T col_data[107136], int32_T ii_data[107136]);

static __global__ void pointpillarsDetect_kernel28(
    const real32_T predOcc[107136], const int8_T varargout_6_data[107136],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const int32_T b, real_T yCen_data[107136], real_T anchorMask_data[107136],
    real32_T confScore_data[107136]);

static __global__ void pointpillarsDetect_kernel29(
    const real_T anchorMask_data[107136], const cell_wrap_40 newAnchors_data[2],
    const real32_T fv2[321408], const int16_T col_data[107136],
    const int32_T ii_data[107136], const int8_T varargout_6_data[107136],
    const int32_T b, real32_T xGt_data[107136]);

static __global__ void pointpillarsDetect_kernel3(const real_T yy[53568],
                                                  const real_T xx[53568],
                                                  real_T tmp[107136]);

static __global__ void pointpillarsDetect_kernel30(
    const real_T yCen_data[107136], const cell_wrap_40 newAnchors_data[2],
    const real32_T fv2[321408], const int16_T col_data[107136],
    const int32_T ii_data[107136], const int8_T varargout_6_data[107136],
    const int32_T b, real32_T yGt_data[107136]);

static __global__ void pointpillarsDetect_kernel31(
    const cell_wrap_40 newAnchors_data[2], const real32_T fv2[321408],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const int8_T varargout_6_data[107136], const int32_T b,
    real32_T zGt_data[107136]);

static __global__ void pointpillarsDetect_kernel32(
    const cell_wrap_40 newAnchors_data[2], const real32_T fv1[321408],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const int8_T varargout_6_data[107136], const int32_T b,
    real32_T lGt_data[107136]);

static __global__ void pointpillarsDetect_kernel33(
    const cell_wrap_40 newAnchors_data[2], const real32_T fv1[321408],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const int8_T varargout_6_data[107136], const int32_T b,
    real32_T wGt_data[107136]);

static __global__ void pointpillarsDetect_kernel34(
    const cell_wrap_40 newAnchors_data[2], const real32_T fv1[321408],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const int8_T varargout_6_data[107136], const int32_T b,
    real32_T hGt_data[107136]);

static __global__ void pointpillarsDetect_kernel35(
    const real32_T this_workspace_predHeading[107136],
    const int8_T varargout_6_data[107136], const int16_T col_data[107136],
    const int32_T ii_data[107136], const int32_T b, real32_T hdGt_data[107136]);

static __global__ void pointpillarsDetect_kernel36(const int32_T rowIdx,
                                                   real32_T hdGt_data[107136]);

static __global__ void pointpillarsDetect_kernel37(const int32_T rowIdx,
                                                   real32_T hdGt_data[107136]);

static __global__ void pointpillarsDetect_kernel38(real32_T predAngle[107136]);

static __global__ void pointpillarsDetect_kernel39(
    const cell_wrap_40 newAnchors_data[2], const real32_T predAngle[107136],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const real32_T hdGt_data[107136], const int8_T varargout_6_data[107136],
    const int32_T b, real32_T angGt_data[107136]);

static __global__ void pointpillarsDetect_kernel4(const real_T tmp[107136],
                                                  const int32_T initAuxVar,
                                                  real_T tmp_data[214272]);

static __global__ void pointpillarsDetect_kernel40(const int32_T rowIdx,
                                                   real32_T angGt_data[107136]);

static __global__ void pointpillarsDetect_kernel41(const int32_T rowIdx,
                                                   real32_T angGt_data[107136]);

static __global__ void pointpillarsDetect_kernel42(
    const real32_T c_this_workspace_predClassifica[107136],
    const int8_T varargout_6_data[107136], const int16_T col_data[107136],
    const int32_T ii_data[107136], const int32_T b,
    real32_T varargout_1_data[107136]);

static __global__ void
pointpillarsDetect_kernel43(const real32_T varargout_1_data[107136],
                            const int32_T b, real_T anchorMask_data[107136]);

static __global__ void
pointpillarsDetect_kernel44(const int32_T b, real_T anchorMask_data[107136]);

static __global__ void
pointpillarsDetect_kernel45(const real_T anchorMask_data[107136],
                            const int32_T b, real_T yCen_data[107136]);

static __global__ void pointpillarsDetect_kernel46(const real_T threshold,
                                                   const int32_T rowIdx,
                                                   real_T yCen_data[107136]);

static __global__ void pointpillarsDetect_kernel47(const real_T threshold,
                                                   const int32_T rowIdx,
                                                   real_T yCen_data[107136]);

static __global__ void pointpillarsDetect_kernel48(
    const real_T yCen_data[107136], const real32_T confScore_data[107136],
    const real32_T angGt_data[107136], const real32_T hGt_data[107136],
    const real32_T wGt_data[107136], const real32_T lGt_data[107136],
    const real32_T zGt_data[107136], const real32_T yGt_data[107136],
    const int32_T boxPreds_size_dim0, const real32_T xGt_data[107136],
    const int32_T b, real32_T boxPreds_data[964224]);

static __global__ void pointpillarsDetect_kernel49(
    const real32_T boxPreds_data[964224], const int32_T boxPreds_size_dim0,
    const int32_T boxPreds_size, boolean_T anchorMask_data[107136]);

static __global__ void
pointpillarsDetect_kernel5(real_T anchors3dMod_data[749952]);

static __global__ void
pointpillarsDetect_kernel50(const int32_T boxPreds_size_dim0, const int32_T b,
                            real32_T boxPreds_data[964224]);

static __global__ void
pointpillarsDetect_kernel51(const real32_T boxPreds_data[964224],
                            const int32_T boxPreds_size_dim0,
                            const int32_T ii_data[107136], const int32_T b,
                            real32_T hdGt_data[107136], real32_T scores_data[]);

static __global__ void pointpillarsDetect_kernel52(
    const real32_T scores_data[], const int32_T scores_size,
    real_T anchorMask_data[107136], real32_T varargout_1_data[107136]);

static __global__ void pointpillarsDetect_kernel53(
    const real32_T boxPreds_data[964224], const int32_T boxPreds_size_dim0_dup0,
    const int32_T ii_data[107136], const real_T anchorMask_data[107136],
    const int32_T boxPreds_size_dim0, const int32_T idx_size,
    real32_T b_boxPreds_data[535680], int8_T iv23_dim0, int8_T iv23_dim1,
    int8_T iv23_dim2, int8_T iv23_dim3, int8_T iv23_dim4);

static __global__ void
pointpillarsDetect_kernel54(const real32_T hdGt_data[107136],
                            const real_T anchorMask_data[107136],
                            const int32_T b, real32_T varargout_1_data[107136]);

static __global__ void
pointpillarsDetect_kernel55(const boolean_T selectedIndex_data[107136],
                            const int32_T selectedIndex_size,
                            boolean_T anchorMask_data[107136]);

static __global__ void pointpillarsDetect_kernel56(const int32_T b,
                                                   real_T bboxes_data[]);

static __global__ void
pointpillarsDetect_kernel57(const boolean_T anchorMask_data[107136],
                            boolean_T b_anchorMask_data[107136]);

static __global__ void
pointpillarsDetect_kernel6(const real_T tmp_data[214272],
                           real_T anchors3dMod_data[749952]);

static __global__ void
pointpillarsDetect_kernel7(const real_T anchors_data[10],
                           real_T anchorsDim_data[535680]);

static __global__ void
pointpillarsDetect_kernel8(const real_T anchorsDim_data[535680],
                           real_T anchors3dMod_data[749952]);

static __global__ void
pointpillarsDetect_kernel9(const real_T anchorsDim_data[535680],
                           real_T anchors3dMod_data[749952]);

// Function Definitions
static void b_error(const mxArray *m, emlrtMCInfo *location)
{
  const mxArray *pArray;
  pArray = m;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, nullptr, 1, &pArray, "error",
                        true, location);
}

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(
      hipMemcpy(cpu->data, gpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(real32_T),
                 hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
}

static void gpuEmxMemcpyGpuToCpu_uint8_T(emxArray_uint8_T *cpu,
                                         emxArray_uint8_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(uint8_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
}

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_boolean_T));
}

static __global__ __launch_bounds__(32, 1) void pointpillarsDetect_kernel1(
    real_T anchors_data[10], real_T dv1_dim0, real_T dv1_dim1, real_T dv1_dim2,
    real_T dv1_dim3, real_T dv1_dim4, real_T dv1_dim5, real_T dv1_dim6,
    real_T dv1_dim7, real_T dv1_dim8, real_T dv1_dim9)
{
  __shared__ real_T dv1_shared[10];
  uint64_T gThreadId;
  int32_T ibcol;
  int32_T ibmat;
  if (mwGetThreadIndexWithinBlock() == 0U) {
    dv1_shared[0] = dv1_dim0;
    dv1_shared[1] = dv1_dim1;
    dv1_shared[2] = dv1_dim2;
    dv1_shared[3] = dv1_dim3;
    dv1_shared[4] = dv1_dim4;
    dv1_shared[5] = dv1_dim5;
    dv1_shared[6] = dv1_dim6;
    dv1_shared[7] = dv1_dim7;
    dv1_shared[8] = dv1_dim8;
    dv1_shared[9] = dv1_dim9;
  }
  __syncthreads();
  gThreadId = mwGetGlobalThreadIndex();
  ibcol = static_cast<int32_T>(gThreadId % 2UL);
  ibmat =
      static_cast<int32_T>((gThreadId - static_cast<uint64_T>(ibcol)) / 2UL);
  if ((ibmat < 5) && (ibcol < 2)) {
    anchors_data[ibcol + 2 * ibmat] = dv1_shared[ibcol + (ibmat << 1)];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel10(
    const real_T anchorsDim_data[535680], real_T anchors3dMod_data[749952])
{
  int32_T ibmat;
  ibmat = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibmat < 107136) {
    anchors3dMod_data[ibmat + 642816] = anchorsDim_data[ibmat + 428544];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel11(
    const real_T anchors3dMod_data[749952], real_T anchorMask_data[107136])
{
  uint64_T gThreadId;
  int32_T ibcol;
  int32_T ibmat;
  gThreadId = mwGetGlobalThreadIndex();
  ibcol = static_cast<int32_T>(gThreadId % 53568UL);
  ibmat = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(ibcol)) /
                               53568UL);
  if ((ibmat < 2) && (ibcol < 53568)) {
    anchorMask_data[ibcol + 53568 * ibmat] =
        anchors3dMod_data[((ibcol << 1) + 107136 * (4 - ibmat)) + 1];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel12(
    const real_T anchorMask_data[107136], real_T anchors3dMod_data[749952])
{
  uint64_T gThreadId;
  int32_T ibcol;
  int32_T ibmat;
  gThreadId = mwGetGlobalThreadIndex();
  ibcol = static_cast<int32_T>(gThreadId % 53568UL);
  ibmat = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(ibcol)) /
                               53568UL);
  if ((ibmat < 2) && (ibcol < 53568)) {
    anchors3dMod_data[((ibcol << 1) + 107136 * (ibmat + 3)) + 1] =
        anchorMask_data[ibcol + 53568 * ibmat];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel13(
    const real_T anchors3dMod_data[749952], real_T tmp2_data[214272],
    real_T tmp_data[214272])
{
  uint64_T gThreadId;
  int32_T ibcol;
  int32_T ibmat;
  gThreadId = mwGetGlobalThreadIndex();
  ibcol = static_cast<int32_T>(gThreadId % 107136UL);
  ibmat = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(ibcol)) /
                               107136UL);
  if ((ibmat < 2) && (ibcol < 107136)) {
    real_T d;
    real_T d1;
    d = anchors3dMod_data[ibcol + 107136 * ibmat];
    d1 = anchors3dMod_data[ibcol + 107136 * (ibmat + 3)];
    tmp_data[ibcol + 107136 * ibmat] = d - d1 / 2.0;
    tmp2_data[ibcol + 107136 * ibmat] = d + d1 / 2.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel14(
    const emxArray_uint8_T b, const int32_T b_b, emxArray_uint8_T b_value)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    b_value.data[ibmat] = b.data[ibmat];
  }
}

static __global__ __launch_bounds__(224, 1) void pointpillarsDetect_kernel15(
    real32_T pillarIndices[24000])
{
  int32_T ibmat;
  ibmat = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibmat < 24000) {
    pillarIndices[ibmat] = 0.0F;
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel16(
    real32_T dlPillarFeatures_Data[10800000])
{
  int32_T ibmat;
  ibmat = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibmat < 10800000) {
    dlPillarFeatures_Data[ibmat] = 0.0F;
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel17(
    real_T sparseVoxelMap[214272])
{
  int32_T ibmat;
  ibmat = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibmat < 214272) {
    sparseVoxelMap[ibmat] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel18(
    const real32_T pillarIndices[24000], const int32_T b,
    int32_T iv1_data[12000])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    iv1_data[ibmat] =
        static_cast<int32_T>(pillarIndices[ibmat]) +
        432 * (static_cast<int32_T>(pillarIndices[ibmat + 12000]) - 1);
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel19(
    const real_T anchorMask_data[107136], boolean_T b_anchorMask_data[107136])
{
  int32_T ibmat;
  ibmat = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibmat < 107136) {
    b_anchorMask_data[ibmat] = (anchorMask_data[ibmat] > 1.0);
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel2(
    const real_T dv2[248], real_T xx[53568], real_T yy[53568])
{
  uint64_T gThreadId;
  int32_T ibcol;
  int32_T ibmat;
  gThreadId = mwGetGlobalThreadIndex();
  ibmat = static_cast<int32_T>(gThreadId % 216UL);
  ibcol =
      static_cast<int32_T>((gThreadId - static_cast<uint64_T>(ibmat)) / 216UL);
  if ((ibcol < 248) && (ibmat < 216)) {
    yy[ibmat + 216 * ibcol] = dv2[ibcol];
    xx[ibmat + 216 * ibcol] =
        0.32000000000000006 * static_cast<real_T>(ibmat) + 0.16;
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel20(
    const boolean_T anchorMask_data[107136],
    boolean_T b_anchorMask_data[107136])
{
  int32_T ibmat;
  ibmat = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibmat < 107136) {
    b_anchorMask_data[ibmat] = anchorMask_data[ibmat];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel21(
    const boolean_T anchorMask_data[107136], const real_T threshold,
    const real32_T predOcc[107136], boolean_T x[107136])
{
  int32_T ibmat;
  ibmat = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibmat < 107136) {
    x[ibmat] = ((static_cast<real_T>(predOcc[ibmat]) > threshold) &&
                anchorMask_data[ibmat]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel22(
    const int32_T ii_data[107136], const int32_T b, int32_T v1_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    v1_data[ibmat] = ii_data[ibmat] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel23(
    const int32_T v1_data[107136], const int32_T b, int16_T vk_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    vk_data[ibmat] = static_cast<int16_T>(v1_data[ibmat] / 53568);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel24(
    const int16_T vk_data[107136], const int32_T b, int32_T v1_data[107136],
    int8_T varargout_6_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    int16_T i;
    ibmat = static_cast<int32_T>(idx);
    i = vk_data[ibmat];
    varargout_6_data[ibmat] = static_cast<int8_T>(static_cast<int32_T>(i) + 1);
    v1_data[ibmat] -= static_cast<int32_T>(i) * 53568;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel25(
    const int32_T v1_data[107136], const int32_T b, int16_T vk_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    vk_data[ibmat] = static_cast<int16_T>(v1_data[ibmat] / 216);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel26(
    const int16_T vk_data[107136], const int32_T b, int32_T v1_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    v1_data[ibmat] -= static_cast<int32_T>(vk_data[ibmat]) * 216;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel27(
    const int16_T vk_data[107136], const int32_T v1_data[107136],
    const int32_T b, int16_T col_data[107136], int32_T ii_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    ii_data[ibmat] = v1_data[ibmat] + 1;
    col_data[ibmat] =
        static_cast<int16_T>(static_cast<int32_T>(vk_data[ibmat]) + 1);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel28(
    const real32_T predOcc[107136], const int8_T varargout_6_data[107136],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const int32_T b, real_T yCen_data[107136], real_T anchorMask_data[107136],
    real32_T confScore_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    int32_T k;
    int16_T i;
    k = static_cast<int32_T>(idx);
    ibmat = ii_data[k];
    i = col_data[k];
    confScore_data[k] =
        predOcc[((ibmat + 216 * (static_cast<int32_T>(i) - 1)) +
                 53568 * (static_cast<int32_T>(varargout_6_data[k]) - 1)) -
                1];
    anchorMask_data[k] = 0.32 * (static_cast<real_T>(ibmat) - 1.0) + 0.16;
    yCen_data[k] = (0.32 * (static_cast<real_T>(i) - 1.0) - 39.68) + 0.16;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel29(
    const real_T anchorMask_data[107136], const cell_wrap_40 newAnchors_data[2],
    const real32_T fv2[321408], const int16_T col_data[107136],
    const int32_T ii_data[107136], const int8_T varargout_6_data[107136],
    const int32_T b, real32_T xGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    int8_T i1;
    k = static_cast<int32_T>(idx);
    i1 = varargout_6_data[k];
    xGt_data[k] =
        fv2[((ii_data[k] + 216 * (static_cast<int32_T>(col_data[k]) - 1)) +
             53568 * (static_cast<int32_T>(i1) - 1)) -
            1] *
            static_cast<real32_T>(
                sqrt(newAnchors_data[static_cast<int32_T>(i1) - 1].f1[0] *
                         newAnchors_data[static_cast<int32_T>(i1) - 1].f1[0] +
                     newAnchors_data[static_cast<int32_T>(i1) - 1].f1[1] *
                         newAnchors_data[static_cast<int32_T>(i1) - 1].f1[1])) +
        static_cast<real32_T>(anchorMask_data[k]);
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel3(
    const real_T yy[53568], const real_T xx[53568], real_T tmp[107136])
{
  int32_T ibcol;
  ibcol = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibcol < 53568) {
    tmp[ibcol] = xx[ibcol];
    tmp[ibcol + 53568] = yy[ibcol];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel30(
    const real_T yCen_data[107136], const cell_wrap_40 newAnchors_data[2],
    const real32_T fv2[321408], const int16_T col_data[107136],
    const int32_T ii_data[107136], const int8_T varargout_6_data[107136],
    const int32_T b, real32_T yGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    int8_T i1;
    k = static_cast<int32_T>(idx);
    i1 = varargout_6_data[k];
    yGt_data[k] =
        fv2[((ii_data[k] + 216 * (static_cast<int32_T>(col_data[k]) - 1)) +
             53568 * (static_cast<int32_T>(i1) - 1)) +
            107135] *
            static_cast<real32_T>(
                sqrt(newAnchors_data[static_cast<int32_T>(i1) - 1].f1[0] *
                         newAnchors_data[static_cast<int32_T>(i1) - 1].f1[0] +
                     newAnchors_data[static_cast<int32_T>(i1) - 1].f1[1] *
                         newAnchors_data[static_cast<int32_T>(i1) - 1].f1[1])) +
        static_cast<real32_T>(yCen_data[k]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel31(
    const cell_wrap_40 newAnchors_data[2], const real32_T fv2[321408],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const int8_T varargout_6_data[107136], const int32_T b,
    real32_T zGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    int8_T i1;
    k = static_cast<int32_T>(idx);
    i1 = varargout_6_data[k];
    zGt_data[k] =
        fv2[((ii_data[k] + 216 * (static_cast<int32_T>(col_data[k]) - 1)) +
             53568 * (static_cast<int32_T>(i1) - 1)) +
            214271] *
            static_cast<real32_T>(
                newAnchors_data[static_cast<int32_T>(i1) - 1].f1[2]) +
        static_cast<real32_T>(
            newAnchors_data[static_cast<int32_T>(i1) - 1].f1[3]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel32(
    const cell_wrap_40 newAnchors_data[2], const real32_T fv1[321408],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const int8_T varargout_6_data[107136], const int32_T b,
    real32_T lGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    int8_T i1;
    k = static_cast<int32_T>(idx);
    i1 = varargout_6_data[k];
    lGt_data[k] =
        expf(fv1[((ii_data[k] + 216 * (static_cast<int32_T>(col_data[k]) - 1)) +
                  53568 * (static_cast<int32_T>(i1) - 1)) -
                 1]) *
        static_cast<real32_T>(
            newAnchors_data[static_cast<int32_T>(i1) - 1].f1[0]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel33(
    const cell_wrap_40 newAnchors_data[2], const real32_T fv1[321408],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const int8_T varargout_6_data[107136], const int32_T b,
    real32_T wGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    int8_T i1;
    k = static_cast<int32_T>(idx);
    i1 = varargout_6_data[k];
    wGt_data[k] =
        expf(fv1[((ii_data[k] + 216 * (static_cast<int32_T>(col_data[k]) - 1)) +
                  53568 * (static_cast<int32_T>(i1) - 1)) +
                 107135]) *
        static_cast<real32_T>(
            newAnchors_data[static_cast<int32_T>(i1) - 1].f1[1]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel34(
    const cell_wrap_40 newAnchors_data[2], const real32_T fv1[321408],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const int8_T varargout_6_data[107136], const int32_T b,
    real32_T hGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    int8_T i1;
    k = static_cast<int32_T>(idx);
    i1 = varargout_6_data[k];
    hGt_data[k] =
        expf(fv1[((ii_data[k] + 216 * (static_cast<int32_T>(col_data[k]) - 1)) +
                  53568 * (static_cast<int32_T>(i1) - 1)) +
                 214271]) *
        static_cast<real32_T>(
            newAnchors_data[static_cast<int32_T>(i1) - 1].f1[2]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel35(
    const real32_T this_workspace_predHeading[107136],
    const int8_T varargout_6_data[107136], const int16_T col_data[107136],
    const int32_T ii_data[107136], const int32_T b, real32_T hdGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    hdGt_data[k] = this_workspace_predHeading
        [((ii_data[k] + 216 * (static_cast<int32_T>(col_data[k]) - 1)) +
          53568 * (static_cast<int32_T>(varargout_6_data[k]) - 1)) -
         1];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel36(
    const int32_T rowIdx, real32_T hdGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    if (hdGt_data[ibmat] >= 0.5F) {
      hdGt_data[ibmat] = 1.0F;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel37(
    const int32_T rowIdx, real32_T hdGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    if (hdGt_data[ibmat] < 0.5F) {
      hdGt_data[ibmat] = -1.0F;
    }
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel38(
    real32_T predAngle[107136])
{
  int32_T ibmat;
  ibmat = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibmat < 107136) {
    real32_T f;
    f = predAngle[ibmat];
    if ((f > 1.0F) || (f < -1.0F)) {
      predAngle[ibmat] = 0.0F;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel39(
    const cell_wrap_40 newAnchors_data[2], const real32_T predAngle[107136],
    const int16_T col_data[107136], const int32_T ii_data[107136],
    const real32_T hdGt_data[107136], const int8_T varargout_6_data[107136],
    const int32_T b, real32_T angGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    int8_T i1;
    k = static_cast<int32_T>(idx);
    i1 = varargout_6_data[k];
    angGt_data[k] =
        hdGt_data[k] *
            asinf(predAngle[((ii_data[k] +
                              216 * (static_cast<int32_T>(col_data[k]) - 1)) +
                             53568 * (static_cast<int32_T>(i1) - 1)) -
                            1]) +
        static_cast<real32_T>(
            newAnchors_data[static_cast<int32_T>(i1) - 1].f1[4]);
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel4(
    const real_T tmp[107136], const int32_T initAuxVar, real_T tmp_data[214272])
{
  uint64_T gThreadId;
  int32_T iacol;
  int32_T ibmat;
  gThreadId = mwGetGlobalThreadIndex();
  ibmat = static_cast<int32_T>(gThreadId % 53568UL);
  iacol = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(ibmat)) /
                               53568UL);
  if ((iacol < 2) && (ibmat < 53568)) {
    real_T numPillars;
    int32_T colIdx;
    int32_T rowIdx;
    colIdx = (initAuxVar + iacol) + 1;
    rowIdx = ibmat * 2 + -1;
    numPillars = tmp[ibmat + 53568 * iacol];
    tmp_data[(rowIdx + 107136 * colIdx) + 1] = numPillars;
    tmp_data[(rowIdx + 107136 * colIdx) + 2] = numPillars;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel40(
    const int32_T rowIdx, real32_T angGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    real32_T f;
    ibmat = static_cast<int32_T>(idx);
    f = angGt_data[ibmat];
    if (static_cast<real_T>(f) > 3.1415926535897931) {
      f -= 6.28318548F;
      angGt_data[ibmat] = f;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel41(
    const int32_T rowIdx, real32_T angGt_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    real32_T f;
    ibmat = static_cast<int32_T>(idx);
    f = angGt_data[ibmat];
    if (static_cast<real_T>(f) < -3.1415926535897931) {
      f += 6.28318548F;
      angGt_data[ibmat] = f;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel42(
    const real32_T c_this_workspace_predClassifica[107136],
    const int8_T varargout_6_data[107136], const int16_T col_data[107136],
    const int32_T ii_data[107136], const int32_T b,
    real32_T varargout_1_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    varargout_1_data[k] = c_this_workspace_predClassifica
        [((ii_data[k] + 216 * (static_cast<int32_T>(col_data[k]) - 1)) +
          53568 * (static_cast<int32_T>(varargout_6_data[k]) - 1)) -
         1];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel43(
    const real32_T varargout_1_data[107136], const int32_T b,
    real_T anchorMask_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    anchorMask_data[ibmat] = -static_cast<real_T>(varargout_1_data[ibmat]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel44(
    const int32_T b, real_T anchorMask_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    anchorMask_data[k] = exp(anchorMask_data[k]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel45(
    const real_T anchorMask_data[107136], const int32_T b,
    real_T yCen_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    yCen_data[ibmat] = 1.0 / (anchorMask_data[ibmat] + 1.0);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel46(
    const real_T threshold, const int32_T rowIdx, real_T yCen_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    if (yCen_data[ibmat] >= threshold) {
      yCen_data[ibmat] = 1.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel47(
    const real_T threshold, const int32_T rowIdx, real_T yCen_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(rowIdx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    if (yCen_data[ibmat] < threshold) {
      yCen_data[ibmat] = 0.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel48(
    const real_T yCen_data[107136], const real32_T confScore_data[107136],
    const real32_T angGt_data[107136], const real32_T hGt_data[107136],
    const real32_T wGt_data[107136], const real32_T lGt_data[107136],
    const real32_T zGt_data[107136], const real32_T yGt_data[107136],
    const int32_T boxPreds_size_dim0, const real32_T xGt_data[107136],
    const int32_T b, real32_T boxPreds_data[964224])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    boxPreds_data[ibmat] = xGt_data[ibmat];
    boxPreds_data[ibmat + boxPreds_size_dim0] = yGt_data[ibmat];
    boxPreds_data[ibmat + boxPreds_size_dim0 * 2] = 0.5F * zGt_data[ibmat];
    boxPreds_data[ibmat + boxPreds_size_dim0 * 3] = lGt_data[ibmat];
    boxPreds_data[ibmat + boxPreds_size_dim0 * 4] = wGt_data[ibmat];
    boxPreds_data[ibmat + boxPreds_size_dim0 * 5] = hGt_data[ibmat];
    boxPreds_data[ibmat + boxPreds_size_dim0 * 6] = angGt_data[ibmat];
    boxPreds_data[ibmat + boxPreds_size_dim0 * 7] = confScore_data[ibmat];
    boxPreds_data[ibmat + boxPreds_size_dim0 * 8] =
        static_cast<real32_T>(yCen_data[ibmat]);
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel49(
    const real32_T boxPreds_data[964224], const int32_T boxPreds_size_dim0,
    const int32_T boxPreds_size, boolean_T anchorMask_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(boxPreds_size);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    anchorMask_data[ibmat] =
        (static_cast<real_T>(boxPreds_data[ibmat + boxPreds_size_dim0 * 8]) >
         0.15);
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel5(
    real_T anchors3dMod_data[749952])
{
  int32_T ibmat;
  ibmat = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibmat < 749952) {
    anchors3dMod_data[ibmat] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel50(
    const int32_T boxPreds_size_dim0, const int32_T b,
    real32_T boxPreds_data[964224])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    boxPreds_data[ibmat + boxPreds_size_dim0 * 6] *= 57.2957802F;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel51(
    const real32_T boxPreds_data[964224], const int32_T boxPreds_size_dim0,
    const int32_T ii_data[107136], const int32_T b, real32_T hdGt_data[107136],
    real32_T scores_data[])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibcol;
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    ibcol = ii_data[ibmat];
    scores_data[ibmat] = boxPreds_data[(ibcol + boxPreds_size_dim0 * 7) - 1];
    hdGt_data[ibmat] = boxPreds_data[(ibcol + boxPreds_size_dim0 * 8) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel52(
    const real32_T scores_data[], const int32_T scores_size,
    real_T anchorMask_data[107136], real32_T varargout_1_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(scores_size);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    varargout_1_data[ibmat] = scores_data[ibmat];
    anchorMask_data[ibmat] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel53(
    const real32_T boxPreds_data[964224], const int32_T boxPreds_size_dim0_dup0,
    const int32_T ii_data[107136], const real_T anchorMask_data[107136],
    const int32_T boxPreds_size_dim0, const int32_T idx_size,
    real32_T b_boxPreds_data[535680], int8_T iv23_dim0, int8_T iv23_dim1,
    int8_T iv23_dim2, int8_T iv23_dim3, int8_T iv23_dim4)
{
  __shared__ int8_T iv23_shared[5];
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  if (mwGetThreadIndexWithinBlock() == 0U) {
    iv23_shared[0] = iv23_dim0;
    iv23_shared[1] = iv23_dim1;
    iv23_shared[2] = iv23_dim2;
    iv23_shared[3] = iv23_dim3;
    iv23_shared[4] = iv23_dim4;
  }
  __syncthreads();
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = 5UL * (static_cast<uint64_T>(idx_size) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibcol;
    int32_T ibmat;
    ibcol = static_cast<int32_T>(idx % (static_cast<uint64_T>(idx_size) + 1UL));
    ibmat = static_cast<int32_T>((idx - static_cast<uint64_T>(ibcol)) /
                                 (static_cast<uint64_T>(idx_size) + 1UL));
    b_boxPreds_data[ibcol + boxPreds_size_dim0 * ibmat] = boxPreds_data
        [(ii_data[static_cast<int32_T>(anchorMask_data[ibcol]) - 1] +
          boxPreds_size_dim0_dup0 * static_cast<int32_T>(iv23_shared[ibmat])) -
         1];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel54(
    const real32_T hdGt_data[107136], const real_T anchorMask_data[107136],
    const int32_T b, real32_T varargout_1_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    varargout_1_data[ibmat] =
        hdGt_data[static_cast<int32_T>(anchorMask_data[ibmat]) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel55(
    const boolean_T selectedIndex_data[107136],
    const int32_T selectedIndex_size, boolean_T anchorMask_data[107136])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(selectedIndex_size);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    anchorMask_data[ibmat] = selectedIndex_data[ibmat];
  }
}

static __global__ __launch_bounds__(1024, 1) void pointpillarsDetect_kernel56(
    const int32_T b, real_T bboxes_data[])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ibmat;
    ibmat = static_cast<int32_T>(idx);
    bboxes_data[ibmat] = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel57(
    const boolean_T anchorMask_data[107136],
    boolean_T b_anchorMask_data[107136])
{
  uint64_T gThreadId;
  int32_T ibcol;
  int32_T ibmat;
  int32_T k;
  gThreadId = mwGetGlobalThreadIndex();
  ibcol = static_cast<int32_T>(gThreadId % 2UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(ibcol)) / 2UL;
  ibmat = static_cast<int32_T>(gThreadId % 216UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(ibmat)) / 216UL;
  k = static_cast<int32_T>(gThreadId);
  if ((k < 248) && (ibmat < 216) && (ibcol < 2)) {
    b_anchorMask_data[(ibmat + 216 * k) + 53568 * ibcol] =
        anchorMask_data[(ibcol + 2 * ibmat) + 432 * k];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel6(
    const real_T tmp_data[214272], real_T anchors3dMod_data[749952])
{
  uint64_T gThreadId;
  int32_T ibcol;
  int32_T ibmat;
  gThreadId = mwGetGlobalThreadIndex();
  ibcol = static_cast<int32_T>(gThreadId % 107136UL);
  ibmat = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(ibcol)) /
                               107136UL);
  if ((ibmat < 2) && (ibcol < 107136)) {
    anchors3dMod_data[ibcol + 107136 * ibmat] =
        tmp_data[ibcol + 107136 * ibmat];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel7(
    const real_T anchors_data[10], real_T anchorsDim_data[535680])
{
  uint64_T gThreadId;
  int32_T itilerow;
  int32_T jcol;
  int32_T k;
  gThreadId = mwGetGlobalThreadIndex();
  k = static_cast<int32_T>(gThreadId % 2UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(k)) / 2UL;
  itilerow = static_cast<int32_T>(gThreadId % 53568UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(itilerow)) / 53568UL;
  jcol = static_cast<int32_T>(gThreadId);
  if ((jcol < 5) && (itilerow < 53568) && (k < 2)) {
    anchorsDim_data[(jcol * 107136 + (itilerow << 1)) + k] =
        anchors_data[(jcol << 1) + k];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel8(
    const real_T anchorsDim_data[535680], real_T anchors3dMod_data[749952])
{
  int32_T ibmat;
  ibmat = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (ibmat < 107136) {
    anchors3dMod_data[ibmat + 214272] = anchorsDim_data[ibmat + 321408];
  }
}

static __global__ __launch_bounds__(512, 1) void pointpillarsDetect_kernel9(
    const real_T anchorsDim_data[535680], real_T anchors3dMod_data[749952])
{
  uint64_T gThreadId;
  int32_T ibcol;
  int32_T ibmat;
  gThreadId = mwGetGlobalThreadIndex();
  ibcol = static_cast<int32_T>(gThreadId % 107136UL);
  ibmat = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(ibcol)) /
                               107136UL);
  if ((ibmat < 3) && (ibcol < 107136)) {
    anchors3dMod_data[ibcol + 107136 * (ibmat + 3)] =
        anchorsDim_data[ibcol + 107136 * ibmat];
  }
}

void MWScatterLayer_detector0_01::cleanupLayer()
{
}

void MWResize2DLayer_detector0_01::cleanupLayer()
{
}

void MWResize2DLayer_detector0_02::cleanupLayer()
{
}

void MWScatterLayer_detector0_01::createCustomLayer(
    MWCudnnTarget::MWTargetNetworkImpl *targetImpl, MWTensorBase *b,
    MWTensorBase *c, const char_T *d, int32_T e)
{
  setupTensors(2, 1, b, c, d, e);
  setupLayer(targetImpl);
}

void MWResize2DLayer_detector0_01::createCustomLayer(
    MWCudnnTarget::MWTargetNetworkImpl *targetImpl, MWTensorBase *b,
    const char_T *c, int32_T d)
{
  setupTensors(1, 1, b, c, d);
  setupLayer(targetImpl);
}

void MWResize2DLayer_detector0_02::createCustomLayer(
    MWCudnnTarget::MWTargetNetworkImpl *targetImpl, MWTensorBase *b,
    const char_T *c, int32_T d)
{
  setupTensors(1, 1, b, c, d);
  setupLayer(targetImpl);
}

void MWResize2DLayer_detector0_01::predict()
{
  reorderInputData(getInputTensor(0), 0);
  coder::internal::l_dlnetwork_layerPredictWithCol(
      m_ntwkImpl->getPermuteBuffer(0), m_ntwkImpl->getPermuteBuffer(1));
  reorderOutputData(getOutputTensor(0), 1);
}

void MWResize2DLayer_detector0_02::predict()
{
  reorderInputData(getInputTensor(0), 0);
  coder::internal::m_dlnetwork_layerPredictWithCol(
      m_ntwkImpl->getPermuteBuffer(0), m_ntwkImpl->getPermuteBuffer(1));
  reorderOutputData(getOutputTensor(0), 1);
}

void MWScatterLayer_detector0_01::predict()
{
  reorderInputData(getInputTensor(0), 0);
  reorderInputData(getInputTensor(1), 1);
  coder::internal::l_dlnetwork_layerPredictWithCol(
      m_ntwkImpl->getPermuteBuffer(0), m_ntwkImpl->getPermuteBuffer(1),
      m_ntwkImpl->getPermuteBuffer(2));
  reorderOutputData(getOutputTensor(0), 2);
}

void MWScatterLayer_detector0_01::propagateSize()
{
  int32_T batchSize;
  MWTensorBase::DIMSLABEL b_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL c_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL d_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL layout[5]{MWTensorBase::S, MWTensorBase::N,
                                    MWTensorBase::C, MWTensorBase::W,
                                    MWTensorBase::H};
  batchSize = getInputTensor(0)->getBatchSize();
  resizeOutputTensor(432, 496, 64, batchSize, 1, 0);
  setupInputDescriptors(getInputTensor(0), b_layout, layout);
  setupInputDescriptors(getInputTensor(1), c_layout, layout);
  setupOutputDescriptors(getOutputTensor(0), layout, d_layout);
}

void MWResize2DLayer_detector0_01::propagateSize()
{
  int32_T batchSize;
  MWTensorBase::DIMSLABEL b_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL c_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL layout[5]{MWTensorBase::S, MWTensorBase::N,
                                    MWTensorBase::C, MWTensorBase::W,
                                    MWTensorBase::H};
  batchSize = getInputTensor(0)->getBatchSize();
  resizeOutputTensor(216, 248, 128, batchSize, 1, 0);
  setupInputDescriptors(getInputTensor(0), b_layout, layout);
  setupOutputDescriptors(getOutputTensor(0), layout, c_layout);
}

void MWResize2DLayer_detector0_02::propagateSize()
{
  int32_T batchSize;
  MWTensorBase::DIMSLABEL b_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL c_layout[5]{MWTensorBase::S, MWTensorBase::N,
                                      MWTensorBase::C, MWTensorBase::H,
                                      MWTensorBase::W};
  MWTensorBase::DIMSLABEL layout[5]{MWTensorBase::S, MWTensorBase::N,
                                    MWTensorBase::C, MWTensorBase::W,
                                    MWTensorBase::H};
  batchSize = getInputTensor(0)->getBatchSize();
  resizeOutputTensor(216, 248, 128, batchSize, 1, 0);
  setupInputDescriptors(getInputTensor(0), b_layout, layout);
  setupOutputDescriptors(getOutputTensor(0), layout, c_layout);
}

void pointpillarsDetect(pointpillarsDetectStackData *SD,
                        const emxArray_real32_T *dataLoc,
                        const emxArray_real32_T *dataInt, real_T threshold,
                        real_T cpu_bboxes_data[], int32_T bboxes_size[2],
                        real32_T cpu_scores_data[], int32_T scores_size[1],
                        coder::categorical *labels)
{
  static const real_T cpu_dv2[248]{-39.52,
                                   -39.2,
                                   -38.88,
                                   -38.56,
                                   -38.24,
                                   -37.92,
                                   -37.6,
                                   -37.28,
                                   -36.96,
                                   -36.64,
                                   -36.32,
                                   -36.0,
                                   -35.68,
                                   -35.36,
                                   -35.04,
                                   -34.72,
                                   -34.4,
                                   -34.08,
                                   -33.76,
                                   -33.44,
                                   -33.12,
                                   -32.8,
                                   -32.480000000000004,
                                   -32.160000000000004,
                                   -31.84,
                                   -31.52,
                                   -31.2,
                                   -30.88,
                                   -30.560000000000002,
                                   -30.240000000000002,
                                   -29.92,
                                   -29.6,
                                   -29.28,
                                   -28.96,
                                   -28.64,
                                   -28.32,
                                   -28.0,
                                   -27.68,
                                   -27.36,
                                   -27.04,
                                   -26.72,
                                   -26.400000000000002,
                                   -26.080000000000002,
                                   -25.76,
                                   -25.44,
                                   -25.12,
                                   -24.8,
                                   -24.48,
                                   -24.16,
                                   -23.84,
                                   -23.52,
                                   -23.2,
                                   -22.88,
                                   -22.56,
                                   -22.240000000000002,
                                   -21.92,
                                   -21.6,
                                   -21.28,
                                   -20.96,
                                   -20.64,
                                   -20.32,
                                   -20.0,
                                   -19.68,
                                   -19.36,
                                   -19.04,
                                   -18.72,
                                   -18.400000000000002,
                                   -18.080000000000002,
                                   -17.76,
                                   -17.44,
                                   -17.12,
                                   -16.8,
                                   -16.48,
                                   -16.16,
                                   -15.84,
                                   -15.52,
                                   -15.200000000000001,
                                   -14.88,
                                   -14.56,
                                   -14.24,
                                   -13.92,
                                   -13.6,
                                   -13.280000000000001,
                                   -12.96,
                                   -12.64,
                                   -12.32,
                                   -12.0,
                                   -11.68,
                                   -11.36,
                                   -11.040000000000001,
                                   -10.72,
                                   -10.4,
                                   -10.08,
                                   -9.76,
                                   -9.44,
                                   -9.120000000000001,
                                   -8.8,
                                   -8.48,
                                   -8.16,
                                   -7.84,
                                   -7.5200000000000005,
                                   -7.2,
                                   -6.88,
                                   -6.5600000000000005,
                                   -6.24,
                                   -5.92,
                                   -5.6000000000000005,
                                   -5.28,
                                   -4.96,
                                   -4.64,
                                   -4.32,
                                   -4.0,
                                   -3.68,
                                   -3.36,
                                   -3.04,
                                   -2.72,
                                   -2.4,
                                   -2.08,
                                   -1.76,
                                   -1.44,
                                   -1.12,
                                   -0.8,
                                   -0.48,
                                   -0.16,
                                   0.16,
                                   0.48,
                                   0.8,
                                   1.12,
                                   1.44,
                                   1.76,
                                   2.08,
                                   2.4,
                                   2.72,
                                   3.04,
                                   3.36,
                                   3.68,
                                   4.0,
                                   4.32,
                                   4.64,
                                   4.96,
                                   5.28,
                                   5.6000000000000005,
                                   5.92,
                                   6.24,
                                   6.5600000000000005,
                                   6.88,
                                   7.2,
                                   7.5200000000000005,
                                   7.84,
                                   8.16,
                                   8.48,
                                   8.8,
                                   9.120000000000001,
                                   9.44,
                                   9.76,
                                   10.08,
                                   10.4,
                                   10.72,
                                   11.040000000000001,
                                   11.36,
                                   11.68,
                                   12.0,
                                   12.32,
                                   12.64,
                                   12.96,
                                   13.280000000000001,
                                   13.6,
                                   13.92,
                                   14.24,
                                   14.56,
                                   14.88,
                                   15.200000000000001,
                                   15.52,
                                   15.84,
                                   16.16,
                                   16.48,
                                   16.8,
                                   17.12,
                                   17.44,
                                   17.76,
                                   18.080000000000002,
                                   18.400000000000002,
                                   18.72,
                                   19.04,
                                   19.36,
                                   19.68,
                                   20.0,
                                   20.32,
                                   20.64,
                                   20.96,
                                   21.28,
                                   21.6,
                                   21.92,
                                   22.240000000000002,
                                   22.56,
                                   22.88,
                                   23.2,
                                   23.52,
                                   23.84,
                                   24.16,
                                   24.48,
                                   24.8,
                                   25.12,
                                   25.44,
                                   25.76,
                                   26.080000000000002,
                                   26.400000000000002,
                                   26.72,
                                   27.04,
                                   27.36,
                                   27.68,
                                   28.0,
                                   28.32,
                                   28.64,
                                   28.96,
                                   29.28,
                                   29.6,
                                   29.92,
                                   30.240000000000002,
                                   30.560000000000002,
                                   30.88,
                                   31.2,
                                   31.52,
                                   31.84,
                                   32.160000000000004,
                                   32.480000000000004,
                                   32.8,
                                   33.12,
                                   33.44,
                                   33.76,
                                   34.08,
                                   34.4,
                                   34.72,
                                   35.04,
                                   35.36,
                                   35.68,
                                   36.0,
                                   36.32,
                                   36.64,
                                   36.96,
                                   37.28,
                                   37.6,
                                   37.92,
                                   38.24,
                                   38.56,
                                   38.88,
                                   39.2,
                                   39.52};
  static const real_T dv1[10]{0.824900632523975,
                              0.824900632523975,
                              0.93545500918179725,
                              0.93545500918179725,
                              1.7943301367067914,
                              1.7943301367067914,
                              -1.78,
                              -1.78,
                              0.0,
                              1.5707963267948966};
  static const int32_T iv[2]{1, 15};
  static const char_T u[15]{'M', 'A', 'T', 'L', 'A', 'B', ':', 'p',
                            'm', 'a', 'x', 's', 'i', 'z', 'e'};
  static const int8_T iv27[7]{0, 1, 2, 3, 4, 5, 8};
  static const int8_T iv23[5]{0, 1, 3, 4, 6};
  static const int8_T dv4[3]{2, 3, 1};
  cell_wrap_40 cpu_newAnchors_data[2];
  cell_wrap_40(*gpu_newAnchors_data)[2];
  coder::b_pointCloud ptCloud;
  coder::pointCloud ptCloudUnorg;
  dim3 block;
  dim3 grid;
  emxArray_boolean_T gpu_indices;
  emxArray_boolean_T *cpu_indices;
  emxArray_real32_T gpu_intensity;
  emxArray_real32_T gpu_location;
  emxArray_real32_T gpu_normals;
  emxArray_real32_T gpu_rangeData;
  emxArray_real32_T *cpu_intensity;
  emxArray_real32_T *cpu_location;
  emxArray_real32_T *cpu_normals;
  emxArray_real32_T *cpu_rangeData;
  emxArray_real32_T *dataIn;
  emxArray_uint8_T gpu_b;
  emxArray_uint8_T gpu_value;
  emxArray_uint8_T *cpu_b;
  emxArray_uint8_T *cpu_value;
  const mxArray *m;
  const mxArray *y;
  real_T(*gpu_anchors3dMod_data)[749952];
  real_T(*gpu_anchorsDim_data)[535680];
  real_T(*gpu_sparseVoxelMap)[214272];
  real_T(*gpu_tmp2_data)[214272];
  real_T(*gpu_tmp_data)[214272];
  real_T(*gpu_anchorMask_data)[107136];
  real_T(*gpu_tmp)[107136];
  real_T(*gpu_yCen_data)[107136];
  real_T(*gpu_xx)[53568];
  real_T(*gpu_yy)[53568];
  real_T(*gpu_anchors_data)[10];
  real_T numPillars;
  real_T *gpu_bboxes_data;
  int32_T(*gpu_ii_data)[107136];
  int32_T(*gpu_v1_data)[107136];
  int32_T cpu_iv1_data[12000];
  int32_T(*gpu_iv1_data)[12000];
  int32_T boxPreds_size[2];
  int32_T outsize[2];
  int32_T colIdx;
  int32_T iacol;
  int32_T ibcol;
  int32_T ibmat;
  int32_T itilerow;
  int32_T jcol;
  int32_T nx;
  int32_T rowIdx;
  real32_T(*gpu_dlPillarFeatures_Data)[10800000];
  real32_T(*gpu_boxPreds_data)[964224];
  real32_T(*b_gpu_boxPreds_data)[535680];
  real32_T(*gpu_fv1)[321408];
  real32_T(*gpu_fv2)[321408];
  real32_T(*c_gpu_this_workspace_predClassi)[107136];
  real32_T(*gpu_angGt_data)[107136];
  real32_T(*gpu_confScore_data)[107136];
  real32_T(*gpu_hGt_data)[107136];
  real32_T(*gpu_hdGt_data)[107136];
  real32_T(*gpu_lGt_data)[107136];
  real32_T(*gpu_predAngle)[107136];
  real32_T(*gpu_predOcc)[107136];
  real32_T(*gpu_this_workspace_predHeading)[107136];
  real32_T(*gpu_varargout_1_data)[107136];
  real32_T(*gpu_wGt_data)[107136];
  real32_T(*gpu_xGt_data)[107136];
  real32_T(*gpu_yGt_data)[107136];
  real32_T(*gpu_zGt_data)[107136];
  real32_T cpu_pillarIndices[24000];
  real32_T(*gpu_pillarIndices)[24000];
  real32_T *gpu_scores_data;
  uint32_T numPoints[2];
  int16_T(*gpu_col_data)[107136];
  int16_T(*gpu_vk_data)[107136];
  int8_T(*gpu_varargout_6_data)[107136];
  int8_T input_sizes_idx_1;
  boolean_T(*b_gpu_anchorMask_data)[107136];
  boolean_T(*c_gpu_anchorMask_data)[107136];
  boolean_T(*gpu_selectedIndex_data)[107136];
  boolean_T(*gpu_x)[107136];
  boolean_T empty_non_axis_sizes;
  boolean_T exitg1;
  boolean_T indices_outdatedOnCpu;
  boolean_T indices_outdatedOnGpu;
  boolean_T intensity_outdatedOnCpu;
  boolean_T location_outdatedOnCpu;
  boolean_T location_outdatedOnGpu;
  boolean_T normals_outdatedOnCpu;
  boolean_T normals_outdatedOnGpu;
  boolean_T rangeData_outdatedOnCpu;
  boolean_T rangeData_outdatedOnGpu;
  boolean_T validLaunchParams;
  boolean_T value_outdatedOnCpu;
  boolean_T value_outdatedOnGpu;
  if (!c_gpuConstsCopied_pointpillarsD) {
    c_gpuConstsCopied_pointpillarsD = true;
    checkCudaError(hipMemcpy(*global_gpu_dv2, cpu_dv2, sizeof(real_T[248]),
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  checkCudaError(mwCudaMalloc(&gpu_selectedIndex_data, 107136UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_boxPreds_data, 2142720UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_boxPreds_data, 3856896UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_varargout_1_data, 428544UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_angGt_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_hdGt_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_hGt_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_wGt_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_lGt_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_zGt_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_yGt_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_xGt_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_confScore_data, 428544UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_yCen_data, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_col_data, 214272UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_varargout_6_data, 107136UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_vk_data, 214272UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_v1_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_ii_data, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_x, 107136UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&c_gpu_anchorMask_data, 107136UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_newAnchors_data, 80UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_anchorMask_data, 107136UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_iv1_data, 48000UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_sparseVoxelMap, 1714176UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_predAngle, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_this_workspace_predHeading, 428544UL),
                 __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&c_gpu_this_workspace_predClassi, 428544UL),
                 __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_predOcc, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_fv2, 1285632UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_fv1, 1285632UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_dlPillarFeatures_Data, 43200000UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_pillarIndices, 96000UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_tmp2_data, 1714176UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_anchorMask_data, 857088UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_anchorsDim_data, 4285440UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_anchors3dMod_data, 5999616UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_tmp_data, 1714176UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_tmp, 857088UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_xx, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_yy, 428544UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_anchors_data, 80UL), __FILE__, __LINE__);
  gpuEmxReset_uint8_T(&gpu_b);
  gpuEmxReset_real32_T(&gpu_rangeData);
  gpuEmxReset_real32_T(&gpu_intensity);
  gpuEmxReset_real32_T(&gpu_normals);
  gpuEmxReset_uint8_T(&gpu_value);
  gpuEmxReset_real32_T(&gpu_location);
  gpuEmxReset_boolean_T(&gpu_indices);
  checkCudaError(mwCudaMalloc(&gpu_scores_data, 107136U * sizeof(real32_T)),
                 __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_bboxes_data, 964224U * sizeof(real_T)),
                 __FILE__, __LINE__);
  rangeData_outdatedOnCpu = false;
  rangeData_outdatedOnGpu = false;
  intensity_outdatedOnCpu = false;
  empty_non_axis_sizes = false;
  normals_outdatedOnCpu = false;
  normals_outdatedOnGpu = false;
  value_outdatedOnCpu = false;
  value_outdatedOnGpu = false;
  location_outdatedOnCpu = false;
  location_outdatedOnGpu = false;
  indices_outdatedOnCpu = false;
  indices_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInitStruct_pointCloud(&ptCloud, &i_emlrtRTEI, true);
  emxInitStruct_pointCloud1(&ptCloudUnorg, &g_emlrtRTEI, true);
  emxInit_boolean_T(&cpu_indices, 1, &h_emlrtRTEI, true);
  emxInit_real32_T(&cpu_location, 2, &h_emlrtRTEI, true);
  emxInit_uint8_T(&cpu_value, 2, &f_emlrtRTEI, true);
  emxInit_real32_T(&cpu_normals, 2, &h_emlrtRTEI, true);
  emxInit_real32_T(&cpu_intensity, 2, &h_emlrtRTEI, true);
  emxInit_real32_T(&cpu_rangeData, 2, &h_emlrtRTEI, true);
  emxInit_uint8_T(&cpu_b, 2, &j_emlrtRTEI, true);
  emxInit_real32_T(&dataIn, 2, &g_emlrtRTEI, true);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emlrtPushHeapReferenceStackR2021a(
      emlrtRootTLSGlobal, false, &ptCloud,
      (void *)static_cast<void (*)(coder::b_pointCloud *)>(
          &coder::handle_matlabCodegenDestructor),
      nullptr, nullptr, nullptr);
  ptCloud.matlabCodegenIsDeleted = true;
  //  Predict the output of network and extract the confidence, x, y,
  //  width, height, and class.
  //  load the deep learning network for prediction
  if (!pointPillarObj_not_empty) {
    coder::internal::dlnetwork_setup(&gobj_0);
    gobj_0.matlabCodegenIsDeleted = false;
    pointPillarObj.Network = &gobj_0;
    pointpillarsDetect_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        *gpu_anchors_data, dv1[0], dv1[1], dv1[2], dv1[3], dv1[4], dv1[5],
        dv1[6], dv1[7], dv1[8], dv1[9]);
    pointpillarsDetect_kernel2<<<dim3(105U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *global_gpu_dv2, *gpu_xx, *gpu_yy);
    pointpillarsDetect_kernel3<<<dim3(105U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_yy, *gpu_xx, *gpu_tmp);
    pointpillarsDetect_kernel4<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_tmp, -1, *gpu_tmp_data);
    pointpillarsDetect_kernel5<<<dim3(1465U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_anchors3dMod_data);
    pointpillarsDetect_kernel6<<<dim3(419U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_tmp_data, *gpu_anchors3dMod_data);
    pointpillarsDetect_kernel7<<<dim3(1047U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_anchors_data, *gpu_anchorsDim_data);
    pointpillarsDetect_kernel8<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_anchorsDim_data, *gpu_anchors3dMod_data);
    pointpillarsDetect_kernel9<<<dim3(628U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_anchorsDim_data, *gpu_anchors3dMod_data);
    pointpillarsDetect_kernel10<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_anchorsDim_data, *gpu_anchors3dMod_data);
    pointpillarsDetect_kernel11<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_anchors3dMod_data, *gpu_anchorMask_data);
    pointpillarsDetect_kernel12<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_anchorMask_data, *gpu_anchors3dMod_data);
    pointpillarsDetect_kernel13<<<dim3(419U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_anchors3dMod_data, *gpu_tmp2_data, *gpu_tmp_data);
    empty_non_axis_sizes = true;
    rangeData_outdatedOnCpu = true;
    pointPillarObj.AnchorsBEV.size[0] = 107136;
    pointPillarObj.AnchorsBEV.size[1] = 4;
    for (iacol = 0; iacol < 214272; iacol++) {
      if (empty_non_axis_sizes) {
        checkCudaError(hipMemcpy(SD->f3.cpu_tmp_data, *gpu_tmp_data, 1714176UL,
                                  hipMemcpyDeviceToHost),
                       __FILE__, __LINE__);
      }
      empty_non_axis_sizes = false;
      pointPillarObj.AnchorsBEV.data[iacol] = SD->f3.cpu_tmp_data[iacol];
    }
    for (iacol = 0; iacol < 214272; iacol++) {
      if (rangeData_outdatedOnCpu) {
        checkCudaError(hipMemcpy(SD->f3.cpu_tmp2_data, *gpu_tmp2_data,
                                  1714176UL, hipMemcpyDeviceToHost),
                       __FILE__, __LINE__);
      }
      rangeData_outdatedOnCpu = false;
      pointPillarObj.AnchorsBEV.data[iacol + 214272] =
          SD->f3.cpu_tmp2_data[iacol];
    }
    pointPillarObj_not_empty = true;
  }
  ptCloud.HasKdtreeConstructed = false;
  ptCloud.HasLocationHandleAllocated = false;
  ibmat = ptCloud.Location->size[0] * ptCloud.Location->size[1];
  ptCloud.Location->size[0] = dataLoc->size[0];
  ptCloud.Location->size[1] = 3;
  emxEnsureCapacity_real32_T(ptCloud.Location, ibmat, &emlrtRTEI);
  for (ibmat = 0; ibmat < dataLoc->size[0] * 3; ibmat++) {
    ptCloud.Location->data[ibmat] = dataLoc->data[ibmat];
  }
  ptCloud.Color->size[0] = 0;
  ptCloud.Color->size[1] = 0;
  ptCloud.Normal->size[0] = 0;
  ptCloud.Normal->size[1] = 0;
  ibmat = ptCloud.Intensity->size[0];
  ptCloud.Intensity->size[0] = dataInt->size[0];
  emxEnsureCapacity_real32_T(ptCloud.Intensity, ibmat, &b_emlrtRTEI);
  for (ibmat = 0; ibmat < dataInt->size[0]; ibmat++) {
    ptCloud.Intensity->data[ibmat] = dataInt->data[ibmat];
  }
  ptCloud.RangeData->size[0] = 0;
  ptCloud.RangeData->size[1] = 0;
  ptCloud.PointCloudArrayData->size[0] = 1;
  ptCloud.PointCloudArrayData->size[1] = 1;
  ptCloud.Kdtree = nullptr;
  ptCloud.LocationHandle = nullptr;
  ptCloud.matlabCodegenIsDeleted = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emlrtPushHeapReferenceStackR2021a(
      emlrtRootTLSGlobal, false, &ptCloudUnorg,
      (void *)static_cast<void (*)(coder::pointCloud *)>(
          &coder::handle_matlabCodegenDestructor),
      nullptr, nullptr, nullptr);
  ptCloudUnorg.matlabCodegenIsDeleted = true;
  coder::vision::internal::codegen::gpu::d_PointCloudImpl_extractValidPo(
      ptCloud.Location, cpu_indices, &indices_outdatedOnCpu, &gpu_indices,
      &indices_outdatedOnGpu);
  coder::vision::internal::codegen::gpu::PointCloudImpl_subsetImpl(
      ptCloud.Location, ptCloud.Color, ptCloud.Normal, ptCloud.Intensity,
      ptCloud.RangeData, cpu_indices, &gpu_indices, &indices_outdatedOnGpu,
      cpu_location, &location_outdatedOnCpu, &gpu_location,
      &location_outdatedOnGpu, cpu_value, &value_outdatedOnCpu, &gpu_value,
      &value_outdatedOnGpu, cpu_normals, &normals_outdatedOnCpu, &gpu_normals,
      &normals_outdatedOnGpu, cpu_intensity, &intensity_outdatedOnCpu,
      &gpu_intensity, &empty_non_axis_sizes, cpu_rangeData,
      &rangeData_outdatedOnCpu, &gpu_rangeData, &rangeData_outdatedOnGpu);
  ptCloudUnorg.HasKdtreeConstructed = false;
  ptCloudUnorg.HasLocationHandleAllocated = false;
  ibmat = ptCloudUnorg.Location->size[0] * ptCloudUnorg.Location->size[1];
  ptCloudUnorg.Location->size[0] = cpu_location->size[0];
  ptCloudUnorg.Location->size[1] = 3;
  emxEnsureCapacity_real32_T(ptCloudUnorg.Location, ibmat, &emlrtRTEI);
  for (ibmat = 0; ibmat < cpu_location->size[0] * 3; ibmat++) {
    if (location_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_location, &gpu_location);
    }
    location_outdatedOnCpu = false;
    ptCloudUnorg.Location->data[ibmat] = cpu_location->data[ibmat];
  }
  for (ibmat = 0; ibmat < 2; ibmat++) {
    numPoints[ibmat] = static_cast<uint32_T>(cpu_value->size[ibmat]);
  }
  empty_non_axis_sizes = true;
  nx = 0;
  exitg1 = false;
  while ((!exitg1) && (nx < 2)) {
    if (static_cast<int32_T>(numPoints[nx]) != (nx << 1) + 1) {
      empty_non_axis_sizes = false;
      exitg1 = true;
    } else {
      nx++;
    }
  }
  if (empty_non_axis_sizes) {
    numPoints[0] = static_cast<uint32_T>(ptCloudUnorg.Location->size[0]);
    outsize[0] = cpu_value->size[0] * static_cast<int32_T>(numPoints[0]);
    if (!(outsize[0] == static_cast<real_T>(cpu_value->size[0]) *
                            static_cast<real_T>(numPoints[0]))) {
      y = nullptr;
      m = emlrtCreateCharArray(2, &iv[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 15, m, &u[0]);
      emlrtAssign(&y, m);
      b_error(y, &emlrtMCI);
    }
    empty_non_axis_sizes = false;
    ibmat = cpu_b->size[0] * cpu_b->size[1];
    cpu_b->size[0] = outsize[0];
    cpu_b->size[1] = 3;
    emxEnsureCapacity_uint8_T(cpu_b, ibmat, &e_emlrtRTEI);
    colIdx = cpu_value->size[0];
    rowIdx = static_cast<int32_T>(numPoints[0]);
    for (jcol = 0; jcol < 3; jcol++) {
      iacol = jcol * colIdx;
      ibmat = jcol * (colIdx * rowIdx);
      for (itilerow = 0; itilerow < rowIdx; itilerow++) {
        ibcol = ibmat + itilerow * colIdx;
        for (nx = 0; nx < colIdx; nx++) {
          if (value_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_uint8_T(cpu_value, &gpu_value);
          }
          value_outdatedOnCpu = false;
          cpu_b->data[ibcol + nx] = cpu_value->data[iacol + nx];
          empty_non_axis_sizes = true;
        }
      }
    }
    ibmat = cpu_value->size[0] * cpu_value->size[1];
    cpu_value->size[0] = cpu_b->size[0];
    cpu_value->size[1] = 3;
    emxEnsureCapacity_uint8_T(cpu_value, ibmat, &f_emlrtRTEI);
    nx = cpu_b->size[0] * 3 - 1;
    mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_uint8_T(cpu_b, &gpu_b, !empty_non_axis_sizes);
    gpuEmxEnsureCapacity_uint8_T(cpu_value, &gpu_value, true);
    if (empty_non_axis_sizes) {
      gpuEmxMemcpyCpuToGpu_uint8_T(&gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillarsDetect_kernel14<<<grid, block>>>(gpu_b, nx, gpu_value);
    }
    value_outdatedOnCpu = true;
  }
  ibmat = ptCloudUnorg.Color->size[0] * ptCloudUnorg.Color->size[1];
  ptCloudUnorg.Color->size[0] = cpu_value->size[0];
  ptCloudUnorg.Color->size[1] = 3;
  emxEnsureCapacity_uint8_T(ptCloudUnorg.Color, ibmat, &c_emlrtRTEI);
  for (ibmat = 0; ibmat < cpu_value->size[0] * 3; ibmat++) {
    if (value_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_uint8_T(cpu_value, &gpu_value);
    }
    value_outdatedOnCpu = false;
    ptCloudUnorg.Color->data[ibmat] = cpu_value->data[ibmat];
  }
  ibmat = ptCloudUnorg.Normal->size[0] * ptCloudUnorg.Normal->size[1];
  ptCloudUnorg.Normal->size[0] = cpu_normals->size[0];
  ptCloudUnorg.Normal->size[1] = 3;
  emxEnsureCapacity_real32_T(ptCloudUnorg.Normal, ibmat, &d_emlrtRTEI);
  for (ibmat = 0; ibmat < cpu_normals->size[0] * 3; ibmat++) {
    if (normals_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_normals, &gpu_normals);
    }
    normals_outdatedOnCpu = false;
    ptCloudUnorg.Normal->data[ibmat] = cpu_normals->data[ibmat];
  }
  ibmat = ptCloudUnorg.Intensity->size[0] * ptCloudUnorg.Intensity->size[1];
  ptCloudUnorg.Intensity->size[0] = cpu_intensity->size[0];
  ptCloudUnorg.Intensity->size[1] = 1;
  emxEnsureCapacity_real32_T(ptCloudUnorg.Intensity, ibmat, &b_emlrtRTEI);
  for (ibmat = 0; ibmat < cpu_intensity->size[0]; ibmat++) {
    if (intensity_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_intensity, &gpu_intensity);
    }
    intensity_outdatedOnCpu = false;
    ptCloudUnorg.Intensity->data[ibmat] = cpu_intensity->data[ibmat];
  }
  ptCloudUnorg.Kdtree = nullptr;
  ptCloudUnorg.LocationHandle = nullptr;
  ptCloudUnorg.matlabCodegenIsDeleted = false;
  if (ptCloudUnorg.Location->size[0] != 0) {
    iacol = ptCloudUnorg.Location->size[0];
  } else if ((ptCloudUnorg.Intensity->size[0] != 0) &&
             (ptCloudUnorg.Intensity->size[1] != 0)) {
    iacol = ptCloudUnorg.Intensity->size[0];
  } else {
    iacol = 0;
    if (ptCloudUnorg.Intensity->size[0] > 0) {
      iacol = ptCloudUnorg.Intensity->size[0];
    }
  }
  empty_non_axis_sizes = (iacol == 0);
  if (empty_non_axis_sizes || (ptCloudUnorg.Location->size[0] != 0)) {
    input_sizes_idx_1 = 3;
  } else {
    input_sizes_idx_1 = 0;
  }
  if (empty_non_axis_sizes || ((ptCloudUnorg.Intensity->size[0] != 0) &&
                               (ptCloudUnorg.Intensity->size[1] != 0))) {
    colIdx = ptCloudUnorg.Intensity->size[1];
  } else {
    colIdx = 0;
  }
  ibmat = dataIn->size[0] * dataIn->size[1];
  dataIn->size[0] = iacol;
  dataIn->size[1] = input_sizes_idx_1 + colIdx;
  emxEnsureCapacity_real32_T(dataIn, ibmat, &g_emlrtRTEI);
  rowIdx = input_sizes_idx_1;
  for (ibmat = 0; ibmat < rowIdx; ibmat++) {
    for (ibcol = 0; ibcol < iacol; ibcol++) {
      dataIn->data[ibcol + dataIn->size[0] * ibmat] =
          ptCloudUnorg.Location->data[ibcol + iacol * ibmat];
    }
  }
  for (ibmat = 0; ibmat < colIdx; ibmat++) {
    for (ibcol = 0; ibcol < iacol; ibcol++) {
      dataIn->data[ibcol + dataIn->size[0] * (ibmat + input_sizes_idx_1)] =
          ptCloudUnorg.Intensity->data[ibcol + iacol * ibmat];
    }
  }
  pointpillarsDetect_kernel15<<<dim3(108U, 1U, 1U), dim3(224U, 1U, 1U)>>>(
      *gpu_pillarIndices);
  pointpillarsDetect_kernel16<<<dim3(21094U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_dlPillarFeatures_Data);
  checkCudaError(hipMemcpy(SD->f3.cpu_dlPillarFeatures_Data,
                            *gpu_dlPillarFeatures_Data, 43200000UL,
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(cpu_pillarIndices, *gpu_pillarIndices, 96000UL,
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  createPillarsImplSingle(&SD->f3.cpu_dlPillarFeatures_Data[0],
                          &cpu_pillarIndices[0], &dataIn->data[0],
                          dataIn->size[0], 100, 12000, 0.16F, 0.16F, 0.0F,
                          69.12F, -39.68F, 39.68F, -5.0F, 5.0F);
  checkCudaError(hipMemcpy(*gpu_pillarIndices, cpu_pillarIndices, 96000UL,
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(*gpu_dlPillarFeatures_Data,
                            SD->f3.cpu_dlPillarFeatures_Data, 43200000UL,
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  coder::internal::dlnetwork_predict(
      pointPillarObj.Network, *gpu_pillarIndices, *gpu_dlPillarFeatures_Data,
      *gpu_fv1, *gpu_fv2, *gpu_predOcc, *c_gpu_this_workspace_predClassi,
      *gpu_this_workspace_predHeading, *gpu_predAngle);
  pointpillarsDetect_kernel17<<<dim3(419U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_sparseVoxelMap);
  empty_non_axis_sizes = false;
  rangeData_outdatedOnGpu = true;
  iacol = 0;
  for (nx = 0; nx < 24000; nx++) {
    if (cpu_pillarIndices[nx] != 0.0F) {
      iacol++;
    }
  }
  numPillars = static_cast<real_T>(iacol) / 2.0;
  if (numPillars < 1.0) {
    ibmat = 0;
  } else {
    ibmat = static_cast<int32_T>(numPillars);
  }
  nx = ibmat;
  mwGetLaunchParameters1D(computeNumIters(ibmat - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel18<<<grid, block>>>(*gpu_pillarIndices, ibmat - 1,
                                                 *gpu_iv1_data);
  }
  rangeData_outdatedOnCpu = true;
  for (ibmat = 0; ibmat < nx; ibmat++) {
    if (rangeData_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_sparseVoxelMap, *gpu_sparseVoxelMap,
                                1714176UL, hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    if (rangeData_outdatedOnCpu) {
      checkCudaError(hipMemcpy(cpu_iv1_data, *gpu_iv1_data, 48000UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    rangeData_outdatedOnCpu = false;
    SD->f3.cpu_sparseVoxelMap[cpu_iv1_data[ibmat] - 1] = 1.0;
    rangeData_outdatedOnGpu = false;
    empty_non_axis_sizes = true;
  }
  if (empty_non_axis_sizes) {
    checkCudaError(hipMemcpy(*gpu_sparseVoxelMap, SD->f3.cpu_sparseVoxelMap,
                              1714176UL, hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  mwCallThrustScanNDEdge(&(*gpu_sparseVoxelMap)[0], 432, false, 214272);
  for (ibmat = 0; ibmat < 2; ibmat++) {
    outsize[ibmat] = (ibmat << 6) + 432;
  }
  mwCallThrustScanNDOther(&(*gpu_sparseVoxelMap)[0], 2, &outsize[0], 1, false,
                          214272, false);
  rangeData_outdatedOnGpu = true;
  empty_non_axis_sizes = false;
  for (nx = 0; nx < 107136; nx++) {
    real_T xmaxGrid;
    real_T ymaxGrid;
    real_T yminGrid;
    numPillars =
        std::fmax(std::floor(pointPillarObj.AnchorsBEV.data[nx] / 0.16), 1.0);
    yminGrid = std::fmax(
        std::floor((pointPillarObj.AnchorsBEV.data[nx + 107136] - -39.68) /
                   0.16),
        1.0);
    xmaxGrid = std::fmin(
        std::floor(pointPillarObj.AnchorsBEV.data[nx + 214272] / 0.16), 432.0);
    ymaxGrid = std::fmin(
        std::floor((pointPillarObj.AnchorsBEV.data[nx + 321408] - -39.68) /
                   0.16),
        496.0);
    if (rangeData_outdatedOnGpu) {
      checkCudaError(hipMemcpy(SD->f3.cpu_sparseVoxelMap, *gpu_sparseVoxelMap,
                                1714176UL, hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    rangeData_outdatedOnGpu = false;
    SD->f3.cpu_anchorMask_data[nx] =
        ((SD->f3
              .cpu_sparseVoxelMap[(static_cast<int32_T>(xmaxGrid) +
                                   432 * (static_cast<int32_T>(ymaxGrid) - 1)) -
                                  1] -
          SD->f3
              .cpu_sparseVoxelMap[(static_cast<int32_T>(xmaxGrid) +
                                   432 * (static_cast<int32_T>(yminGrid) - 1)) -
                                  1]) -
         SD->f3
             .cpu_sparseVoxelMap[(static_cast<int32_T>(numPillars) +
                                  432 * (static_cast<int32_T>(ymaxGrid) - 1)) -
                                 1]) +
        SD->f3.cpu_sparseVoxelMap[(static_cast<int32_T>(numPillars) +
                                   432 * (static_cast<int32_T>(yminGrid) - 1)) -
                                  1];
    empty_non_axis_sizes = true;
  }
  if (empty_non_axis_sizes) {
    checkCudaError(hipMemcpy(*gpu_anchorMask_data, SD->f3.cpu_anchorMask_data,
                              857088UL, hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  pointpillarsDetect_kernel19<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_anchorMask_data, *b_gpu_anchorMask_data);
  rangeData_outdatedOnCpu = false;
  for (iacol = 0; iacol < 2; iacol++) {
    numPillars = static_cast<real_T>(iacol) + 1.0;
    for (ibmat = 0; ibmat < 5; ibmat++) {
      cpu_newAnchors_data[static_cast<int32_T>(numPillars) - 1].f1[ibmat] =
          dv1[iacol + (ibmat << 1)];
      rangeData_outdatedOnCpu = true;
    }
  }
  empty_non_axis_sizes = true;
  colIdx = 0;
  nx = 0;
  exitg1 = false;
  while ((!exitg1) && (nx + 1 < 4)) {
    if (colIdx > dv4[nx]) {
      empty_non_axis_sizes = false;
      exitg1 = true;
    } else {
      colIdx = dv4[nx];
      nx++;
    }
  }
  if (empty_non_axis_sizes) {
    pointpillarsDetect_kernel20<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *b_gpu_anchorMask_data, *c_gpu_anchorMask_data);
  } else {
    pointpillarsDetect_kernel57<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *b_gpu_anchorMask_data, *c_gpu_anchorMask_data);
  }
  pointpillarsDetect_kernel21<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *c_gpu_anchorMask_data, threshold, *gpu_predOcc, *gpu_x);
  empty_non_axis_sizes = true;
  rowIdx = 0;
  rangeData_outdatedOnGpu = false;
  iacol = 1;
  exitg1 = false;
  while ((!exitg1) && (iacol < 107137)) {
    if (empty_non_axis_sizes) {
      checkCudaError(
          hipMemcpy(SD->f3.cpu_x, *gpu_x, 107136UL, hipMemcpyDeviceToHost),
          __FILE__, __LINE__);
    }
    empty_non_axis_sizes = false;
    if (SD->f3.cpu_x[iacol - 1]) {
      rowIdx++;
      SD->f3.cpu_ii_data[rowIdx - 1] = iacol;
      rangeData_outdatedOnGpu = true;
      if (rowIdx >= 107136) {
        exitg1 = true;
      } else {
        iacol++;
      }
    } else {
      iacol++;
    }
  }
  if (rowIdx < 1) {
    itilerow = 0;
  } else {
    itilerow = rowIdx;
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  if (rangeData_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_ii_data, SD->f3.cpu_ii_data, 428544UL,
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel22<<<grid, block>>>(*gpu_ii_data, itilerow - 1,
                                                 *gpu_v1_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel23<<<grid, block>>>(*gpu_v1_data, itilerow - 1,
                                                 *gpu_vk_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel24<<<grid, block>>>(
        *gpu_vk_data, itilerow - 1, *gpu_v1_data, *gpu_varargout_6_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel25<<<grid, block>>>(*gpu_v1_data, itilerow - 1,
                                                 *gpu_vk_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel26<<<grid, block>>>(*gpu_vk_data, itilerow - 1,
                                                 *gpu_v1_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel27<<<grid, block>>>(
        *gpu_vk_data, *gpu_v1_data, itilerow - 1, *gpu_col_data, *gpu_ii_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel28<<<grid, block>>>(
        *gpu_predOcc, *gpu_varargout_6_data, *gpu_col_data, *gpu_ii_data,
        itilerow - 1, *gpu_yCen_data, *gpu_anchorMask_data,
        *gpu_confScore_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  if (rangeData_outdatedOnCpu) {
    checkCudaError(hipMemcpy(*gpu_newAnchors_data, cpu_newAnchors_data, 80UL,
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel29<<<grid, block>>>(
        *gpu_anchorMask_data, *gpu_newAnchors_data, *gpu_fv2, *gpu_col_data,
        *gpu_ii_data, *gpu_varargout_6_data, itilerow - 1, *gpu_xGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel30<<<grid, block>>>(
        *gpu_yCen_data, *gpu_newAnchors_data, *gpu_fv2, *gpu_col_data,
        *gpu_ii_data, *gpu_varargout_6_data, itilerow - 1, *gpu_yGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel31<<<grid, block>>>(
        *gpu_newAnchors_data, *gpu_fv2, *gpu_col_data, *gpu_ii_data,
        *gpu_varargout_6_data, itilerow - 1, *gpu_zGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel32<<<grid, block>>>(
        *gpu_newAnchors_data, *gpu_fv1, *gpu_col_data, *gpu_ii_data,
        *gpu_varargout_6_data, itilerow - 1, *gpu_lGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel33<<<grid, block>>>(
        *gpu_newAnchors_data, *gpu_fv1, *gpu_col_data, *gpu_ii_data,
        *gpu_varargout_6_data, itilerow - 1, *gpu_wGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel34<<<grid, block>>>(
        *gpu_newAnchors_data, *gpu_fv1, *gpu_col_data, *gpu_ii_data,
        *gpu_varargout_6_data, itilerow - 1, *gpu_hGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel35<<<grid, block>>>(
        *gpu_this_workspace_predHeading, *gpu_varargout_6_data, *gpu_col_data,
        *gpu_ii_data, itilerow - 1, *gpu_hdGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel36<<<grid, block>>>(itilerow - 1, *gpu_hdGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel37<<<grid, block>>>(itilerow - 1, *gpu_hdGt_data);
  }
  pointpillarsDetect_kernel38<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_predAngle);
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel39<<<grid, block>>>(
        *gpu_newAnchors_data, *gpu_predAngle, *gpu_col_data, *gpu_ii_data,
        *gpu_hdGt_data, *gpu_varargout_6_data, itilerow - 1, *gpu_angGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel40<<<grid, block>>>(itilerow - 1, *gpu_angGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel41<<<grid, block>>>(itilerow - 1, *gpu_angGt_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel42<<<grid, block>>>(
        *c_gpu_this_workspace_predClassi, *gpu_varargout_6_data, *gpu_col_data,
        *gpu_ii_data, itilerow - 1, *gpu_varargout_1_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel43<<<grid, block>>>(
        *gpu_varargout_1_data, itilerow - 1, *gpu_anchorMask_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel44<<<grid, block>>>(itilerow - 1,
                                                 *gpu_anchorMask_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel45<<<grid, block>>>(*gpu_anchorMask_data,
                                                 itilerow - 1, *gpu_yCen_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel46<<<grid, block>>>(threshold, itilerow - 1,
                                                 *gpu_yCen_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel47<<<grid, block>>>(threshold, itilerow - 1,
                                                 *gpu_yCen_data);
  }
  mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    pointpillarsDetect_kernel48<<<grid, block>>>(
        *gpu_yCen_data, *gpu_confScore_data, *gpu_angGt_data, *gpu_hGt_data,
        *gpu_wGt_data, *gpu_lGt_data, *gpu_zGt_data, *gpu_yGt_data, itilerow,
        *gpu_xGt_data, itilerow - 1, *gpu_boxPreds_data);
  }
  if (itilerow != 0) {
    mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillarsDetect_kernel49<<<grid, block>>>(
          *gpu_boxPreds_data, itilerow, itilerow - 1, *b_gpu_anchorMask_data);
    }
    normals_outdatedOnGpu = true;
    rowIdx = 0;
    rangeData_outdatedOnGpu = false;
    jcol = itilerow;
    iacol = 1;
    exitg1 = false;
    while ((!exitg1) && (iacol <= itilerow)) {
      if (normals_outdatedOnGpu) {
        checkCudaError(hipMemcpy(SD->f3.b_cpu_anchorMask_data,
                                  *b_gpu_anchorMask_data, 107136UL,
                                  hipMemcpyDeviceToHost),
                       __FILE__, __LINE__);
      }
      normals_outdatedOnGpu = false;
      if (SD->f3.b_cpu_anchorMask_data[iacol - 1]) {
        rowIdx++;
        SD->f3.cpu_ii_data[rowIdx - 1] = iacol;
        rangeData_outdatedOnGpu = true;
        if (rowIdx >= itilerow) {
          exitg1 = true;
        } else {
          iacol++;
        }
      } else {
        iacol++;
      }
    }
    if (itilerow == 1) {
      if (rowIdx == 0) {
        rangeData_outdatedOnGpu = false;
        jcol = 0;
      }
    } else if (rowIdx < 1) {
      jcol = 0;
    } else {
      jcol = rowIdx;
    }
    mwGetLaunchParameters1D(computeNumIters(itilerow - 1), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      pointpillarsDetect_kernel50<<<grid, block>>>(itilerow, itilerow - 1,
                                                   *gpu_boxPreds_data);
    }
    value_outdatedOnGpu = true;
    if (jcol != 0) {
      int32_T selectedIndex_size[1];
      scores_size[0] = jcol;
      mwGetLaunchParameters1D(computeNumIters(jcol - 1), &grid, &block,
                              2147483647U);
      if (rangeData_outdatedOnGpu) {
        checkCudaError(hipMemcpy(*gpu_ii_data, SD->f3.cpu_ii_data, 428544UL,
                                  hipMemcpyHostToDevice),
                       __FILE__, __LINE__);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillarsDetect_kernel51<<<grid, block>>>(
            *gpu_boxPreds_data, itilerow, *gpu_ii_data, jcol - 1,
            *gpu_hdGt_data, gpu_scores_data);
      }
      rangeData_outdatedOnCpu = true;
      colIdx = 2;
      if (jcol != 1) {
        colIdx = 1;
      }
      outsize[0] = jcol;
      outsize[1] = 1;
      nx = scores_size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block, 2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillarsDetect_kernel52<<<grid, block>>>(
            gpu_scores_data, nx, *gpu_anchorMask_data, *gpu_varargout_1_data);
      }
      gpu_thrust_sort_with_index(&(*gpu_varargout_1_data)[0],
                                 &(*gpu_anchorMask_data)[0], 2, &outsize[0],
                                 colIdx, 'd', false);
      empty_non_axis_sizes = true;
      boxPreds_size[0] = jcol;
      boxPreds_size[1] = 5;
      mwGetLaunchParameters1D(computeNumIters(4, jcol - 1), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillarsDetect_kernel53<<<grid, block>>>(
            *gpu_boxPreds_data, itilerow, *gpu_ii_data, *gpu_anchorMask_data,
            jcol, jcol - 1, *b_gpu_boxPreds_data, iv23[0], iv23[1], iv23[2],
            iv23[3], iv23[4]);
      }
      mwGetLaunchParameters1D(computeNumIters(jcol - 1), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillarsDetect_kernel54<<<grid, block>>>(
            *gpu_hdGt_data, *gpu_anchorMask_data, jcol - 1,
            *gpu_varargout_1_data);
      }
      checkCudaError(hipMemcpy(SD->f3.b_cpu_boxPreds_data,
                                *b_gpu_boxPreds_data, 2142720UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
      checkCudaError(hipMemcpy(SD->f3.cpu_varargout_1_data,
                                *gpu_varargout_1_data, 428544UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
      coder::b_selectStrongestBboxCodegen(
          SD, SD->f3.b_cpu_boxPreds_data, boxPreds_size,
          SD->f3.cpu_varargout_1_data, SD->f3.cpu_selectedIndex_data,
          selectedIndex_size);
      nx = selectedIndex_size[0];
      mwGetLaunchParameters1D(computeNumIters(selectedIndex_size[0] - 1), &grid,
                              &block, 2147483647U);
      checkCudaError(hipMemcpy(*gpu_selectedIndex_data,
                                SD->f3.cpu_selectedIndex_data, 107136UL,
                                hipMemcpyHostToDevice),
                     __FILE__, __LINE__);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillarsDetect_kernel55<<<grid, block>>>(*gpu_selectedIndex_data,
                                                     selectedIndex_size[0] - 1,
                                                     *b_gpu_anchorMask_data);
      }
      normals_outdatedOnGpu = true;
      for (ibmat = 0; ibmat < nx; ibmat++) {
        if (empty_non_axis_sizes) {
          checkCudaError(hipMemcpy(SD->f3.cpu_anchorMask_data,
                                    *gpu_anchorMask_data, 857088UL,
                                    hipMemcpyDeviceToHost),
                         __FILE__, __LINE__);
        }
        empty_non_axis_sizes = false;
        if (normals_outdatedOnGpu) {
          checkCudaError(hipMemcpy(SD->f3.b_cpu_anchorMask_data,
                                    *b_gpu_anchorMask_data, 107136UL,
                                    hipMemcpyDeviceToHost),
                         __FILE__, __LINE__);
        }
        normals_outdatedOnGpu = false;
        SD->f3.cpu_selectedIndex_data[static_cast<int32_T>(
                                          SD->f3.cpu_anchorMask_data[ibmat]) -
                                      1] = SD->f3.b_cpu_anchorMask_data[ibmat];
      }
      rowIdx = selectedIndex_size[0] - 1;
      iacol = 0;
      colIdx = 0;
      for (ibmat = 0; ibmat <= rowIdx; ibmat++) {
        if (SD->f3.cpu_selectedIndex_data[ibmat]) {
          iacol++;
          if (rangeData_outdatedOnCpu) {
            checkCudaError(hipMemcpy(cpu_scores_data, gpu_scores_data,
                                      static_cast<uint32_T>(scores_size[0]) *
                                          sizeof(real32_T),
                                      hipMemcpyDeviceToHost),
                           __FILE__, __LINE__);
          }
          cpu_scores_data[colIdx] = cpu_scores_data[ibmat];
          rangeData_outdatedOnCpu = false;
          colIdx++;
        }
      }
      scores_size[0] = iacol;
      nx = selectedIndex_size[0];
      rowIdx = 0;
      jcol = selectedIndex_size[0];
      iacol = 1;
      exitg1 = false;
      while ((!exitg1) && (iacol <= nx)) {
        if (SD->f3.cpu_selectedIndex_data[iacol - 1]) {
          rowIdx++;
          SD->f3.cpu_ii_data[rowIdx - 1] = iacol;
          if (rowIdx >= nx) {
            exitg1 = true;
          } else {
            iacol++;
          }
        } else {
          iacol++;
        }
      }
      if (selectedIndex_size[0] == 1) {
        if (rowIdx == 0) {
          jcol = 0;
        }
      } else if (rowIdx < 1) {
        jcol = 0;
      } else {
        jcol = rowIdx;
      }
      bboxes_size[0] = jcol;
      bboxes_size[1] = 9;
      nx = jcol * 9 - 1;
      mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block, 2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        pointpillarsDetect_kernel56<<<grid, block>>>(nx, gpu_bboxes_data);
      }
      empty_non_axis_sizes = true;
      for (ibmat = 0; ibmat < 7; ibmat++) {
        for (ibcol = 0; ibcol < jcol; ibcol++) {
          if (empty_non_axis_sizes) {
            checkCudaError(
                hipMemcpy(cpu_bboxes_data, gpu_bboxes_data,
                           static_cast<uint32_T>(bboxes_size[0] * 9) *
                               sizeof(real_T),
                           hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          if (value_outdatedOnGpu) {
            checkCudaError(hipMemcpy(SD->f3.cpu_boxPreds_data,
                                      *gpu_boxPreds_data, 3856896UL,
                                      hipMemcpyDeviceToHost),
                           __FILE__, __LINE__);
          }
          value_outdatedOnGpu = false;
          cpu_bboxes_data[ibcol + bboxes_size[0] * iv27[ibmat]] =
              SD->f3.cpu_boxPreds_data[(SD->f3.cpu_ii_data[ibcol] +
                                        itilerow * ibmat) -
                                       1];
          empty_non_axis_sizes = false;
        }
      }
      coder::internal::c_returnCategoricalLabels(
          SD, static_cast<real_T>(bboxes_size[0]), labels->codes.data,
          labels->codes.size, labels->categoryNames.data,
          labels->categoryNames.size);
    } else {
      empty_non_axis_sizes = false;
      bboxes_size[0] = 0;
      bboxes_size[1] = 9;
      rangeData_outdatedOnCpu = false;
      scores_size[0] = 0;
      coder::internal::c_returnCategoricalLabels(SD, labels->codes.size,
                                                 labels->categoryNames.data,
                                                 labels->categoryNames.size);
    }
  } else {
    empty_non_axis_sizes = false;
    bboxes_size[0] = 0;
    bboxes_size[1] = 9;
    rangeData_outdatedOnCpu = false;
    scores_size[0] = 0;
    coder::internal::c_returnCategoricalLabels(SD, labels->codes.size,
                                               labels->categoryNames.data,
                                               labels->categoryNames.size);
  }
  coder::handle_matlabCodegenDestructor(&ptCloudUnorg);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  coder::handle_matlabCodegenDestructor(&ptCloud);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  emxFree_real32_T(&dataIn);
  emxFree_uint8_T(&cpu_b);
  emxFree_real32_T(&cpu_rangeData);
  emxFree_real32_T(&cpu_intensity);
  emxFree_real32_T(&cpu_normals);
  emxFree_uint8_T(&cpu_value);
  emxFree_real32_T(&cpu_location);
  emxFree_boolean_T(&cpu_indices);
  emxFreeStruct_pointCloud1(&ptCloudUnorg);
  emxFreeStruct_pointCloud(&ptCloud);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (empty_non_axis_sizes) {
    checkCudaError(
        hipMemcpy(cpu_bboxes_data, gpu_bboxes_data,
                   static_cast<uint32_T>(bboxes_size[0] * 9) * sizeof(real_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  if (rangeData_outdatedOnCpu) {
    checkCudaError(
        hipMemcpy(cpu_scores_data, gpu_scores_data,
                   static_cast<uint32_T>(scores_size[0]) * sizeof(real32_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  checkCudaError(mwCudaFree(gpu_bboxes_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_scores_data), __FILE__, __LINE__);
  gpuEmxFree_boolean_T(&gpu_indices);
  gpuEmxFree_real32_T(&gpu_location);
  gpuEmxFree_uint8_T(&gpu_value);
  gpuEmxFree_real32_T(&gpu_normals);
  gpuEmxFree_real32_T(&gpu_intensity);
  gpuEmxFree_real32_T(&gpu_rangeData);
  gpuEmxFree_uint8_T(&gpu_b);
  checkCudaError(mwCudaFree(*gpu_anchors_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_yy), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_xx), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_tmp), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_tmp_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_anchors3dMod_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_anchorsDim_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_anchorMask_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_tmp2_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_pillarIndices), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_dlPillarFeatures_Data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_fv1), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_fv2), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_predOcc), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*c_gpu_this_workspace_predClassi), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaFree(*gpu_this_workspace_predHeading), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaFree(*gpu_predAngle), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_sparseVoxelMap), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_iv1_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_anchorMask_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_newAnchors_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*c_gpu_anchorMask_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_x), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_ii_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_v1_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_vk_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_varargout_6_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_col_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_yCen_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_confScore_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_xGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_yGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_zGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_lGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_wGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_hGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_hdGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_angGt_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_varargout_1_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_boxPreds_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_boxPreds_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_selectedIndex_data), __FILE__, __LINE__);
}

void pointpillarsDetect_delete()
{
  if (!gobj_0.matlabCodegenIsDeleted) {
    gobj_0.matlabCodegenIsDeleted = true;
    coder::internal::dlnetwork_delete(&gobj_0);
  }
}

void pointpillarsDetect_emx_init()
{
  c_emxInitStruct_pointPillarsObj(&pointPillarObj);
}

void pointpillarsDetect_init()
{
  pointPillarObj_not_empty = false;
}

void pointpillarsDetect_new()
{
  gobj_0.matlabCodegenIsDeleted = true;
}

// End of code generation (pointpillarsDetect.cu)
