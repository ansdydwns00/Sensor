#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// selectStrongestBboxMulticlass.cu
//
// Code generation for function 'selectStrongestBboxMulticlass'
//

// Include files
#include "selectStrongestBboxMulticlass.h"
#include "overlapPolygonRotatedRects.h"
#include "pointpillarsDetect_types.h"
#include "rt_nonfinite.h"
#include <cfloat>
#include <cmath>

// Function Declarations
static real32_T rt_remf_snf(real32_T u0, real32_T u1);

// Function Definitions
static real32_T rt_remf_snf(real32_T u0, real32_T u1)
{
  real32_T y;
  if (std::isnan(u0) || std::isnan(u1) || std::isinf(u0)) {
    y = rtNaNF;
  } else if (std::isinf(u1)) {
    y = u0;
  } else if ((u1 != 0.0F) && (u1 != std::trunc(u1))) {
    real32_T q;
    q = std::abs(u0 / u1);
    if (!(std::abs(q - std::floor(q + 0.5F)) > FLT_EPSILON * q)) {
      y = 0.0F * u0;
    } else {
      y = std::fmod(u0, u1);
    }
  } else {
    y = std::fmod(u0, u1);
  }
  return y;
}

namespace coder {
void b_selectStrongestBboxCodegen(pointpillarsDetectStackData *SD,
                                  const real32_T bbox_data[],
                                  const int32_T bbox_size[2],
                                  const real32_T label_data[],
                                  boolean_T selectedIndex_data[],
                                  int32_T selectedIndex_size[1])
{
  real_T overlapPoly_data[50];
  real_T overlapedPolyPts_data[48];
  int32_T u_size[2];
  int32_T v_size[2];
  int32_T bbox_size_idx_0;
  int32_T i;
  int32_T inidx;
  int32_T npages;
  int32_T vidx;
  int32_T x_size_idx_0;
  real32_T absx;
  real32_T xmin;
  int8_T n;
  selectedIndex_size[0] = bbox_size[0];
  for (int32_T j{0}; j < bbox_size[0]; j++) {
    selectedIndex_data[j] = true;
  }
  for (int32_T j{0}; j < (bbox_size[0] << 2); j++) {
    SD->f1.X_data[j] = 0.0F;
    SD->f1.Y_data[j] = 0.0F;
  }
  for (int32_T k{0}; k < bbox_size[0]; k++) {
    SD->f1.r_data[k] = bbox_data[k + bbox_size[0] * 4];
    xmin = bbox_data[k + bbox_size[0] * 4];
    SD->f1.b_x_data[k] = xmin;
    if (std::isinf(xmin) || std::isnan(xmin)) {
      SD->f1.b_x_data[k] = rtNaNF;
    } else {
      xmin = rt_remf_snf(xmin, 360.0F);
      absx = std::abs(xmin);
      if (absx > 180.0F) {
        if (xmin > 0.0F) {
          xmin -= 360.0F;
        } else {
          xmin += 360.0F;
        }
        absx = std::abs(xmin);
      }
      if (absx <= 45.0F) {
        xmin *= 0.0174532924F;
        n = 0;
      } else if (absx <= 135.0F) {
        if (xmin > 0.0F) {
          xmin = 0.0174532924F * (xmin - 90.0F);
          n = 1;
        } else {
          xmin = 0.0174532924F * (xmin + 90.0F);
          n = -1;
        }
      } else if (xmin > 0.0F) {
        xmin = 0.0174532924F * (xmin - 180.0F);
        n = 2;
      } else {
        xmin = 0.0174532924F * (xmin + 180.0F);
        n = -2;
      }
      if (n == 0) {
        SD->f1.b_x_data[k] = std::cos(xmin);
      } else if (n == 1) {
        SD->f1.b_x_data[k] = -std::sin(xmin);
      } else if (n == -1) {
        SD->f1.b_x_data[k] = std::sin(xmin);
      } else {
        SD->f1.b_x_data[k] = -std::cos(xmin);
      }
    }
    xmin = bbox_data[k + bbox_size[0] * 4];
    SD->f1.c_x_data[k] = xmin;
    if (std::isinf(xmin) || std::isnan(xmin)) {
      SD->f1.c_x_data[k] = rtNaNF;
    } else {
      xmin = rt_remf_snf(xmin, 360.0F);
      absx = std::abs(xmin);
      if (absx > 180.0F) {
        if (xmin > 0.0F) {
          xmin -= 360.0F;
        } else {
          xmin += 360.0F;
        }
        absx = std::abs(xmin);
      }
      if (absx <= 45.0F) {
        xmin *= 0.0174532924F;
        n = 0;
      } else if (absx <= 135.0F) {
        if (xmin > 0.0F) {
          xmin = 0.0174532924F * (xmin - 90.0F);
          n = 1;
        } else {
          xmin = 0.0174532924F * (xmin + 90.0F);
          n = -1;
        }
      } else if (xmin > 0.0F) {
        xmin = 0.0174532924F * (xmin - 180.0F);
        n = 2;
      } else {
        xmin = 0.0174532924F * (xmin + 180.0F);
        n = -2;
      }
      if (n == 0) {
        SD->f1.c_x_data[k] = std::sin(xmin);
      } else if (n == 1) {
        SD->f1.c_x_data[k] = std::cos(xmin);
      } else if (n == -1) {
        SD->f1.c_x_data[k] = -std::cos(xmin);
      } else {
        SD->f1.c_x_data[k] = -std::sin(xmin);
      }
    }
  }
  npages = bbox_size[0];
  vidx = bbox_size[0];
  inidx = bbox_size[0];
  for (int32_T j{0}; j < vidx; j++) {
    xmin = bbox_data[j + bbox_size[0] * 2];
    SD->f1.bbox_data[j] = xmin / 2.0F;
    SD->f1.bbox_data[j + npages] = xmin / 2.0F;
    SD->f1.x_data[j] = SD->f1.b_x_data[j];
    SD->f1.x_data[j + inidx] = SD->f1.c_x_data[j];
  }
  u_size[0] = bbox_size[0];
  for (int32_T j{0}; j < (npages << 1); j++) {
    SD->f1.u_data[j] = SD->f1.bbox_data[j] * SD->f1.x_data[j];
  }
  for (int32_T k{0}; k < bbox_size[0]; k++) {
    xmin = bbox_data[k + bbox_size[0] * 4];
    SD->f1.b_x_data[k] = xmin;
    if (std::isinf(xmin) || std::isnan(xmin)) {
      SD->f1.b_x_data[k] = rtNaNF;
    } else {
      xmin = rt_remf_snf(xmin, 360.0F);
      absx = std::abs(xmin);
      if (absx > 180.0F) {
        if (xmin > 0.0F) {
          xmin -= 360.0F;
        } else {
          xmin += 360.0F;
        }
        absx = std::abs(xmin);
      }
      if (absx <= 45.0F) {
        xmin *= 0.0174532924F;
        n = 0;
      } else if (absx <= 135.0F) {
        if (xmin > 0.0F) {
          xmin = 0.0174532924F * (xmin - 90.0F);
          n = 1;
        } else {
          xmin = 0.0174532924F * (xmin + 90.0F);
          n = -1;
        }
      } else if (xmin > 0.0F) {
        xmin = 0.0174532924F * (xmin - 180.0F);
        n = 2;
      } else {
        xmin = 0.0174532924F * (xmin + 180.0F);
        n = -2;
      }
      if (n == 0) {
        SD->f1.b_x_data[k] = std::sin(xmin);
      } else if (n == 1) {
        SD->f1.b_x_data[k] = std::cos(xmin);
      } else if (n == -1) {
        SD->f1.b_x_data[k] = -std::cos(xmin);
      } else {
        SD->f1.b_x_data[k] = -std::sin(xmin);
      }
    }
    if (std::isinf(SD->f1.r_data[k]) || std::isnan(SD->f1.r_data[k])) {
      SD->f1.r_data[k] = rtNaNF;
    } else {
      xmin = rt_remf_snf(SD->f1.r_data[k], 360.0F);
      absx = std::abs(xmin);
      if (absx > 180.0F) {
        if (xmin > 0.0F) {
          xmin -= 360.0F;
        } else {
          xmin += 360.0F;
        }
        absx = std::abs(xmin);
      }
      if (absx <= 45.0F) {
        xmin *= 0.0174532924F;
        n = 0;
      } else if (absx <= 135.0F) {
        if (xmin > 0.0F) {
          xmin = 0.0174532924F * (xmin - 90.0F);
          n = 1;
        } else {
          xmin = 0.0174532924F * (xmin + 90.0F);
          n = -1;
        }
      } else if (xmin > 0.0F) {
        xmin = 0.0174532924F * (xmin - 180.0F);
        n = 2;
      } else {
        xmin = 0.0174532924F * (xmin + 180.0F);
        n = -2;
      }
      if (n == 0) {
        SD->f1.r_data[k] = std::cos(xmin);
      } else if (n == 1) {
        SD->f1.r_data[k] = -std::sin(xmin);
      } else if (n == -1) {
        SD->f1.r_data[k] = std::sin(xmin);
      } else {
        SD->f1.r_data[k] = -std::cos(xmin);
      }
    }
  }
  bbox_size_idx_0 = bbox_size[0];
  vidx = bbox_size[0];
  x_size_idx_0 = bbox_size[0];
  for (int32_T j{0}; j < vidx; j++) {
    xmin = bbox_data[j + bbox_size[0] * 3];
    SD->f1.bbox_data[j] = xmin / 2.0F;
    SD->f1.bbox_data[j + bbox_size_idx_0] = xmin / 2.0F;
    SD->f1.x_data[j] = -SD->f1.b_x_data[j];
    SD->f1.x_data[j + x_size_idx_0] = SD->f1.r_data[j];
  }
  v_size[0] = bbox_size[0];
  for (int32_T j{0}; j < (bbox_size_idx_0 << 1); j++) {
    SD->f1.v_data[j] = SD->f1.bbox_data[j] * SD->f1.x_data[j];
  }
  npages = bbox_size[0];
  for (int32_T j{0}; j < npages; j++) {
    SD->f1.X_data[4 * j] = (bbox_data[j] - SD->f1.u_data[j]) - SD->f1.v_data[j];
  }
  npages = bbox_size[0];
  for (int32_T j{0}; j < npages; j++) {
    SD->f1.Y_data[4 * j] =
        (bbox_data[j + bbox_size[0]] - SD->f1.u_data[j + u_size[0]]) -
        SD->f1.v_data[j + v_size[0]];
  }
  npages = bbox_size[0];
  for (int32_T j{0}; j < npages; j++) {
    SD->f1.X_data[4 * j + 1] =
        (bbox_data[j] + SD->f1.u_data[j]) - SD->f1.v_data[j];
  }
  npages = bbox_size[0];
  for (int32_T j{0}; j < npages; j++) {
    SD->f1.Y_data[4 * j + 1] =
        (bbox_data[j + bbox_size[0]] + SD->f1.u_data[j + u_size[0]]) -
        SD->f1.v_data[j + v_size[0]];
  }
  npages = bbox_size[0];
  for (int32_T j{0}; j < npages; j++) {
    SD->f1.X_data[4 * j + 2] =
        (bbox_data[j] + SD->f1.u_data[j]) + SD->f1.v_data[j];
  }
  npages = bbox_size[0];
  for (int32_T j{0}; j < npages; j++) {
    SD->f1.Y_data[4 * j + 2] =
        (bbox_data[j + bbox_size[0]] + SD->f1.u_data[j + u_size[0]]) +
        SD->f1.v_data[j + v_size[0]];
  }
  npages = bbox_size[0];
  for (int32_T j{0}; j < npages; j++) {
    SD->f1.X_data[4 * j + 3] =
        (bbox_data[j] - SD->f1.u_data[j]) + SD->f1.v_data[j];
  }
  npages = bbox_size[0];
  for (int32_T j{0}; j < npages; j++) {
    SD->f1.Y_data[4 * j + 3] =
        (bbox_data[j + bbox_size[0]] - SD->f1.u_data[j + u_size[0]]) +
        SD->f1.v_data[j + v_size[0]];
  }
  npages = bbox_size[0];
  for (i = 0; i < npages; i++) {
    inidx = i << 2;
    vidx = i << 2;
    SD->f1.r_data[i] = std::abs(
        (((SD->f1.X_data[vidx] - SD->f1.X_data[vidx + 3]) *
              (SD->f1.Y_data[vidx] + SD->f1.Y_data[vidx + 3]) / 2.0F +
          (SD->f1.X_data[inidx + 1] - SD->f1.X_data[inidx]) *
              (SD->f1.Y_data[inidx + 1] + SD->f1.Y_data[inidx]) / 2.0F) +
         (SD->f1.X_data[inidx + 2] - SD->f1.X_data[inidx + 1]) *
             (SD->f1.Y_data[inidx + 2] + SD->f1.Y_data[inidx + 1]) / 2.0F) +
        (SD->f1.X_data[inidx + 3] - SD->f1.X_data[inidx + 2]) *
            (SD->f1.Y_data[inidx + 3] + SD->f1.Y_data[inidx + 2]) / 2.0F);
  }
  inidx = -1;
  i = bbox_size[0];
  for (x_size_idx_0 = 0; x_size_idx_0 < i; x_size_idx_0++) {
    real32_T xIndices1_data[4];
    real32_T yIndices1_data[4];
    inidx = x_size_idx_0;
    xIndices1_data[0] = SD->f1.X_data[4 * x_size_idx_0];
    yIndices1_data[0] = SD->f1.Y_data[4 * x_size_idx_0];
    xIndices1_data[1] = SD->f1.X_data[4 * x_size_idx_0 + 1];
    yIndices1_data[1] = SD->f1.Y_data[4 * x_size_idx_0 + 1];
    xIndices1_data[2] = SD->f1.X_data[4 * x_size_idx_0 + 2];
    yIndices1_data[2] = SD->f1.Y_data[4 * x_size_idx_0 + 2];
    xIndices1_data[3] = SD->f1.X_data[4 * x_size_idx_0 + 3];
    yIndices1_data[3] = SD->f1.Y_data[4 * x_size_idx_0 + 3];
    if (std::isnan(label_data[x_size_idx_0])) {
      selectedIndex_data[x_size_idx_0] = false;
    } else {
      if (selectedIndex_data[x_size_idx_0]) {
        vidx = 2;
      } else {
        vidx = 1;
      }
      if (vidx != 1) {
        vidx = bbox_size[0] - x_size_idx_0;
        for (int32_T j{0}; j <= vidx - 2; j++) {
          bbox_size_idx_0 = (x_size_idx_0 + j) + 1;
          if (selectedIndex_data[bbox_size_idx_0] &&
              (!(label_data[bbox_size_idx_0] != label_data[x_size_idx_0]))) {
            real32_T xIndices2_data[4];
            real32_T yIndices2_data[4];
            real32_T b_xmin;
            real32_T b_ymax;
            real32_T b_ymin;
            real32_T xmax;
            real32_T ymax;
            real32_T ymin;
            xIndices2_data[0] = SD->f1.X_data[4 * bbox_size_idx_0];
            yIndices2_data[0] = SD->f1.Y_data[4 * bbox_size_idx_0];
            xIndices2_data[1] = SD->f1.X_data[4 * bbox_size_idx_0 + 1];
            yIndices2_data[1] = SD->f1.Y_data[4 * bbox_size_idx_0 + 1];
            xIndices2_data[2] = SD->f1.X_data[4 * bbox_size_idx_0 + 2];
            yIndices2_data[2] = SD->f1.Y_data[4 * bbox_size_idx_0 + 2];
            xIndices2_data[3] = SD->f1.X_data[4 * bbox_size_idx_0 + 3];
            yIndices2_data[3] = SD->f1.Y_data[4 * bbox_size_idx_0 + 3];
            xmin = SD->f1.X_data[4 * x_size_idx_0];
            absx = xmin;
            ymin = SD->f1.Y_data[4 * x_size_idx_0];
            ymax = ymin;
            b_xmin = SD->f1.X_data[4 * bbox_size_idx_0];
            xmax = SD->f1.X_data[4 * bbox_size_idx_0];
            b_ymin = SD->f1.Y_data[4 * bbox_size_idx_0];
            b_ymax = SD->f1.Y_data[4 * bbox_size_idx_0];
            for (npages = 0; npages < 3; npages++) {
              boolean_T p;
              if (std::isnan(xIndices1_data[npages + 1])) {
                p = false;
              } else if (std::isnan(xmin)) {
                p = true;
              } else {
                p = (xmin > SD->f1.X_data[(npages + 4 * x_size_idx_0) + 1]);
              }
              if (p) {
                xmin = SD->f1.X_data[(npages + 4 * x_size_idx_0) + 1];
              }
              if (std::isnan(xIndices1_data[npages + 1])) {
                p = false;
              } else if (std::isnan(absx)) {
                p = true;
              } else {
                p = (absx < SD->f1.X_data[(npages + 4 * x_size_idx_0) + 1]);
              }
              if (p) {
                absx = SD->f1.X_data[(npages + 4 * x_size_idx_0) + 1];
              }
              if (std::isnan(yIndices1_data[npages + 1])) {
                p = false;
              } else if (std::isnan(ymin)) {
                p = true;
              } else {
                p = (ymin > SD->f1.Y_data[(npages + 4 * x_size_idx_0) + 1]);
              }
              if (p) {
                ymin = SD->f1.Y_data[(npages + 4 * x_size_idx_0) + 1];
              }
              if (std::isnan(yIndices1_data[npages + 1])) {
                p = false;
              } else if (std::isnan(ymax)) {
                p = true;
              } else {
                p = (ymax < SD->f1.Y_data[(npages + 4 * x_size_idx_0) + 1]);
              }
              if (p) {
                ymax = SD->f1.Y_data[(npages + 4 * x_size_idx_0) + 1];
              }
              if (std::isnan(xIndices2_data[npages + 1])) {
                p = false;
              } else if (std::isnan(b_xmin)) {
                p = true;
              } else {
                p = (b_xmin >
                     SD->f1.X_data[(npages + 4 * bbox_size_idx_0) + 1]);
              }
              if (p) {
                b_xmin = SD->f1.X_data[(npages + 4 * bbox_size_idx_0) + 1];
              }
              if (std::isnan(xIndices2_data[npages + 1])) {
                p = false;
              } else if (std::isnan(xmax)) {
                p = true;
              } else {
                p = (xmax < SD->f1.X_data[(npages + 4 * bbox_size_idx_0) + 1]);
              }
              if (p) {
                xmax = SD->f1.X_data[(npages + 4 * bbox_size_idx_0) + 1];
              }
              if (std::isnan(yIndices2_data[npages + 1])) {
                p = false;
              } else if (std::isnan(b_ymin)) {
                p = true;
              } else {
                p = (b_ymin >
                     SD->f1.Y_data[(npages + 4 * bbox_size_idx_0) + 1]);
              }
              if (p) {
                b_ymin = SD->f1.Y_data[(npages + 4 * bbox_size_idx_0) + 1];
              }
              if (std::isnan(yIndices2_data[npages + 1])) {
                p = false;
              } else if (std::isnan(b_ymax)) {
                p = true;
              } else {
                p = (b_ymax <
                     SD->f1.Y_data[(npages + 4 * bbox_size_idx_0) + 1]);
              }
              if (p) {
                b_ymax = SD->f1.Y_data[(npages + 4 * bbox_size_idx_0) + 1];
              }
            }
            if ((!(std::fmin(absx, xmax) - std::fmax(xmin, b_xmin) <= 0.0F)) &&
                (!(std::fmin(ymax, b_ymax) - std::fmax(ymin, b_ymin) <=
                   0.0F))) {
              vision::internal::detector::c_overlapPolygonRotatedRects_fi(
                  xIndices1_data, yIndices1_data, xIndices2_data,
                  yIndices2_data, overlapedPolyPts_data, u_size);
              if (u_size[1] != 0) {
                real_T s;
                vision::internal::detector::d_overlapPolygonRotatedRects_fi(
                    overlapedPolyPts_data, u_size, overlapPoly_data, v_size);
                npages = v_size[1];
                s = (overlapPoly_data[0] -
                     overlapPoly_data[2 * (v_size[1] - 1)]) *
                    (overlapPoly_data[1] +
                     overlapPoly_data[2 * (v_size[1] - 1) + 1]) /
                    2.0;
                for (int32_T k{0}; k <= npages - 2; k++) {
                  s += (overlapPoly_data[2 * (k + 1)] -
                        overlapPoly_data[2 * k]) *
                       (overlapPoly_data[2 * (k + 1) + 1] +
                        overlapPoly_data[2 * k + 1]) /
                       2.0;
                }
                if (static_cast<real32_T>(std::abs(s)) /
                        std::fmin(SD->f1.r_data[x_size_idx_0],
                                  SD->f1.r_data[bbox_size_idx_0]) >
                    0.1) {
                  selectedIndex_data[bbox_size_idx_0] = false;
                }
              }
            }
          }
        }
      }
    }
  }
  if (inidx + 2 > bbox_size[0]) {
    inidx = 0;
    npages = 0;
  } else {
    inidx++;
    npages = bbox_size[0];
  }
  vidx = npages - inidx;
  for (int32_T j{0}; j < vidx; j++) {
    selectedIndex_data[inidx + j] = false;
  }
}

} // namespace coder

// End of code generation (selectStrongestBboxMulticlass.cu)
