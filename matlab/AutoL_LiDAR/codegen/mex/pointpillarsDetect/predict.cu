#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// predict.cu
//
// Code generation for function 'predict'
//

// Include files
#include "predict.h"
#include "dlnetwork.h"
#include "pointpillarsDetect_data.h"
#include "pointpillarsDetect_mexutil.h"
#include "pointpillarsDetect_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"

// Function Declarations
static __global__ void
dlnetwork_predict_kernel75(const real32_T varargin_1_Data[24000],
                           real32_T inputDataT_f1[24000]);

static __global__ void
dlnetwork_predict_kernel76(const real32_T varargin_2_Data[10800000],
                           real32_T inputDataT_f2[10800000]);

static __global__ void dlnetwork_predict_kernel77(
    const real32_T outputData_f2[321408], const real32_T outputData_f1[321408],
    real32_T varargout_2_Data[321408], real32_T varargout_1_Data[321408]);

static __global__ void dlnetwork_predict_kernel78(
    const real32_T outputData_f6[107136], const real32_T outputData_f5[107136],
    const real32_T outputData_f4[107136], const real32_T outputData_f3[107136],
    real32_T varargout_6_Data[107136], real32_T varargout_5_Data[107136],
    real32_T varargout_4_Data[107136], real32_T varargout_3_Data[107136]);

// Function Definitions
static __global__ __launch_bounds__(224, 1) void dlnetwork_predict_kernel75(
    const real32_T varargin_1_Data[24000], real32_T inputDataT_f1[24000])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T i1;
  gThreadId = mwGetGlobalThreadIndex();
  i1 = static_cast<int32_T>(gThreadId % 2UL);
  i = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(i1)) / 2UL);
  if ((i < 12000) && (i1 < 2)) {
    inputDataT_f1[i1 + (i << 1)] = varargin_1_Data[i + 12000 * i1];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel76(
    const real32_T varargin_2_Data[10800000], real32_T inputDataT_f2[10800000])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  gThreadId = mwGetGlobalThreadIndex();
  i = static_cast<int32_T>(gThreadId % 100UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(i)) / 100UL;
  i1 = static_cast<int32_T>(gThreadId % 12000UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(i1)) / 12000UL;
  p = static_cast<int32_T>(gThreadId);
  if ((p < 9) && (i1 < 12000) && (i < 100)) {
    inputDataT_f2[(i + 100 * i1) + 1200000 * p] =
        varargin_2_Data[(i1 + 12000 * i) + 1200000 * p];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel77(
    const real32_T outputData_f2[321408], const real32_T outputData_f1[321408],
    real32_T varargout_2_Data[321408], real32_T varargout_1_Data[321408])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  gThreadId = mwGetGlobalThreadIndex();
  i = static_cast<int32_T>(gThreadId % 216UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(i)) / 216UL;
  i1 = static_cast<int32_T>(gThreadId % 248UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(i1)) / 248UL;
  p = static_cast<int32_T>(gThreadId);
  if ((p < 6) && (i1 < 248) && (i < 216)) {
    varargout_1_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f1[(i1 + 248 * i) + 53568 * p];
    varargout_2_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f2[(i1 + 248 * i) + 53568 * p];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel78(
    const real32_T outputData_f6[107136], const real32_T outputData_f5[107136],
    const real32_T outputData_f4[107136], const real32_T outputData_f3[107136],
    real32_T varargout_6_Data[107136], real32_T varargout_5_Data[107136],
    real32_T varargout_4_Data[107136], real32_T varargout_3_Data[107136])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  gThreadId = mwGetGlobalThreadIndex();
  i = static_cast<int32_T>(gThreadId % 216UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(i)) / 216UL;
  i1 = static_cast<int32_T>(gThreadId % 248UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(i1)) / 248UL;
  p = static_cast<int32_T>(gThreadId);
  if ((p < 2) && (i1 < 248) && (i < 216)) {
    varargout_3_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f3[(i1 + 248 * i) + 53568 * p];
    varargout_4_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f4[(i1 + 248 * i) + 53568 * p];
    varargout_5_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f5[(i1 + 248 * i) + 53568 * p];
    varargout_6_Data[(i + 216 * i1) + 53568 * p] =
        outputData_f6[(i1 + 248 * i) + 53568 * p];
  }
}

namespace coder {
namespace internal {
void dlnetwork_predict(detector0_0 *obj, const real32_T varargin_1_Data[24000],
                       const real32_T varargin_2_Data[10800000],
                       real32_T varargout_1_Data[321408],
                       real32_T varargout_2_Data[321408],
                       real32_T varargout_3_Data[107136],
                       real32_T varargout_4_Data[107136],
                       real32_T varargout_5_Data[107136],
                       real32_T varargout_6_Data[107136])
{
  real32_T(*gpu_inputDataT_f2)[10800000];
  real32_T(*gpu_outputData_f1)[321408];
  real32_T(*gpu_outputData_f2)[321408];
  real32_T(*gpu_outputData_f3)[107136];
  real32_T(*gpu_outputData_f4)[107136];
  real32_T(*gpu_outputData_f5)[107136];
  real32_T(*gpu_outputData_f6)[107136];
  real32_T(*gpu_inputDataT_f1)[24000];
  checkCudaError(mwCudaMalloc(&gpu_outputData_f6, 428544UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_outputData_f5, 428544UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_outputData_f4, 428544UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_outputData_f3, 428544UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_outputData_f2, 1285632UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_outputData_f1, 1285632UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_inputDataT_f2, 43200000UL), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_inputDataT_f1, 96000UL), __FILE__, __LINE__);
  dlnetwork_predict_kernel75<<<dim3(108U, 1U, 1U), dim3(224U, 1U, 1U)>>>(
      varargin_1_Data, *gpu_inputDataT_f1);
  dlnetwork_predict_kernel76<<<dim3(21094U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      varargin_2_Data, *gpu_inputDataT_f2);
  checkCudaError(hipMemcpy(obj->getInputDataPointer(0), *gpu_inputDataT_f1,
                            obj->getLayerOutputSize(3, 0),
                            hipMemcpyDeviceToDevice),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(obj->getInputDataPointer(1), *gpu_inputDataT_f2,
                            obj->getLayerOutputSize(0, 0),
                            hipMemcpyDeviceToDevice),
                 __FILE__, __LINE__);
  try {
    obj->activations(56);
  } catch (std::runtime_error const &err) {
    checkRunTimeError(err.what(), __FILE__, __LINE__);
  } catch (...) {
    checkRunTimeError("", __FILE__, __LINE__);
  }
  checkCudaError(hipMemcpy(*gpu_outputData_f1, obj->getLayerOutput(56, 0),
                            obj->getLayerOutputSize(56, 0),
                            hipMemcpyDeviceToDevice),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(*gpu_outputData_f2, obj->getLayerOutput(53, 0),
                            obj->getLayerOutputSize(53, 0),
                            hipMemcpyDeviceToDevice),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(*gpu_outputData_f3, obj->getLayerOutput(55, 0),
                            obj->getLayerOutputSize(55, 0),
                            hipMemcpyDeviceToDevice),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(*gpu_outputData_f4, obj->getLayerOutput(49, 0),
                            obj->getLayerOutputSize(49, 0),
                            hipMemcpyDeviceToDevice),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(*gpu_outputData_f5, obj->getLayerOutput(52, 0),
                            obj->getLayerOutputSize(52, 0),
                            hipMemcpyDeviceToDevice),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(*gpu_outputData_f6, obj->getLayerOutput(50, 0),
                            obj->getLayerOutputSize(50, 0),
                            hipMemcpyDeviceToDevice),
                 __FILE__, __LINE__);
  dlnetwork_predict_kernel77<<<dim3(628U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_outputData_f2, *gpu_outputData_f1, varargout_2_Data,
      varargout_1_Data);
  dlnetwork_predict_kernel78<<<dim3(210U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_outputData_f6, *gpu_outputData_f5, *gpu_outputData_f4,
      *gpu_outputData_f3, varargout_6_Data, varargout_5_Data, varargout_4_Data,
      varargout_3_Data);
  checkCudaError(mwCudaFree(*gpu_inputDataT_f1), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_inputDataT_f2), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_outputData_f1), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_outputData_f2), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_outputData_f3), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_outputData_f4), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_outputData_f5), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_outputData_f6), __FILE__, __LINE__);
}

} // namespace internal
} // namespace coder

// End of code generation (predict.cu)
