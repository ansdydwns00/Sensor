#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// dlnetwork.cu
//
// Code generation for function 'dlnetwork'
//

// Include files
#include "dlnetwork.h"
#include "pointpillarsDetect.h"
#include "pointpillarsDetect_data.h"
#include "pointpillarsDetect_internal_types.h"
#include "pointpillarsDetect_mexutil.h"
#include "pointpillarsDetect_types.h"
#include "rt_nonfinite.h"
#include "MWBatchNormalizationLayer.hpp"
#include "MWCNNLayer.hpp"
#include "MWConcatenationLayer.hpp"
#include "MWConvLayer.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWCudnnCustomLayerBase.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWFusedConvActivationLayer.hpp"
#include "MWInputLayer.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWMaxPoolingLayer.hpp"
#include "MWOutputLayer.hpp"
#include "MWSigmoidLayer.hpp"
#include "MWSplittingLayer.hpp"
#include "MWTensor.hpp"
#include "MWTensorBase.hpp"
#include "stdlib.h"
#include "string.h"
#include <cstring>

// Variable Definitions
static boolean_T c_gpuConstsCopied_dlnetwork_lay;

static boolean_T d_gpuConstsCopied_dlnetwork_lay;

// Function Declarations
static __global__ void
c_dlnetwork_layerPredictWithCol(const real32_T maps_data[24000],
                                const int32_T trueCount, const int32_T b,
                                int32_T tmp_data[12000]);

static void checkCleanupCudaError(hipError_t errCode, const char_T *file,
                                  uint32_T b_line);

static void checkCleanupRunTimeError(const char_T *errMsg, const char_T *file,
                                     uint32_T b_line);

static __global__ void
d_dlnetwork_layerPredictWithCol(const real32_T varargin_1[1714176],
                                const real_T xq[216], real32_T out[3428352]);

static __global__ void
e_dlnetwork_layerPredictWithCol(const real32_T out[3428352],
                                real32_T in[3428352]);

static __global__ void
f_dlnetwork_layerPredictWithCol(const real32_T in[3428352],
                                const real_T xq[248], real32_T out[6856704]);

static __global__ void
g_dlnetwork_layerPredictWithCol(const real32_T out[6856704],
                                real32_T varargout_1[6856704]);

static __global__ void
h_dlnetwork_layerPredictWithCol(const real32_T varargin_1[428544],
                                const real_T xq[216], real32_T out[1714176]);

static __global__ void
i_dlnetwork_layerPredictWithCol(const real32_T out[1714176],
                                real32_T in[1714176]);

static __global__ void
j_dlnetwork_layerPredictWithCol(const real32_T in[1714176],
                                const real_T xq[248], real32_T out[6856704]);

static __global__ void
k_dlnetwork_layerPredictWithCol(const real32_T out[6856704],
                                real32_T varargout_1[6856704]);

// Function Definitions
void detector0_0::allocate()
{
  targetImpl->allocatePermuteBuffers(13713408, 3);
  targetImpl->allocate(76800000, 4);
  for (int32_T idx{0}; idx < 57; idx++) {
    layers[idx]->allocate();
  }
  (static_cast<MWTensor<real32_T> *>(inputTensors[0]))
      ->setData(layers[3]->getLayerOutput(0));
  (static_cast<MWTensor<real32_T> *>(inputTensors[1]))
      ->setData(layers[0]->getLayerOutput(0));
}

void detector0_0::cleanup()
{
  try {
    isInitialized = false;
    deallocate();
    for (int32_T idx{0}; idx < 57; idx++) {
      layers[idx]->cleanup();
    }
    if (targetImpl) {
      targetImpl->cleanup();
    }
    checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
  } catch (std::runtime_error const &err) {
    checkCleanupRunTimeError(err.what(), __FILE__, __LINE__);
  } catch (...) {
    checkCleanupRunTimeError("", __FILE__, __LINE__);
  }
}

void detector0_0::deallocate()
{
  targetImpl->deallocate();
  for (int32_T idx{0}; idx < 57; idx++) {
    layers[idx]->deallocate();
  }
}

void detector0_0::postsetup()
{
  targetImpl->postSetup(layers, numLayers);
}

void detector0_0::resetState()
{
}

void detector0_0::setSize()
{
  for (int32_T idx{0}; idx < 57; idx++) {
    layers[idx]->propagateSize();
  }
  allocate();
  postsetup();
}

void detector0_0::setup()
{
  if (isInitialized) {
    resetState();
  } else {
    targetImpl->preSetup();
    targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(layers[0]))
        ->createInputLayer(targetImpl, inputTensors[1], "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[1]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[0]->getOutputTensor(0), 1, 1, 9, 64, 1, 1, 0,
            0, 0, 0, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_pillars_conv2d_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_pillars_conv2d_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWMaxPoolingLayer *>(layers[2]))
        ->createMaxPoolingLayer<real32_T, real32_T>(
            targetImpl, layers[1]->getOutputTensor(0), 1, 100, 1, 100, 0, 0, 0,
            0, 0, 0, "FLOAT", 1, "SSCB", 0);
    (static_cast<MWInputLayer *>(layers[3]))
        ->createInputLayer(targetImpl, inputTensors[0], "SSCB", 2);
    (static_cast<MWScatterLayer_detector0_01 *>(layers[4]))
        ->createCustomLayer(targetImpl, layers[2]->getOutputTensor(0),
                            layers[3]->getOutputTensor(0), "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[5]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[4]->getOutputTensor(0), 3, 3, 64, 64, 2, 2, 0,
            1, 0, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_conv2d0_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_conv2d0_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[6]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[5]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn0_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn0_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn0_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block1_bn0_trainedVariance.bin",
            64, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[7]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[6]->getOutputTensor(0), 3, 3, 64, 64, 1, 1, 1,
            1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_conv2d1_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_conv2d1_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWBatchNormalizationLayer *>(layers[8]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[7]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn1_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn1_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn1_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block1_bn1_trainedVariance.bin",
            64, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[9]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[8]->getOutputTensor(0), 3, 3, 64, 64, 1, 1, 1,
            1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_conv2d2_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_conv2d2_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[10]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[9]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn2_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn2_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn2_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block1_bn2_trainedVariance.bin",
            64, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[11]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[10]->getOutputTensor(0), 3, 3, 64, 64, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_conv2d3_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_conv2d3_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWBatchNormalizationLayer *>(layers[12]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[11]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn3_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn3_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block1_bn3_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block1_bn3_trainedVariance.bin",
            64, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[13]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[12]->getOutputTensor(0), 3, 3, 64, 128, 2, 2,
            0, 1, 0, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d0_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d0_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[14]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[13]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn0_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn0_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn0_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block2_bn0_trainedVariance.bin",
            128, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[15]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[14]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d1_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d1_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[16]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[15]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn1_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn1_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn1_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block2_bn1_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWFusedConvActivationLayer *>(layers[17]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[16]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d2_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d2_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[18]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[17]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn2_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn2_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn2_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block2_bn2_trainedVariance.bin",
            128, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[19]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[18]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d3_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d3_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[20]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[19]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn3_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn3_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn3_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block2_bn3_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWFusedConvActivationLayer *>(layers[21]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[20]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d4_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d4_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[22]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[21]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn4_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn4_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn4_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block2_bn4_trainedVariance.bin",
            128, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[23]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[22]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d5_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_conv2d5_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[24]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[23]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn5_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn5_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block2_bn5_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block2_bn5_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWFusedConvActivationLayer *>(layers[25]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[12]->getOutputTensor(0), 3, 3, 64, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up1_conv2dt_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up1_conv2dt_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            0);
    (static_cast<MWBatchNormalizationLayer *>(layers[26]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[25]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up1_bn_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up1_bn_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up1_bn_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up1_bn_trainedVariance.bin",
            128, "SSCB", 0);
    (static_cast<MWResize2DLayer_detector0_01 *>(layers[27]))
        ->createCustomLayer(targetImpl, layers[24]->getOutputTensor(0), "SSCB",
                            1);
    (static_cast<MWFusedConvActivationLayer *>(layers[28]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[27]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up2_conv2dt_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up2_conv2dt_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            3);
    (static_cast<MWBatchNormalizationLayer *>(layers[29]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[28]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up2_bn_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up2_bn_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up2_bn_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up2_bn_trainedVariance.bin",
            128, "SSCB", 3);
    (static_cast<MWFusedConvActivationLayer *>(layers[30]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[24]->getOutputTensor(0), 3, 3, 128, 128, 2, 2,
            0, 1, 0, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d0_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d0_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWBatchNormalizationLayer *>(layers[31]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[30]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn0_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn0_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn0_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block3_bn0_trainedVariance.bin",
            128, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[32]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[31]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d1_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d1_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[33]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[32]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn1_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn1_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn1_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block3_bn1_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWFusedConvActivationLayer *>(layers[34]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[33]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d2_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d2_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWBatchNormalizationLayer *>(layers[35]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[34]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn2_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn2_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn2_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block3_bn2_trainedVariance.bin",
            128, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[36]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[35]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d3_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d3_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[37]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[36]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn3_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn3_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn3_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block3_bn3_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWFusedConvActivationLayer *>(layers[38]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[37]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d4_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d4_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWBatchNormalizationLayer *>(layers[39]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[38]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn4_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn4_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn4_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block3_bn4_trainedVariance.bin",
            128, "SSCB", 1);
    (static_cast<MWFusedConvActivationLayer *>(layers[40]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[39]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d5_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_conv2d5_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[41]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[40]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn5_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn5_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_block3_bn5_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/"
            "cnn_detector0_0_cnn_block3_bn5_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWResize2DLayer_detector0_02 *>(layers[42]))
        ->createCustomLayer(targetImpl, layers[41]->getOutputTensor(0), "SSCB",
                            1);
    (static_cast<MWFusedConvActivationLayer *>(layers[43]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[42]->getOutputTensor(0), 3, 3, 128, 128, 1, 1,
            1, 1, 1, 1, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up3_conv2dt_w.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up3_conv2dt_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            2);
    (static_cast<MWBatchNormalizationLayer *>(layers[44]))
        ->createBatchNormalizationLayer(
            targetImpl, layers[43]->getOutputTensor(0), 0.001F,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up3_bn_offset.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up3_bn_scale.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up3_bn_trainedMean.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_cnn_up3_bn_trainedVariance.bin",
            128, "SSCB", 2);
    (static_cast<MWConcatenationLayer *>(layers[45]))
        ->createConcatenationLayer(
            targetImpl, 3, layers[26]->getOutputTensor(0),
            layers[29]->getOutputTensor(0), layers[44]->getOutputTensor(0), 3,
            "C", "SSCB", 1);
    (static_cast<MWConvLayer *>(layers[46]))
        ->createConvLayer(
            targetImpl, layers[45]->getOutputTensor(0), 1, 1, 384, 20, 1, 1, 0,
            0, 0, 0, 1, 1, 1,
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_angle_conv2d_w_fused.bin",
            "/home/aiv/YongJun_ws/matlab/AutoL_LiDAR_ROS/codegen/mex/"
            "pointpillarsDetect/cnn_detector0_0_angle_conv2d_b_fused.bin",
            "SSCB", 0);
    (static_cast<MWSplittingLayer *>(layers[47]))
        ->createSplittingLayer(targetImpl, layers[46]->getOutputTensor(0), 6, 2,
                               2, 2, 2, 6, 6, 6, "SSCB", "SSCB", "SSCB", "SSCB",
                               "SSCB", "SSCB", 0, 0, 0, 0, 0, 0);
    (static_cast<MWSigmoidLayer *>(layers[48]))
        ->createSigmoidLayer(targetImpl, layers[47]->getOutputTensor(2), "SSCB",
                             0);
    (static_cast<MWOutputLayer *>(layers[49]))
        ->createOutputLayer(targetImpl, layers[48]->getOutputTensor(0), "SSCB",
                            0);
    (static_cast<MWOutputLayer *>(layers[50]))
        ->createOutputLayer(targetImpl, layers[47]->getOutputTensor(0), "SSCB",
                            0);
    (static_cast<MWSigmoidLayer *>(layers[51]))
        ->createSigmoidLayer(targetImpl, layers[47]->getOutputTensor(1), "SSCB",
                             0);
    (static_cast<MWOutputLayer *>(layers[52]))
        ->createOutputLayer(targetImpl, layers[51]->getOutputTensor(0), "SSCB",
                            0);
    (static_cast<MWOutputLayer *>(layers[53]))
        ->createOutputLayer(targetImpl, layers[47]->getOutputTensor(4), "SSCB",
                            0);
    (static_cast<MWSigmoidLayer *>(layers[54]))
        ->createSigmoidLayer(targetImpl, layers[47]->getOutputTensor(3), "SSCB",
                             0);
    (static_cast<MWOutputLayer *>(layers[55]))
        ->createOutputLayer(targetImpl, layers[54]->getOutputTensor(0), "SSCB",
                            0);
    (static_cast<MWOutputLayer *>(layers[56]))
        ->createOutputLayer(targetImpl, layers[47]->getOutputTensor(5), "SSCB",
                            0);
    outputTensors[0] = layers[49]->getOutputTensor(0);
    outputTensors[1] = layers[50]->getOutputTensor(0);
    outputTensors[2] = layers[52]->getOutputTensor(0);
    outputTensors[3] = layers[53]->getOutputTensor(0);
    outputTensors[4] = layers[55]->getOutputTensor(0);
    outputTensors[5] = layers[56]->getOutputTensor(0);
    setSize();
  }
  isInitialized = true;
}

static __global__
    __launch_bounds__(1024, 1) void c_dlnetwork_layerPredictWithCol(
        const real32_T maps_data[24000], const int32_T trueCount,
        const int32_T b, int32_T tmp_data[12000])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T maps_size_idx_0;
    maps_size_idx_0 = static_cast<int32_T>(idx);
    tmp_data[maps_size_idx_0] =
        static_cast<int32_T>((maps_data[maps_size_idx_0 + trueCount] * 432.0F +
                              maps_data[maps_size_idx_0]) +
                             1.0F) -
        1;
  }
}

static void checkCleanupCudaError(hipError_t errCode, const char_T *file,
                                  uint32_T b_line)
{
  emlrtRTEInfo rtInfo;
  if ((errCode != hipSuccess) && (errCode != hipErrorDeinitialized)) {
    rtInfo = createEmlrtInfoStruct(file, b_line);
    emlrtCUDAWarning((uint32_T)errCode, (char_T *)hipGetErrorName(errCode),
                     (char_T *)hipGetErrorString(errCode), &rtInfo);
  }
}

static void checkCleanupRunTimeError(const char_T *errMsg, const char_T *file,
                                     uint32_T b_line)
{
  emlrtRTEInfo rtInfo;
  uint32_T errMsgLen;
  rtInfo = createEmlrtInfoStruct(file, b_line);
  errMsgLen = (uint32_T)strlen(errMsg);
  emlrtErrorWithMessageIdR2018a(emlrtRootTLSGlobal, &rtInfo,
                                "dnn_core:cnncodegen:CleanupRunTimeError",
                                "dnn_core:cnncodegen:CleanupRunTimeError", 3,
                                mxCHAR_CLASS, errMsgLen, errMsg);
}

static __global__
    __launch_bounds__(512, 1) void d_dlnetwork_layerPredictWithCol(
        const real32_T varargin_1[1714176], const real_T xq[216],
        real32_T out[3428352])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T j;
  gThreadId = mwGetGlobalThreadIndex();
  j = static_cast<int32_T>(gThreadId % 15872UL);
  i = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(j)) / 15872UL);
  if ((i < 216) && (j < 15872)) {
    real_T d;
    int32_T idx;
    int32_T nextIdx;
    d = xq[i];
    idx = static_cast<int32_T>(floor(d));
    nextIdx = static_cast<int32_T>(ceil(d));
    if (idx != nextIdx) {
      real32_T slope;
      slope = (varargin_1[(idx + 108 * j) - 1] -
               varargin_1[(nextIdx + 108 * j) - 1]) /
              static_cast<real32_T>(idx - nextIdx);
      out[i + 216 * j] = slope * static_cast<real32_T>(d) +
                         (varargin_1[(idx + 108 * j) - 1] -
                          slope * ((static_cast<real32_T>(idx) - 1.0F) + 1.0F));
    } else {
      out[i + 216 * j] = varargin_1[(idx + 108 * j) - 1];
    }
  }
}

static __global__ __launch_bounds__(
    512, 1) void e_dlnetwork_layerPredictWithCol(const real32_T out[3428352],
                                                 real32_T in[3428352])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T idx;
  int32_T nextIdx;
  gThreadId = mwGetGlobalThreadIndex();
  nextIdx = static_cast<int32_T>(gThreadId % 216UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(nextIdx)) / 216UL;
  idx = static_cast<int32_T>(gThreadId % 124UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(idx)) / 124UL;
  i = static_cast<int32_T>(gThreadId);
  if ((i < 128) && (idx < 124) && (nextIdx < 216)) {
    in[(idx + 124 * nextIdx) + 26784 * i] =
        out[(nextIdx + 216 * idx) + 26784 * i];
  }
}

static __global__
    __launch_bounds__(512, 1) void f_dlnetwork_layerPredictWithCol(
        const real32_T in[3428352], const real_T xq[248], real32_T out[6856704])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T j;
  gThreadId = mwGetGlobalThreadIndex();
  j = static_cast<int32_T>(gThreadId % 27648UL);
  i = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(j)) / 27648UL);
  if ((i < 248) && (j < 27648)) {
    real_T d;
    int32_T idx;
    int32_T nextIdx;
    d = xq[i];
    idx = static_cast<int32_T>(floor(d));
    nextIdx = static_cast<int32_T>(ceil(d));
    if (idx != nextIdx) {
      real32_T slope;
      slope = (in[(idx + 124 * j) - 1] - in[(nextIdx + 124 * j) - 1]) /
              static_cast<real32_T>(idx - nextIdx);
      out[i + 248 * j] = slope * static_cast<real32_T>(d) +
                         (in[(idx + 124 * j) - 1] -
                          slope * ((static_cast<real32_T>(idx) - 1.0F) + 1.0F));
    } else {
      out[i + 248 * j] = in[(idx + 124 * j) - 1];
    }
  }
}

static __global__ __launch_bounds__(
    512, 1) void g_dlnetwork_layerPredictWithCol(const real32_T out[6856704],
                                                 real32_T varargout_1[6856704])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T idx;
  int32_T nextIdx;
  gThreadId = mwGetGlobalThreadIndex();
  nextIdx = static_cast<int32_T>(gThreadId % 248UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(nextIdx)) / 248UL;
  idx = static_cast<int32_T>(gThreadId % 216UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(idx)) / 216UL;
  i = static_cast<int32_T>(gThreadId);
  if ((i < 128) && (idx < 216) && (nextIdx < 248)) {
    varargout_1[(idx + 216 * nextIdx) + 53568 * i] =
        out[(nextIdx + 248 * idx) + 53568 * i];
  }
}

static __global__
    __launch_bounds__(512, 1) void h_dlnetwork_layerPredictWithCol(
        const real32_T varargin_1[428544], const real_T xq[216],
        real32_T out[1714176])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T j;
  gThreadId = mwGetGlobalThreadIndex();
  j = static_cast<int32_T>(gThreadId % 7936UL);
  i = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(j)) / 7936UL);
  if ((i < 216) && (j < 7936)) {
    real_T d;
    int32_T idx;
    int32_T nextIdx;
    d = xq[i];
    idx = static_cast<int32_T>(floor(d));
    nextIdx = static_cast<int32_T>(ceil(d));
    if (idx != nextIdx) {
      real32_T slope;
      slope = (varargin_1[(idx + 54 * j) - 1] -
               varargin_1[(nextIdx + 54 * j) - 1]) /
              static_cast<real32_T>(idx - nextIdx);
      out[i + 216 * j] = slope * static_cast<real32_T>(d) +
                         (varargin_1[(idx + 54 * j) - 1] -
                          slope * ((static_cast<real32_T>(idx) - 1.0F) + 1.0F));
    } else {
      out[i + 216 * j] = varargin_1[(idx + 54 * j) - 1];
    }
  }
}

static __global__ __launch_bounds__(
    512, 1) void i_dlnetwork_layerPredictWithCol(const real32_T out[1714176],
                                                 real32_T in[1714176])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T idx;
  int32_T nextIdx;
  gThreadId = mwGetGlobalThreadIndex();
  nextIdx = static_cast<int32_T>(gThreadId % 216UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(nextIdx)) / 216UL;
  idx = static_cast<int32_T>(gThreadId % 62UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(idx)) / 62UL;
  i = static_cast<int32_T>(gThreadId);
  if ((i < 128) && (idx < 62) && (nextIdx < 216)) {
    in[(idx + 62 * nextIdx) + 13392 * i] =
        out[(nextIdx + 216 * idx) + 13392 * i];
  }
}

static __global__
    __launch_bounds__(512, 1) void j_dlnetwork_layerPredictWithCol(
        const real32_T in[1714176], const real_T xq[248], real32_T out[6856704])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T j;
  gThreadId = mwGetGlobalThreadIndex();
  j = static_cast<int32_T>(gThreadId % 27648UL);
  i = static_cast<int32_T>((gThreadId - static_cast<uint64_T>(j)) / 27648UL);
  if ((i < 248) && (j < 27648)) {
    real_T d;
    int32_T idx;
    int32_T nextIdx;
    d = xq[i];
    idx = static_cast<int32_T>(floor(d));
    nextIdx = static_cast<int32_T>(ceil(d));
    if (idx != nextIdx) {
      real32_T slope;
      slope = (in[(idx + 62 * j) - 1] - in[(nextIdx + 62 * j) - 1]) /
              static_cast<real32_T>(idx - nextIdx);
      out[i + 248 * j] = slope * static_cast<real32_T>(d) +
                         (in[(idx + 62 * j) - 1] -
                          slope * ((static_cast<real32_T>(idx) - 1.0F) + 1.0F));
    } else {
      out[i + 248 * j] = in[(idx + 62 * j) - 1];
    }
  }
}

static __global__ __launch_bounds__(
    512, 1) void k_dlnetwork_layerPredictWithCol(const real32_T out[6856704],
                                                 real32_T varargout_1[6856704])
{
  uint64_T gThreadId;
  int32_T i;
  int32_T idx;
  int32_T nextIdx;
  gThreadId = mwGetGlobalThreadIndex();
  nextIdx = static_cast<int32_T>(gThreadId % 248UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(nextIdx)) / 248UL;
  idx = static_cast<int32_T>(gThreadId % 216UL);
  gThreadId = (gThreadId - static_cast<uint64_T>(idx)) / 216UL;
  i = static_cast<int32_T>(gThreadId);
  if ((i < 128) && (idx < 216) && (nextIdx < 248)) {
    varargout_1[(idx + 216 * nextIdx) + 53568 * i] =
        out[(nextIdx + 248 * idx) + 53568 * i];
  }
}

void detector0_0::activations(int32_T layerIdx)
{
  for (int32_T idx{0}; idx <= layerIdx; idx++) {
    layers[idx]->predict();
  }
}

detector0_0::detector0_0()
{
  numLayers = 57;
  isInitialized = false;
  targetImpl = 0;
  layers[0] = new MWInputLayer;
  layers[0]->setName("pillars|input");
  layers[1] = new MWFusedConvActivationLayer;
  layers[1]->setName("pillars|conv2d_pillars|relu");
  layers[2] = new MWMaxPoolingLayer;
  layers[2]->setName("pillars|reshape");
  layers[3] = new MWInputLayer;
  layers[3]->setName("pillars|indices|reshape");
  layers[4] = new MWScatterLayer_detector0_01;
  layers[4]->setName("pillars|scatter_nd");
  layers[5] = new MWFusedConvActivationLayer;
  layers[5]->setName("cnn|block1|conv2d0_cnn|block1|conv2d0_relu");
  layers[6] = new MWBatchNormalizationLayer;
  layers[6]->setName("cnn|block1|bn0");
  layers[6]->setInPlaceIndex(0, 0);
  layers[7] = new MWFusedConvActivationLayer;
  layers[7]->setName("cnn|block1|conv2d1_cnn|block1|conv2d1_relu");
  layers[8] = new MWBatchNormalizationLayer;
  layers[8]->setName("cnn|block1|bn1");
  layers[8]->setInPlaceIndex(0, 0);
  layers[9] = new MWFusedConvActivationLayer;
  layers[9]->setName("cnn|block1|conv2d2_cnn|block1|conv2d2_relu");
  layers[10] = new MWBatchNormalizationLayer;
  layers[10]->setName("cnn|block1|bn2");
  layers[10]->setInPlaceIndex(0, 0);
  layers[11] = new MWFusedConvActivationLayer;
  layers[11]->setName("cnn|block1|conv2d3_cnn|block1|conv2d3_relu");
  layers[12] = new MWBatchNormalizationLayer;
  layers[12]->setName("cnn|block1|bn3");
  layers[12]->setInPlaceIndex(0, 0);
  layers[13] = new MWFusedConvActivationLayer;
  layers[13]->setName("cnn|block2|conv2d0_cnn|block2|conv2d0_relu");
  layers[14] = new MWBatchNormalizationLayer;
  layers[14]->setName("cnn|block2|bn0");
  layers[14]->setInPlaceIndex(0, 0);
  layers[15] = new MWFusedConvActivationLayer;
  layers[15]->setName("cnn|block2|conv2d1_cnn|block2|conv2d1_relu");
  layers[16] = new MWBatchNormalizationLayer;
  layers[16]->setName("cnn|block2|bn1");
  layers[16]->setInPlaceIndex(0, 0);
  layers[17] = new MWFusedConvActivationLayer;
  layers[17]->setName("cnn|block2|conv2d2_cnn|block2|conv2d2_relu");
  layers[18] = new MWBatchNormalizationLayer;
  layers[18]->setName("cnn|block2|bn2");
  layers[18]->setInPlaceIndex(0, 0);
  layers[19] = new MWFusedConvActivationLayer;
  layers[19]->setName("cnn|block2|conv2d3_cnn|block2|conv2d3_relu");
  layers[20] = new MWBatchNormalizationLayer;
  layers[20]->setName("cnn|block2|bn3");
  layers[20]->setInPlaceIndex(0, 0);
  layers[21] = new MWFusedConvActivationLayer;
  layers[21]->setName("cnn|block2|conv2d4_cnn|block2|conv2d4_relu");
  layers[22] = new MWBatchNormalizationLayer;
  layers[22]->setName("cnn|block2|bn4");
  layers[22]->setInPlaceIndex(0, 0);
  layers[23] = new MWFusedConvActivationLayer;
  layers[23]->setName("cnn|block2|conv2d5_cnn|block2|conv2d5_relu");
  layers[24] = new MWBatchNormalizationLayer;
  layers[24]->setName("cnn|block2|bn5");
  layers[24]->setInPlaceIndex(0, 0);
  layers[25] = new MWFusedConvActivationLayer;
  layers[25]->setName("cnn|up1|conv2dt_cnn|up1|conv2dt_relu");
  layers[26] = new MWBatchNormalizationLayer;
  layers[26]->setName("cnn|up1|bn");
  layers[26]->setInPlaceIndex(0, 0);
  layers[27] = new MWResize2DLayer_detector0_01;
  layers[27]->setName("cnn|add|resize|2");
  layers[28] = new MWFusedConvActivationLayer;
  layers[28]->setName("cnn|up2|conv2dt_cnn|up2|conv2dt_relu");
  layers[29] = new MWBatchNormalizationLayer;
  layers[29]->setName("cnn|up2|bn");
  layers[29]->setInPlaceIndex(0, 0);
  layers[30] = new MWFusedConvActivationLayer;
  layers[30]->setName("cnn|block3|conv2d0_cnn|block3|conv2d0_relu");
  layers[31] = new MWBatchNormalizationLayer;
  layers[31]->setName("cnn|block3|bn0");
  layers[31]->setInPlaceIndex(0, 0);
  layers[32] = new MWFusedConvActivationLayer;
  layers[32]->setName("cnn|block3|conv2d1_cnn|block3|conv2d1_relu");
  layers[33] = new MWBatchNormalizationLayer;
  layers[33]->setName("cnn|block3|bn1");
  layers[33]->setInPlaceIndex(0, 0);
  layers[34] = new MWFusedConvActivationLayer;
  layers[34]->setName("cnn|block3|conv2d2_cnn|block3|conv2d2_relu");
  layers[35] = new MWBatchNormalizationLayer;
  layers[35]->setName("cnn|block3|bn2");
  layers[35]->setInPlaceIndex(0, 0);
  layers[36] = new MWFusedConvActivationLayer;
  layers[36]->setName("cnn|block3|conv2d3_cnn|block3|conv2d3_relu");
  layers[37] = new MWBatchNormalizationLayer;
  layers[37]->setName("cnn|block3|bn3");
  layers[37]->setInPlaceIndex(0, 0);
  layers[38] = new MWFusedConvActivationLayer;
  layers[38]->setName("cnn|block3|conv2d4_cnn|block3|conv2d4_relu");
  layers[39] = new MWBatchNormalizationLayer;
  layers[39]->setName("cnn|block3|bn4");
  layers[39]->setInPlaceIndex(0, 0);
  layers[40] = new MWFusedConvActivationLayer;
  layers[40]->setName("cnn|block3|conv2d5_cnn|block3|conv2d5_relu");
  layers[41] = new MWBatchNormalizationLayer;
  layers[41]->setName("cnn|block3|bn5");
  layers[41]->setInPlaceIndex(0, 0);
  layers[42] = new MWResize2DLayer_detector0_02;
  layers[42]->setName("cnn|add|resize|3");
  layers[43] = new MWFusedConvActivationLayer;
  layers[43]->setName("cnn|up3|conv2dt_cnn|up3|conv2dt_relu");
  layers[44] = new MWBatchNormalizationLayer;
  layers[44]->setName("cnn|up3|bn");
  layers[44]->setInPlaceIndex(0, 0);
  layers[45] = new MWConcatenationLayer;
  layers[45]->setName("cnn|concatenate");
  layers[46] = new MWConvLayer;
  layers[46]->setName("angle|conv2d_fused");
  layers[47] = new MWSplittingLayer;
  layers[47]->setName("angle|conv2d_fused_channelSplit");
  layers[47]->setInPlaceIndex(0, 0);
  layers[47]->setInPlaceIndex(1, 0);
  layers[47]->setInPlaceIndex(2, 0);
  layers[47]->setInPlaceIndex(3, 0);
  layers[47]->setInPlaceIndex(4, 0);
  layers[47]->setInPlaceIndex(5, 0);
  layers[48] = new MWSigmoidLayer;
  layers[48]->setName("activation");
  layers[48]->setInPlaceIndex(0, 0);
  layers[49] = new MWOutputLayer;
  layers[49]->setName("output_activation");
  layers[49]->setInPlaceIndex(0, 0);
  layers[50] = new MWOutputLayer;
  layers[50]->setName("output_angle|conv2d");
  layers[50]->setInPlaceIndex(0, 0);
  layers[51] = new MWSigmoidLayer;
  layers[51]->setName("heading|conv2dSigmoid");
  layers[51]->setInPlaceIndex(0, 0);
  layers[52] = new MWOutputLayer;
  layers[52]->setName("output_heading|conv2dSigmoid");
  layers[52]->setInPlaceIndex(0, 0);
  layers[53] = new MWOutputLayer;
  layers[53]->setName("output_loc|conv2d");
  layers[53]->setInPlaceIndex(0, 0);
  layers[54] = new MWSigmoidLayer;
  layers[54]->setName("occupancy|conv2dSigmoid");
  layers[54]->setInPlaceIndex(0, 0);
  layers[55] = new MWOutputLayer;
  layers[55]->setName("output_occupancy|conv2dSigmoid");
  layers[55]->setInPlaceIndex(0, 0);
  layers[56] = new MWOutputLayer;
  layers[56]->setName("output_size|conv2d");
  layers[56]->setInPlaceIndex(0, 0);
  targetImpl = new MWCudnnTarget::MWTargetNetworkImpl;
  inputTensors[0] = new MWTensor<real32_T>;
  inputTensors[0]->setHeight(12000);
  inputTensors[0]->setWidth(2);
  inputTensors[0]->setChannels(1);
  inputTensors[0]->setBatchSize(1);
  inputTensors[0]->setSequenceLength(1);
  inputTensors[1] = new MWTensor<real32_T>;
  inputTensors[1]->setHeight(12000);
  inputTensors[1]->setWidth(100);
  inputTensors[1]->setChannels(9);
  inputTensors[1]->setBatchSize(1);
  inputTensors[1]->setSequenceLength(1);
}

detector0_0::~detector0_0()
{
  emlrtSetIsInDestructor((emlrtCTX *)emlrtRootTLSGlobal, true);
  try {
    if (isInitialized) {
      cleanup();
    }
    for (int32_T idx{0}; idx < 57; idx++) {
      delete layers[idx];
    }
    if (targetImpl) {
      delete targetImpl;
    }
    delete inputTensors[0];
    delete inputTensors[1];
  } catch (...) {
  }
  emlrtSetIsInDestructor((emlrtCTX *)emlrtRootTLSGlobal, false);
}

int32_T detector0_0::getBatchSize()
{
  return inputTensors[0]->getBatchSize();
}

real32_T *detector0_0::getInputDataPointer(int32_T b_index)
{
  return (static_cast<MWTensor<real32_T> *>(inputTensors[b_index]))->getData();
}

real32_T *detector0_0::getInputDataPointer()
{
  return (static_cast<MWTensor<real32_T> *>(inputTensors[0]))->getData();
}

real32_T *detector0_0::getLayerOutput(int32_T layerIndex, int32_T portIndex)
{
  return layers[layerIndex]->getLayerOutput(portIndex);
}

int32_T detector0_0::getLayerOutputSize(int32_T layerIndex, int32_T portIndex)
{
  return static_cast<uint32_T>(
             layers[layerIndex]->getOutputTensor(portIndex)->getNumElements()) *
         sizeof(real32_T);
}

real32_T *detector0_0::getOutputDataPointer(int32_T b_index)
{
  return (static_cast<MWTensor<real32_T> *>(outputTensors[b_index]))->getData();
}

real32_T *detector0_0::getOutputDataPointer()
{
  return (static_cast<MWTensor<real32_T> *>(outputTensors[0]))->getData();
}

int32_T detector0_0::getOutputSequenceLength(int32_T layerIndex,
                                             int32_T portIndex)
{
  return layers[layerIndex]->getOutputTensor(portIndex)->getSequenceLength();
}

namespace coder {
namespace internal {
void dlnetwork_delete(detector0_0 *obj)
{
  if (obj->isInitialized) {
    obj->cleanup();
  }
}

void dlnetwork_setup(detector0_0 *obj)
{
  try {
    obj->setup();
  } catch (std::runtime_error const &err) {
    obj->cleanup();
    checkRunTimeError(err.what(), __FILE__, __LINE__);
  } catch (...) {
    obj->cleanup();
    checkRunTimeError("", __FILE__, __LINE__);
  }
}

void l_dlnetwork_layerPredictWithCol(const real32_T gpu_varargin_1[768000],
                                     const real32_T gpu_varargin_2[24000],
                                     real32_T gpu_varargout_1[13713408])
{
  dim3 block;
  dim3 grid;
  int32_T cpu_tmp_data[12000];
  int32_T(*gpu_tmp_data)[12000];
  int32_T partialTrueCount;
  int32_T trueCount;
  real32_T(*cpu_varargout_1)[13713408];
  real32_T(*cpu_varargin_1)[768000];
  real32_T cpu_maps_data[24000];
  real32_T(*cpu_varargin_2)[24000];
  real32_T(*gpu_maps_data)[24000];
  int16_T tmp_data[12000];
  boolean_T indices[12000];
  boolean_T b;
  boolean_T validLaunchParams;
  boolean_T varargin_1_outdatedOnCpu;
  boolean_T varargin_2_outdatedOnCpu;
  mwCudaMalloc(&gpu_tmp_data, 48000UL);
  mwCudaMalloc(&gpu_maps_data, 96000UL);
  cpu_varargout_1 = (real32_T(*)[13713408])malloc(54853632U);
  cpu_varargin_2 = (real32_T(*)[24000])malloc(96000U);
  cpu_varargin_1 = (real32_T(*)[768000])malloc(3072000U);
  varargin_2_outdatedOnCpu = true;
  varargin_1_outdatedOnCpu = true;
  std::memset(&(*cpu_varargout_1)[0], 0, 13713408U * sizeof(real32_T));
  trueCount = 0;
  for (int32_T i{0}; i < 12000; i++) {
    if (varargin_2_outdatedOnCpu) {
      hipMemcpy(*cpu_varargin_2, gpu_varargin_2, 96000UL,
                 hipMemcpyDeviceToHost);
    }
    varargin_2_outdatedOnCpu = false;
    b = ((*cpu_varargin_2)[i] >= 1.0F);
    indices[i] = b;
    if (b) {
      trueCount++;
    }
  }
  partialTrueCount = 0;
  for (int32_T i{0}; i < 12000; i++) {
    if (indices[i]) {
      cpu_tmp_data[partialTrueCount] = i;
      partialTrueCount++;
    }
  }
  b = false;
  for (partialTrueCount = 0; partialTrueCount < 2; partialTrueCount++) {
    for (int32_T i{0}; i < trueCount; i++) {
      if (varargin_2_outdatedOnCpu) {
        hipMemcpy(*cpu_varargin_2, gpu_varargin_2, 96000UL,
                   hipMemcpyDeviceToHost);
      }
      varargin_2_outdatedOnCpu = false;
      cpu_maps_data[i + trueCount * partialTrueCount] =
          (*cpu_varargin_2)[cpu_tmp_data[i] + 12000 * partialTrueCount] - 1.0F;
      b = true;
    }
  }
  mwGetLaunchParameters1D(computeNumIters(trueCount - 1), &grid, &block,
                          2147483647U);
  if (b) {
    hipMemcpy(*gpu_maps_data, cpu_maps_data, 96000UL, hipMemcpyHostToDevice);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    c_dlnetwork_layerPredictWithCol<<<grid, block>>>(
        *gpu_maps_data, trueCount, trueCount - 1, *gpu_tmp_data);
  }
  b = true;
  partialTrueCount = 0;
  for (int32_T i{0}; i < 12000; i++) {
    if (indices[i]) {
      tmp_data[partialTrueCount] = static_cast<int16_T>(i);
      partialTrueCount++;
    }
  }
  for (partialTrueCount = 0; partialTrueCount < 64; partialTrueCount++) {
    for (int32_T i{0}; i < trueCount; i++) {
      if (b) {
        hipMemcpy(cpu_tmp_data, *gpu_tmp_data, 48000UL,
                   hipMemcpyDeviceToHost);
      }
      b = false;
      if (varargin_1_outdatedOnCpu) {
        hipMemcpy(*cpu_varargin_1, gpu_varargin_1, 3072000UL,
                   hipMemcpyDeviceToHost);
      }
      varargin_1_outdatedOnCpu = false;
      (*cpu_varargout_1)[cpu_tmp_data[i] + 214272 * partialTrueCount] =
          (*cpu_varargin_1)[tmp_data[i] + 12000 * partialTrueCount];
    }
  }
  hipMemcpy(gpu_varargout_1, *cpu_varargout_1, 54853632UL,
             hipMemcpyHostToDevice);
  free(*cpu_varargin_1);
  free(*cpu_varargin_2);
  free(*cpu_varargout_1);
  mwCudaFree(*gpu_maps_data);
  mwCudaFree(*gpu_tmp_data);
}

void l_dlnetwork_layerPredictWithCol(const real32_T varargin_1[1714176],
                                     real32_T varargout_1[6856704])
{
  static const real_T b_cpu_xq[248]{
      1.0,    1.25,   1.75,   2.25,   2.75,   3.25,   3.75,   4.25,   4.75,
      5.25,   5.75,   6.25,   6.75,   7.25,   7.75,   8.25,   8.75,   9.25,
      9.75,   10.25,  10.75,  11.25,  11.75,  12.25,  12.75,  13.25,  13.75,
      14.25,  14.75,  15.25,  15.75,  16.25,  16.75,  17.25,  17.75,  18.25,
      18.75,  19.25,  19.75,  20.25,  20.75,  21.25,  21.75,  22.25,  22.75,
      23.25,  23.75,  24.25,  24.75,  25.25,  25.75,  26.25,  26.75,  27.25,
      27.75,  28.25,  28.75,  29.25,  29.75,  30.25,  30.75,  31.25,  31.75,
      32.25,  32.75,  33.25,  33.75,  34.25,  34.75,  35.25,  35.75,  36.25,
      36.75,  37.25,  37.75,  38.25,  38.75,  39.25,  39.75,  40.25,  40.75,
      41.25,  41.75,  42.25,  42.75,  43.25,  43.75,  44.25,  44.75,  45.25,
      45.75,  46.25,  46.75,  47.25,  47.75,  48.25,  48.75,  49.25,  49.75,
      50.25,  50.75,  51.25,  51.75,  52.25,  52.75,  53.25,  53.75,  54.25,
      54.75,  55.25,  55.75,  56.25,  56.75,  57.25,  57.75,  58.25,  58.75,
      59.25,  59.75,  60.25,  60.75,  61.25,  61.75,  62.25,  62.75,  63.25,
      63.75,  64.25,  64.75,  65.25,  65.75,  66.25,  66.75,  67.25,  67.75,
      68.25,  68.75,  69.25,  69.75,  70.25,  70.75,  71.25,  71.75,  72.25,
      72.75,  73.25,  73.75,  74.25,  74.75,  75.25,  75.75,  76.25,  76.75,
      77.25,  77.75,  78.25,  78.75,  79.25,  79.75,  80.25,  80.75,  81.25,
      81.75,  82.25,  82.75,  83.25,  83.75,  84.25,  84.75,  85.25,  85.75,
      86.25,  86.75,  87.25,  87.75,  88.25,  88.75,  89.25,  89.75,  90.25,
      90.75,  91.25,  91.75,  92.25,  92.75,  93.25,  93.75,  94.25,  94.75,
      95.25,  95.75,  96.25,  96.75,  97.25,  97.75,  98.25,  98.75,  99.25,
      99.75,  100.25, 100.75, 101.25, 101.75, 102.25, 102.75, 103.25, 103.75,
      104.25, 104.75, 105.25, 105.75, 106.25, 106.75, 107.25, 107.75, 108.25,
      108.75, 109.25, 109.75, 110.25, 110.75, 111.25, 111.75, 112.25, 112.75,
      113.25, 113.75, 114.25, 114.75, 115.25, 115.75, 116.25, 116.75, 117.25,
      117.75, 118.25, 118.75, 119.25, 119.75, 120.25, 120.75, 121.25, 121.75,
      122.25, 122.75, 123.25, 123.75, 124.0};
  static const real_T cpu_xq[216]{
      1.0,    1.25,   1.75,   2.25,   2.75,   3.25,   3.75,   4.25,   4.75,
      5.25,   5.75,   6.25,   6.75,   7.25,   7.75,   8.25,   8.75,   9.25,
      9.75,   10.25,  10.75,  11.25,  11.75,  12.25,  12.75,  13.25,  13.75,
      14.25,  14.75,  15.25,  15.75,  16.25,  16.75,  17.25,  17.75,  18.25,
      18.75,  19.25,  19.75,  20.25,  20.75,  21.25,  21.75,  22.25,  22.75,
      23.25,  23.75,  24.25,  24.75,  25.25,  25.75,  26.25,  26.75,  27.25,
      27.75,  28.25,  28.75,  29.25,  29.75,  30.25,  30.75,  31.25,  31.75,
      32.25,  32.75,  33.25,  33.75,  34.25,  34.75,  35.25,  35.75,  36.25,
      36.75,  37.25,  37.75,  38.25,  38.75,  39.25,  39.75,  40.25,  40.75,
      41.25,  41.75,  42.25,  42.75,  43.25,  43.75,  44.25,  44.75,  45.25,
      45.75,  46.25,  46.75,  47.25,  47.75,  48.25,  48.75,  49.25,  49.75,
      50.25,  50.75,  51.25,  51.75,  52.25,  52.75,  53.25,  53.75,  54.25,
      54.75,  55.25,  55.75,  56.25,  56.75,  57.25,  57.75,  58.25,  58.75,
      59.25,  59.75,  60.25,  60.75,  61.25,  61.75,  62.25,  62.75,  63.25,
      63.75,  64.25,  64.75,  65.25,  65.75,  66.25,  66.75,  67.25,  67.75,
      68.25,  68.75,  69.25,  69.75,  70.25,  70.75,  71.25,  71.75,  72.25,
      72.75,  73.25,  73.75,  74.25,  74.75,  75.25,  75.75,  76.25,  76.75,
      77.25,  77.75,  78.25,  78.75,  79.25,  79.75,  80.25,  80.75,  81.25,
      81.75,  82.25,  82.75,  83.25,  83.75,  84.25,  84.75,  85.25,  85.75,
      86.25,  86.75,  87.25,  87.75,  88.25,  88.75,  89.25,  89.75,  90.25,
      90.75,  91.25,  91.75,  92.25,  92.75,  93.25,  93.75,  94.25,  94.75,
      95.25,  95.75,  96.25,  96.75,  97.25,  97.75,  98.25,  98.75,  99.25,
      99.75,  100.25, 100.75, 101.25, 101.75, 102.25, 102.75, 103.25, 103.75,
      104.25, 104.75, 105.25, 105.75, 106.25, 106.75, 107.25, 107.75, 108.0};
  real32_T(*b_gpu_out)[6856704];
  real32_T(*gpu_in)[3428352];
  real32_T(*gpu_out)[3428352];
  if (!c_gpuConstsCopied_dlnetwork_lay) {
    c_gpuConstsCopied_dlnetwork_lay = true;
    hipMemcpy(*global_gpu_xq, cpu_xq, sizeof(real_T[216]),
               hipMemcpyHostToDevice);
    hipMemcpy(*b_global_gpu_xq, b_cpu_xq, sizeof(real_T[248]),
               hipMemcpyHostToDevice);
  }
  mwCudaMalloc(&b_gpu_out, 27426816UL);
  mwCudaMalloc(&gpu_in, 13713408UL);
  mwCudaMalloc(&gpu_out, 13713408UL);
  d_dlnetwork_layerPredictWithCol<<<dim3(6696U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      varargin_1, *global_gpu_xq, *gpu_out);
  e_dlnetwork_layerPredictWithCol<<<dim3(6696U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_out, *gpu_in);
  f_dlnetwork_layerPredictWithCol<<<dim3(13392U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_in, *b_global_gpu_xq, *b_gpu_out);
  g_dlnetwork_layerPredictWithCol<<<dim3(13392U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *b_gpu_out, varargout_1);
  mwCudaFree(*gpu_out);
  mwCudaFree(*gpu_in);
  mwCudaFree(*b_gpu_out);
}

void m_dlnetwork_layerPredictWithCol(const real32_T varargin_1[428544],
                                     real32_T varargout_1[6856704])
{
  static const real_T b_cpu_xq[248]{
      1.0,    1.0,    1.125,  1.375,  1.625,  1.875,  2.125,  2.375,  2.625,
      2.875,  3.125,  3.375,  3.625,  3.875,  4.125,  4.375,  4.625,  4.875,
      5.125,  5.375,  5.625,  5.875,  6.125,  6.375,  6.625,  6.875,  7.125,
      7.375,  7.625,  7.875,  8.125,  8.375,  8.625,  8.875,  9.125,  9.375,
      9.625,  9.875,  10.125, 10.375, 10.625, 10.875, 11.125, 11.375, 11.625,
      11.875, 12.125, 12.375, 12.625, 12.875, 13.125, 13.375, 13.625, 13.875,
      14.125, 14.375, 14.625, 14.875, 15.125, 15.375, 15.625, 15.875, 16.125,
      16.375, 16.625, 16.875, 17.125, 17.375, 17.625, 17.875, 18.125, 18.375,
      18.625, 18.875, 19.125, 19.375, 19.625, 19.875, 20.125, 20.375, 20.625,
      20.875, 21.125, 21.375, 21.625, 21.875, 22.125, 22.375, 22.625, 22.875,
      23.125, 23.375, 23.625, 23.875, 24.125, 24.375, 24.625, 24.875, 25.125,
      25.375, 25.625, 25.875, 26.125, 26.375, 26.625, 26.875, 27.125, 27.375,
      27.625, 27.875, 28.125, 28.375, 28.625, 28.875, 29.125, 29.375, 29.625,
      29.875, 30.125, 30.375, 30.625, 30.875, 31.125, 31.375, 31.625, 31.875,
      32.125, 32.375, 32.625, 32.875, 33.125, 33.375, 33.625, 33.875, 34.125,
      34.375, 34.625, 34.875, 35.125, 35.375, 35.625, 35.875, 36.125, 36.375,
      36.625, 36.875, 37.125, 37.375, 37.625, 37.875, 38.125, 38.375, 38.625,
      38.875, 39.125, 39.375, 39.625, 39.875, 40.125, 40.375, 40.625, 40.875,
      41.125, 41.375, 41.625, 41.875, 42.125, 42.375, 42.625, 42.875, 43.125,
      43.375, 43.625, 43.875, 44.125, 44.375, 44.625, 44.875, 45.125, 45.375,
      45.625, 45.875, 46.125, 46.375, 46.625, 46.875, 47.125, 47.375, 47.625,
      47.875, 48.125, 48.375, 48.625, 48.875, 49.125, 49.375, 49.625, 49.875,
      50.125, 50.375, 50.625, 50.875, 51.125, 51.375, 51.625, 51.875, 52.125,
      52.375, 52.625, 52.875, 53.125, 53.375, 53.625, 53.875, 54.125, 54.375,
      54.625, 54.875, 55.125, 55.375, 55.625, 55.875, 56.125, 56.375, 56.625,
      56.875, 57.125, 57.375, 57.625, 57.875, 58.125, 58.375, 58.625, 58.875,
      59.125, 59.375, 59.625, 59.875, 60.125, 60.375, 60.625, 60.875, 61.125,
      61.375, 61.625, 61.875, 62.0,   62.0};
  static const real_T cpu_xq[216]{
      1.0,    1.0,    1.125,  1.375,  1.625,  1.875,  2.125,  2.375,  2.625,
      2.875,  3.125,  3.375,  3.625,  3.875,  4.125,  4.375,  4.625,  4.875,
      5.125,  5.375,  5.625,  5.875,  6.125,  6.375,  6.625,  6.875,  7.125,
      7.375,  7.625,  7.875,  8.125,  8.375,  8.625,  8.875,  9.125,  9.375,
      9.625,  9.875,  10.125, 10.375, 10.625, 10.875, 11.125, 11.375, 11.625,
      11.875, 12.125, 12.375, 12.625, 12.875, 13.125, 13.375, 13.625, 13.875,
      14.125, 14.375, 14.625, 14.875, 15.125, 15.375, 15.625, 15.875, 16.125,
      16.375, 16.625, 16.875, 17.125, 17.375, 17.625, 17.875, 18.125, 18.375,
      18.625, 18.875, 19.125, 19.375, 19.625, 19.875, 20.125, 20.375, 20.625,
      20.875, 21.125, 21.375, 21.625, 21.875, 22.125, 22.375, 22.625, 22.875,
      23.125, 23.375, 23.625, 23.875, 24.125, 24.375, 24.625, 24.875, 25.125,
      25.375, 25.625, 25.875, 26.125, 26.375, 26.625, 26.875, 27.125, 27.375,
      27.625, 27.875, 28.125, 28.375, 28.625, 28.875, 29.125, 29.375, 29.625,
      29.875, 30.125, 30.375, 30.625, 30.875, 31.125, 31.375, 31.625, 31.875,
      32.125, 32.375, 32.625, 32.875, 33.125, 33.375, 33.625, 33.875, 34.125,
      34.375, 34.625, 34.875, 35.125, 35.375, 35.625, 35.875, 36.125, 36.375,
      36.625, 36.875, 37.125, 37.375, 37.625, 37.875, 38.125, 38.375, 38.625,
      38.875, 39.125, 39.375, 39.625, 39.875, 40.125, 40.375, 40.625, 40.875,
      41.125, 41.375, 41.625, 41.875, 42.125, 42.375, 42.625, 42.875, 43.125,
      43.375, 43.625, 43.875, 44.125, 44.375, 44.625, 44.875, 45.125, 45.375,
      45.625, 45.875, 46.125, 46.375, 46.625, 46.875, 47.125, 47.375, 47.625,
      47.875, 48.125, 48.375, 48.625, 48.875, 49.125, 49.375, 49.625, 49.875,
      50.125, 50.375, 50.625, 50.875, 51.125, 51.375, 51.625, 51.875, 52.125,
      52.375, 52.625, 52.875, 53.125, 53.375, 53.625, 53.875, 54.0,   54.0};
  real32_T(*b_gpu_out)[6856704];
  real32_T(*gpu_in)[1714176];
  real32_T(*gpu_out)[1714176];
  if (!d_gpuConstsCopied_dlnetwork_lay) {
    d_gpuConstsCopied_dlnetwork_lay = true;
    hipMemcpy(*c_global_gpu_xq, cpu_xq, sizeof(real_T[216]),
               hipMemcpyHostToDevice);
    hipMemcpy(*d_global_gpu_xq, b_cpu_xq, sizeof(real_T[248]),
               hipMemcpyHostToDevice);
  }
  mwCudaMalloc(&b_gpu_out, 27426816UL);
  mwCudaMalloc(&gpu_in, 6856704UL);
  mwCudaMalloc(&gpu_out, 6856704UL);
  h_dlnetwork_layerPredictWithCol<<<dim3(3348U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      varargin_1, *c_global_gpu_xq, *gpu_out);
  i_dlnetwork_layerPredictWithCol<<<dim3(3348U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_out, *gpu_in);
  j_dlnetwork_layerPredictWithCol<<<dim3(13392U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_in, *d_global_gpu_xq, *b_gpu_out);
  k_dlnetwork_layerPredictWithCol<<<dim3(13392U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *b_gpu_out, varargout_1);
  mwCudaFree(*gpu_out);
  mwCudaFree(*gpu_in);
  mwCudaFree(*b_gpu_out);
}

} // namespace internal
} // namespace coder

// End of code generation (dlnetwork.cu)
